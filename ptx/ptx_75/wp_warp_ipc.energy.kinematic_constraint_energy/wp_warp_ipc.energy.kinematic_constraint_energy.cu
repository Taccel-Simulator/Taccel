#include "hip/hip_runtime.h"

#define WP_TILE_BLOCK_DIM 256
#define WP_NO_CRT
#include "builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx, dim)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)

extern "C" {
}

struct COOMatrix3x3_0df4b45d
{
    wp::array_t<wp::int32> rows;
    wp::array_t<wp::int32> cols;
    wp::array_t<wp::mat_t<3,3,wp::float64>> vals;
    wp::int32 capacity;
    wp::int32 size;
    wp::int32 n_rows;
    wp::int32 n_cols;


    COOMatrix3x3_0df4b45d() = default;
    CUDA_CALLABLE COOMatrix3x3_0df4b45d(wp::array_t<wp::int32> const& rows,
    wp::array_t<wp::int32> const& cols = {},
    wp::array_t<wp::mat_t<3,3,wp::float64>> const& vals = {},
    wp::int32 const& capacity = {},
    wp::int32 const& size = {},
    wp::int32 const& n_rows = {},
    wp::int32 const& n_cols = {})
        : rows{rows}
        , cols{cols}
        , vals{vals}
        , capacity{capacity}
        , size{size}
        , n_rows{n_rows}
        , n_cols{n_cols}

    {
    }

    CUDA_CALLABLE COOMatrix3x3_0df4b45d& operator += (const COOMatrix3x3_0df4b45d& rhs)
    {    capacity += rhs.capacity;
    size += rhs.size;
    n_rows += rhs.n_rows;
    n_cols += rhs.n_cols;

        return *this;}

};

static CUDA_CALLABLE void adj_COOMatrix3x3_0df4b45d(wp::array_t<wp::int32> const&,
    wp::array_t<wp::int32> const&,
    wp::array_t<wp::mat_t<3,3,wp::float64>> const&,
    wp::int32 const&,
    wp::int32 const&,
    wp::int32 const&,
    wp::int32 const&,
    wp::array_t<wp::int32> & adj_rows,
    wp::array_t<wp::int32> & adj_cols,
    wp::array_t<wp::mat_t<3,3,wp::float64>> & adj_vals,
    wp::int32 & adj_capacity,
    wp::int32 & adj_size,
    wp::int32 & adj_n_rows,
    wp::int32 & adj_n_cols,
    COOMatrix3x3_0df4b45d & adj_ret)
{
    adj_rows = adj_ret.rows;
    adj_cols = adj_ret.cols;
    adj_vals = adj_ret.vals;
    adj_capacity += adj_ret.capacity;
    adj_size += adj_ret.size;
    adj_n_rows += adj_ret.n_rows;
    adj_n_cols += adj_ret.n_cols;
}

CUDA_CALLABLE void adj_atomic_add(COOMatrix3x3_0df4b45d* p, COOMatrix3x3_0df4b45d t)
{
    wp::adj_atomic_add(&p->rows, t.rows);
    wp::adj_atomic_add(&p->cols, t.cols);
    wp::adj_atomic_add(&p->vals, t.vals);
    wp::adj_atomic_add(&p->capacity, t.capacity);
    wp::adj_atomic_add(&p->size, t.size);
    wp::adj_atomic_add(&p->n_rows, t.n_rows);
    wp::adj_atomic_add(&p->n_cols, t.n_cols);
}



// /home/changyu/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/wp_math.py:145
static CUDA_CALLABLE wp::float64 sqr_0(
    wp::float64 var_x)
{
    //---------
    // primal vars
    wp::float64 var_0;
    //---------
    // forward
    // def sqr(x: wp.float64):                                                                <L 146>
    // return x * x                                                                           <L 147>
    var_0 = wp::mul(var_x, var_x);
    return var_0;
}


// /home/changyu/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/wp_math.py:145
static CUDA_CALLABLE void adj_sqr_0(
    wp::float64 var_x,
    wp::float64 & adj_x,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::float64 var_0;
    //---------
    // dual vars
    wp::float64 adj_0 = {};
    //---------
    // forward
    // def sqr(x: wp.float64):                                                                <L 146>
    // return x * x                                                                           <L 147>
    var_0 = wp::mul(var_x, var_x);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_0 += adj_ret;
    wp::adj_mul(var_x, var_x, adj_x, adj_x, adj_0);
    // adj: return x * x                                                                      <L 147>
    // adj: def sqr(x: wp.float64):                                                           <L 146>
    return;
}


// /home/changyu/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/matrix.py:152
static CUDA_CALLABLE void COOMatrix3x3_atomic_add_0(
    COOMatrix3x3_0df4b45d var_mat,
    wp::int32 var_block_index,
    wp::mat_t<3,3,wp::float64> var_v)
{
    //---------
    // primal vars
    const bool var_0 = true;
    const wp::int32 var_1 = 0;
    bool var_2;
    wp::int32* var_3;
    bool var_4;
    wp::int32 var_5;
    bool var_6;
    const wp::str var_7 = "[COOMatrix OFB Error]\tblock_index: %d, size: %d!!!!!\n";
    wp::int32* var_8;
    wp::int32 var_9;
    wp::array_t<wp::mat_t<3,3,wp::float64>>* var_10;
    wp::mat_t<3,3,wp::float64> var_11;
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_12;
    //---------
    // forward
    // def COOMatrix3x3_atomic_add(mat: COOMatrix3x3, block_index: wp.int32, v: wp.mat33d):       <L 153>
    // if ENABLE_COO_OFB:                                                                     <L 154>
    // if block_index < 0 or block_index >= mat.size:                                         <L 155>
    var_2 = (var_block_index < var_1);
    var_3 = &(var_mat.size);
    var_5 = wp::load(var_3);
    var_4 = (var_block_index >= var_5);
    var_6 = var_2 || var_4;
    if (var_6) {
        // wp.printf(                                                                         <L 156>
        // "[COOMatrix OFB Error]\tblock_index: %d, size: %d!!!!!\n",                         <L 157>
        // block_index,                                                                       <L 158>
        // mat.size,                                                                          <L 159>
        var_8 = &(var_mat.size);
        var_9 = wp::load(var_8);
        printf(var_7, var_block_index, var_9);
        // return                                                                             <L 161>
        return;
    }
    // wp.atomic_add(mat.vals, block_index, v)                                                <L 162>
    var_10 = &(var_mat.vals);
    var_12 = wp::load(var_10);
    var_11 = wp::atomic_add(var_12, var_block_index, var_v);
}


// /home/changyu/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/matrix.py:152
static CUDA_CALLABLE void adj_COOMatrix3x3_atomic_add_0(
    COOMatrix3x3_0df4b45d var_mat,
    wp::int32 var_block_index,
    wp::mat_t<3,3,wp::float64> var_v,
    COOMatrix3x3_0df4b45d & adj_mat,
    wp::int32 & adj_block_index,
    wp::mat_t<3,3,wp::float64> & adj_v)
{
    //---------
    // primal vars
    const bool var_0 = true;
    const wp::int32 var_1 = 0;
    bool var_2;
    wp::int32* var_3;
    bool var_4;
    wp::int32 var_5;
    bool var_6;
    const wp::str var_7 = "[COOMatrix OFB Error]\tblock_index: %d, size: %d!!!!!\n";
    wp::int32* var_8;
    wp::int32 var_9;
    wp::array_t<wp::mat_t<3,3,wp::float64>>* var_10;
    wp::mat_t<3,3,wp::float64> var_11;
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_12;
    //---------
    // dual vars
    bool adj_0 = {};
    wp::int32 adj_1 = {};
    bool adj_2 = {};
    wp::int32 adj_3 = {};
    bool adj_4 = {};
    wp::int32 adj_5 = {};
    bool adj_6 = {};
    wp::str adj_7 = {};
    wp::int32 adj_8 = {};
    wp::int32 adj_9 = {};
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_10 = {};
    wp::mat_t<3,3,wp::float64> adj_11 = {};
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_12 = {};
    //---------
    // forward
    // def COOMatrix3x3_atomic_add(mat: COOMatrix3x3, block_index: wp.int32, v: wp.mat33d):       <L 153>
    // if ENABLE_COO_OFB:                                                                     <L 154>
    // if block_index < 0 or block_index >= mat.size:                                         <L 155>
    var_2 = (var_block_index < var_1);
    var_3 = &(var_mat.size);
    var_5 = wp::load(var_3);
    var_4 = (var_block_index >= var_5);
    var_6 = var_2 || var_4;
    if (var_6) {
        // wp.printf(                                                                         <L 156>
        // "[COOMatrix OFB Error]\tblock_index: %d, size: %d!!!!!\n",                         <L 157>
        // block_index,                                                                       <L 158>
        // mat.size,                                                                          <L 159>
        var_8 = &(var_mat.size);
        var_9 = wp::load(var_8);
        printf(var_7, var_block_index, var_9);
        // return                                                                             <L 161>
        goto label0;
    }
    // wp.atomic_add(mat.vals, block_index, v)                                                <L 162>
    var_10 = &(var_mat.vals);
    var_12 = wp::load(var_10);
    // var_11 = wp::atomic_add(var_12, var_block_index, var_v);
    //---------
    // reverse
    wp::adj_atomic_add(var_12, var_block_index, var_v, adj_10, adj_block_index, adj_v, adj_11);
    wp::adj_load(var_10, adj_10, adj_12);
    adj_mat.vals = adj_10;
    // adj: wp.atomic_add(mat.vals, block_index, v)                                           <L 162>
    if (var_6) {
        label0:;
        // adj: return                                                                        <L 161>
        adj_printf(var_7, var_block_index, var_9, adj_7, adj_block_index, adj_8);
        wp::adj_load(var_8, adj_8, adj_9);
        adj_mat.size = adj_8;
        // adj: mat.size,                                                                     <L 159>
        // adj: block_index,                                                                  <L 158>
        // adj: "[COOMatrix OFB Error]\tblock_index: %d, size: %d!!!!!\n",                    <L 157>
        // adj: wp.printf(                                                                    <L 156>
    }
    wp::adj_load(var_3, adj_3, adj_5);
    adj_mat.size = adj_3;
    // adj: if block_index < 0 or block_index >= mat.size:                                    <L 155>
    // adj: if ENABLE_COO_OFB:                                                                <L 154>
    // adj: def COOMatrix3x3_atomic_add(mat: COOMatrix3x3, block_index: wp.int32, v: wp.mat33d):  <L 153>
    return;
}



extern "C" __global__ void compute_soft_kinematic_energy_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::float64> var_soft_energy,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::float64> var_soft_verts_mass)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        bool var_2;
        bool var_3;
        wp::float64* var_4;
        wp::float64 var_5;
        wp::float64 var_6;
        const wp::float32 var_7 = 0.5;
        wp::float64 var_8;
        wp::float64 var_9;
        wp::float64 var_10;
        wp::int32 var_11;
        wp::vec_t<3,wp::float64>* var_12;
        wp::vec_t<3,wp::float64>* var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64> var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::float64 var_17;
        wp::float64 var_18;
        wp::float64 var_19;
        //---------
        // forward
        // def compute_soft_kinematic_energy(                                                     <L 86>
        // tid = wp.tid()                                                                         <L 95>
        var_0 = builtin_tid1d();
        // has_constraint = soft_has_constraint[tid]                                              <L 96>
        var_1 = wp::address(var_soft_has_constraint, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // mass = soft_verts_mass[tid]                                                            <L 97>
        var_4 = wp::address(var_soft_verts_mass, var_0);
        var_6 = wp::load(var_4);
        var_5 = wp::copy(var_6);
        // if has_constraint:                                                                     <L 98>
        if (var_2) {
            // soft_energy[tid] += wp.float64(0.5) * weight * mass * wp.length_sq(x[tid + affine_verts_num] - soft_target_dof[tid])       <L 99>
            var_8 = wp::float64(var_7);
            var_9 = wp::mul(var_8, var_weight);
            var_10 = wp::mul(var_9, var_5);
            var_11 = wp::add(var_0, var_affine_verts_num);
            var_12 = wp::address(var_x, var_11);
            var_13 = wp::address(var_soft_target_dof, var_0);
            var_15 = wp::load(var_12);
            var_16 = wp::load(var_13);
            var_14 = wp::sub(var_15, var_16);
            var_17 = wp::length_sq(var_14);
            var_18 = wp::mul(var_10, var_17);
            var_19 = wp::atomic_add(var_soft_energy, var_0, var_18);
        }
    }
}



extern "C" __global__ void compute_soft_kinematic_energy_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::float64> var_soft_energy,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<bool> adj_soft_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_soft_target_dof,
    wp::float64 adj_weight,
    wp::array_t<wp::float64> adj_soft_energy,
    wp::int32 adj_affine_verts_num,
    wp::array_t<wp::float64> adj_soft_verts_mass)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        bool var_2;
        bool var_3;
        wp::float64* var_4;
        wp::float64 var_5;
        wp::float64 var_6;
        const wp::float32 var_7 = 0.5;
        wp::float64 var_8;
        wp::float64 var_9;
        wp::float64 var_10;
        wp::int32 var_11;
        wp::vec_t<3,wp::float64>* var_12;
        wp::vec_t<3,wp::float64>* var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64> var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::float64 var_17;
        wp::float64 var_18;
        wp::float64 var_19;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        bool adj_1 = {};
        bool adj_2 = {};
        bool adj_3 = {};
        wp::float64 adj_4 = {};
        wp::float64 adj_5 = {};
        wp::float64 adj_6 = {};
        wp::float32 adj_7 = {};
        wp::float64 adj_8 = {};
        wp::float64 adj_9 = {};
        wp::float64 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::vec_t<3,wp::float64> adj_12 = {};
        wp::vec_t<3,wp::float64> adj_13 = {};
        wp::vec_t<3,wp::float64> adj_14 = {};
        wp::vec_t<3,wp::float64> adj_15 = {};
        wp::vec_t<3,wp::float64> adj_16 = {};
        wp::float64 adj_17 = {};
        wp::float64 adj_18 = {};
        wp::float64 adj_19 = {};
        //---------
        // forward
        // def compute_soft_kinematic_energy(                                                     <L 86>
        // tid = wp.tid()                                                                         <L 95>
        var_0 = builtin_tid1d();
        // has_constraint = soft_has_constraint[tid]                                              <L 96>
        var_1 = wp::address(var_soft_has_constraint, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // mass = soft_verts_mass[tid]                                                            <L 97>
        var_4 = wp::address(var_soft_verts_mass, var_0);
        var_6 = wp::load(var_4);
        var_5 = wp::copy(var_6);
        // if has_constraint:                                                                     <L 98>
        if (var_2) {
            // soft_energy[tid] += wp.float64(0.5) * weight * mass * wp.length_sq(x[tid + affine_verts_num] - soft_target_dof[tid])       <L 99>
            var_8 = wp::float64(var_7);
            var_9 = wp::mul(var_8, var_weight);
            var_10 = wp::mul(var_9, var_5);
            var_11 = wp::add(var_0, var_affine_verts_num);
            var_12 = wp::address(var_x, var_11);
            var_13 = wp::address(var_soft_target_dof, var_0);
            var_15 = wp::load(var_12);
            var_16 = wp::load(var_13);
            var_14 = wp::sub(var_15, var_16);
            var_17 = wp::length_sq(var_14);
            var_18 = wp::mul(var_10, var_17);
            // var_19 = wp::atomic_add(var_soft_energy, var_0, var_18);
        }
        //---------
        // reverse
        if (var_2) {
            wp::adj_atomic_add(var_soft_energy, var_0, var_18, adj_soft_energy, adj_0, adj_18, adj_19);
            wp::adj_mul(var_10, var_17, adj_10, adj_17, adj_18);
            wp::adj_length_sq(var_14, adj_14, adj_17);
            wp::adj_sub(var_15, var_16, adj_12, adj_13, adj_14);
            wp::adj_load(var_13, adj_13, adj_16);
            wp::adj_load(var_12, adj_12, adj_15);
            wp::adj_address(var_soft_target_dof, var_0, adj_soft_target_dof, adj_0, adj_13);
            wp::adj_address(var_x, var_11, adj_x, adj_11, adj_12);
            wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_11);
            wp::adj_mul(var_9, var_5, adj_9, adj_5, adj_10);
            wp::adj_mul(var_8, var_weight, adj_8, adj_weight, adj_9);
            wp::adj_float64(var_7, adj_7, adj_8);
            // adj: soft_energy[tid] += wp.float64(0.5) * weight * mass * wp.length_sq(x[tid + affine_verts_num] - soft_target_dof[tid])  <L 99>
        }
        // adj: if has_constraint:                                                                <L 98>
        wp::adj_copy(var_6, adj_4, adj_5);
        wp::adj_load(var_4, adj_4, adj_6);
        wp::adj_address(var_soft_verts_mass, var_0, adj_soft_verts_mass, adj_0, adj_4);
        // adj: mass = soft_verts_mass[tid]                                                       <L 97>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_soft_has_constraint, var_0, adj_soft_has_constraint, adj_0, adj_1);
        // adj: has_constraint = soft_has_constraint[tid]                                         <L 96>
        // adj: tid = wp.tid()                                                                    <L 95>
        // adj: def compute_soft_kinematic_energy(                                                <L 86>
        continue;
    }
}



extern "C" __global__ void compute_affine_kinematic_grad_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_grad,
    wp::array_t<wp::float64> var_mass_body,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32* var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 1;
        bool var_5;
        wp::int32 var_6;
        wp::int32* var_7;
        wp::int32* var_8;
        wp::int32 var_9;
        const wp::int32 var_10 = 2;
        bool var_11;
        wp::int32 var_12;
        bool var_13;
        wp::float64* var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        bool* var_17;
        bool var_18;
        bool var_19;
        wp::float64 var_20;
        wp::vec_t<12,wp::float64>* var_21;
        wp::vec_t<12,wp::float64>* var_22;
        wp::vec_t<12,wp::float64> var_23;
        wp::vec_t<12,wp::float64> var_24;
        wp::vec_t<12,wp::float64> var_25;
        wp::vec_t<12,wp::float64> var_26;
        wp::vec_t<12,wp::float64> var_27;
        //---------
        // forward
        // def compute_affine_kinematic_grad(                                                     <L 143>
        // tid = wp.tid()                                                                         <L 153>
        var_0 = builtin_tid1d();
        // if env_states[body_env_id[tid]] == ENV_STATE_INVALID or env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED:       <L 154>
        var_1 = wp::address(var_body_env_id, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::address(var_env_states, var_3);
        var_6 = wp::load(var_2);
        var_5 = (var_6 == var_4);
        var_7 = wp::address(var_body_env_id, var_0);
        var_9 = wp::load(var_7);
        var_8 = wp::address(var_env_states, var_9);
        var_12 = wp::load(var_8);
        var_11 = (var_12 == var_10);
        var_13 = var_5 || var_11;
        if (var_13) {
            // return                                                                             <L 155>
            continue;
        }
        // mass = mass_body[tid]                                                                  <L 156>
        var_14 = wp::address(var_mass_body, var_0);
        var_16 = wp::load(var_14);
        var_15 = wp::copy(var_16);
        // has_constraint = affine_has_constraint[tid]                                            <L 157>
        var_17 = wp::address(var_affine_has_constraint, var_0);
        var_19 = wp::load(var_17);
        var_18 = wp::copy(var_19);
        // if has_constraint:                                                                     <L 158>
        if (var_18) {
            // wp.atomic_add(affine_grad, tid, weight * mass * (y[tid] - affine_target_dof[tid]))       <L 159>
            var_20 = wp::mul(var_weight, var_15);
            var_21 = wp::address(var_y, var_0);
            var_22 = wp::address(var_affine_target_dof, var_0);
            var_24 = wp::load(var_21);
            var_25 = wp::load(var_22);
            var_23 = wp::sub(var_24, var_25);
            var_26 = wp::mul(var_20, var_23);
            var_27 = wp::atomic_add(var_affine_grad, var_0, var_26);
        }
    }
}



extern "C" __global__ void compute_affine_kinematic_grad_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_grad,
    wp::array_t<wp::float64> var_mass_body,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y,
    wp::array_t<bool> adj_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_target_dof,
    wp::float64 adj_weight,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_grad,
    wp::array_t<wp::float64> adj_mass_body,
    wp::array_t<wp::int32> adj_body_env_id,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32* var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 1;
        bool var_5;
        wp::int32 var_6;
        wp::int32* var_7;
        wp::int32* var_8;
        wp::int32 var_9;
        const wp::int32 var_10 = 2;
        bool var_11;
        wp::int32 var_12;
        bool var_13;
        wp::float64* var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        bool* var_17;
        bool var_18;
        bool var_19;
        wp::float64 var_20;
        wp::vec_t<12,wp::float64>* var_21;
        wp::vec_t<12,wp::float64>* var_22;
        wp::vec_t<12,wp::float64> var_23;
        wp::vec_t<12,wp::float64> var_24;
        wp::vec_t<12,wp::float64> var_25;
        wp::vec_t<12,wp::float64> var_26;
        wp::vec_t<12,wp::float64> var_27;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::int32 adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        bool adj_11 = {};
        wp::int32 adj_12 = {};
        bool adj_13 = {};
        wp::float64 adj_14 = {};
        wp::float64 adj_15 = {};
        wp::float64 adj_16 = {};
        bool adj_17 = {};
        bool adj_18 = {};
        bool adj_19 = {};
        wp::float64 adj_20 = {};
        wp::vec_t<12,wp::float64> adj_21 = {};
        wp::vec_t<12,wp::float64> adj_22 = {};
        wp::vec_t<12,wp::float64> adj_23 = {};
        wp::vec_t<12,wp::float64> adj_24 = {};
        wp::vec_t<12,wp::float64> adj_25 = {};
        wp::vec_t<12,wp::float64> adj_26 = {};
        wp::vec_t<12,wp::float64> adj_27 = {};
        //---------
        // forward
        // def compute_affine_kinematic_grad(                                                     <L 143>
        // tid = wp.tid()                                                                         <L 153>
        var_0 = builtin_tid1d();
        // if env_states[body_env_id[tid]] == ENV_STATE_INVALID or env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED:       <L 154>
        var_1 = wp::address(var_body_env_id, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::address(var_env_states, var_3);
        var_6 = wp::load(var_2);
        var_5 = (var_6 == var_4);
        var_7 = wp::address(var_body_env_id, var_0);
        var_9 = wp::load(var_7);
        var_8 = wp::address(var_env_states, var_9);
        var_12 = wp::load(var_8);
        var_11 = (var_12 == var_10);
        var_13 = var_5 || var_11;
        if (var_13) {
            // return                                                                             <L 155>
            goto label0;
        }
        // mass = mass_body[tid]                                                                  <L 156>
        var_14 = wp::address(var_mass_body, var_0);
        var_16 = wp::load(var_14);
        var_15 = wp::copy(var_16);
        // has_constraint = affine_has_constraint[tid]                                            <L 157>
        var_17 = wp::address(var_affine_has_constraint, var_0);
        var_19 = wp::load(var_17);
        var_18 = wp::copy(var_19);
        // if has_constraint:                                                                     <L 158>
        if (var_18) {
            // wp.atomic_add(affine_grad, tid, weight * mass * (y[tid] - affine_target_dof[tid]))       <L 159>
            var_20 = wp::mul(var_weight, var_15);
            var_21 = wp::address(var_y, var_0);
            var_22 = wp::address(var_affine_target_dof, var_0);
            var_24 = wp::load(var_21);
            var_25 = wp::load(var_22);
            var_23 = wp::sub(var_24, var_25);
            var_26 = wp::mul(var_20, var_23);
            // var_27 = wp::atomic_add(var_affine_grad, var_0, var_26);
        }
        //---------
        // reverse
        if (var_18) {
            wp::adj_atomic_add(var_affine_grad, var_0, var_26, adj_affine_grad, adj_0, adj_26, adj_27);
            wp::adj_mul(var_20, var_23, adj_20, adj_23, adj_26);
            wp::adj_sub(var_24, var_25, adj_21, adj_22, adj_23);
            wp::adj_load(var_22, adj_22, adj_25);
            wp::adj_load(var_21, adj_21, adj_24);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_22);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_21);
            wp::adj_mul(var_weight, var_15, adj_weight, adj_15, adj_20);
            // adj: wp.atomic_add(affine_grad, tid, weight * mass * (y[tid] - affine_target_dof[tid]))  <L 159>
        }
        // adj: if has_constraint:                                                                <L 158>
        wp::adj_copy(var_19, adj_17, adj_18);
        wp::adj_load(var_17, adj_17, adj_19);
        wp::adj_address(var_affine_has_constraint, var_0, adj_affine_has_constraint, adj_0, adj_17);
        // adj: has_constraint = affine_has_constraint[tid]                                       <L 157>
        wp::adj_copy(var_16, adj_14, adj_15);
        wp::adj_load(var_14, adj_14, adj_16);
        wp::adj_address(var_mass_body, var_0, adj_mass_body, adj_0, adj_14);
        // adj: mass = mass_body[tid]                                                             <L 156>
        if (var_13) {
            label0:;
            // adj: return                                                                        <L 155>
        }
        wp::adj_load(var_8, adj_8, adj_12);
        wp::adj_address(var_env_states, var_9, adj_env_states, adj_7, adj_8);
        wp::adj_load(var_7, adj_7, adj_9);
        wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_7);
        wp::adj_load(var_2, adj_2, adj_6);
        wp::adj_address(var_env_states, var_3, adj_env_states, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_1);
        // adj: if env_states[body_env_id[tid]] == ENV_STATE_INVALID or env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED:  <L 154>
        // adj: tid = wp.tid()                                                                    <L 153>
        // adj: def compute_affine_kinematic_grad(                                                <L 143>
        continue;
    }
}



extern "C" __global__ void compute_affine_kinematic_energy_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::float64> var_affine_energy,
    wp::array_t<wp::float64> var_mass_body)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::float64* var_1;
        wp::float64 var_2;
        wp::float64 var_3;
        bool* var_4;
        bool var_5;
        bool var_6;
        const wp::int32 var_7 = 0;
        const wp::float32 var_8 = 0.5;
        wp::float64 var_9;
        wp::float64 var_10;
        wp::float64 var_11;
        wp::vec_t<12,wp::float64>* var_12;
        wp::float64 var_13;
        wp::vec_t<12,wp::float64> var_14;
        wp::vec_t<12,wp::float64>* var_15;
        wp::float64 var_16;
        wp::vec_t<12,wp::float64> var_17;
        wp::float64 var_18;
        wp::float64 var_19;
        wp::float64 var_20;
        wp::float64 var_21;
        const wp::int32 var_22 = 1;
        const wp::float32 var_23 = 0.5;
        wp::float64 var_24;
        wp::float64 var_25;
        wp::float64 var_26;
        wp::vec_t<12,wp::float64>* var_27;
        wp::float64 var_28;
        wp::vec_t<12,wp::float64> var_29;
        wp::vec_t<12,wp::float64>* var_30;
        wp::float64 var_31;
        wp::vec_t<12,wp::float64> var_32;
        wp::float64 var_33;
        wp::float64 var_34;
        wp::float64 var_35;
        wp::float64 var_36;
        const wp::int32 var_37 = 2;
        const wp::float32 var_38 = 0.5;
        wp::float64 var_39;
        wp::float64 var_40;
        wp::float64 var_41;
        wp::vec_t<12,wp::float64>* var_42;
        wp::float64 var_43;
        wp::vec_t<12,wp::float64> var_44;
        wp::vec_t<12,wp::float64>* var_45;
        wp::float64 var_46;
        wp::vec_t<12,wp::float64> var_47;
        wp::float64 var_48;
        wp::float64 var_49;
        wp::float64 var_50;
        wp::float64 var_51;
        const wp::int32 var_52 = 3;
        const wp::float32 var_53 = 0.5;
        wp::float64 var_54;
        wp::float64 var_55;
        wp::float64 var_56;
        wp::vec_t<12,wp::float64>* var_57;
        wp::float64 var_58;
        wp::vec_t<12,wp::float64> var_59;
        wp::vec_t<12,wp::float64>* var_60;
        wp::float64 var_61;
        wp::vec_t<12,wp::float64> var_62;
        wp::float64 var_63;
        wp::float64 var_64;
        wp::float64 var_65;
        wp::float64 var_66;
        const wp::int32 var_67 = 4;
        const wp::float32 var_68 = 0.5;
        wp::float64 var_69;
        wp::float64 var_70;
        wp::float64 var_71;
        wp::vec_t<12,wp::float64>* var_72;
        wp::float64 var_73;
        wp::vec_t<12,wp::float64> var_74;
        wp::vec_t<12,wp::float64>* var_75;
        wp::float64 var_76;
        wp::vec_t<12,wp::float64> var_77;
        wp::float64 var_78;
        wp::float64 var_79;
        wp::float64 var_80;
        wp::float64 var_81;
        const wp::int32 var_82 = 5;
        const wp::float32 var_83 = 0.5;
        wp::float64 var_84;
        wp::float64 var_85;
        wp::float64 var_86;
        wp::vec_t<12,wp::float64>* var_87;
        wp::float64 var_88;
        wp::vec_t<12,wp::float64> var_89;
        wp::vec_t<12,wp::float64>* var_90;
        wp::float64 var_91;
        wp::vec_t<12,wp::float64> var_92;
        wp::float64 var_93;
        wp::float64 var_94;
        wp::float64 var_95;
        wp::float64 var_96;
        const wp::int32 var_97 = 6;
        const wp::float32 var_98 = 0.5;
        wp::float64 var_99;
        wp::float64 var_100;
        wp::float64 var_101;
        wp::vec_t<12,wp::float64>* var_102;
        wp::float64 var_103;
        wp::vec_t<12,wp::float64> var_104;
        wp::vec_t<12,wp::float64>* var_105;
        wp::float64 var_106;
        wp::vec_t<12,wp::float64> var_107;
        wp::float64 var_108;
        wp::float64 var_109;
        wp::float64 var_110;
        wp::float64 var_111;
        const wp::int32 var_112 = 7;
        const wp::float32 var_113 = 0.5;
        wp::float64 var_114;
        wp::float64 var_115;
        wp::float64 var_116;
        wp::vec_t<12,wp::float64>* var_117;
        wp::float64 var_118;
        wp::vec_t<12,wp::float64> var_119;
        wp::vec_t<12,wp::float64>* var_120;
        wp::float64 var_121;
        wp::vec_t<12,wp::float64> var_122;
        wp::float64 var_123;
        wp::float64 var_124;
        wp::float64 var_125;
        wp::float64 var_126;
        const wp::int32 var_127 = 8;
        const wp::float32 var_128 = 0.5;
        wp::float64 var_129;
        wp::float64 var_130;
        wp::float64 var_131;
        wp::vec_t<12,wp::float64>* var_132;
        wp::float64 var_133;
        wp::vec_t<12,wp::float64> var_134;
        wp::vec_t<12,wp::float64>* var_135;
        wp::float64 var_136;
        wp::vec_t<12,wp::float64> var_137;
        wp::float64 var_138;
        wp::float64 var_139;
        wp::float64 var_140;
        wp::float64 var_141;
        const wp::int32 var_142 = 9;
        const wp::float32 var_143 = 0.5;
        wp::float64 var_144;
        wp::float64 var_145;
        wp::float64 var_146;
        wp::vec_t<12,wp::float64>* var_147;
        wp::float64 var_148;
        wp::vec_t<12,wp::float64> var_149;
        wp::vec_t<12,wp::float64>* var_150;
        wp::float64 var_151;
        wp::vec_t<12,wp::float64> var_152;
        wp::float64 var_153;
        wp::float64 var_154;
        wp::float64 var_155;
        wp::float64 var_156;
        const wp::int32 var_157 = 10;
        const wp::float32 var_158 = 0.5;
        wp::float64 var_159;
        wp::float64 var_160;
        wp::float64 var_161;
        wp::vec_t<12,wp::float64>* var_162;
        wp::float64 var_163;
        wp::vec_t<12,wp::float64> var_164;
        wp::vec_t<12,wp::float64>* var_165;
        wp::float64 var_166;
        wp::vec_t<12,wp::float64> var_167;
        wp::float64 var_168;
        wp::float64 var_169;
        wp::float64 var_170;
        wp::float64 var_171;
        const wp::int32 var_172 = 11;
        const wp::float32 var_173 = 0.5;
        wp::float64 var_174;
        wp::float64 var_175;
        wp::float64 var_176;
        wp::vec_t<12,wp::float64>* var_177;
        wp::float64 var_178;
        wp::vec_t<12,wp::float64> var_179;
        wp::vec_t<12,wp::float64>* var_180;
        wp::float64 var_181;
        wp::vec_t<12,wp::float64> var_182;
        wp::float64 var_183;
        wp::float64 var_184;
        wp::float64 var_185;
        wp::float64 var_186;
        //---------
        // forward
        // def compute_affine_kinematic_energy(                                                   <L 69>
        // tid = wp.tid()                                                                         <L 77>
        var_0 = builtin_tid1d();
        // mass = mass_body[tid]                                                                  <L 78>
        var_1 = wp::address(var_mass_body, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // has_constraint = affine_has_constraint[tid]                                            <L 79>
        var_4 = wp::address(var_affine_has_constraint, var_0);
        var_6 = wp::load(var_4);
        var_5 = wp::copy(var_6);
        // if has_constraint:                                                                     <L 80>
        if (var_5) {
            // for d in range(12):                                                                <L 81>
            // affine_energy[tid] += wp.float64(0.5) * weight * mass * sqr(y[tid][d] - affine_target_dof[tid][d])       <L 82>
            var_9 = wp::float64(var_8);
            var_10 = wp::mul(var_9, var_weight);
            var_11 = wp::mul(var_10, var_2);
            var_12 = wp::address(var_y, var_0);
            var_14 = wp::load(var_12);
            var_13 = wp::extract(var_14, var_7);
            var_15 = wp::address(var_affine_target_dof, var_0);
            var_17 = wp::load(var_15);
            var_16 = wp::extract(var_17, var_7);
            var_18 = wp::sub(var_13, var_16);
            var_19 = sqr_0(var_18);
            var_20 = wp::mul(var_11, var_19);
            var_21 = wp::atomic_add(var_affine_energy, var_0, var_20);
            var_24 = wp::float64(var_23);
            var_25 = wp::mul(var_24, var_weight);
            var_26 = wp::mul(var_25, var_2);
            var_27 = wp::address(var_y, var_0);
            var_29 = wp::load(var_27);
            var_28 = wp::extract(var_29, var_22);
            var_30 = wp::address(var_affine_target_dof, var_0);
            var_32 = wp::load(var_30);
            var_31 = wp::extract(var_32, var_22);
            var_33 = wp::sub(var_28, var_31);
            var_34 = sqr_0(var_33);
            var_35 = wp::mul(var_26, var_34);
            var_36 = wp::atomic_add(var_affine_energy, var_0, var_35);
            var_39 = wp::float64(var_38);
            var_40 = wp::mul(var_39, var_weight);
            var_41 = wp::mul(var_40, var_2);
            var_42 = wp::address(var_y, var_0);
            var_44 = wp::load(var_42);
            var_43 = wp::extract(var_44, var_37);
            var_45 = wp::address(var_affine_target_dof, var_0);
            var_47 = wp::load(var_45);
            var_46 = wp::extract(var_47, var_37);
            var_48 = wp::sub(var_43, var_46);
            var_49 = sqr_0(var_48);
            var_50 = wp::mul(var_41, var_49);
            var_51 = wp::atomic_add(var_affine_energy, var_0, var_50);
            var_54 = wp::float64(var_53);
            var_55 = wp::mul(var_54, var_weight);
            var_56 = wp::mul(var_55, var_2);
            var_57 = wp::address(var_y, var_0);
            var_59 = wp::load(var_57);
            var_58 = wp::extract(var_59, var_52);
            var_60 = wp::address(var_affine_target_dof, var_0);
            var_62 = wp::load(var_60);
            var_61 = wp::extract(var_62, var_52);
            var_63 = wp::sub(var_58, var_61);
            var_64 = sqr_0(var_63);
            var_65 = wp::mul(var_56, var_64);
            var_66 = wp::atomic_add(var_affine_energy, var_0, var_65);
            var_69 = wp::float64(var_68);
            var_70 = wp::mul(var_69, var_weight);
            var_71 = wp::mul(var_70, var_2);
            var_72 = wp::address(var_y, var_0);
            var_74 = wp::load(var_72);
            var_73 = wp::extract(var_74, var_67);
            var_75 = wp::address(var_affine_target_dof, var_0);
            var_77 = wp::load(var_75);
            var_76 = wp::extract(var_77, var_67);
            var_78 = wp::sub(var_73, var_76);
            var_79 = sqr_0(var_78);
            var_80 = wp::mul(var_71, var_79);
            var_81 = wp::atomic_add(var_affine_energy, var_0, var_80);
            var_84 = wp::float64(var_83);
            var_85 = wp::mul(var_84, var_weight);
            var_86 = wp::mul(var_85, var_2);
            var_87 = wp::address(var_y, var_0);
            var_89 = wp::load(var_87);
            var_88 = wp::extract(var_89, var_82);
            var_90 = wp::address(var_affine_target_dof, var_0);
            var_92 = wp::load(var_90);
            var_91 = wp::extract(var_92, var_82);
            var_93 = wp::sub(var_88, var_91);
            var_94 = sqr_0(var_93);
            var_95 = wp::mul(var_86, var_94);
            var_96 = wp::atomic_add(var_affine_energy, var_0, var_95);
            var_99 = wp::float64(var_98);
            var_100 = wp::mul(var_99, var_weight);
            var_101 = wp::mul(var_100, var_2);
            var_102 = wp::address(var_y, var_0);
            var_104 = wp::load(var_102);
            var_103 = wp::extract(var_104, var_97);
            var_105 = wp::address(var_affine_target_dof, var_0);
            var_107 = wp::load(var_105);
            var_106 = wp::extract(var_107, var_97);
            var_108 = wp::sub(var_103, var_106);
            var_109 = sqr_0(var_108);
            var_110 = wp::mul(var_101, var_109);
            var_111 = wp::atomic_add(var_affine_energy, var_0, var_110);
            var_114 = wp::float64(var_113);
            var_115 = wp::mul(var_114, var_weight);
            var_116 = wp::mul(var_115, var_2);
            var_117 = wp::address(var_y, var_0);
            var_119 = wp::load(var_117);
            var_118 = wp::extract(var_119, var_112);
            var_120 = wp::address(var_affine_target_dof, var_0);
            var_122 = wp::load(var_120);
            var_121 = wp::extract(var_122, var_112);
            var_123 = wp::sub(var_118, var_121);
            var_124 = sqr_0(var_123);
            var_125 = wp::mul(var_116, var_124);
            var_126 = wp::atomic_add(var_affine_energy, var_0, var_125);
            var_129 = wp::float64(var_128);
            var_130 = wp::mul(var_129, var_weight);
            var_131 = wp::mul(var_130, var_2);
            var_132 = wp::address(var_y, var_0);
            var_134 = wp::load(var_132);
            var_133 = wp::extract(var_134, var_127);
            var_135 = wp::address(var_affine_target_dof, var_0);
            var_137 = wp::load(var_135);
            var_136 = wp::extract(var_137, var_127);
            var_138 = wp::sub(var_133, var_136);
            var_139 = sqr_0(var_138);
            var_140 = wp::mul(var_131, var_139);
            var_141 = wp::atomic_add(var_affine_energy, var_0, var_140);
            var_144 = wp::float64(var_143);
            var_145 = wp::mul(var_144, var_weight);
            var_146 = wp::mul(var_145, var_2);
            var_147 = wp::address(var_y, var_0);
            var_149 = wp::load(var_147);
            var_148 = wp::extract(var_149, var_142);
            var_150 = wp::address(var_affine_target_dof, var_0);
            var_152 = wp::load(var_150);
            var_151 = wp::extract(var_152, var_142);
            var_153 = wp::sub(var_148, var_151);
            var_154 = sqr_0(var_153);
            var_155 = wp::mul(var_146, var_154);
            var_156 = wp::atomic_add(var_affine_energy, var_0, var_155);
            var_159 = wp::float64(var_158);
            var_160 = wp::mul(var_159, var_weight);
            var_161 = wp::mul(var_160, var_2);
            var_162 = wp::address(var_y, var_0);
            var_164 = wp::load(var_162);
            var_163 = wp::extract(var_164, var_157);
            var_165 = wp::address(var_affine_target_dof, var_0);
            var_167 = wp::load(var_165);
            var_166 = wp::extract(var_167, var_157);
            var_168 = wp::sub(var_163, var_166);
            var_169 = sqr_0(var_168);
            var_170 = wp::mul(var_161, var_169);
            var_171 = wp::atomic_add(var_affine_energy, var_0, var_170);
            var_174 = wp::float64(var_173);
            var_175 = wp::mul(var_174, var_weight);
            var_176 = wp::mul(var_175, var_2);
            var_177 = wp::address(var_y, var_0);
            var_179 = wp::load(var_177);
            var_178 = wp::extract(var_179, var_172);
            var_180 = wp::address(var_affine_target_dof, var_0);
            var_182 = wp::load(var_180);
            var_181 = wp::extract(var_182, var_172);
            var_183 = wp::sub(var_178, var_181);
            var_184 = sqr_0(var_183);
            var_185 = wp::mul(var_176, var_184);
            var_186 = wp::atomic_add(var_affine_energy, var_0, var_185);
        }
    }
}



extern "C" __global__ void compute_affine_kinematic_energy_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::float64> var_affine_energy,
    wp::array_t<wp::float64> var_mass_body,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y,
    wp::array_t<bool> adj_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_target_dof,
    wp::float64 adj_weight,
    wp::array_t<wp::float64> adj_affine_energy,
    wp::array_t<wp::float64> adj_mass_body)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::float64* var_1;
        wp::float64 var_2;
        wp::float64 var_3;
        bool* var_4;
        bool var_5;
        bool var_6;
        const wp::int32 var_7 = 0;
        const wp::float32 var_8 = 0.5;
        wp::float64 var_9;
        wp::float64 var_10;
        wp::float64 var_11;
        wp::vec_t<12,wp::float64>* var_12;
        wp::float64 var_13;
        wp::vec_t<12,wp::float64> var_14;
        wp::vec_t<12,wp::float64>* var_15;
        wp::float64 var_16;
        wp::vec_t<12,wp::float64> var_17;
        wp::float64 var_18;
        wp::float64 var_19;
        wp::float64 var_20;
        wp::float64 var_21;
        const wp::int32 var_22 = 1;
        const wp::float32 var_23 = 0.5;
        wp::float64 var_24;
        wp::float64 var_25;
        wp::float64 var_26;
        wp::vec_t<12,wp::float64>* var_27;
        wp::float64 var_28;
        wp::vec_t<12,wp::float64> var_29;
        wp::vec_t<12,wp::float64>* var_30;
        wp::float64 var_31;
        wp::vec_t<12,wp::float64> var_32;
        wp::float64 var_33;
        wp::float64 var_34;
        wp::float64 var_35;
        wp::float64 var_36;
        const wp::int32 var_37 = 2;
        const wp::float32 var_38 = 0.5;
        wp::float64 var_39;
        wp::float64 var_40;
        wp::float64 var_41;
        wp::vec_t<12,wp::float64>* var_42;
        wp::float64 var_43;
        wp::vec_t<12,wp::float64> var_44;
        wp::vec_t<12,wp::float64>* var_45;
        wp::float64 var_46;
        wp::vec_t<12,wp::float64> var_47;
        wp::float64 var_48;
        wp::float64 var_49;
        wp::float64 var_50;
        wp::float64 var_51;
        const wp::int32 var_52 = 3;
        const wp::float32 var_53 = 0.5;
        wp::float64 var_54;
        wp::float64 var_55;
        wp::float64 var_56;
        wp::vec_t<12,wp::float64>* var_57;
        wp::float64 var_58;
        wp::vec_t<12,wp::float64> var_59;
        wp::vec_t<12,wp::float64>* var_60;
        wp::float64 var_61;
        wp::vec_t<12,wp::float64> var_62;
        wp::float64 var_63;
        wp::float64 var_64;
        wp::float64 var_65;
        wp::float64 var_66;
        const wp::int32 var_67 = 4;
        const wp::float32 var_68 = 0.5;
        wp::float64 var_69;
        wp::float64 var_70;
        wp::float64 var_71;
        wp::vec_t<12,wp::float64>* var_72;
        wp::float64 var_73;
        wp::vec_t<12,wp::float64> var_74;
        wp::vec_t<12,wp::float64>* var_75;
        wp::float64 var_76;
        wp::vec_t<12,wp::float64> var_77;
        wp::float64 var_78;
        wp::float64 var_79;
        wp::float64 var_80;
        wp::float64 var_81;
        const wp::int32 var_82 = 5;
        const wp::float32 var_83 = 0.5;
        wp::float64 var_84;
        wp::float64 var_85;
        wp::float64 var_86;
        wp::vec_t<12,wp::float64>* var_87;
        wp::float64 var_88;
        wp::vec_t<12,wp::float64> var_89;
        wp::vec_t<12,wp::float64>* var_90;
        wp::float64 var_91;
        wp::vec_t<12,wp::float64> var_92;
        wp::float64 var_93;
        wp::float64 var_94;
        wp::float64 var_95;
        wp::float64 var_96;
        const wp::int32 var_97 = 6;
        const wp::float32 var_98 = 0.5;
        wp::float64 var_99;
        wp::float64 var_100;
        wp::float64 var_101;
        wp::vec_t<12,wp::float64>* var_102;
        wp::float64 var_103;
        wp::vec_t<12,wp::float64> var_104;
        wp::vec_t<12,wp::float64>* var_105;
        wp::float64 var_106;
        wp::vec_t<12,wp::float64> var_107;
        wp::float64 var_108;
        wp::float64 var_109;
        wp::float64 var_110;
        wp::float64 var_111;
        const wp::int32 var_112 = 7;
        const wp::float32 var_113 = 0.5;
        wp::float64 var_114;
        wp::float64 var_115;
        wp::float64 var_116;
        wp::vec_t<12,wp::float64>* var_117;
        wp::float64 var_118;
        wp::vec_t<12,wp::float64> var_119;
        wp::vec_t<12,wp::float64>* var_120;
        wp::float64 var_121;
        wp::vec_t<12,wp::float64> var_122;
        wp::float64 var_123;
        wp::float64 var_124;
        wp::float64 var_125;
        wp::float64 var_126;
        const wp::int32 var_127 = 8;
        const wp::float32 var_128 = 0.5;
        wp::float64 var_129;
        wp::float64 var_130;
        wp::float64 var_131;
        wp::vec_t<12,wp::float64>* var_132;
        wp::float64 var_133;
        wp::vec_t<12,wp::float64> var_134;
        wp::vec_t<12,wp::float64>* var_135;
        wp::float64 var_136;
        wp::vec_t<12,wp::float64> var_137;
        wp::float64 var_138;
        wp::float64 var_139;
        wp::float64 var_140;
        wp::float64 var_141;
        const wp::int32 var_142 = 9;
        const wp::float32 var_143 = 0.5;
        wp::float64 var_144;
        wp::float64 var_145;
        wp::float64 var_146;
        wp::vec_t<12,wp::float64>* var_147;
        wp::float64 var_148;
        wp::vec_t<12,wp::float64> var_149;
        wp::vec_t<12,wp::float64>* var_150;
        wp::float64 var_151;
        wp::vec_t<12,wp::float64> var_152;
        wp::float64 var_153;
        wp::float64 var_154;
        wp::float64 var_155;
        wp::float64 var_156;
        const wp::int32 var_157 = 10;
        const wp::float32 var_158 = 0.5;
        wp::float64 var_159;
        wp::float64 var_160;
        wp::float64 var_161;
        wp::vec_t<12,wp::float64>* var_162;
        wp::float64 var_163;
        wp::vec_t<12,wp::float64> var_164;
        wp::vec_t<12,wp::float64>* var_165;
        wp::float64 var_166;
        wp::vec_t<12,wp::float64> var_167;
        wp::float64 var_168;
        wp::float64 var_169;
        wp::float64 var_170;
        wp::float64 var_171;
        const wp::int32 var_172 = 11;
        const wp::float32 var_173 = 0.5;
        wp::float64 var_174;
        wp::float64 var_175;
        wp::float64 var_176;
        wp::vec_t<12,wp::float64>* var_177;
        wp::float64 var_178;
        wp::vec_t<12,wp::float64> var_179;
        wp::vec_t<12,wp::float64>* var_180;
        wp::float64 var_181;
        wp::vec_t<12,wp::float64> var_182;
        wp::float64 var_183;
        wp::float64 var_184;
        wp::float64 var_185;
        wp::float64 var_186;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::float64 adj_1 = {};
        wp::float64 adj_2 = {};
        wp::float64 adj_3 = {};
        bool adj_4 = {};
        bool adj_5 = {};
        bool adj_6 = {};
        wp::int32 adj_7 = {};
        wp::float32 adj_8 = {};
        wp::float64 adj_9 = {};
        wp::float64 adj_10 = {};
        wp::float64 adj_11 = {};
        wp::vec_t<12,wp::float64> adj_12 = {};
        wp::float64 adj_13 = {};
        wp::vec_t<12,wp::float64> adj_14 = {};
        wp::vec_t<12,wp::float64> adj_15 = {};
        wp::float64 adj_16 = {};
        wp::vec_t<12,wp::float64> adj_17 = {};
        wp::float64 adj_18 = {};
        wp::float64 adj_19 = {};
        wp::float64 adj_20 = {};
        wp::float64 adj_21 = {};
        wp::int32 adj_22 = {};
        wp::float32 adj_23 = {};
        wp::float64 adj_24 = {};
        wp::float64 adj_25 = {};
        wp::float64 adj_26 = {};
        wp::vec_t<12,wp::float64> adj_27 = {};
        wp::float64 adj_28 = {};
        wp::vec_t<12,wp::float64> adj_29 = {};
        wp::vec_t<12,wp::float64> adj_30 = {};
        wp::float64 adj_31 = {};
        wp::vec_t<12,wp::float64> adj_32 = {};
        wp::float64 adj_33 = {};
        wp::float64 adj_34 = {};
        wp::float64 adj_35 = {};
        wp::float64 adj_36 = {};
        wp::int32 adj_37 = {};
        wp::float32 adj_38 = {};
        wp::float64 adj_39 = {};
        wp::float64 adj_40 = {};
        wp::float64 adj_41 = {};
        wp::vec_t<12,wp::float64> adj_42 = {};
        wp::float64 adj_43 = {};
        wp::vec_t<12,wp::float64> adj_44 = {};
        wp::vec_t<12,wp::float64> adj_45 = {};
        wp::float64 adj_46 = {};
        wp::vec_t<12,wp::float64> adj_47 = {};
        wp::float64 adj_48 = {};
        wp::float64 adj_49 = {};
        wp::float64 adj_50 = {};
        wp::float64 adj_51 = {};
        wp::int32 adj_52 = {};
        wp::float32 adj_53 = {};
        wp::float64 adj_54 = {};
        wp::float64 adj_55 = {};
        wp::float64 adj_56 = {};
        wp::vec_t<12,wp::float64> adj_57 = {};
        wp::float64 adj_58 = {};
        wp::vec_t<12,wp::float64> adj_59 = {};
        wp::vec_t<12,wp::float64> adj_60 = {};
        wp::float64 adj_61 = {};
        wp::vec_t<12,wp::float64> adj_62 = {};
        wp::float64 adj_63 = {};
        wp::float64 adj_64 = {};
        wp::float64 adj_65 = {};
        wp::float64 adj_66 = {};
        wp::int32 adj_67 = {};
        wp::float32 adj_68 = {};
        wp::float64 adj_69 = {};
        wp::float64 adj_70 = {};
        wp::float64 adj_71 = {};
        wp::vec_t<12,wp::float64> adj_72 = {};
        wp::float64 adj_73 = {};
        wp::vec_t<12,wp::float64> adj_74 = {};
        wp::vec_t<12,wp::float64> adj_75 = {};
        wp::float64 adj_76 = {};
        wp::vec_t<12,wp::float64> adj_77 = {};
        wp::float64 adj_78 = {};
        wp::float64 adj_79 = {};
        wp::float64 adj_80 = {};
        wp::float64 adj_81 = {};
        wp::int32 adj_82 = {};
        wp::float32 adj_83 = {};
        wp::float64 adj_84 = {};
        wp::float64 adj_85 = {};
        wp::float64 adj_86 = {};
        wp::vec_t<12,wp::float64> adj_87 = {};
        wp::float64 adj_88 = {};
        wp::vec_t<12,wp::float64> adj_89 = {};
        wp::vec_t<12,wp::float64> adj_90 = {};
        wp::float64 adj_91 = {};
        wp::vec_t<12,wp::float64> adj_92 = {};
        wp::float64 adj_93 = {};
        wp::float64 adj_94 = {};
        wp::float64 adj_95 = {};
        wp::float64 adj_96 = {};
        wp::int32 adj_97 = {};
        wp::float32 adj_98 = {};
        wp::float64 adj_99 = {};
        wp::float64 adj_100 = {};
        wp::float64 adj_101 = {};
        wp::vec_t<12,wp::float64> adj_102 = {};
        wp::float64 adj_103 = {};
        wp::vec_t<12,wp::float64> adj_104 = {};
        wp::vec_t<12,wp::float64> adj_105 = {};
        wp::float64 adj_106 = {};
        wp::vec_t<12,wp::float64> adj_107 = {};
        wp::float64 adj_108 = {};
        wp::float64 adj_109 = {};
        wp::float64 adj_110 = {};
        wp::float64 adj_111 = {};
        wp::int32 adj_112 = {};
        wp::float32 adj_113 = {};
        wp::float64 adj_114 = {};
        wp::float64 adj_115 = {};
        wp::float64 adj_116 = {};
        wp::vec_t<12,wp::float64> adj_117 = {};
        wp::float64 adj_118 = {};
        wp::vec_t<12,wp::float64> adj_119 = {};
        wp::vec_t<12,wp::float64> adj_120 = {};
        wp::float64 adj_121 = {};
        wp::vec_t<12,wp::float64> adj_122 = {};
        wp::float64 adj_123 = {};
        wp::float64 adj_124 = {};
        wp::float64 adj_125 = {};
        wp::float64 adj_126 = {};
        wp::int32 adj_127 = {};
        wp::float32 adj_128 = {};
        wp::float64 adj_129 = {};
        wp::float64 adj_130 = {};
        wp::float64 adj_131 = {};
        wp::vec_t<12,wp::float64> adj_132 = {};
        wp::float64 adj_133 = {};
        wp::vec_t<12,wp::float64> adj_134 = {};
        wp::vec_t<12,wp::float64> adj_135 = {};
        wp::float64 adj_136 = {};
        wp::vec_t<12,wp::float64> adj_137 = {};
        wp::float64 adj_138 = {};
        wp::float64 adj_139 = {};
        wp::float64 adj_140 = {};
        wp::float64 adj_141 = {};
        wp::int32 adj_142 = {};
        wp::float32 adj_143 = {};
        wp::float64 adj_144 = {};
        wp::float64 adj_145 = {};
        wp::float64 adj_146 = {};
        wp::vec_t<12,wp::float64> adj_147 = {};
        wp::float64 adj_148 = {};
        wp::vec_t<12,wp::float64> adj_149 = {};
        wp::vec_t<12,wp::float64> adj_150 = {};
        wp::float64 adj_151 = {};
        wp::vec_t<12,wp::float64> adj_152 = {};
        wp::float64 adj_153 = {};
        wp::float64 adj_154 = {};
        wp::float64 adj_155 = {};
        wp::float64 adj_156 = {};
        wp::int32 adj_157 = {};
        wp::float32 adj_158 = {};
        wp::float64 adj_159 = {};
        wp::float64 adj_160 = {};
        wp::float64 adj_161 = {};
        wp::vec_t<12,wp::float64> adj_162 = {};
        wp::float64 adj_163 = {};
        wp::vec_t<12,wp::float64> adj_164 = {};
        wp::vec_t<12,wp::float64> adj_165 = {};
        wp::float64 adj_166 = {};
        wp::vec_t<12,wp::float64> adj_167 = {};
        wp::float64 adj_168 = {};
        wp::float64 adj_169 = {};
        wp::float64 adj_170 = {};
        wp::float64 adj_171 = {};
        wp::int32 adj_172 = {};
        wp::float32 adj_173 = {};
        wp::float64 adj_174 = {};
        wp::float64 adj_175 = {};
        wp::float64 adj_176 = {};
        wp::vec_t<12,wp::float64> adj_177 = {};
        wp::float64 adj_178 = {};
        wp::vec_t<12,wp::float64> adj_179 = {};
        wp::vec_t<12,wp::float64> adj_180 = {};
        wp::float64 adj_181 = {};
        wp::vec_t<12,wp::float64> adj_182 = {};
        wp::float64 adj_183 = {};
        wp::float64 adj_184 = {};
        wp::float64 adj_185 = {};
        wp::float64 adj_186 = {};
        //---------
        // forward
        // def compute_affine_kinematic_energy(                                                   <L 69>
        // tid = wp.tid()                                                                         <L 77>
        var_0 = builtin_tid1d();
        // mass = mass_body[tid]                                                                  <L 78>
        var_1 = wp::address(var_mass_body, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // has_constraint = affine_has_constraint[tid]                                            <L 79>
        var_4 = wp::address(var_affine_has_constraint, var_0);
        var_6 = wp::load(var_4);
        var_5 = wp::copy(var_6);
        // if has_constraint:                                                                     <L 80>
        if (var_5) {
            // for d in range(12):                                                                <L 81>
            // affine_energy[tid] += wp.float64(0.5) * weight * mass * sqr(y[tid][d] - affine_target_dof[tid][d])       <L 82>
            var_9 = wp::float64(var_8);
            var_10 = wp::mul(var_9, var_weight);
            var_11 = wp::mul(var_10, var_2);
            var_12 = wp::address(var_y, var_0);
            var_14 = wp::load(var_12);
            var_13 = wp::extract(var_14, var_7);
            var_15 = wp::address(var_affine_target_dof, var_0);
            var_17 = wp::load(var_15);
            var_16 = wp::extract(var_17, var_7);
            var_18 = wp::sub(var_13, var_16);
            var_19 = sqr_0(var_18);
            var_20 = wp::mul(var_11, var_19);
            // var_21 = wp::atomic_add(var_affine_energy, var_0, var_20);
            var_24 = wp::float64(var_23);
            var_25 = wp::mul(var_24, var_weight);
            var_26 = wp::mul(var_25, var_2);
            var_27 = wp::address(var_y, var_0);
            var_29 = wp::load(var_27);
            var_28 = wp::extract(var_29, var_22);
            var_30 = wp::address(var_affine_target_dof, var_0);
            var_32 = wp::load(var_30);
            var_31 = wp::extract(var_32, var_22);
            var_33 = wp::sub(var_28, var_31);
            var_34 = sqr_0(var_33);
            var_35 = wp::mul(var_26, var_34);
            // var_36 = wp::atomic_add(var_affine_energy, var_0, var_35);
            var_39 = wp::float64(var_38);
            var_40 = wp::mul(var_39, var_weight);
            var_41 = wp::mul(var_40, var_2);
            var_42 = wp::address(var_y, var_0);
            var_44 = wp::load(var_42);
            var_43 = wp::extract(var_44, var_37);
            var_45 = wp::address(var_affine_target_dof, var_0);
            var_47 = wp::load(var_45);
            var_46 = wp::extract(var_47, var_37);
            var_48 = wp::sub(var_43, var_46);
            var_49 = sqr_0(var_48);
            var_50 = wp::mul(var_41, var_49);
            // var_51 = wp::atomic_add(var_affine_energy, var_0, var_50);
            var_54 = wp::float64(var_53);
            var_55 = wp::mul(var_54, var_weight);
            var_56 = wp::mul(var_55, var_2);
            var_57 = wp::address(var_y, var_0);
            var_59 = wp::load(var_57);
            var_58 = wp::extract(var_59, var_52);
            var_60 = wp::address(var_affine_target_dof, var_0);
            var_62 = wp::load(var_60);
            var_61 = wp::extract(var_62, var_52);
            var_63 = wp::sub(var_58, var_61);
            var_64 = sqr_0(var_63);
            var_65 = wp::mul(var_56, var_64);
            // var_66 = wp::atomic_add(var_affine_energy, var_0, var_65);
            var_69 = wp::float64(var_68);
            var_70 = wp::mul(var_69, var_weight);
            var_71 = wp::mul(var_70, var_2);
            var_72 = wp::address(var_y, var_0);
            var_74 = wp::load(var_72);
            var_73 = wp::extract(var_74, var_67);
            var_75 = wp::address(var_affine_target_dof, var_0);
            var_77 = wp::load(var_75);
            var_76 = wp::extract(var_77, var_67);
            var_78 = wp::sub(var_73, var_76);
            var_79 = sqr_0(var_78);
            var_80 = wp::mul(var_71, var_79);
            // var_81 = wp::atomic_add(var_affine_energy, var_0, var_80);
            var_84 = wp::float64(var_83);
            var_85 = wp::mul(var_84, var_weight);
            var_86 = wp::mul(var_85, var_2);
            var_87 = wp::address(var_y, var_0);
            var_89 = wp::load(var_87);
            var_88 = wp::extract(var_89, var_82);
            var_90 = wp::address(var_affine_target_dof, var_0);
            var_92 = wp::load(var_90);
            var_91 = wp::extract(var_92, var_82);
            var_93 = wp::sub(var_88, var_91);
            var_94 = sqr_0(var_93);
            var_95 = wp::mul(var_86, var_94);
            // var_96 = wp::atomic_add(var_affine_energy, var_0, var_95);
            var_99 = wp::float64(var_98);
            var_100 = wp::mul(var_99, var_weight);
            var_101 = wp::mul(var_100, var_2);
            var_102 = wp::address(var_y, var_0);
            var_104 = wp::load(var_102);
            var_103 = wp::extract(var_104, var_97);
            var_105 = wp::address(var_affine_target_dof, var_0);
            var_107 = wp::load(var_105);
            var_106 = wp::extract(var_107, var_97);
            var_108 = wp::sub(var_103, var_106);
            var_109 = sqr_0(var_108);
            var_110 = wp::mul(var_101, var_109);
            // var_111 = wp::atomic_add(var_affine_energy, var_0, var_110);
            var_114 = wp::float64(var_113);
            var_115 = wp::mul(var_114, var_weight);
            var_116 = wp::mul(var_115, var_2);
            var_117 = wp::address(var_y, var_0);
            var_119 = wp::load(var_117);
            var_118 = wp::extract(var_119, var_112);
            var_120 = wp::address(var_affine_target_dof, var_0);
            var_122 = wp::load(var_120);
            var_121 = wp::extract(var_122, var_112);
            var_123 = wp::sub(var_118, var_121);
            var_124 = sqr_0(var_123);
            var_125 = wp::mul(var_116, var_124);
            // var_126 = wp::atomic_add(var_affine_energy, var_0, var_125);
            var_129 = wp::float64(var_128);
            var_130 = wp::mul(var_129, var_weight);
            var_131 = wp::mul(var_130, var_2);
            var_132 = wp::address(var_y, var_0);
            var_134 = wp::load(var_132);
            var_133 = wp::extract(var_134, var_127);
            var_135 = wp::address(var_affine_target_dof, var_0);
            var_137 = wp::load(var_135);
            var_136 = wp::extract(var_137, var_127);
            var_138 = wp::sub(var_133, var_136);
            var_139 = sqr_0(var_138);
            var_140 = wp::mul(var_131, var_139);
            // var_141 = wp::atomic_add(var_affine_energy, var_0, var_140);
            var_144 = wp::float64(var_143);
            var_145 = wp::mul(var_144, var_weight);
            var_146 = wp::mul(var_145, var_2);
            var_147 = wp::address(var_y, var_0);
            var_149 = wp::load(var_147);
            var_148 = wp::extract(var_149, var_142);
            var_150 = wp::address(var_affine_target_dof, var_0);
            var_152 = wp::load(var_150);
            var_151 = wp::extract(var_152, var_142);
            var_153 = wp::sub(var_148, var_151);
            var_154 = sqr_0(var_153);
            var_155 = wp::mul(var_146, var_154);
            // var_156 = wp::atomic_add(var_affine_energy, var_0, var_155);
            var_159 = wp::float64(var_158);
            var_160 = wp::mul(var_159, var_weight);
            var_161 = wp::mul(var_160, var_2);
            var_162 = wp::address(var_y, var_0);
            var_164 = wp::load(var_162);
            var_163 = wp::extract(var_164, var_157);
            var_165 = wp::address(var_affine_target_dof, var_0);
            var_167 = wp::load(var_165);
            var_166 = wp::extract(var_167, var_157);
            var_168 = wp::sub(var_163, var_166);
            var_169 = sqr_0(var_168);
            var_170 = wp::mul(var_161, var_169);
            // var_171 = wp::atomic_add(var_affine_energy, var_0, var_170);
            var_174 = wp::float64(var_173);
            var_175 = wp::mul(var_174, var_weight);
            var_176 = wp::mul(var_175, var_2);
            var_177 = wp::address(var_y, var_0);
            var_179 = wp::load(var_177);
            var_178 = wp::extract(var_179, var_172);
            var_180 = wp::address(var_affine_target_dof, var_0);
            var_182 = wp::load(var_180);
            var_181 = wp::extract(var_182, var_172);
            var_183 = wp::sub(var_178, var_181);
            var_184 = sqr_0(var_183);
            var_185 = wp::mul(var_176, var_184);
            // var_186 = wp::atomic_add(var_affine_energy, var_0, var_185);
        }
        //---------
        // reverse
        if (var_5) {
            wp::adj_atomic_add(var_affine_energy, var_0, var_185, adj_affine_energy, adj_0, adj_185, adj_186);
            wp::adj_mul(var_176, var_184, adj_176, adj_184, adj_185);
            adj_sqr_0(var_183, adj_183, adj_184);
            wp::adj_sub(var_178, var_181, adj_178, adj_181, adj_183);
            wp::adj_extract(var_182, var_172, adj_180, adj_172, adj_181);
            wp::adj_load(var_180, adj_180, adj_182);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_180);
            wp::adj_extract(var_179, var_172, adj_177, adj_172, adj_178);
            wp::adj_load(var_177, adj_177, adj_179);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_177);
            wp::adj_mul(var_175, var_2, adj_175, adj_2, adj_176);
            wp::adj_mul(var_174, var_weight, adj_174, adj_weight, adj_175);
            wp::adj_float64(var_173, adj_173, adj_174);
            wp::adj_atomic_add(var_affine_energy, var_0, var_170, adj_affine_energy, adj_0, adj_170, adj_171);
            wp::adj_mul(var_161, var_169, adj_161, adj_169, adj_170);
            adj_sqr_0(var_168, adj_168, adj_169);
            wp::adj_sub(var_163, var_166, adj_163, adj_166, adj_168);
            wp::adj_extract(var_167, var_157, adj_165, adj_157, adj_166);
            wp::adj_load(var_165, adj_165, adj_167);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_165);
            wp::adj_extract(var_164, var_157, adj_162, adj_157, adj_163);
            wp::adj_load(var_162, adj_162, adj_164);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_162);
            wp::adj_mul(var_160, var_2, adj_160, adj_2, adj_161);
            wp::adj_mul(var_159, var_weight, adj_159, adj_weight, adj_160);
            wp::adj_float64(var_158, adj_158, adj_159);
            wp::adj_atomic_add(var_affine_energy, var_0, var_155, adj_affine_energy, adj_0, adj_155, adj_156);
            wp::adj_mul(var_146, var_154, adj_146, adj_154, adj_155);
            adj_sqr_0(var_153, adj_153, adj_154);
            wp::adj_sub(var_148, var_151, adj_148, adj_151, adj_153);
            wp::adj_extract(var_152, var_142, adj_150, adj_142, adj_151);
            wp::adj_load(var_150, adj_150, adj_152);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_150);
            wp::adj_extract(var_149, var_142, adj_147, adj_142, adj_148);
            wp::adj_load(var_147, adj_147, adj_149);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_147);
            wp::adj_mul(var_145, var_2, adj_145, adj_2, adj_146);
            wp::adj_mul(var_144, var_weight, adj_144, adj_weight, adj_145);
            wp::adj_float64(var_143, adj_143, adj_144);
            wp::adj_atomic_add(var_affine_energy, var_0, var_140, adj_affine_energy, adj_0, adj_140, adj_141);
            wp::adj_mul(var_131, var_139, adj_131, adj_139, adj_140);
            adj_sqr_0(var_138, adj_138, adj_139);
            wp::adj_sub(var_133, var_136, adj_133, adj_136, adj_138);
            wp::adj_extract(var_137, var_127, adj_135, adj_127, adj_136);
            wp::adj_load(var_135, adj_135, adj_137);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_135);
            wp::adj_extract(var_134, var_127, adj_132, adj_127, adj_133);
            wp::adj_load(var_132, adj_132, adj_134);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_132);
            wp::adj_mul(var_130, var_2, adj_130, adj_2, adj_131);
            wp::adj_mul(var_129, var_weight, adj_129, adj_weight, adj_130);
            wp::adj_float64(var_128, adj_128, adj_129);
            wp::adj_atomic_add(var_affine_energy, var_0, var_125, adj_affine_energy, adj_0, adj_125, adj_126);
            wp::adj_mul(var_116, var_124, adj_116, adj_124, adj_125);
            adj_sqr_0(var_123, adj_123, adj_124);
            wp::adj_sub(var_118, var_121, adj_118, adj_121, adj_123);
            wp::adj_extract(var_122, var_112, adj_120, adj_112, adj_121);
            wp::adj_load(var_120, adj_120, adj_122);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_120);
            wp::adj_extract(var_119, var_112, adj_117, adj_112, adj_118);
            wp::adj_load(var_117, adj_117, adj_119);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_117);
            wp::adj_mul(var_115, var_2, adj_115, adj_2, adj_116);
            wp::adj_mul(var_114, var_weight, adj_114, adj_weight, adj_115);
            wp::adj_float64(var_113, adj_113, adj_114);
            wp::adj_atomic_add(var_affine_energy, var_0, var_110, adj_affine_energy, adj_0, adj_110, adj_111);
            wp::adj_mul(var_101, var_109, adj_101, adj_109, adj_110);
            adj_sqr_0(var_108, adj_108, adj_109);
            wp::adj_sub(var_103, var_106, adj_103, adj_106, adj_108);
            wp::adj_extract(var_107, var_97, adj_105, adj_97, adj_106);
            wp::adj_load(var_105, adj_105, adj_107);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_105);
            wp::adj_extract(var_104, var_97, adj_102, adj_97, adj_103);
            wp::adj_load(var_102, adj_102, adj_104);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_102);
            wp::adj_mul(var_100, var_2, adj_100, adj_2, adj_101);
            wp::adj_mul(var_99, var_weight, adj_99, adj_weight, adj_100);
            wp::adj_float64(var_98, adj_98, adj_99);
            wp::adj_atomic_add(var_affine_energy, var_0, var_95, adj_affine_energy, adj_0, adj_95, adj_96);
            wp::adj_mul(var_86, var_94, adj_86, adj_94, adj_95);
            adj_sqr_0(var_93, adj_93, adj_94);
            wp::adj_sub(var_88, var_91, adj_88, adj_91, adj_93);
            wp::adj_extract(var_92, var_82, adj_90, adj_82, adj_91);
            wp::adj_load(var_90, adj_90, adj_92);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_90);
            wp::adj_extract(var_89, var_82, adj_87, adj_82, adj_88);
            wp::adj_load(var_87, adj_87, adj_89);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_87);
            wp::adj_mul(var_85, var_2, adj_85, adj_2, adj_86);
            wp::adj_mul(var_84, var_weight, adj_84, adj_weight, adj_85);
            wp::adj_float64(var_83, adj_83, adj_84);
            wp::adj_atomic_add(var_affine_energy, var_0, var_80, adj_affine_energy, adj_0, adj_80, adj_81);
            wp::adj_mul(var_71, var_79, adj_71, adj_79, adj_80);
            adj_sqr_0(var_78, adj_78, adj_79);
            wp::adj_sub(var_73, var_76, adj_73, adj_76, adj_78);
            wp::adj_extract(var_77, var_67, adj_75, adj_67, adj_76);
            wp::adj_load(var_75, adj_75, adj_77);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_75);
            wp::adj_extract(var_74, var_67, adj_72, adj_67, adj_73);
            wp::adj_load(var_72, adj_72, adj_74);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_72);
            wp::adj_mul(var_70, var_2, adj_70, adj_2, adj_71);
            wp::adj_mul(var_69, var_weight, adj_69, adj_weight, adj_70);
            wp::adj_float64(var_68, adj_68, adj_69);
            wp::adj_atomic_add(var_affine_energy, var_0, var_65, adj_affine_energy, adj_0, adj_65, adj_66);
            wp::adj_mul(var_56, var_64, adj_56, adj_64, adj_65);
            adj_sqr_0(var_63, adj_63, adj_64);
            wp::adj_sub(var_58, var_61, adj_58, adj_61, adj_63);
            wp::adj_extract(var_62, var_52, adj_60, adj_52, adj_61);
            wp::adj_load(var_60, adj_60, adj_62);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_60);
            wp::adj_extract(var_59, var_52, adj_57, adj_52, adj_58);
            wp::adj_load(var_57, adj_57, adj_59);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_57);
            wp::adj_mul(var_55, var_2, adj_55, adj_2, adj_56);
            wp::adj_mul(var_54, var_weight, adj_54, adj_weight, adj_55);
            wp::adj_float64(var_53, adj_53, adj_54);
            wp::adj_atomic_add(var_affine_energy, var_0, var_50, adj_affine_energy, adj_0, adj_50, adj_51);
            wp::adj_mul(var_41, var_49, adj_41, adj_49, adj_50);
            adj_sqr_0(var_48, adj_48, adj_49);
            wp::adj_sub(var_43, var_46, adj_43, adj_46, adj_48);
            wp::adj_extract(var_47, var_37, adj_45, adj_37, adj_46);
            wp::adj_load(var_45, adj_45, adj_47);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_45);
            wp::adj_extract(var_44, var_37, adj_42, adj_37, adj_43);
            wp::adj_load(var_42, adj_42, adj_44);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_42);
            wp::adj_mul(var_40, var_2, adj_40, adj_2, adj_41);
            wp::adj_mul(var_39, var_weight, adj_39, adj_weight, adj_40);
            wp::adj_float64(var_38, adj_38, adj_39);
            wp::adj_atomic_add(var_affine_energy, var_0, var_35, adj_affine_energy, adj_0, adj_35, adj_36);
            wp::adj_mul(var_26, var_34, adj_26, adj_34, adj_35);
            adj_sqr_0(var_33, adj_33, adj_34);
            wp::adj_sub(var_28, var_31, adj_28, adj_31, adj_33);
            wp::adj_extract(var_32, var_22, adj_30, adj_22, adj_31);
            wp::adj_load(var_30, adj_30, adj_32);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_30);
            wp::adj_extract(var_29, var_22, adj_27, adj_22, adj_28);
            wp::adj_load(var_27, adj_27, adj_29);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_27);
            wp::adj_mul(var_25, var_2, adj_25, adj_2, adj_26);
            wp::adj_mul(var_24, var_weight, adj_24, adj_weight, adj_25);
            wp::adj_float64(var_23, adj_23, adj_24);
            wp::adj_atomic_add(var_affine_energy, var_0, var_20, adj_affine_energy, adj_0, adj_20, adj_21);
            wp::adj_mul(var_11, var_19, adj_11, adj_19, adj_20);
            adj_sqr_0(var_18, adj_18, adj_19);
            wp::adj_sub(var_13, var_16, adj_13, adj_16, adj_18);
            wp::adj_extract(var_17, var_7, adj_15, adj_7, adj_16);
            wp::adj_load(var_15, adj_15, adj_17);
            wp::adj_address(var_affine_target_dof, var_0, adj_affine_target_dof, adj_0, adj_15);
            wp::adj_extract(var_14, var_7, adj_12, adj_7, adj_13);
            wp::adj_load(var_12, adj_12, adj_14);
            wp::adj_address(var_y, var_0, adj_y, adj_0, adj_12);
            wp::adj_mul(var_10, var_2, adj_10, adj_2, adj_11);
            wp::adj_mul(var_9, var_weight, adj_9, adj_weight, adj_10);
            wp::adj_float64(var_8, adj_8, adj_9);
            // adj: affine_energy[tid] += wp.float64(0.5) * weight * mass * sqr(y[tid][d] - affine_target_dof[tid][d])  <L 82>
            // adj: for d in range(12):                                                           <L 81>
        }
        // adj: if has_constraint:                                                                <L 80>
        wp::adj_copy(var_6, adj_4, adj_5);
        wp::adj_load(var_4, adj_4, adj_6);
        wp::adj_address(var_affine_has_constraint, var_0, adj_affine_has_constraint, adj_0, adj_4);
        // adj: has_constraint = affine_has_constraint[tid]                                       <L 79>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_mass_body, var_0, adj_mass_body, adj_0, adj_1);
        // adj: mass = mass_body[tid]                                                             <L 78>
        // adj: tid = wp.tid()                                                                    <L 77>
        // adj: def compute_affine_kinematic_energy(                                              <L 69>
        continue;
    }
}



extern "C" __global__ void compute_soft_kinematic_grad_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_grad,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32 var_1;
        wp::int32* var_2;
        wp::int32* var_3;
        wp::int32 var_4;
        const wp::int32 var_5 = 1;
        bool var_6;
        wp::int32 var_7;
        wp::int32 var_8;
        wp::int32* var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        const wp::int32 var_12 = 2;
        bool var_13;
        wp::int32 var_14;
        bool var_15;
        wp::float64* var_16;
        wp::float64 var_17;
        wp::float64 var_18;
        bool* var_19;
        bool var_20;
        bool var_21;
        wp::float64 var_22;
        wp::int32 var_23;
        wp::vec_t<3,wp::float64>* var_24;
        wp::vec_t<3,wp::float64>* var_25;
        wp::vec_t<3,wp::float64> var_26;
        wp::vec_t<3,wp::float64> var_27;
        wp::vec_t<3,wp::float64> var_28;
        wp::vec_t<3,wp::float64> var_29;
        wp::vec_t<3,wp::float64> var_30;
        //---------
        // forward
        // def compute_soft_kinematic_grad(                                                       <L 163>
        // tid = wp.tid()                                                                         <L 174>
        var_0 = builtin_tid1d();
        // if env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID or env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED:       <L 175>
        var_1 = wp::add(var_0, var_affine_verts_num);
        var_2 = wp::address(var_node2env, var_1);
        var_4 = wp::load(var_2);
        var_3 = wp::address(var_env_states, var_4);
        var_7 = wp::load(var_3);
        var_6 = (var_7 == var_5);
        var_8 = wp::add(var_0, var_affine_verts_num);
        var_9 = wp::address(var_node2env, var_8);
        var_11 = wp::load(var_9);
        var_10 = wp::address(var_env_states, var_11);
        var_14 = wp::load(var_10);
        var_13 = (var_14 == var_12);
        var_15 = var_6 || var_13;
        if (var_15) {
            // return                                                                             <L 176>
            continue;
        }
        // mass = soft_verts_mass[tid]                                                            <L 177>
        var_16 = wp::address(var_soft_verts_mass, var_0);
        var_18 = wp::load(var_16);
        var_17 = wp::copy(var_18);
        // has_constraint = soft_has_constraint[tid]                                              <L 178>
        var_19 = wp::address(var_soft_has_constraint, var_0);
        var_21 = wp::load(var_19);
        var_20 = wp::copy(var_21);
        // if has_constraint:                                                                     <L 179>
        if (var_20) {
            // wp.atomic_add(                                                                     <L 180>
            // soft_grad,                                                                         <L 181>
            // tid,                                                                               <L 182>
            // weight * mass * (x[tid + affine_verts_num] - soft_target_dof[tid]),                <L 183>
            var_22 = wp::mul(var_weight, var_17);
            var_23 = wp::add(var_0, var_affine_verts_num);
            var_24 = wp::address(var_x, var_23);
            var_25 = wp::address(var_soft_target_dof, var_0);
            var_27 = wp::load(var_24);
            var_28 = wp::load(var_25);
            var_26 = wp::sub(var_27, var_28);
            var_29 = wp::mul(var_22, var_26);
            var_30 = wp::atomic_add(var_soft_grad, var_0, var_29);
        }
    }
}



extern "C" __global__ void compute_soft_kinematic_grad_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_target_dof,
    wp::float64 var_weight,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_grad,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<bool> adj_soft_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_soft_target_dof,
    wp::float64 adj_weight,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_soft_grad,
    wp::int32 adj_affine_verts_num,
    wp::array_t<wp::float64> adj_soft_verts_mass,
    wp::array_t<wp::int32> adj_node2env,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32 var_1;
        wp::int32* var_2;
        wp::int32* var_3;
        wp::int32 var_4;
        const wp::int32 var_5 = 1;
        bool var_6;
        wp::int32 var_7;
        wp::int32 var_8;
        wp::int32* var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        const wp::int32 var_12 = 2;
        bool var_13;
        wp::int32 var_14;
        bool var_15;
        wp::float64* var_16;
        wp::float64 var_17;
        wp::float64 var_18;
        bool* var_19;
        bool var_20;
        bool var_21;
        wp::float64 var_22;
        wp::int32 var_23;
        wp::vec_t<3,wp::float64>* var_24;
        wp::vec_t<3,wp::float64>* var_25;
        wp::vec_t<3,wp::float64> var_26;
        wp::vec_t<3,wp::float64> var_27;
        wp::vec_t<3,wp::float64> var_28;
        wp::vec_t<3,wp::float64> var_29;
        wp::vec_t<3,wp::float64> var_30;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        wp::int32 adj_5 = {};
        bool adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int32 adj_12 = {};
        bool adj_13 = {};
        wp::int32 adj_14 = {};
        bool adj_15 = {};
        wp::float64 adj_16 = {};
        wp::float64 adj_17 = {};
        wp::float64 adj_18 = {};
        bool adj_19 = {};
        bool adj_20 = {};
        bool adj_21 = {};
        wp::float64 adj_22 = {};
        wp::int32 adj_23 = {};
        wp::vec_t<3,wp::float64> adj_24 = {};
        wp::vec_t<3,wp::float64> adj_25 = {};
        wp::vec_t<3,wp::float64> adj_26 = {};
        wp::vec_t<3,wp::float64> adj_27 = {};
        wp::vec_t<3,wp::float64> adj_28 = {};
        wp::vec_t<3,wp::float64> adj_29 = {};
        wp::vec_t<3,wp::float64> adj_30 = {};
        //---------
        // forward
        // def compute_soft_kinematic_grad(                                                       <L 163>
        // tid = wp.tid()                                                                         <L 174>
        var_0 = builtin_tid1d();
        // if env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID or env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED:       <L 175>
        var_1 = wp::add(var_0, var_affine_verts_num);
        var_2 = wp::address(var_node2env, var_1);
        var_4 = wp::load(var_2);
        var_3 = wp::address(var_env_states, var_4);
        var_7 = wp::load(var_3);
        var_6 = (var_7 == var_5);
        var_8 = wp::add(var_0, var_affine_verts_num);
        var_9 = wp::address(var_node2env, var_8);
        var_11 = wp::load(var_9);
        var_10 = wp::address(var_env_states, var_11);
        var_14 = wp::load(var_10);
        var_13 = (var_14 == var_12);
        var_15 = var_6 || var_13;
        if (var_15) {
            // return                                                                             <L 176>
            goto label0;
        }
        // mass = soft_verts_mass[tid]                                                            <L 177>
        var_16 = wp::address(var_soft_verts_mass, var_0);
        var_18 = wp::load(var_16);
        var_17 = wp::copy(var_18);
        // has_constraint = soft_has_constraint[tid]                                              <L 178>
        var_19 = wp::address(var_soft_has_constraint, var_0);
        var_21 = wp::load(var_19);
        var_20 = wp::copy(var_21);
        // if has_constraint:                                                                     <L 179>
        if (var_20) {
            // wp.atomic_add(                                                                     <L 180>
            // soft_grad,                                                                         <L 181>
            // tid,                                                                               <L 182>
            // weight * mass * (x[tid + affine_verts_num] - soft_target_dof[tid]),                <L 183>
            var_22 = wp::mul(var_weight, var_17);
            var_23 = wp::add(var_0, var_affine_verts_num);
            var_24 = wp::address(var_x, var_23);
            var_25 = wp::address(var_soft_target_dof, var_0);
            var_27 = wp::load(var_24);
            var_28 = wp::load(var_25);
            var_26 = wp::sub(var_27, var_28);
            var_29 = wp::mul(var_22, var_26);
            // var_30 = wp::atomic_add(var_soft_grad, var_0, var_29);
        }
        //---------
        // reverse
        if (var_20) {
            wp::adj_atomic_add(var_soft_grad, var_0, var_29, adj_soft_grad, adj_0, adj_29, adj_30);
            wp::adj_mul(var_22, var_26, adj_22, adj_26, adj_29);
            wp::adj_sub(var_27, var_28, adj_24, adj_25, adj_26);
            wp::adj_load(var_25, adj_25, adj_28);
            wp::adj_load(var_24, adj_24, adj_27);
            wp::adj_address(var_soft_target_dof, var_0, adj_soft_target_dof, adj_0, adj_25);
            wp::adj_address(var_x, var_23, adj_x, adj_23, adj_24);
            wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_23);
            wp::adj_mul(var_weight, var_17, adj_weight, adj_17, adj_22);
            // adj: weight * mass * (x[tid + affine_verts_num] - soft_target_dof[tid]),           <L 183>
            // adj: tid,                                                                          <L 182>
            // adj: soft_grad,                                                                    <L 181>
            // adj: wp.atomic_add(                                                                <L 180>
        }
        // adj: if has_constraint:                                                                <L 179>
        wp::adj_copy(var_21, adj_19, adj_20);
        wp::adj_load(var_19, adj_19, adj_21);
        wp::adj_address(var_soft_has_constraint, var_0, adj_soft_has_constraint, adj_0, adj_19);
        // adj: has_constraint = soft_has_constraint[tid]                                         <L 178>
        wp::adj_copy(var_18, adj_16, adj_17);
        wp::adj_load(var_16, adj_16, adj_18);
        wp::adj_address(var_soft_verts_mass, var_0, adj_soft_verts_mass, adj_0, adj_16);
        // adj: mass = soft_verts_mass[tid]                                                       <L 177>
        if (var_15) {
            label0:;
            // adj: return                                                                        <L 176>
        }
        wp::adj_load(var_10, adj_10, adj_14);
        wp::adj_address(var_env_states, var_11, adj_env_states, adj_9, adj_10);
        wp::adj_load(var_9, adj_9, adj_11);
        wp::adj_address(var_node2env, var_8, adj_node2env, adj_8, adj_9);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_8);
        wp::adj_load(var_3, adj_3, adj_7);
        wp::adj_address(var_env_states, var_4, adj_env_states, adj_2, adj_3);
        wp::adj_load(var_2, adj_2, adj_4);
        wp::adj_address(var_node2env, var_1, adj_node2env, adj_1, adj_2);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_1);
        // adj: if env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID or env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED:  <L 175>
        // adj: tid = wp.tid()                                                                    <L 174>
        // adj: def compute_soft_kinematic_grad(                                                  <L 163>
        continue;
    }
}



extern "C" __global__ void compute_affine_kinematic_hess_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_affine_has_constraint,
    wp::float64 var_weight,
    COOMatrix3x3_0df4b45d var_hess_affine_diag,
    wp::array_t<wp::float64> var_mass_body,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32* var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 1;
        bool var_5;
        wp::int32 var_6;
        wp::int32* var_7;
        wp::int32* var_8;
        wp::int32 var_9;
        const wp::int32 var_10 = 2;
        bool var_11;
        wp::int32 var_12;
        bool var_13;
        wp::float64* var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        const wp::float32 var_17 = 0.0;
        wp::float64 var_18;
        bool* var_19;
        bool var_20;
        bool var_21;
        wp::float64 var_22;
        wp::mat_t<3,3,wp::float64> var_23;
        const wp::int32 var_24 = 0;
        const wp::int32 var_25 = 16;
        wp::int32 var_26;
        const wp::int32 var_27 = 4;
        wp::int32 var_28;
        wp::int32 var_29;
        wp::int32 var_30;
        const wp::int32 var_31 = 1;
        const wp::int32 var_32 = 16;
        wp::int32 var_33;
        const wp::int32 var_34 = 4;
        wp::int32 var_35;
        wp::int32 var_36;
        wp::int32 var_37;
        const wp::int32 var_38 = 2;
        const wp::int32 var_39 = 16;
        wp::int32 var_40;
        const wp::int32 var_41 = 4;
        wp::int32 var_42;
        wp::int32 var_43;
        wp::int32 var_44;
        const wp::int32 var_45 = 3;
        const wp::int32 var_46 = 16;
        wp::int32 var_47;
        const wp::int32 var_48 = 4;
        wp::int32 var_49;
        wp::int32 var_50;
        wp::int32 var_51;
        //---------
        // forward
        // def compute_affine_kinematic_hess(                                                     <L 254>
        // tid = wp.tid()                                                                         <L 262>
        var_0 = builtin_tid1d();
        // if env_states[body_env_id[tid]] == ENV_STATE_INVALID or env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED:       <L 263>
        var_1 = wp::address(var_body_env_id, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::address(var_env_states, var_3);
        var_6 = wp::load(var_2);
        var_5 = (var_6 == var_4);
        var_7 = wp::address(var_body_env_id, var_0);
        var_9 = wp::load(var_7);
        var_8 = wp::address(var_env_states, var_9);
        var_12 = wp::load(var_8);
        var_11 = (var_12 == var_10);
        var_13 = var_5 || var_11;
        if (var_13) {
            // return                                                                             <L 264>
            continue;
        }
        // mass = mass_body[tid]                                                                  <L 265>
        var_14 = wp::address(var_mass_body, var_0);
        var_16 = wp::load(var_14);
        var_15 = wp::copy(var_16);
        // _0 = wp.float64(0.0)                                                                   <L 266>
        var_18 = wp::float64(var_17);
        // has_constraint = affine_has_constraint[tid]                                            <L 267>
        var_19 = wp::address(var_affine_has_constraint, var_0);
        var_21 = wp::load(var_19);
        var_20 = wp::copy(var_21);
        // weighted_mass = mass * weight                                                          <L 268>
        var_22 = wp::mul(var_15, var_weight);
        // mat3 = wp.mat33d(weighted_mass, _0, _0, _0, weighted_mass, _0, _0, _0, weighted_mass)       <L 269>
        var_23 = wp::mat_t<3,3,wp::float64>(var_22, var_18, var_18, var_18, var_22, var_18, var_18, var_18, var_22);
        // if has_constraint:                                                                     <L 270>
        if (var_20) {
            // for bi in range(4):                                                                <L 271>
            // matrix.COOMatrix3x3_atomic_add(hess_affine_diag, tid * 16 + 4 * bi + bi, mat3)       <L 272>
            var_26 = wp::mul(var_0, var_25);
            var_28 = wp::mul(var_27, var_24);
            var_29 = wp::add(var_26, var_28);
            var_30 = wp::add(var_29, var_24);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_30, var_23);
            var_33 = wp::mul(var_0, var_32);
            var_35 = wp::mul(var_34, var_31);
            var_36 = wp::add(var_33, var_35);
            var_37 = wp::add(var_36, var_31);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_37, var_23);
            var_40 = wp::mul(var_0, var_39);
            var_42 = wp::mul(var_41, var_38);
            var_43 = wp::add(var_40, var_42);
            var_44 = wp::add(var_43, var_38);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_44, var_23);
            var_47 = wp::mul(var_0, var_46);
            var_49 = wp::mul(var_48, var_45);
            var_50 = wp::add(var_47, var_49);
            var_51 = wp::add(var_50, var_45);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_51, var_23);
        }
    }
}



extern "C" __global__ void compute_affine_kinematic_hess_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_affine_has_constraint,
    wp::float64 var_weight,
    COOMatrix3x3_0df4b45d var_hess_affine_diag,
    wp::array_t<wp::float64> var_mass_body,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<bool> adj_affine_has_constraint,
    wp::float64 adj_weight,
    COOMatrix3x3_0df4b45d adj_hess_affine_diag,
    wp::array_t<wp::float64> adj_mass_body,
    wp::array_t<wp::int32> adj_body_env_id,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32* var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 1;
        bool var_5;
        wp::int32 var_6;
        wp::int32* var_7;
        wp::int32* var_8;
        wp::int32 var_9;
        const wp::int32 var_10 = 2;
        bool var_11;
        wp::int32 var_12;
        bool var_13;
        wp::float64* var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        const wp::float32 var_17 = 0.0;
        wp::float64 var_18;
        bool* var_19;
        bool var_20;
        bool var_21;
        wp::float64 var_22;
        wp::mat_t<3,3,wp::float64> var_23;
        const wp::int32 var_24 = 0;
        const wp::int32 var_25 = 16;
        wp::int32 var_26;
        const wp::int32 var_27 = 4;
        wp::int32 var_28;
        wp::int32 var_29;
        wp::int32 var_30;
        const wp::int32 var_31 = 1;
        const wp::int32 var_32 = 16;
        wp::int32 var_33;
        const wp::int32 var_34 = 4;
        wp::int32 var_35;
        wp::int32 var_36;
        wp::int32 var_37;
        const wp::int32 var_38 = 2;
        const wp::int32 var_39 = 16;
        wp::int32 var_40;
        const wp::int32 var_41 = 4;
        wp::int32 var_42;
        wp::int32 var_43;
        wp::int32 var_44;
        const wp::int32 var_45 = 3;
        const wp::int32 var_46 = 16;
        wp::int32 var_47;
        const wp::int32 var_48 = 4;
        wp::int32 var_49;
        wp::int32 var_50;
        wp::int32 var_51;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::int32 adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        bool adj_11 = {};
        wp::int32 adj_12 = {};
        bool adj_13 = {};
        wp::float64 adj_14 = {};
        wp::float64 adj_15 = {};
        wp::float64 adj_16 = {};
        wp::float32 adj_17 = {};
        wp::float64 adj_18 = {};
        bool adj_19 = {};
        bool adj_20 = {};
        bool adj_21 = {};
        wp::float64 adj_22 = {};
        wp::mat_t<3,3,wp::float64> adj_23 = {};
        wp::int32 adj_24 = {};
        wp::int32 adj_25 = {};
        wp::int32 adj_26 = {};
        wp::int32 adj_27 = {};
        wp::int32 adj_28 = {};
        wp::int32 adj_29 = {};
        wp::int32 adj_30 = {};
        wp::int32 adj_31 = {};
        wp::int32 adj_32 = {};
        wp::int32 adj_33 = {};
        wp::int32 adj_34 = {};
        wp::int32 adj_35 = {};
        wp::int32 adj_36 = {};
        wp::int32 adj_37 = {};
        wp::int32 adj_38 = {};
        wp::int32 adj_39 = {};
        wp::int32 adj_40 = {};
        wp::int32 adj_41 = {};
        wp::int32 adj_42 = {};
        wp::int32 adj_43 = {};
        wp::int32 adj_44 = {};
        wp::int32 adj_45 = {};
        wp::int32 adj_46 = {};
        wp::int32 adj_47 = {};
        wp::int32 adj_48 = {};
        wp::int32 adj_49 = {};
        wp::int32 adj_50 = {};
        wp::int32 adj_51 = {};
        //---------
        // forward
        // def compute_affine_kinematic_hess(                                                     <L 254>
        // tid = wp.tid()                                                                         <L 262>
        var_0 = builtin_tid1d();
        // if env_states[body_env_id[tid]] == ENV_STATE_INVALID or env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED:       <L 263>
        var_1 = wp::address(var_body_env_id, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::address(var_env_states, var_3);
        var_6 = wp::load(var_2);
        var_5 = (var_6 == var_4);
        var_7 = wp::address(var_body_env_id, var_0);
        var_9 = wp::load(var_7);
        var_8 = wp::address(var_env_states, var_9);
        var_12 = wp::load(var_8);
        var_11 = (var_12 == var_10);
        var_13 = var_5 || var_11;
        if (var_13) {
            // return                                                                             <L 264>
            goto label0;
        }
        // mass = mass_body[tid]                                                                  <L 265>
        var_14 = wp::address(var_mass_body, var_0);
        var_16 = wp::load(var_14);
        var_15 = wp::copy(var_16);
        // _0 = wp.float64(0.0)                                                                   <L 266>
        var_18 = wp::float64(var_17);
        // has_constraint = affine_has_constraint[tid]                                            <L 267>
        var_19 = wp::address(var_affine_has_constraint, var_0);
        var_21 = wp::load(var_19);
        var_20 = wp::copy(var_21);
        // weighted_mass = mass * weight                                                          <L 268>
        var_22 = wp::mul(var_15, var_weight);
        // mat3 = wp.mat33d(weighted_mass, _0, _0, _0, weighted_mass, _0, _0, _0, weighted_mass)       <L 269>
        var_23 = wp::mat_t<3,3,wp::float64>(var_22, var_18, var_18, var_18, var_22, var_18, var_18, var_18, var_22);
        // if has_constraint:                                                                     <L 270>
        if (var_20) {
            // for bi in range(4):                                                                <L 271>
            // matrix.COOMatrix3x3_atomic_add(hess_affine_diag, tid * 16 + 4 * bi + bi, mat3)       <L 272>
            var_26 = wp::mul(var_0, var_25);
            var_28 = wp::mul(var_27, var_24);
            var_29 = wp::add(var_26, var_28);
            var_30 = wp::add(var_29, var_24);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_30, var_23);
            var_33 = wp::mul(var_0, var_32);
            var_35 = wp::mul(var_34, var_31);
            var_36 = wp::add(var_33, var_35);
            var_37 = wp::add(var_36, var_31);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_37, var_23);
            var_40 = wp::mul(var_0, var_39);
            var_42 = wp::mul(var_41, var_38);
            var_43 = wp::add(var_40, var_42);
            var_44 = wp::add(var_43, var_38);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_44, var_23);
            var_47 = wp::mul(var_0, var_46);
            var_49 = wp::mul(var_48, var_45);
            var_50 = wp::add(var_47, var_49);
            var_51 = wp::add(var_50, var_45);
            COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_51, var_23);
        }
        //---------
        // reverse
        if (var_20) {
            adj_COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_51, var_23, adj_hess_affine_diag, adj_51, adj_23);
            wp::adj_add(var_50, var_45, adj_50, adj_45, adj_51);
            wp::adj_add(var_47, var_49, adj_47, adj_49, adj_50);
            wp::adj_mul(var_48, var_45, adj_48, adj_45, adj_49);
            wp::adj_mul(var_0, var_46, adj_0, adj_46, adj_47);
            adj_COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_44, var_23, adj_hess_affine_diag, adj_44, adj_23);
            wp::adj_add(var_43, var_38, adj_43, adj_38, adj_44);
            wp::adj_add(var_40, var_42, adj_40, adj_42, adj_43);
            wp::adj_mul(var_41, var_38, adj_41, adj_38, adj_42);
            wp::adj_mul(var_0, var_39, adj_0, adj_39, adj_40);
            adj_COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_37, var_23, adj_hess_affine_diag, adj_37, adj_23);
            wp::adj_add(var_36, var_31, adj_36, adj_31, adj_37);
            wp::adj_add(var_33, var_35, adj_33, adj_35, adj_36);
            wp::adj_mul(var_34, var_31, adj_34, adj_31, adj_35);
            wp::adj_mul(var_0, var_32, adj_0, adj_32, adj_33);
            adj_COOMatrix3x3_atomic_add_0(var_hess_affine_diag, var_30, var_23, adj_hess_affine_diag, adj_30, adj_23);
            wp::adj_add(var_29, var_24, adj_29, adj_24, adj_30);
            wp::adj_add(var_26, var_28, adj_26, adj_28, adj_29);
            wp::adj_mul(var_27, var_24, adj_27, adj_24, adj_28);
            wp::adj_mul(var_0, var_25, adj_0, adj_25, adj_26);
            // adj: matrix.COOMatrix3x3_atomic_add(hess_affine_diag, tid * 16 + 4 * bi + bi, mat3)  <L 272>
            // adj: for bi in range(4):                                                           <L 271>
        }
        // adj: if has_constraint:                                                                <L 270>
        wp::adj_mat_t(var_22, var_18, var_18, var_18, var_22, var_18, var_18, var_18, var_22, adj_22, adj_18, adj_18, adj_18, adj_22, adj_18, adj_18, adj_18, adj_22, adj_23);
        // adj: mat3 = wp.mat33d(weighted_mass, _0, _0, _0, weighted_mass, _0, _0, _0, weighted_mass)  <L 269>
        wp::adj_mul(var_15, var_weight, adj_15, adj_weight, adj_22);
        // adj: weighted_mass = mass * weight                                                     <L 268>
        wp::adj_copy(var_21, adj_19, adj_20);
        wp::adj_load(var_19, adj_19, adj_21);
        wp::adj_address(var_affine_has_constraint, var_0, adj_affine_has_constraint, adj_0, adj_19);
        // adj: has_constraint = affine_has_constraint[tid]                                       <L 267>
        wp::adj_float64(var_17, adj_17, adj_18);
        // adj: _0 = wp.float64(0.0)                                                              <L 266>
        wp::adj_copy(var_16, adj_14, adj_15);
        wp::adj_load(var_14, adj_14, adj_16);
        wp::adj_address(var_mass_body, var_0, adj_mass_body, adj_0, adj_14);
        // adj: mass = mass_body[tid]                                                             <L 265>
        if (var_13) {
            label0:;
            // adj: return                                                                        <L 264>
        }
        wp::adj_load(var_8, adj_8, adj_12);
        wp::adj_address(var_env_states, var_9, adj_env_states, adj_7, adj_8);
        wp::adj_load(var_7, adj_7, adj_9);
        wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_7);
        wp::adj_load(var_2, adj_2, adj_6);
        wp::adj_address(var_env_states, var_3, adj_env_states, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_1);
        // adj: if env_states[body_env_id[tid]] == ENV_STATE_INVALID or env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED:  <L 263>
        // adj: tid = wp.tid()                                                                    <L 262>
        // adj: def compute_affine_kinematic_hess(                                                <L 254>
        continue;
    }
}



extern "C" __global__ void init_affine_kinematic_target_kernel_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_kinematic_target_pose,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_target_dof,
    wp::array_t<wp::vec_t<3,wp::float64>> var_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ABD_centers)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        const bool var_2 = false;
        bool var_3;
        bool var_4;
        wp::vec_t<12,wp::float64>* var_5;
        wp::vec_t<12,wp::float64> var_6;
        wp::vec_t<12,wp::float64> var_7;
        wp::vec_t<3,wp::float64>* var_8;
        wp::vec_t<3,wp::float64> var_9;
        wp::vec_t<3,wp::float64> var_10;
        const wp::float32 var_11 = 0.0;
        wp::float64 var_12;
        const wp::float32 var_13 = 1.0;
        wp::float64 var_14;
        wp::vec_t<3,wp::float64>* var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::vec_t<3,wp::float64> var_18;
        wp::vec_t<3,wp::float64> var_19;
        wp::vec_t<3,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::vec_t<3,wp::float64> var_22;
        wp::vec_t<3,wp::float64> var_23;
        const wp::int32 var_24 = 0;
        wp::float64 var_25;
        const wp::int32 var_26 = 1;
        wp::float64 var_27;
        const wp::int32 var_28 = 2;
        wp::float64 var_29;
        const wp::int32 var_30 = 3;
        wp::float64 var_31;
        const wp::int32 var_32 = 4;
        wp::float64 var_33;
        const wp::int32 var_34 = 5;
        wp::float64 var_35;
        const wp::int32 var_36 = 6;
        wp::float64 var_37;
        const wp::int32 var_38 = 7;
        wp::float64 var_39;
        const wp::int32 var_40 = 8;
        wp::float64 var_41;
        wp::mat_t<3,3,wp::float64> var_42;
        const wp::int32 var_43 = 9;
        wp::float64 var_44;
        const wp::int32 var_45 = 10;
        wp::float64 var_46;
        const wp::int32 var_47 = 11;
        wp::float64 var_48;
        wp::vec_t<3,wp::float64> var_49;
        wp::vec_t<3,wp::float64> var_50;
        wp::vec_t<3,wp::float64> var_51;
        wp::vec_t<3,wp::float64> var_52;
        wp::vec_t<3,wp::float64> var_53;
        wp::vec_t<3,wp::float64> var_54;
        wp::vec_t<3,wp::float64> var_55;
        wp::vec_t<3,wp::float64> var_56;
        wp::vec_t<3,wp::float64> var_57;
        wp::vec_t<3,wp::float64> var_58;
        wp::vec_t<3,wp::float64> var_59;
        wp::vec_t<3,wp::float64> var_60;
        wp::vec_t<3,wp::float64> var_61;
        wp::vec_t<3,wp::float64> var_62;
        wp::vec_t<3,wp::float64> var_63;
        wp::vec_t<3,wp::float64> var_64;
        wp::vec_t<3,wp::float64> var_65;
        const wp::int32 var_66 = 0;
        wp::float64 var_67;
        const wp::int32 var_68 = 1;
        wp::float64 var_69;
        const wp::int32 var_70 = 2;
        wp::float64 var_71;
        const wp::int32 var_72 = 0;
        wp::float64 var_73;
        const wp::int32 var_74 = 1;
        wp::float64 var_75;
        const wp::int32 var_76 = 2;
        wp::float64 var_77;
        const wp::int32 var_78 = 0;
        wp::float64 var_79;
        const wp::int32 var_80 = 1;
        wp::float64 var_81;
        const wp::int32 var_82 = 2;
        wp::float64 var_83;
        const wp::int32 var_84 = 0;
        wp::float64 var_85;
        const wp::int32 var_86 = 1;
        wp::float64 var_87;
        const wp::int32 var_88 = 2;
        wp::float64 var_89;
        wp::vec_t<12,wp::float64> var_90;
        //---------
        // forward
        // def init_affine_kinematic_target_kernel(                                               <L 16>
        // tid = wp.tid()                                                                         <L 23>
        var_0 = builtin_tid1d();
        // if affine_has_constraint[tid] == False:                                                <L 24>
        var_1 = wp::address(var_affine_has_constraint, var_0);
        var_4 = wp::load(var_1);
        var_3 = (var_4 == var_2);
        if (var_3) {
            // return                                                                             <L 25>
            continue;
        }
        // target_state = affine_kinematic_target_pose[tid]                                       <L 26>
        var_5 = wp::address(var_affine_kinematic_target_pose, var_0);
        var_7 = wp::load(var_5);
        var_6 = wp::copy(var_7);
        // virtual_center = virtual_object_centers[tid]                                           <L 27>
        var_8 = wp::address(var_virtual_object_centers, var_0);
        var_10 = wp::load(var_8);
        var_9 = wp::copy(var_10);
        // _0 = wp.float64(0.0)                                                                   <L 28>
        var_12 = wp::float64(var_11);
        // _1 = wp.float64(1.0)                                                                   <L 29>
        var_14 = wp::float64(var_13);
        // rest_O = ABD_centers[tid]                                                              <L 30>
        var_15 = wp::address(var_ABD_centers, var_0);
        var_17 = wp::load(var_15);
        var_16 = wp::copy(var_17);
        // rest_A = rest_O + wp.vec3d(_1, _0, _0)                                                 <L 31>
        var_18 = wp::vec_t<3,wp::float64>(var_14, var_12, var_12);
        var_19 = wp::add(var_16, var_18);
        // rest_B = rest_O + wp.vec3d(_0, _1, _0)                                                 <L 32>
        var_20 = wp::vec_t<3,wp::float64>(var_12, var_14, var_12);
        var_21 = wp::add(var_16, var_20);
        // rest_C = rest_O + wp.vec3d(_0, _0, _1)                                                 <L 33>
        var_22 = wp::vec_t<3,wp::float64>(var_12, var_12, var_14);
        var_23 = wp::add(var_16, var_22);
        // R = wp.mat33d(                                                                         <L 34>
        // target_state[0],                                                                       <L 35>
        var_25 = wp::extract(var_6, var_24);
        // target_state[1],                                                                       <L 36>
        var_27 = wp::extract(var_6, var_26);
        // target_state[2],                                                                       <L 37>
        var_29 = wp::extract(var_6, var_28);
        // target_state[3],                                                                       <L 38>
        var_31 = wp::extract(var_6, var_30);
        // target_state[4],                                                                       <L 39>
        var_33 = wp::extract(var_6, var_32);
        // target_state[5],                                                                       <L 40>
        var_35 = wp::extract(var_6, var_34);
        // target_state[6],                                                                       <L 41>
        var_37 = wp::extract(var_6, var_36);
        // target_state[7],                                                                       <L 42>
        var_39 = wp::extract(var_6, var_38);
        // target_state[8],                                                                       <L 43>
        var_41 = wp::extract(var_6, var_40);
        var_42 = wp::mat_t<3,3,wp::float64>(var_25, var_27, var_29, var_31, var_33, var_35, var_37, var_39, var_41);
        // t = wp.vec3d(target_state[9], target_state[10], target_state[11])                      <L 45>
        var_44 = wp::extract(var_6, var_43);
        var_46 = wp::extract(var_6, var_45);
        var_48 = wp::extract(var_6, var_47);
        var_49 = wp::vec_t<3,wp::float64>(var_44, var_46, var_48);
        // O = R @ (rest_O - virtual_center) + virtual_center + t                                 <L 46>
        var_50 = wp::sub(var_16, var_9);
        var_51 = wp::mul(var_42, var_50);
        var_52 = wp::add(var_51, var_9);
        var_53 = wp::add(var_52, var_49);
        // A = R @ (rest_A - virtual_center) + virtual_center + t                                 <L 47>
        var_54 = wp::sub(var_19, var_9);
        var_55 = wp::mul(var_42, var_54);
        var_56 = wp::add(var_55, var_9);
        var_57 = wp::add(var_56, var_49);
        // B = R @ (rest_B - virtual_center) + virtual_center + t                                 <L 48>
        var_58 = wp::sub(var_21, var_9);
        var_59 = wp::mul(var_42, var_58);
        var_60 = wp::add(var_59, var_9);
        var_61 = wp::add(var_60, var_49);
        // C = R @ (rest_C - virtual_center) + virtual_center + t                                 <L 49>
        var_62 = wp::sub(var_23, var_9);
        var_63 = wp::mul(var_42, var_62);
        var_64 = wp::add(var_63, var_9);
        var_65 = wp::add(var_64, var_49);
        // affine_target_dof[tid] = vec12d(O[0], O[1], O[2], A[0], A[1], A[2], B[0], B[1], B[2], C[0], C[1], C[2])       <L 50>
        var_67 = wp::extract(var_53, var_66);
        var_69 = wp::extract(var_53, var_68);
        var_71 = wp::extract(var_53, var_70);
        var_73 = wp::extract(var_57, var_72);
        var_75 = wp::extract(var_57, var_74);
        var_77 = wp::extract(var_57, var_76);
        var_79 = wp::extract(var_61, var_78);
        var_81 = wp::extract(var_61, var_80);
        var_83 = wp::extract(var_61, var_82);
        var_85 = wp::extract(var_65, var_84);
        var_87 = wp::extract(var_65, var_86);
        var_89 = wp::extract(var_65, var_88);
        var_90 = wp::vec_t<12,wp::float64>({var_67, var_69, var_71, var_73, var_75, var_77, var_79, var_81, var_83, var_85, var_87, var_89});
        wp::array_store(var_affine_target_dof, var_0, var_90);
    }
}



extern "C" __global__ void init_affine_kinematic_target_kernel_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_kinematic_target_pose,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_target_dof,
    wp::array_t<wp::vec_t<3,wp::float64>> var_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ABD_centers,
    wp::array_t<bool> adj_affine_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_kinematic_target_pose,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_target_dof,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_ABD_centers)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        const bool var_2 = false;
        bool var_3;
        bool var_4;
        wp::vec_t<12,wp::float64>* var_5;
        wp::vec_t<12,wp::float64> var_6;
        wp::vec_t<12,wp::float64> var_7;
        wp::vec_t<3,wp::float64>* var_8;
        wp::vec_t<3,wp::float64> var_9;
        wp::vec_t<3,wp::float64> var_10;
        const wp::float32 var_11 = 0.0;
        wp::float64 var_12;
        const wp::float32 var_13 = 1.0;
        wp::float64 var_14;
        wp::vec_t<3,wp::float64>* var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::vec_t<3,wp::float64> var_18;
        wp::vec_t<3,wp::float64> var_19;
        wp::vec_t<3,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::vec_t<3,wp::float64> var_22;
        wp::vec_t<3,wp::float64> var_23;
        const wp::int32 var_24 = 0;
        wp::float64 var_25;
        const wp::int32 var_26 = 1;
        wp::float64 var_27;
        const wp::int32 var_28 = 2;
        wp::float64 var_29;
        const wp::int32 var_30 = 3;
        wp::float64 var_31;
        const wp::int32 var_32 = 4;
        wp::float64 var_33;
        const wp::int32 var_34 = 5;
        wp::float64 var_35;
        const wp::int32 var_36 = 6;
        wp::float64 var_37;
        const wp::int32 var_38 = 7;
        wp::float64 var_39;
        const wp::int32 var_40 = 8;
        wp::float64 var_41;
        wp::mat_t<3,3,wp::float64> var_42;
        const wp::int32 var_43 = 9;
        wp::float64 var_44;
        const wp::int32 var_45 = 10;
        wp::float64 var_46;
        const wp::int32 var_47 = 11;
        wp::float64 var_48;
        wp::vec_t<3,wp::float64> var_49;
        wp::vec_t<3,wp::float64> var_50;
        wp::vec_t<3,wp::float64> var_51;
        wp::vec_t<3,wp::float64> var_52;
        wp::vec_t<3,wp::float64> var_53;
        wp::vec_t<3,wp::float64> var_54;
        wp::vec_t<3,wp::float64> var_55;
        wp::vec_t<3,wp::float64> var_56;
        wp::vec_t<3,wp::float64> var_57;
        wp::vec_t<3,wp::float64> var_58;
        wp::vec_t<3,wp::float64> var_59;
        wp::vec_t<3,wp::float64> var_60;
        wp::vec_t<3,wp::float64> var_61;
        wp::vec_t<3,wp::float64> var_62;
        wp::vec_t<3,wp::float64> var_63;
        wp::vec_t<3,wp::float64> var_64;
        wp::vec_t<3,wp::float64> var_65;
        const wp::int32 var_66 = 0;
        wp::float64 var_67;
        const wp::int32 var_68 = 1;
        wp::float64 var_69;
        const wp::int32 var_70 = 2;
        wp::float64 var_71;
        const wp::int32 var_72 = 0;
        wp::float64 var_73;
        const wp::int32 var_74 = 1;
        wp::float64 var_75;
        const wp::int32 var_76 = 2;
        wp::float64 var_77;
        const wp::int32 var_78 = 0;
        wp::float64 var_79;
        const wp::int32 var_80 = 1;
        wp::float64 var_81;
        const wp::int32 var_82 = 2;
        wp::float64 var_83;
        const wp::int32 var_84 = 0;
        wp::float64 var_85;
        const wp::int32 var_86 = 1;
        wp::float64 var_87;
        const wp::int32 var_88 = 2;
        wp::float64 var_89;
        wp::vec_t<12,wp::float64> var_90;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        bool adj_1 = {};
        bool adj_2 = {};
        bool adj_3 = {};
        bool adj_4 = {};
        wp::vec_t<12,wp::float64> adj_5 = {};
        wp::vec_t<12,wp::float64> adj_6 = {};
        wp::vec_t<12,wp::float64> adj_7 = {};
        wp::vec_t<3,wp::float64> adj_8 = {};
        wp::vec_t<3,wp::float64> adj_9 = {};
        wp::vec_t<3,wp::float64> adj_10 = {};
        wp::float32 adj_11 = {};
        wp::float64 adj_12 = {};
        wp::float32 adj_13 = {};
        wp::float64 adj_14 = {};
        wp::vec_t<3,wp::float64> adj_15 = {};
        wp::vec_t<3,wp::float64> adj_16 = {};
        wp::vec_t<3,wp::float64> adj_17 = {};
        wp::vec_t<3,wp::float64> adj_18 = {};
        wp::vec_t<3,wp::float64> adj_19 = {};
        wp::vec_t<3,wp::float64> adj_20 = {};
        wp::vec_t<3,wp::float64> adj_21 = {};
        wp::vec_t<3,wp::float64> adj_22 = {};
        wp::vec_t<3,wp::float64> adj_23 = {};
        wp::int32 adj_24 = {};
        wp::float64 adj_25 = {};
        wp::int32 adj_26 = {};
        wp::float64 adj_27 = {};
        wp::int32 adj_28 = {};
        wp::float64 adj_29 = {};
        wp::int32 adj_30 = {};
        wp::float64 adj_31 = {};
        wp::int32 adj_32 = {};
        wp::float64 adj_33 = {};
        wp::int32 adj_34 = {};
        wp::float64 adj_35 = {};
        wp::int32 adj_36 = {};
        wp::float64 adj_37 = {};
        wp::int32 adj_38 = {};
        wp::float64 adj_39 = {};
        wp::int32 adj_40 = {};
        wp::float64 adj_41 = {};
        wp::mat_t<3,3,wp::float64> adj_42 = {};
        wp::int32 adj_43 = {};
        wp::float64 adj_44 = {};
        wp::int32 adj_45 = {};
        wp::float64 adj_46 = {};
        wp::int32 adj_47 = {};
        wp::float64 adj_48 = {};
        wp::vec_t<3,wp::float64> adj_49 = {};
        wp::vec_t<3,wp::float64> adj_50 = {};
        wp::vec_t<3,wp::float64> adj_51 = {};
        wp::vec_t<3,wp::float64> adj_52 = {};
        wp::vec_t<3,wp::float64> adj_53 = {};
        wp::vec_t<3,wp::float64> adj_54 = {};
        wp::vec_t<3,wp::float64> adj_55 = {};
        wp::vec_t<3,wp::float64> adj_56 = {};
        wp::vec_t<3,wp::float64> adj_57 = {};
        wp::vec_t<3,wp::float64> adj_58 = {};
        wp::vec_t<3,wp::float64> adj_59 = {};
        wp::vec_t<3,wp::float64> adj_60 = {};
        wp::vec_t<3,wp::float64> adj_61 = {};
        wp::vec_t<3,wp::float64> adj_62 = {};
        wp::vec_t<3,wp::float64> adj_63 = {};
        wp::vec_t<3,wp::float64> adj_64 = {};
        wp::vec_t<3,wp::float64> adj_65 = {};
        wp::int32 adj_66 = {};
        wp::float64 adj_67 = {};
        wp::int32 adj_68 = {};
        wp::float64 adj_69 = {};
        wp::int32 adj_70 = {};
        wp::float64 adj_71 = {};
        wp::int32 adj_72 = {};
        wp::float64 adj_73 = {};
        wp::int32 adj_74 = {};
        wp::float64 adj_75 = {};
        wp::int32 adj_76 = {};
        wp::float64 adj_77 = {};
        wp::int32 adj_78 = {};
        wp::float64 adj_79 = {};
        wp::int32 adj_80 = {};
        wp::float64 adj_81 = {};
        wp::int32 adj_82 = {};
        wp::float64 adj_83 = {};
        wp::int32 adj_84 = {};
        wp::float64 adj_85 = {};
        wp::int32 adj_86 = {};
        wp::float64 adj_87 = {};
        wp::int32 adj_88 = {};
        wp::float64 adj_89 = {};
        wp::vec_t<12,wp::float64> adj_90 = {};
        //---------
        // forward
        // def init_affine_kinematic_target_kernel(                                               <L 16>
        // tid = wp.tid()                                                                         <L 23>
        var_0 = builtin_tid1d();
        // if affine_has_constraint[tid] == False:                                                <L 24>
        var_1 = wp::address(var_affine_has_constraint, var_0);
        var_4 = wp::load(var_1);
        var_3 = (var_4 == var_2);
        if (var_3) {
            // return                                                                             <L 25>
            goto label0;
        }
        // target_state = affine_kinematic_target_pose[tid]                                       <L 26>
        var_5 = wp::address(var_affine_kinematic_target_pose, var_0);
        var_7 = wp::load(var_5);
        var_6 = wp::copy(var_7);
        // virtual_center = virtual_object_centers[tid]                                           <L 27>
        var_8 = wp::address(var_virtual_object_centers, var_0);
        var_10 = wp::load(var_8);
        var_9 = wp::copy(var_10);
        // _0 = wp.float64(0.0)                                                                   <L 28>
        var_12 = wp::float64(var_11);
        // _1 = wp.float64(1.0)                                                                   <L 29>
        var_14 = wp::float64(var_13);
        // rest_O = ABD_centers[tid]                                                              <L 30>
        var_15 = wp::address(var_ABD_centers, var_0);
        var_17 = wp::load(var_15);
        var_16 = wp::copy(var_17);
        // rest_A = rest_O + wp.vec3d(_1, _0, _0)                                                 <L 31>
        var_18 = wp::vec_t<3,wp::float64>(var_14, var_12, var_12);
        var_19 = wp::add(var_16, var_18);
        // rest_B = rest_O + wp.vec3d(_0, _1, _0)                                                 <L 32>
        var_20 = wp::vec_t<3,wp::float64>(var_12, var_14, var_12);
        var_21 = wp::add(var_16, var_20);
        // rest_C = rest_O + wp.vec3d(_0, _0, _1)                                                 <L 33>
        var_22 = wp::vec_t<3,wp::float64>(var_12, var_12, var_14);
        var_23 = wp::add(var_16, var_22);
        // R = wp.mat33d(                                                                         <L 34>
        // target_state[0],                                                                       <L 35>
        var_25 = wp::extract(var_6, var_24);
        // target_state[1],                                                                       <L 36>
        var_27 = wp::extract(var_6, var_26);
        // target_state[2],                                                                       <L 37>
        var_29 = wp::extract(var_6, var_28);
        // target_state[3],                                                                       <L 38>
        var_31 = wp::extract(var_6, var_30);
        // target_state[4],                                                                       <L 39>
        var_33 = wp::extract(var_6, var_32);
        // target_state[5],                                                                       <L 40>
        var_35 = wp::extract(var_6, var_34);
        // target_state[6],                                                                       <L 41>
        var_37 = wp::extract(var_6, var_36);
        // target_state[7],                                                                       <L 42>
        var_39 = wp::extract(var_6, var_38);
        // target_state[8],                                                                       <L 43>
        var_41 = wp::extract(var_6, var_40);
        var_42 = wp::mat_t<3,3,wp::float64>(var_25, var_27, var_29, var_31, var_33, var_35, var_37, var_39, var_41);
        // t = wp.vec3d(target_state[9], target_state[10], target_state[11])                      <L 45>
        var_44 = wp::extract(var_6, var_43);
        var_46 = wp::extract(var_6, var_45);
        var_48 = wp::extract(var_6, var_47);
        var_49 = wp::vec_t<3,wp::float64>(var_44, var_46, var_48);
        // O = R @ (rest_O - virtual_center) + virtual_center + t                                 <L 46>
        var_50 = wp::sub(var_16, var_9);
        var_51 = wp::mul(var_42, var_50);
        var_52 = wp::add(var_51, var_9);
        var_53 = wp::add(var_52, var_49);
        // A = R @ (rest_A - virtual_center) + virtual_center + t                                 <L 47>
        var_54 = wp::sub(var_19, var_9);
        var_55 = wp::mul(var_42, var_54);
        var_56 = wp::add(var_55, var_9);
        var_57 = wp::add(var_56, var_49);
        // B = R @ (rest_B - virtual_center) + virtual_center + t                                 <L 48>
        var_58 = wp::sub(var_21, var_9);
        var_59 = wp::mul(var_42, var_58);
        var_60 = wp::add(var_59, var_9);
        var_61 = wp::add(var_60, var_49);
        // C = R @ (rest_C - virtual_center) + virtual_center + t                                 <L 49>
        var_62 = wp::sub(var_23, var_9);
        var_63 = wp::mul(var_42, var_62);
        var_64 = wp::add(var_63, var_9);
        var_65 = wp::add(var_64, var_49);
        // affine_target_dof[tid] = vec12d(O[0], O[1], O[2], A[0], A[1], A[2], B[0], B[1], B[2], C[0], C[1], C[2])       <L 50>
        var_67 = wp::extract(var_53, var_66);
        var_69 = wp::extract(var_53, var_68);
        var_71 = wp::extract(var_53, var_70);
        var_73 = wp::extract(var_57, var_72);
        var_75 = wp::extract(var_57, var_74);
        var_77 = wp::extract(var_57, var_76);
        var_79 = wp::extract(var_61, var_78);
        var_81 = wp::extract(var_61, var_80);
        var_83 = wp::extract(var_61, var_82);
        var_85 = wp::extract(var_65, var_84);
        var_87 = wp::extract(var_65, var_86);
        var_89 = wp::extract(var_65, var_88);
        var_90 = wp::vec_t<12,wp::float64>({var_67, var_69, var_71, var_73, var_75, var_77, var_79, var_81, var_83, var_85, var_87, var_89});
        // wp::array_store(var_affine_target_dof, var_0, var_90);
        //---------
        // reverse
        wp::adj_array_store(var_affine_target_dof, var_0, var_90, adj_affine_target_dof, adj_0, adj_90);
        wp::adj_vec_t({var_67, var_69, var_71, var_73, var_75, var_77, var_79, var_81, var_83, var_85, var_87, var_89}, {&adj_67, &adj_69, &adj_71, &adj_73, &adj_75, &adj_77, &adj_79, &adj_81, &adj_83, &adj_85, &adj_87, &adj_89}, adj_90);
        wp::adj_extract(var_65, var_88, adj_65, adj_88, adj_89);
        wp::adj_extract(var_65, var_86, adj_65, adj_86, adj_87);
        wp::adj_extract(var_65, var_84, adj_65, adj_84, adj_85);
        wp::adj_extract(var_61, var_82, adj_61, adj_82, adj_83);
        wp::adj_extract(var_61, var_80, adj_61, adj_80, adj_81);
        wp::adj_extract(var_61, var_78, adj_61, adj_78, adj_79);
        wp::adj_extract(var_57, var_76, adj_57, adj_76, adj_77);
        wp::adj_extract(var_57, var_74, adj_57, adj_74, adj_75);
        wp::adj_extract(var_57, var_72, adj_57, adj_72, adj_73);
        wp::adj_extract(var_53, var_70, adj_53, adj_70, adj_71);
        wp::adj_extract(var_53, var_68, adj_53, adj_68, adj_69);
        wp::adj_extract(var_53, var_66, adj_53, adj_66, adj_67);
        // adj: affine_target_dof[tid] = vec12d(O[0], O[1], O[2], A[0], A[1], A[2], B[0], B[1], B[2], C[0], C[1], C[2])  <L 50>
        wp::adj_add(var_64, var_49, adj_64, adj_49, adj_65);
        wp::adj_add(var_63, var_9, adj_63, adj_9, adj_64);
        wp::adj_mul(var_42, var_62, adj_42, adj_62, adj_63);
        wp::adj_sub(var_23, var_9, adj_23, adj_9, adj_62);
        // adj: C = R @ (rest_C - virtual_center) + virtual_center + t                            <L 49>
        wp::adj_add(var_60, var_49, adj_60, adj_49, adj_61);
        wp::adj_add(var_59, var_9, adj_59, adj_9, adj_60);
        wp::adj_mul(var_42, var_58, adj_42, adj_58, adj_59);
        wp::adj_sub(var_21, var_9, adj_21, adj_9, adj_58);
        // adj: B = R @ (rest_B - virtual_center) + virtual_center + t                            <L 48>
        wp::adj_add(var_56, var_49, adj_56, adj_49, adj_57);
        wp::adj_add(var_55, var_9, adj_55, adj_9, adj_56);
        wp::adj_mul(var_42, var_54, adj_42, adj_54, adj_55);
        wp::adj_sub(var_19, var_9, adj_19, adj_9, adj_54);
        // adj: A = R @ (rest_A - virtual_center) + virtual_center + t                            <L 47>
        wp::adj_add(var_52, var_49, adj_52, adj_49, adj_53);
        wp::adj_add(var_51, var_9, adj_51, adj_9, adj_52);
        wp::adj_mul(var_42, var_50, adj_42, adj_50, adj_51);
        wp::adj_sub(var_16, var_9, adj_16, adj_9, adj_50);
        // adj: O = R @ (rest_O - virtual_center) + virtual_center + t                            <L 46>
        wp::adj_vec_t(var_44, var_46, var_48, adj_44, adj_46, adj_48, adj_49);
        wp::adj_extract(var_6, var_47, adj_6, adj_47, adj_48);
        wp::adj_extract(var_6, var_45, adj_6, adj_45, adj_46);
        wp::adj_extract(var_6, var_43, adj_6, adj_43, adj_44);
        // adj: t = wp.vec3d(target_state[9], target_state[10], target_state[11])                 <L 45>
        wp::adj_mat_t(var_25, var_27, var_29, var_31, var_33, var_35, var_37, var_39, var_41, adj_25, adj_27, adj_29, adj_31, adj_33, adj_35, adj_37, adj_39, adj_41, adj_42);
        wp::adj_extract(var_6, var_40, adj_6, adj_40, adj_41);
        // adj: target_state[8],                                                                  <L 43>
        wp::adj_extract(var_6, var_38, adj_6, adj_38, adj_39);
        // adj: target_state[7],                                                                  <L 42>
        wp::adj_extract(var_6, var_36, adj_6, adj_36, adj_37);
        // adj: target_state[6],                                                                  <L 41>
        wp::adj_extract(var_6, var_34, adj_6, adj_34, adj_35);
        // adj: target_state[5],                                                                  <L 40>
        wp::adj_extract(var_6, var_32, adj_6, adj_32, adj_33);
        // adj: target_state[4],                                                                  <L 39>
        wp::adj_extract(var_6, var_30, adj_6, adj_30, adj_31);
        // adj: target_state[3],                                                                  <L 38>
        wp::adj_extract(var_6, var_28, adj_6, adj_28, adj_29);
        // adj: target_state[2],                                                                  <L 37>
        wp::adj_extract(var_6, var_26, adj_6, adj_26, adj_27);
        // adj: target_state[1],                                                                  <L 36>
        wp::adj_extract(var_6, var_24, adj_6, adj_24, adj_25);
        // adj: target_state[0],                                                                  <L 35>
        // adj: R = wp.mat33d(                                                                    <L 34>
        wp::adj_add(var_16, var_22, adj_16, adj_22, adj_23);
        wp::adj_vec_t(var_12, var_12, var_14, adj_12, adj_12, adj_14, adj_22);
        // adj: rest_C = rest_O + wp.vec3d(_0, _0, _1)                                            <L 33>
        wp::adj_add(var_16, var_20, adj_16, adj_20, adj_21);
        wp::adj_vec_t(var_12, var_14, var_12, adj_12, adj_14, adj_12, adj_20);
        // adj: rest_B = rest_O + wp.vec3d(_0, _1, _0)                                            <L 32>
        wp::adj_add(var_16, var_18, adj_16, adj_18, adj_19);
        wp::adj_vec_t(var_14, var_12, var_12, adj_14, adj_12, adj_12, adj_18);
        // adj: rest_A = rest_O + wp.vec3d(_1, _0, _0)                                            <L 31>
        wp::adj_copy(var_17, adj_15, adj_16);
        wp::adj_load(var_15, adj_15, adj_17);
        wp::adj_address(var_ABD_centers, var_0, adj_ABD_centers, adj_0, adj_15);
        // adj: rest_O = ABD_centers[tid]                                                         <L 30>
        wp::adj_float64(var_13, adj_13, adj_14);
        // adj: _1 = wp.float64(1.0)                                                              <L 29>
        wp::adj_float64(var_11, adj_11, adj_12);
        // adj: _0 = wp.float64(0.0)                                                              <L 28>
        wp::adj_copy(var_10, adj_8, adj_9);
        wp::adj_load(var_8, adj_8, adj_10);
        wp::adj_address(var_virtual_object_centers, var_0, adj_virtual_object_centers, adj_0, adj_8);
        // adj: virtual_center = virtual_object_centers[tid]                                      <L 27>
        wp::adj_copy(var_7, adj_5, adj_6);
        wp::adj_load(var_5, adj_5, adj_7);
        wp::adj_address(var_affine_kinematic_target_pose, var_0, adj_affine_kinematic_target_pose, adj_0, adj_5);
        // adj: target_state = affine_kinematic_target_pose[tid]                                  <L 26>
        if (var_3) {
            label0:;
            // adj: return                                                                        <L 25>
        }
        wp::adj_load(var_1, adj_1, adj_4);
        wp::adj_address(var_affine_has_constraint, var_0, adj_affine_has_constraint, adj_0, adj_1);
        // adj: if affine_has_constraint[tid] == False:                                           <L 24>
        // adj: tid = wp.tid()                                                                    <L 23>
        // adj: def init_affine_kinematic_target_kernel(                                          <L 16>
        continue;
    }
}



extern "C" __global__ void compute_soft_kinematic_hess_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_soft_has_constraint,
    wp::float64 var_weight,
    COOMatrix3x3_0df4b45d var_hess_soft_diag,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32 var_1;
        wp::int32* var_2;
        wp::int32* var_3;
        wp::int32 var_4;
        const wp::int32 var_5 = 1;
        bool var_6;
        wp::int32 var_7;
        wp::int32 var_8;
        wp::int32* var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        const wp::int32 var_12 = 2;
        bool var_13;
        wp::int32 var_14;
        bool var_15;
        wp::float64* var_16;
        wp::float64 var_17;
        wp::float64 var_18;
        const wp::float32 var_19 = 0.0;
        wp::float64 var_20;
        bool* var_21;
        bool var_22;
        bool var_23;
        wp::float64 var_24;
        wp::mat_t<3,3,wp::float64> var_25;
        //---------
        // forward
        // def compute_soft_kinematic_hess(                                                       <L 276>
        // tid = wp.tid()                                                                         <L 285>
        var_0 = builtin_tid1d();
        // if env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID or env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED:       <L 286>
        var_1 = wp::add(var_0, var_affine_verts_num);
        var_2 = wp::address(var_node2env, var_1);
        var_4 = wp::load(var_2);
        var_3 = wp::address(var_env_states, var_4);
        var_7 = wp::load(var_3);
        var_6 = (var_7 == var_5);
        var_8 = wp::add(var_0, var_affine_verts_num);
        var_9 = wp::address(var_node2env, var_8);
        var_11 = wp::load(var_9);
        var_10 = wp::address(var_env_states, var_11);
        var_14 = wp::load(var_10);
        var_13 = (var_14 == var_12);
        var_15 = var_6 || var_13;
        if (var_15) {
            // return                                                                             <L 287>
            continue;
        }
        // mass = soft_verts_mass[tid]                                                            <L 288>
        var_16 = wp::address(var_soft_verts_mass, var_0);
        var_18 = wp::load(var_16);
        var_17 = wp::copy(var_18);
        // _0 = wp.float64(0.0)                                                                   <L 289>
        var_20 = wp::float64(var_19);
        // has_constraint = soft_has_constraint[tid]                                              <L 290>
        var_21 = wp::address(var_soft_has_constraint, var_0);
        var_23 = wp::load(var_21);
        var_22 = wp::copy(var_23);
        // weighted_mass = mass * weight                                                          <L 291>
        var_24 = wp::mul(var_17, var_weight);
        // mat3 = wp.mat33d(weighted_mass, _0, _0, _0, weighted_mass, _0, _0, _0, weighted_mass)       <L 292>
        var_25 = wp::mat_t<3,3,wp::float64>(var_24, var_20, var_20, var_20, var_24, var_20, var_20, var_20, var_24);
        // if has_constraint:                                                                     <L 293>
        if (var_22) {
            // matrix.COOMatrix3x3_atomic_add(hess_soft_diag, tid, mat3)                          <L 294>
            COOMatrix3x3_atomic_add_0(var_hess_soft_diag, var_0, var_25);
        }
    }
}



extern "C" __global__ void compute_soft_kinematic_hess_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_soft_has_constraint,
    wp::float64 var_weight,
    COOMatrix3x3_0df4b45d var_hess_soft_diag,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<bool> adj_soft_has_constraint,
    wp::float64 adj_weight,
    COOMatrix3x3_0df4b45d adj_hess_soft_diag,
    wp::int32 adj_affine_verts_num,
    wp::array_t<wp::float64> adj_soft_verts_mass,
    wp::array_t<wp::int32> adj_node2env,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32 var_1;
        wp::int32* var_2;
        wp::int32* var_3;
        wp::int32 var_4;
        const wp::int32 var_5 = 1;
        bool var_6;
        wp::int32 var_7;
        wp::int32 var_8;
        wp::int32* var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        const wp::int32 var_12 = 2;
        bool var_13;
        wp::int32 var_14;
        bool var_15;
        wp::float64* var_16;
        wp::float64 var_17;
        wp::float64 var_18;
        const wp::float32 var_19 = 0.0;
        wp::float64 var_20;
        bool* var_21;
        bool var_22;
        bool var_23;
        wp::float64 var_24;
        wp::mat_t<3,3,wp::float64> var_25;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        wp::int32 adj_5 = {};
        bool adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int32 adj_12 = {};
        bool adj_13 = {};
        wp::int32 adj_14 = {};
        bool adj_15 = {};
        wp::float64 adj_16 = {};
        wp::float64 adj_17 = {};
        wp::float64 adj_18 = {};
        wp::float32 adj_19 = {};
        wp::float64 adj_20 = {};
        bool adj_21 = {};
        bool adj_22 = {};
        bool adj_23 = {};
        wp::float64 adj_24 = {};
        wp::mat_t<3,3,wp::float64> adj_25 = {};
        //---------
        // forward
        // def compute_soft_kinematic_hess(                                                       <L 276>
        // tid = wp.tid()                                                                         <L 285>
        var_0 = builtin_tid1d();
        // if env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID or env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED:       <L 286>
        var_1 = wp::add(var_0, var_affine_verts_num);
        var_2 = wp::address(var_node2env, var_1);
        var_4 = wp::load(var_2);
        var_3 = wp::address(var_env_states, var_4);
        var_7 = wp::load(var_3);
        var_6 = (var_7 == var_5);
        var_8 = wp::add(var_0, var_affine_verts_num);
        var_9 = wp::address(var_node2env, var_8);
        var_11 = wp::load(var_9);
        var_10 = wp::address(var_env_states, var_11);
        var_14 = wp::load(var_10);
        var_13 = (var_14 == var_12);
        var_15 = var_6 || var_13;
        if (var_15) {
            // return                                                                             <L 287>
            goto label0;
        }
        // mass = soft_verts_mass[tid]                                                            <L 288>
        var_16 = wp::address(var_soft_verts_mass, var_0);
        var_18 = wp::load(var_16);
        var_17 = wp::copy(var_18);
        // _0 = wp.float64(0.0)                                                                   <L 289>
        var_20 = wp::float64(var_19);
        // has_constraint = soft_has_constraint[tid]                                              <L 290>
        var_21 = wp::address(var_soft_has_constraint, var_0);
        var_23 = wp::load(var_21);
        var_22 = wp::copy(var_23);
        // weighted_mass = mass * weight                                                          <L 291>
        var_24 = wp::mul(var_17, var_weight);
        // mat3 = wp.mat33d(weighted_mass, _0, _0, _0, weighted_mass, _0, _0, _0, weighted_mass)       <L 292>
        var_25 = wp::mat_t<3,3,wp::float64>(var_24, var_20, var_20, var_20, var_24, var_20, var_20, var_20, var_24);
        // if has_constraint:                                                                     <L 293>
        if (var_22) {
            // matrix.COOMatrix3x3_atomic_add(hess_soft_diag, tid, mat3)                          <L 294>
            COOMatrix3x3_atomic_add_0(var_hess_soft_diag, var_0, var_25);
        }
        //---------
        // reverse
        if (var_22) {
            adj_COOMatrix3x3_atomic_add_0(var_hess_soft_diag, var_0, var_25, adj_hess_soft_diag, adj_0, adj_25);
            // adj: matrix.COOMatrix3x3_atomic_add(hess_soft_diag, tid, mat3)                     <L 294>
        }
        // adj: if has_constraint:                                                                <L 293>
        wp::adj_mat_t(var_24, var_20, var_20, var_20, var_24, var_20, var_20, var_20, var_24, adj_24, adj_20, adj_20, adj_20, adj_24, adj_20, adj_20, adj_20, adj_24, adj_25);
        // adj: mat3 = wp.mat33d(weighted_mass, _0, _0, _0, weighted_mass, _0, _0, _0, weighted_mass)  <L 292>
        wp::adj_mul(var_17, var_weight, adj_17, adj_weight, adj_24);
        // adj: weighted_mass = mass * weight                                                     <L 291>
        wp::adj_copy(var_23, adj_21, adj_22);
        wp::adj_load(var_21, adj_21, adj_23);
        wp::adj_address(var_soft_has_constraint, var_0, adj_soft_has_constraint, adj_0, adj_21);
        // adj: has_constraint = soft_has_constraint[tid]                                         <L 290>
        wp::adj_float64(var_19, adj_19, adj_20);
        // adj: _0 = wp.float64(0.0)                                                              <L 289>
        wp::adj_copy(var_18, adj_16, adj_17);
        wp::adj_load(var_16, adj_16, adj_18);
        wp::adj_address(var_soft_verts_mass, var_0, adj_soft_verts_mass, adj_0, adj_16);
        // adj: mass = soft_verts_mass[tid]                                                       <L 288>
        if (var_15) {
            label0:;
            // adj: return                                                                        <L 287>
        }
        wp::adj_load(var_10, adj_10, adj_14);
        wp::adj_address(var_env_states, var_11, adj_env_states, adj_9, adj_10);
        wp::adj_load(var_9, adj_9, adj_11);
        wp::adj_address(var_node2env, var_8, adj_node2env, adj_8, adj_9);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_8);
        wp::adj_load(var_3, adj_3, adj_7);
        wp::adj_address(var_env_states, var_4, adj_env_states, adj_2, adj_3);
        wp::adj_load(var_2, adj_2, adj_4);
        wp::adj_address(var_node2env, var_1, adj_node2env, adj_1, adj_2);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_1);
        // adj: if env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID or env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED:  <L 286>
        // adj: tid = wp.tid()                                                                    <L 285>
        // adj: def compute_soft_kinematic_hess(                                                  <L 276>
        continue;
    }
}

