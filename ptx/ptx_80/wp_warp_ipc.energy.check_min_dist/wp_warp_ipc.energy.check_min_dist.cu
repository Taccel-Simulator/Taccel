#include "hip/hip_runtime.h"

#define WP_TILE_BLOCK_DIM 256
#define WP_NO_CRT
#include "builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx, dim)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)

extern "C" {
}

// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/distance_type.py:4
static CUDA_CALLABLE void solveLdlt2D_0(
    wp::float64 var_A00,
    wp::float64 var_A01,
    wp::float64 var_A11,
    wp::float64 var_b0,
    wp::float64 var_b1,
    wp::float64 & ret_0,
    wp::float64 & ret_1)
{
    //---------
    // primal vars
    wp::float64 var_0;
    wp::float64 var_1;
    wp::float64 var_2;
    wp::float64 var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    wp::float64 var_8;
    wp::float64 var_9;
    wp::float64 var_10;
    wp::float64 var_11;
    wp::float64 var_12;
    //---------
    // forward
    // def solveLdlt2D(A00: wp.float64, A01: wp.float64, A11: wp.float64, b0: wp.float64, b1: wp.float64):       <L 5>
    // D00 = A00                                                                              <L 6>
    var_0 = wp::copy(var_A00);
    // L01 = A01 / D00                                                                        <L 7>
    var_1 = wp::div(var_A01, var_0);
    // D11 = A11 - L01 * L01 * D00                                                            <L 8>
    var_2 = wp::mul(var_1, var_1);
    var_3 = wp::mul(var_2, var_0);
    var_4 = wp::sub(var_A11, var_3);
    // y0 = b0                                                                                <L 10>
    var_5 = wp::copy(var_b0);
    // y1 = b1 - L01 * y0                                                                     <L 11>
    var_6 = wp::mul(var_1, var_5);
    var_7 = wp::sub(var_b1, var_6);
    // x1 = y1 / D11                                                                          <L 13>
    var_8 = wp::div(var_7, var_4);
    // x0 = (y0 - D00 * L01 * x1) / D00                                                       <L 14>
    var_9 = wp::mul(var_0, var_1);
    var_10 = wp::mul(var_9, var_8);
    var_11 = wp::sub(var_5, var_10);
    var_12 = wp::div(var_11, var_0);
    // return x0, x1                                                                          <L 15>
    ret_0 = var_12;
    ret_1 = var_8;
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/distance_type.py:4
static CUDA_CALLABLE void adj_solveLdlt2D_0(
    wp::float64 var_A00,
    wp::float64 var_A01,
    wp::float64 var_A11,
    wp::float64 var_b0,
    wp::float64 var_b1,
    wp::float64 & ret_0,
    wp::float64 & ret_1,
    wp::float64 & adj_A00,
    wp::float64 & adj_A01,
    wp::float64 & adj_A11,
    wp::float64 & adj_b0,
    wp::float64 & adj_b1,
    wp::float64 & adj_ret_0,
    wp::float64 & adj_ret_1)
{
    //---------
    // primal vars
    wp::float64 var_0;
    wp::float64 var_1;
    wp::float64 var_2;
    wp::float64 var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    wp::float64 var_8;
    wp::float64 var_9;
    wp::float64 var_10;
    wp::float64 var_11;
    wp::float64 var_12;
    //---------
    // dual vars
    wp::float64 adj_0 = {};
    wp::float64 adj_1 = {};
    wp::float64 adj_2 = {};
    wp::float64 adj_3 = {};
    wp::float64 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::float64 adj_6 = {};
    wp::float64 adj_7 = {};
    wp::float64 adj_8 = {};
    wp::float64 adj_9 = {};
    wp::float64 adj_10 = {};
    wp::float64 adj_11 = {};
    wp::float64 adj_12 = {};
    //---------
    // forward
    // def solveLdlt2D(A00: wp.float64, A01: wp.float64, A11: wp.float64, b0: wp.float64, b1: wp.float64):       <L 5>
    // D00 = A00                                                                              <L 6>
    var_0 = wp::copy(var_A00);
    // L01 = A01 / D00                                                                        <L 7>
    var_1 = wp::div(var_A01, var_0);
    // D11 = A11 - L01 * L01 * D00                                                            <L 8>
    var_2 = wp::mul(var_1, var_1);
    var_3 = wp::mul(var_2, var_0);
    var_4 = wp::sub(var_A11, var_3);
    // y0 = b0                                                                                <L 10>
    var_5 = wp::copy(var_b0);
    // y1 = b1 - L01 * y0                                                                     <L 11>
    var_6 = wp::mul(var_1, var_5);
    var_7 = wp::sub(var_b1, var_6);
    // x1 = y1 / D11                                                                          <L 13>
    var_8 = wp::div(var_7, var_4);
    // x0 = (y0 - D00 * L01 * x1) / D00                                                       <L 14>
    var_9 = wp::mul(var_0, var_1);
    var_10 = wp::mul(var_9, var_8);
    var_11 = wp::sub(var_5, var_10);
    var_12 = wp::div(var_11, var_0);
    // return x0, x1                                                                          <L 15>
    ret_0 = var_12;
    ret_1 = var_8;
    goto label0;
    //---------
    // reverse
    label0:;
    adj_8 += adj_ret_1;
    adj_12 += adj_ret_0;
    // adj: return x0, x1                                                                     <L 15>
    wp::adj_div(var_11, var_0, var_12, adj_11, adj_0, adj_12);
    wp::adj_sub(var_5, var_10, adj_5, adj_10, adj_11);
    wp::adj_mul(var_9, var_8, adj_9, adj_8, adj_10);
    wp::adj_mul(var_0, var_1, adj_0, adj_1, adj_9);
    // adj: x0 = (y0 - D00 * L01 * x1) / D00                                                  <L 14>
    wp::adj_div(var_7, var_4, var_8, adj_7, adj_4, adj_8);
    // adj: x1 = y1 / D11                                                                     <L 13>
    wp::adj_sub(var_b1, var_6, adj_b1, adj_6, adj_7);
    wp::adj_mul(var_1, var_5, adj_1, adj_5, adj_6);
    // adj: y1 = b1 - L01 * y0                                                                <L 11>
    wp::adj_copy(var_b0, adj_b0, adj_5);
    // adj: y0 = b0                                                                           <L 10>
    wp::adj_sub(var_A11, var_3, adj_A11, adj_3, adj_4);
    wp::adj_mul(var_2, var_0, adj_2, adj_0, adj_3);
    wp::adj_mul(var_1, var_1, adj_1, adj_1, adj_2);
    // adj: D11 = A11 - L01 * L01 * D00                                                       <L 8>
    wp::adj_div(var_A01, var_0, var_1, adj_A01, adj_0, adj_1);
    // adj: L01 = A01 / D00                                                                   <L 7>
    wp::adj_copy(var_A00, adj_A00, adj_0);
    // adj: D00 = A00                                                                         <L 6>
    // adj: def solveLdlt2D(A00: wp.float64, A01: wp.float64, A11: wp.float64, b0: wp.float64, b1: wp.float64):  <L 5>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/distance_type.py:18
static CUDA_CALLABLE wp::int32 point_triangle_distance_type_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_t0,
    wp::vec_t<3,wp::float64> var_t1,
    wp::vec_t<3,wp::float64> var_t2)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::vec_t<4,wp::float64> var_4;
    wp::vec_t<2,wp::float64> var_5;
    const wp::int32 var_6 = 0;
    wp::float64 var_7;
    const wp::int32 var_8 = 0;
    wp::float64 var_9;
    wp::float64 var_10;
    const wp::int32 var_11 = 1;
    wp::float64 var_12;
    const wp::int32 var_13 = 1;
    wp::float64 var_14;
    wp::float64 var_15;
    wp::float64 var_16;
    const wp::int32 var_17 = 2;
    wp::float64 var_18;
    const wp::int32 var_19 = 2;
    wp::float64 var_20;
    wp::float64 var_21;
    wp::float64 var_22;
    const wp::int32 var_23 = 0;
    const wp::int32 var_24 = 0;
    wp::float64 var_25;
    const wp::int32 var_26 = 0;
    wp::float64 var_27;
    wp::float64 var_28;
    const wp::int32 var_29 = 1;
    wp::float64 var_30;
    const wp::int32 var_31 = 1;
    wp::float64 var_32;
    wp::float64 var_33;
    wp::float64 var_34;
    const wp::int32 var_35 = 2;
    wp::float64 var_36;
    const wp::int32 var_37 = 2;
    wp::float64 var_38;
    wp::float64 var_39;
    wp::float64 var_40;
    const wp::int32 var_41 = 2;
    const wp::int32 var_42 = 2;
    wp::float64 var_43;
    const wp::int32 var_44 = 1;
    const wp::int32 var_45 = 0;
    wp::float64 var_46;
    const wp::int32 var_47 = 0;
    wp::float64 var_48;
    wp::float64 var_49;
    const wp::int32 var_50 = 1;
    wp::float64 var_51;
    const wp::int32 var_52 = 1;
    wp::float64 var_53;
    wp::float64 var_54;
    wp::float64 var_55;
    const wp::int32 var_56 = 2;
    wp::float64 var_57;
    const wp::int32 var_58 = 2;
    wp::float64 var_59;
    wp::float64 var_60;
    wp::float64 var_61;
    const wp::int32 var_62 = 3;
    wp::vec_t<3,wp::float64> var_63;
    wp::float64 var_64;
    const wp::int32 var_65 = 0;
    wp::float64 var_66;
    const wp::int32 var_67 = 1;
    const wp::int32 var_68 = 0;
    wp::float64 var_69;
    const wp::int32 var_70 = 1;
    wp::float64 var_71;
    const wp::int32 var_72 = 3;
    wp::float64 var_73;
    const wp::int32 var_74 = 0;
    wp::float64 var_75;
    const wp::int32 var_76 = 1;
    wp::float64 var_77;
    wp::float64 var_78;
    wp::float64 var_79;
    const wp::float32 var_80 = 0.0;
    wp::float64 var_81;
    bool var_82;
    const wp::float32 var_83 = 1.0;
    wp::float64 var_84;
    bool var_85;
    const wp::float32 var_86 = 0.0;
    wp::float64 var_87;
    bool var_88;
    bool var_89;
    const wp::int32 var_90 = 3;
    wp::vec_t<3,wp::float64> var_91;
    wp::vec_t<3,wp::float64> var_92;
    const wp::int32 var_93 = 0;
    wp::float64 var_94;
    const wp::int32 var_95 = 0;
    wp::float64 var_96;
    wp::float64 var_97;
    const wp::int32 var_98 = 1;
    wp::float64 var_99;
    const wp::int32 var_100 = 1;
    wp::float64 var_101;
    wp::float64 var_102;
    wp::float64 var_103;
    const wp::int32 var_104 = 2;
    wp::float64 var_105;
    const wp::int32 var_106 = 2;
    wp::float64 var_107;
    wp::float64 var_108;
    wp::float64 var_109;
    const wp::int32 var_110 = 0;
    const wp::int32 var_111 = 0;
    wp::float64 var_112;
    const wp::int32 var_113 = 0;
    wp::float64 var_114;
    wp::float64 var_115;
    const wp::int32 var_116 = 1;
    wp::float64 var_117;
    const wp::int32 var_118 = 1;
    wp::float64 var_119;
    wp::float64 var_120;
    wp::float64 var_121;
    const wp::int32 var_122 = 2;
    wp::float64 var_123;
    const wp::int32 var_124 = 2;
    wp::float64 var_125;
    wp::float64 var_126;
    wp::float64 var_127;
    const wp::int32 var_128 = 2;
    const wp::int32 var_129 = 2;
    wp::float64 var_130;
    const wp::int32 var_131 = 1;
    const wp::int32 var_132 = 0;
    wp::float64 var_133;
    const wp::int32 var_134 = 0;
    wp::float64 var_135;
    wp::float64 var_136;
    const wp::int32 var_137 = 1;
    wp::float64 var_138;
    const wp::int32 var_139 = 1;
    wp::float64 var_140;
    wp::float64 var_141;
    wp::float64 var_142;
    const wp::int32 var_143 = 2;
    wp::float64 var_144;
    const wp::int32 var_145 = 2;
    wp::float64 var_146;
    wp::float64 var_147;
    wp::float64 var_148;
    const wp::int32 var_149 = 3;
    wp::vec_t<3,wp::float64> var_150;
    wp::float64 var_151;
    const wp::int32 var_152 = 0;
    wp::float64 var_153;
    const wp::int32 var_154 = 1;
    const wp::int32 var_155 = 0;
    wp::float64 var_156;
    const wp::int32 var_157 = 1;
    wp::float64 var_158;
    const wp::int32 var_159 = 3;
    wp::float64 var_160;
    const wp::int32 var_161 = 0;
    wp::float64 var_162;
    const wp::int32 var_163 = 1;
    wp::float64 var_164;
    wp::float64 var_165;
    wp::float64 var_166;
    const wp::float32 var_167 = 0.0;
    wp::float64 var_168;
    bool var_169;
    const wp::float32 var_170 = 1.0;
    wp::float64 var_171;
    bool var_172;
    const wp::float32 var_173 = 0.0;
    wp::float64 var_174;
    bool var_175;
    bool var_176;
    const wp::int32 var_177 = 4;
    wp::vec_t<3,wp::float64> var_178;
    wp::vec_t<3,wp::float64> var_179;
    const wp::int32 var_180 = 0;
    wp::float64 var_181;
    const wp::int32 var_182 = 0;
    wp::float64 var_183;
    wp::float64 var_184;
    const wp::int32 var_185 = 1;
    wp::float64 var_186;
    const wp::int32 var_187 = 1;
    wp::float64 var_188;
    wp::float64 var_189;
    wp::float64 var_190;
    const wp::int32 var_191 = 2;
    wp::float64 var_192;
    const wp::int32 var_193 = 2;
    wp::float64 var_194;
    wp::float64 var_195;
    wp::float64 var_196;
    const wp::int32 var_197 = 0;
    const wp::int32 var_198 = 0;
    wp::float64 var_199;
    const wp::int32 var_200 = 0;
    wp::float64 var_201;
    wp::float64 var_202;
    const wp::int32 var_203 = 1;
    wp::float64 var_204;
    const wp::int32 var_205 = 1;
    wp::float64 var_206;
    wp::float64 var_207;
    wp::float64 var_208;
    const wp::int32 var_209 = 2;
    wp::float64 var_210;
    const wp::int32 var_211 = 2;
    wp::float64 var_212;
    wp::float64 var_213;
    wp::float64 var_214;
    const wp::int32 var_215 = 2;
    const wp::int32 var_216 = 2;
    wp::float64 var_217;
    const wp::int32 var_218 = 1;
    const wp::int32 var_219 = 0;
    wp::float64 var_220;
    const wp::int32 var_221 = 0;
    wp::float64 var_222;
    wp::float64 var_223;
    const wp::int32 var_224 = 1;
    wp::float64 var_225;
    const wp::int32 var_226 = 1;
    wp::float64 var_227;
    wp::float64 var_228;
    wp::float64 var_229;
    const wp::int32 var_230 = 2;
    wp::float64 var_231;
    const wp::int32 var_232 = 2;
    wp::float64 var_233;
    wp::float64 var_234;
    wp::float64 var_235;
    const wp::int32 var_236 = 3;
    wp::vec_t<3,wp::float64> var_237;
    wp::float64 var_238;
    const wp::int32 var_239 = 0;
    wp::float64 var_240;
    const wp::int32 var_241 = 1;
    const wp::int32 var_242 = 0;
    wp::float64 var_243;
    const wp::int32 var_244 = 1;
    wp::float64 var_245;
    const wp::int32 var_246 = 3;
    wp::float64 var_247;
    const wp::int32 var_248 = 0;
    wp::float64 var_249;
    const wp::int32 var_250 = 1;
    wp::float64 var_251;
    wp::float64 var_252;
    wp::float64 var_253;
    const wp::float32 var_254 = 0.0;
    wp::float64 var_255;
    bool var_256;
    const wp::float32 var_257 = 1.0;
    wp::float64 var_258;
    bool var_259;
    const wp::float32 var_260 = 0.0;
    wp::float64 var_261;
    bool var_262;
    bool var_263;
    const wp::int32 var_264 = 5;
    const wp::float32 var_265 = 0.0;
    wp::float64 var_266;
    bool var_267;
    const wp::float32 var_268 = 1.0;
    wp::float64 var_269;
    bool var_270;
    bool var_271;
    const wp::int32 var_272 = 0;
    const wp::float32 var_273 = 0.0;
    wp::float64 var_274;
    bool var_275;
    const wp::float32 var_276 = 1.0;
    wp::float64 var_277;
    bool var_278;
    bool var_279;
    const wp::int32 var_280 = 1;
    const wp::float32 var_281 = 0.0;
    wp::float64 var_282;
    bool var_283;
    const wp::float32 var_284 = 1.0;
    wp::float64 var_285;
    bool var_286;
    bool var_287;
    const wp::int32 var_288 = 2;
    const wp::int32 var_289 = 6;
    wp::vec_t<3,wp::float64> var_290;
    wp::vec_t<3,wp::float64> var_291;
    wp::vec_t<3,wp::float64> var_292;
    wp::vec_t<3,wp::float64> var_293;
    wp::vec_t<3,wp::float64> var_294;
    wp::vec_t<3,wp::float64> var_295;
    //---------
    // forward
    // def point_triangle_distance_type(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d) -> int:       <L 19>
    // basis_row0 = t1 - t0                                                                   <L 20>
    var_0 = wp::sub(var_t1, var_t0);
    // basis_row1 = t2 - t0                                                                   <L 21>
    var_1 = wp::sub(var_t2, var_t0);
    // nVec = wp.cross(basis_row0, basis_row1)                                                <L 22>
    var_2 = wp::cross(var_0, var_1);
    // basis_row1 = wp.cross(basis_row0, nVec)                                                <L 23>
    var_3 = wp::cross(var_0, var_2);
    // sys = wp.vec4d()                                                                       <L 25>
    var_4 = wp::vec_t<4,wp::float64>();
    // rhs = wp.vec2d()                                                                       <L 26>
    var_5 = wp::vec_t<2,wp::float64>();
    // sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]       <L 27>
    var_7 = wp::extract(var_0, var_6);
    var_9 = wp::extract(var_0, var_8);
    var_10 = wp::mul(var_7, var_9);
    var_12 = wp::extract(var_0, var_11);
    var_14 = wp::extract(var_0, var_13);
    var_15 = wp::mul(var_12, var_14);
    var_16 = wp::add(var_10, var_15);
    var_18 = wp::extract(var_0, var_17);
    var_20 = wp::extract(var_0, var_19);
    var_21 = wp::mul(var_18, var_20);
    var_22 = wp::add(var_16, var_21);
    wp::assign_inplace(var_4, var_23, var_22);
    // sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]       <L 28>
    var_25 = wp::extract(var_0, var_24);
    var_27 = wp::extract(var_3, var_26);
    var_28 = wp::mul(var_25, var_27);
    var_30 = wp::extract(var_0, var_29);
    var_32 = wp::extract(var_3, var_31);
    var_33 = wp::mul(var_30, var_32);
    var_34 = wp::add(var_28, var_33);
    var_36 = wp::extract(var_0, var_35);
    var_38 = wp::extract(var_3, var_37);
    var_39 = wp::mul(var_36, var_38);
    var_40 = wp::add(var_34, var_39);
    wp::assign_inplace(var_4, var_41, var_40);
    // sys[1] = sys[2]                                                                        <L 29>
    var_43 = wp::extract(var_4, var_42);
    wp::assign_inplace(var_4, var_44, var_43);
    // sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]       <L 30>
    var_46 = wp::extract(var_3, var_45);
    var_48 = wp::extract(var_3, var_47);
    var_49 = wp::mul(var_46, var_48);
    var_51 = wp::extract(var_3, var_50);
    var_53 = wp::extract(var_3, var_52);
    var_54 = wp::mul(var_51, var_53);
    var_55 = wp::add(var_49, var_54);
    var_57 = wp::extract(var_3, var_56);
    var_59 = wp::extract(var_3, var_58);
    var_60 = wp::mul(var_57, var_59);
    var_61 = wp::add(var_55, var_60);
    wp::assign_inplace(var_4, var_62, var_61);
    // b = p - t0                                                                             <L 32>
    var_63 = wp::sub(var_p, var_t0);
    // rhs[0] = wp.dot(basis_row0, b)                                                         <L 33>
    var_64 = wp::dot(var_0, var_63);
    wp::assign_inplace(var_5, var_65, var_64);
    // rhs[1] = wp.dot(basis_row1, b)                                                         <L 34>
    var_66 = wp::dot(var_3, var_63);
    wp::assign_inplace(var_5, var_67, var_66);
    // param_col00, param_col01 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])         <L 35>
    var_69 = wp::extract(var_4, var_68);
    var_71 = wp::extract(var_4, var_70);
    var_73 = wp::extract(var_4, var_72);
    var_75 = wp::extract(var_5, var_74);
    var_77 = wp::extract(var_5, var_76);
    solveLdlt2D_0(var_69, var_71, var_73, var_75, var_77, var_78, var_79);
    // if param_col00 > wp.float64(0.0) and param_col00 < wp.float64(1.0) and param_col01 >= wp.float64(0.0):       <L 36>
    var_81 = wp::float64(var_80);
    var_82 = (var_78 > var_81);
    var_84 = wp::float64(var_83);
    var_85 = (var_78 < var_84);
    var_87 = wp::float64(var_86);
    var_88 = (var_79 >= var_87);
    var_89 = var_82 && var_85 && var_88;
    if (var_89) {
        // return 3  # PE t0t1                                                                <L 37>
        return var_90;
    }
    if (!var_89) {
        // basis_row0 = t2 - t1                                                               <L 39>
        var_91 = wp::sub(var_t2, var_t1);
        // basis_row1 = wp.cross(basis_row0, nVec)                                            <L 40>
        var_92 = wp::cross(var_91, var_2);
        // sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]       <L 42>
        var_94 = wp::extract(var_91, var_93);
        var_96 = wp::extract(var_91, var_95);
        var_97 = wp::mul(var_94, var_96);
        var_99 = wp::extract(var_91, var_98);
        var_101 = wp::extract(var_91, var_100);
        var_102 = wp::mul(var_99, var_101);
        var_103 = wp::add(var_97, var_102);
        var_105 = wp::extract(var_91, var_104);
        var_107 = wp::extract(var_91, var_106);
        var_108 = wp::mul(var_105, var_107);
        var_109 = wp::add(var_103, var_108);
        wp::assign_inplace(var_4, var_110, var_109);
        // sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]       <L 43>
        var_112 = wp::extract(var_91, var_111);
        var_114 = wp::extract(var_92, var_113);
        var_115 = wp::mul(var_112, var_114);
        var_117 = wp::extract(var_91, var_116);
        var_119 = wp::extract(var_92, var_118);
        var_120 = wp::mul(var_117, var_119);
        var_121 = wp::add(var_115, var_120);
        var_123 = wp::extract(var_91, var_122);
        var_125 = wp::extract(var_92, var_124);
        var_126 = wp::mul(var_123, var_125);
        var_127 = wp::add(var_121, var_126);
        wp::assign_inplace(var_4, var_128, var_127);
        // sys[1] = sys[2]                                                                    <L 44>
        var_130 = wp::extract(var_4, var_129);
        wp::assign_inplace(var_4, var_131, var_130);
        // sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]       <L 45>
        var_133 = wp::extract(var_92, var_132);
        var_135 = wp::extract(var_92, var_134);
        var_136 = wp::mul(var_133, var_135);
        var_138 = wp::extract(var_92, var_137);
        var_140 = wp::extract(var_92, var_139);
        var_141 = wp::mul(var_138, var_140);
        var_142 = wp::add(var_136, var_141);
        var_144 = wp::extract(var_92, var_143);
        var_146 = wp::extract(var_92, var_145);
        var_147 = wp::mul(var_144, var_146);
        var_148 = wp::add(var_142, var_147);
        wp::assign_inplace(var_4, var_149, var_148);
        // b = p - t1                                                                         <L 46>
        var_150 = wp::sub(var_p, var_t1);
        // rhs[0] = wp.dot(basis_row0, b)                                                     <L 47>
        var_151 = wp::dot(var_91, var_150);
        wp::assign_inplace(var_5, var_152, var_151);
        // rhs[1] = wp.dot(basis_row1, b)                                                     <L 48>
        var_153 = wp::dot(var_92, var_150);
        wp::assign_inplace(var_5, var_154, var_153);
        // param_col10, param_col11 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])       <L 49>
        var_156 = wp::extract(var_4, var_155);
        var_158 = wp::extract(var_4, var_157);
        var_160 = wp::extract(var_4, var_159);
        var_162 = wp::extract(var_5, var_161);
        var_164 = wp::extract(var_5, var_163);
        solveLdlt2D_0(var_156, var_158, var_160, var_162, var_164, var_165, var_166);
        // if param_col10 > wp.float64(0.0) and param_col10 < wp.float64(1.0) and param_col11 >= wp.float64(0.0):       <L 50>
        var_168 = wp::float64(var_167);
        var_169 = (var_165 > var_168);
        var_171 = wp::float64(var_170);
        var_172 = (var_165 < var_171);
        var_174 = wp::float64(var_173);
        var_175 = (var_166 >= var_174);
        var_176 = var_169 && var_172 && var_175;
        if (var_176) {
            // return 4  # PE t1t2                                                            <L 51>
            return var_177;
        }
        if (!var_176) {
            // basis_row0 = t0 - t2                                                           <L 53>
            var_178 = wp::sub(var_t0, var_t2);
            // basis_row1 = wp.cross(basis_row0, nVec)                                        <L 54>
            var_179 = wp::cross(var_178, var_2);
            // sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]       <L 55>
            var_181 = wp::extract(var_178, var_180);
            var_183 = wp::extract(var_178, var_182);
            var_184 = wp::mul(var_181, var_183);
            var_186 = wp::extract(var_178, var_185);
            var_188 = wp::extract(var_178, var_187);
            var_189 = wp::mul(var_186, var_188);
            var_190 = wp::add(var_184, var_189);
            var_192 = wp::extract(var_178, var_191);
            var_194 = wp::extract(var_178, var_193);
            var_195 = wp::mul(var_192, var_194);
            var_196 = wp::add(var_190, var_195);
            wp::assign_inplace(var_4, var_197, var_196);
            // sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]       <L 56>
            var_199 = wp::extract(var_178, var_198);
            var_201 = wp::extract(var_179, var_200);
            var_202 = wp::mul(var_199, var_201);
            var_204 = wp::extract(var_178, var_203);
            var_206 = wp::extract(var_179, var_205);
            var_207 = wp::mul(var_204, var_206);
            var_208 = wp::add(var_202, var_207);
            var_210 = wp::extract(var_178, var_209);
            var_212 = wp::extract(var_179, var_211);
            var_213 = wp::mul(var_210, var_212);
            var_214 = wp::add(var_208, var_213);
            wp::assign_inplace(var_4, var_215, var_214);
            // sys[1] = sys[2]                                                                <L 57>
            var_217 = wp::extract(var_4, var_216);
            wp::assign_inplace(var_4, var_218, var_217);
            // sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]       <L 58>
            var_220 = wp::extract(var_179, var_219);
            var_222 = wp::extract(var_179, var_221);
            var_223 = wp::mul(var_220, var_222);
            var_225 = wp::extract(var_179, var_224);
            var_227 = wp::extract(var_179, var_226);
            var_228 = wp::mul(var_225, var_227);
            var_229 = wp::add(var_223, var_228);
            var_231 = wp::extract(var_179, var_230);
            var_233 = wp::extract(var_179, var_232);
            var_234 = wp::mul(var_231, var_233);
            var_235 = wp::add(var_229, var_234);
            wp::assign_inplace(var_4, var_236, var_235);
            // b = p - t2                                                                     <L 59>
            var_237 = wp::sub(var_p, var_t2);
            // rhs[0] = wp.dot(basis_row0, b)                                                 <L 60>
            var_238 = wp::dot(var_178, var_237);
            wp::assign_inplace(var_5, var_239, var_238);
            // rhs[1] = wp.dot(basis_row1, b)                                                 <L 61>
            var_240 = wp::dot(var_179, var_237);
            wp::assign_inplace(var_5, var_241, var_240);
            // param_col20, param_col21 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])       <L 62>
            var_243 = wp::extract(var_4, var_242);
            var_245 = wp::extract(var_4, var_244);
            var_247 = wp::extract(var_4, var_246);
            var_249 = wp::extract(var_5, var_248);
            var_251 = wp::extract(var_5, var_250);
            solveLdlt2D_0(var_243, var_245, var_247, var_249, var_251, var_252, var_253);
            // if param_col20 > wp.float64(0.0) and param_col20 < wp.float64(1.0) and param_col21 >= wp.float64(0.0):       <L 64>
            var_255 = wp::float64(var_254);
            var_256 = (var_252 > var_255);
            var_258 = wp::float64(var_257);
            var_259 = (var_252 < var_258);
            var_261 = wp::float64(var_260);
            var_262 = (var_253 >= var_261);
            var_263 = var_256 && var_259 && var_262;
            if (var_263) {
                // return 5  # PE t2t0                                                        <L 65>
                return var_264;
            }
            if (!var_263) {
                // if param_col00 <= wp.float64(0.0) and param_col20 >= wp.float64(1.0):       <L 67>
                var_266 = wp::float64(var_265);
                var_267 = (var_78 <= var_266);
                var_269 = wp::float64(var_268);
                var_270 = (var_252 >= var_269);
                var_271 = var_267 && var_270;
                if (var_271) {
                    // return 0  # PP t0                                                      <L 68>
                    return var_272;
                }
                if (!var_271) {
                    // elif param_col10 <= wp.float64(0.0) and param_col00 >= wp.float64(1.0):       <L 69>
                    var_274 = wp::float64(var_273);
                    var_275 = (var_165 <= var_274);
                    var_277 = wp::float64(var_276);
                    var_278 = (var_78 >= var_277);
                    var_279 = var_275 && var_278;
                    if (var_279) {
                        // return 1  # PP t1                                                  <L 70>
                        return var_280;
                    }
                    if (!var_279) {
                        // elif param_col20 <= wp.float64(0.0) and param_col10 >= wp.float64(1.0):       <L 71>
                        var_282 = wp::float64(var_281);
                        var_283 = (var_252 <= var_282);
                        var_285 = wp::float64(var_284);
                        var_286 = (var_165 >= var_285);
                        var_287 = var_283 && var_286;
                        if (var_287) {
                            // return 2  # PP t2                                              <L 72>
                            return var_288;
                        }
                        if (!var_287) {
                            // return 6  # PT                                                 <L 74>
                            return var_289;
                        }
                    }
                }
            }
        }
        var_290 = wp::where(var_176, var_91, var_178);
        var_291 = wp::where(var_176, var_92, var_179);
        var_292 = wp::where(var_176, var_150, var_237);
    }
    var_293 = wp::where(var_89, var_0, var_290);
    var_294 = wp::where(var_89, var_3, var_291);
    var_295 = wp::where(var_89, var_63, var_292);
    return {};
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/distance_type.py:18
static CUDA_CALLABLE void adj_point_triangle_distance_type_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_t0,
    wp::vec_t<3,wp::float64> var_t1,
    wp::vec_t<3,wp::float64> var_t2,
    wp::vec_t<3,wp::float64> & adj_p,
    wp::vec_t<3,wp::float64> & adj_t0,
    wp::vec_t<3,wp::float64> & adj_t1,
    wp::vec_t<3,wp::float64> & adj_t2,
    wp::int32 & adj_ret)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::vec_t<4,wp::float64> var_4;
    wp::vec_t<2,wp::float64> var_5;
    const wp::int32 var_6 = 0;
    wp::float64 var_7;
    const wp::int32 var_8 = 0;
    wp::float64 var_9;
    wp::float64 var_10;
    const wp::int32 var_11 = 1;
    wp::float64 var_12;
    const wp::int32 var_13 = 1;
    wp::float64 var_14;
    wp::float64 var_15;
    wp::float64 var_16;
    const wp::int32 var_17 = 2;
    wp::float64 var_18;
    const wp::int32 var_19 = 2;
    wp::float64 var_20;
    wp::float64 var_21;
    wp::float64 var_22;
    const wp::int32 var_23 = 0;
    const wp::int32 var_24 = 0;
    wp::float64 var_25;
    const wp::int32 var_26 = 0;
    wp::float64 var_27;
    wp::float64 var_28;
    const wp::int32 var_29 = 1;
    wp::float64 var_30;
    const wp::int32 var_31 = 1;
    wp::float64 var_32;
    wp::float64 var_33;
    wp::float64 var_34;
    const wp::int32 var_35 = 2;
    wp::float64 var_36;
    const wp::int32 var_37 = 2;
    wp::float64 var_38;
    wp::float64 var_39;
    wp::float64 var_40;
    const wp::int32 var_41 = 2;
    const wp::int32 var_42 = 2;
    wp::float64 var_43;
    const wp::int32 var_44 = 1;
    const wp::int32 var_45 = 0;
    wp::float64 var_46;
    const wp::int32 var_47 = 0;
    wp::float64 var_48;
    wp::float64 var_49;
    const wp::int32 var_50 = 1;
    wp::float64 var_51;
    const wp::int32 var_52 = 1;
    wp::float64 var_53;
    wp::float64 var_54;
    wp::float64 var_55;
    const wp::int32 var_56 = 2;
    wp::float64 var_57;
    const wp::int32 var_58 = 2;
    wp::float64 var_59;
    wp::float64 var_60;
    wp::float64 var_61;
    const wp::int32 var_62 = 3;
    wp::vec_t<3,wp::float64> var_63;
    wp::float64 var_64;
    const wp::int32 var_65 = 0;
    wp::float64 var_66;
    const wp::int32 var_67 = 1;
    const wp::int32 var_68 = 0;
    wp::float64 var_69;
    const wp::int32 var_70 = 1;
    wp::float64 var_71;
    const wp::int32 var_72 = 3;
    wp::float64 var_73;
    const wp::int32 var_74 = 0;
    wp::float64 var_75;
    const wp::int32 var_76 = 1;
    wp::float64 var_77;
    wp::float64 var_78;
    wp::float64 var_79;
    const wp::float32 var_80 = 0.0;
    wp::float64 var_81;
    bool var_82;
    const wp::float32 var_83 = 1.0;
    wp::float64 var_84;
    bool var_85;
    const wp::float32 var_86 = 0.0;
    wp::float64 var_87;
    bool var_88;
    bool var_89;
    const wp::int32 var_90 = 3;
    wp::vec_t<3,wp::float64> var_91;
    wp::vec_t<3,wp::float64> var_92;
    const wp::int32 var_93 = 0;
    wp::float64 var_94;
    const wp::int32 var_95 = 0;
    wp::float64 var_96;
    wp::float64 var_97;
    const wp::int32 var_98 = 1;
    wp::float64 var_99;
    const wp::int32 var_100 = 1;
    wp::float64 var_101;
    wp::float64 var_102;
    wp::float64 var_103;
    const wp::int32 var_104 = 2;
    wp::float64 var_105;
    const wp::int32 var_106 = 2;
    wp::float64 var_107;
    wp::float64 var_108;
    wp::float64 var_109;
    const wp::int32 var_110 = 0;
    const wp::int32 var_111 = 0;
    wp::float64 var_112;
    const wp::int32 var_113 = 0;
    wp::float64 var_114;
    wp::float64 var_115;
    const wp::int32 var_116 = 1;
    wp::float64 var_117;
    const wp::int32 var_118 = 1;
    wp::float64 var_119;
    wp::float64 var_120;
    wp::float64 var_121;
    const wp::int32 var_122 = 2;
    wp::float64 var_123;
    const wp::int32 var_124 = 2;
    wp::float64 var_125;
    wp::float64 var_126;
    wp::float64 var_127;
    const wp::int32 var_128 = 2;
    const wp::int32 var_129 = 2;
    wp::float64 var_130;
    const wp::int32 var_131 = 1;
    const wp::int32 var_132 = 0;
    wp::float64 var_133;
    const wp::int32 var_134 = 0;
    wp::float64 var_135;
    wp::float64 var_136;
    const wp::int32 var_137 = 1;
    wp::float64 var_138;
    const wp::int32 var_139 = 1;
    wp::float64 var_140;
    wp::float64 var_141;
    wp::float64 var_142;
    const wp::int32 var_143 = 2;
    wp::float64 var_144;
    const wp::int32 var_145 = 2;
    wp::float64 var_146;
    wp::float64 var_147;
    wp::float64 var_148;
    const wp::int32 var_149 = 3;
    wp::vec_t<3,wp::float64> var_150;
    wp::float64 var_151;
    const wp::int32 var_152 = 0;
    wp::float64 var_153;
    const wp::int32 var_154 = 1;
    const wp::int32 var_155 = 0;
    wp::float64 var_156;
    const wp::int32 var_157 = 1;
    wp::float64 var_158;
    const wp::int32 var_159 = 3;
    wp::float64 var_160;
    const wp::int32 var_161 = 0;
    wp::float64 var_162;
    const wp::int32 var_163 = 1;
    wp::float64 var_164;
    wp::float64 var_165;
    wp::float64 var_166;
    const wp::float32 var_167 = 0.0;
    wp::float64 var_168;
    bool var_169;
    const wp::float32 var_170 = 1.0;
    wp::float64 var_171;
    bool var_172;
    const wp::float32 var_173 = 0.0;
    wp::float64 var_174;
    bool var_175;
    bool var_176;
    const wp::int32 var_177 = 4;
    wp::vec_t<3,wp::float64> var_178;
    wp::vec_t<3,wp::float64> var_179;
    const wp::int32 var_180 = 0;
    wp::float64 var_181;
    const wp::int32 var_182 = 0;
    wp::float64 var_183;
    wp::float64 var_184;
    const wp::int32 var_185 = 1;
    wp::float64 var_186;
    const wp::int32 var_187 = 1;
    wp::float64 var_188;
    wp::float64 var_189;
    wp::float64 var_190;
    const wp::int32 var_191 = 2;
    wp::float64 var_192;
    const wp::int32 var_193 = 2;
    wp::float64 var_194;
    wp::float64 var_195;
    wp::float64 var_196;
    const wp::int32 var_197 = 0;
    const wp::int32 var_198 = 0;
    wp::float64 var_199;
    const wp::int32 var_200 = 0;
    wp::float64 var_201;
    wp::float64 var_202;
    const wp::int32 var_203 = 1;
    wp::float64 var_204;
    const wp::int32 var_205 = 1;
    wp::float64 var_206;
    wp::float64 var_207;
    wp::float64 var_208;
    const wp::int32 var_209 = 2;
    wp::float64 var_210;
    const wp::int32 var_211 = 2;
    wp::float64 var_212;
    wp::float64 var_213;
    wp::float64 var_214;
    const wp::int32 var_215 = 2;
    const wp::int32 var_216 = 2;
    wp::float64 var_217;
    const wp::int32 var_218 = 1;
    const wp::int32 var_219 = 0;
    wp::float64 var_220;
    const wp::int32 var_221 = 0;
    wp::float64 var_222;
    wp::float64 var_223;
    const wp::int32 var_224 = 1;
    wp::float64 var_225;
    const wp::int32 var_226 = 1;
    wp::float64 var_227;
    wp::float64 var_228;
    wp::float64 var_229;
    const wp::int32 var_230 = 2;
    wp::float64 var_231;
    const wp::int32 var_232 = 2;
    wp::float64 var_233;
    wp::float64 var_234;
    wp::float64 var_235;
    const wp::int32 var_236 = 3;
    wp::vec_t<3,wp::float64> var_237;
    wp::float64 var_238;
    const wp::int32 var_239 = 0;
    wp::float64 var_240;
    const wp::int32 var_241 = 1;
    const wp::int32 var_242 = 0;
    wp::float64 var_243;
    const wp::int32 var_244 = 1;
    wp::float64 var_245;
    const wp::int32 var_246 = 3;
    wp::float64 var_247;
    const wp::int32 var_248 = 0;
    wp::float64 var_249;
    const wp::int32 var_250 = 1;
    wp::float64 var_251;
    wp::float64 var_252;
    wp::float64 var_253;
    const wp::float32 var_254 = 0.0;
    wp::float64 var_255;
    bool var_256;
    const wp::float32 var_257 = 1.0;
    wp::float64 var_258;
    bool var_259;
    const wp::float32 var_260 = 0.0;
    wp::float64 var_261;
    bool var_262;
    bool var_263;
    const wp::int32 var_264 = 5;
    const wp::float32 var_265 = 0.0;
    wp::float64 var_266;
    bool var_267;
    const wp::float32 var_268 = 1.0;
    wp::float64 var_269;
    bool var_270;
    bool var_271;
    const wp::int32 var_272 = 0;
    const wp::float32 var_273 = 0.0;
    wp::float64 var_274;
    bool var_275;
    const wp::float32 var_276 = 1.0;
    wp::float64 var_277;
    bool var_278;
    bool var_279;
    const wp::int32 var_280 = 1;
    const wp::float32 var_281 = 0.0;
    wp::float64 var_282;
    bool var_283;
    const wp::float32 var_284 = 1.0;
    wp::float64 var_285;
    bool var_286;
    bool var_287;
    const wp::int32 var_288 = 2;
    const wp::int32 var_289 = 6;
    wp::vec_t<3,wp::float64> var_290;
    wp::vec_t<3,wp::float64> var_291;
    wp::vec_t<3,wp::float64> var_292;
    wp::vec_t<3,wp::float64> var_293;
    wp::vec_t<3,wp::float64> var_294;
    wp::vec_t<3,wp::float64> var_295;
    //---------
    // dual vars
    wp::vec_t<3,wp::float64> adj_0 = {};
    wp::vec_t<3,wp::float64> adj_1 = {};
    wp::vec_t<3,wp::float64> adj_2 = {};
    wp::vec_t<3,wp::float64> adj_3 = {};
    wp::vec_t<4,wp::float64> adj_4 = {};
    wp::vec_t<2,wp::float64> adj_5 = {};
    wp::int32 adj_6 = {};
    wp::float64 adj_7 = {};
    wp::int32 adj_8 = {};
    wp::float64 adj_9 = {};
    wp::float64 adj_10 = {};
    wp::int32 adj_11 = {};
    wp::float64 adj_12 = {};
    wp::int32 adj_13 = {};
    wp::float64 adj_14 = {};
    wp::float64 adj_15 = {};
    wp::float64 adj_16 = {};
    wp::int32 adj_17 = {};
    wp::float64 adj_18 = {};
    wp::int32 adj_19 = {};
    wp::float64 adj_20 = {};
    wp::float64 adj_21 = {};
    wp::float64 adj_22 = {};
    wp::int32 adj_23 = {};
    wp::int32 adj_24 = {};
    wp::float64 adj_25 = {};
    wp::int32 adj_26 = {};
    wp::float64 adj_27 = {};
    wp::float64 adj_28 = {};
    wp::int32 adj_29 = {};
    wp::float64 adj_30 = {};
    wp::int32 adj_31 = {};
    wp::float64 adj_32 = {};
    wp::float64 adj_33 = {};
    wp::float64 adj_34 = {};
    wp::int32 adj_35 = {};
    wp::float64 adj_36 = {};
    wp::int32 adj_37 = {};
    wp::float64 adj_38 = {};
    wp::float64 adj_39 = {};
    wp::float64 adj_40 = {};
    wp::int32 adj_41 = {};
    wp::int32 adj_42 = {};
    wp::float64 adj_43 = {};
    wp::int32 adj_44 = {};
    wp::int32 adj_45 = {};
    wp::float64 adj_46 = {};
    wp::int32 adj_47 = {};
    wp::float64 adj_48 = {};
    wp::float64 adj_49 = {};
    wp::int32 adj_50 = {};
    wp::float64 adj_51 = {};
    wp::int32 adj_52 = {};
    wp::float64 adj_53 = {};
    wp::float64 adj_54 = {};
    wp::float64 adj_55 = {};
    wp::int32 adj_56 = {};
    wp::float64 adj_57 = {};
    wp::int32 adj_58 = {};
    wp::float64 adj_59 = {};
    wp::float64 adj_60 = {};
    wp::float64 adj_61 = {};
    wp::int32 adj_62 = {};
    wp::vec_t<3,wp::float64> adj_63 = {};
    wp::float64 adj_64 = {};
    wp::int32 adj_65 = {};
    wp::float64 adj_66 = {};
    wp::int32 adj_67 = {};
    wp::int32 adj_68 = {};
    wp::float64 adj_69 = {};
    wp::int32 adj_70 = {};
    wp::float64 adj_71 = {};
    wp::int32 adj_72 = {};
    wp::float64 adj_73 = {};
    wp::int32 adj_74 = {};
    wp::float64 adj_75 = {};
    wp::int32 adj_76 = {};
    wp::float64 adj_77 = {};
    wp::float64 adj_78 = {};
    wp::float64 adj_79 = {};
    wp::float32 adj_80 = {};
    wp::float64 adj_81 = {};
    bool adj_82 = {};
    wp::float32 adj_83 = {};
    wp::float64 adj_84 = {};
    bool adj_85 = {};
    wp::float32 adj_86 = {};
    wp::float64 adj_87 = {};
    bool adj_88 = {};
    bool adj_89 = {};
    wp::int32 adj_90 = {};
    wp::vec_t<3,wp::float64> adj_91 = {};
    wp::vec_t<3,wp::float64> adj_92 = {};
    wp::int32 adj_93 = {};
    wp::float64 adj_94 = {};
    wp::int32 adj_95 = {};
    wp::float64 adj_96 = {};
    wp::float64 adj_97 = {};
    wp::int32 adj_98 = {};
    wp::float64 adj_99 = {};
    wp::int32 adj_100 = {};
    wp::float64 adj_101 = {};
    wp::float64 adj_102 = {};
    wp::float64 adj_103 = {};
    wp::int32 adj_104 = {};
    wp::float64 adj_105 = {};
    wp::int32 adj_106 = {};
    wp::float64 adj_107 = {};
    wp::float64 adj_108 = {};
    wp::float64 adj_109 = {};
    wp::int32 adj_110 = {};
    wp::int32 adj_111 = {};
    wp::float64 adj_112 = {};
    wp::int32 adj_113 = {};
    wp::float64 adj_114 = {};
    wp::float64 adj_115 = {};
    wp::int32 adj_116 = {};
    wp::float64 adj_117 = {};
    wp::int32 adj_118 = {};
    wp::float64 adj_119 = {};
    wp::float64 adj_120 = {};
    wp::float64 adj_121 = {};
    wp::int32 adj_122 = {};
    wp::float64 adj_123 = {};
    wp::int32 adj_124 = {};
    wp::float64 adj_125 = {};
    wp::float64 adj_126 = {};
    wp::float64 adj_127 = {};
    wp::int32 adj_128 = {};
    wp::int32 adj_129 = {};
    wp::float64 adj_130 = {};
    wp::int32 adj_131 = {};
    wp::int32 adj_132 = {};
    wp::float64 adj_133 = {};
    wp::int32 adj_134 = {};
    wp::float64 adj_135 = {};
    wp::float64 adj_136 = {};
    wp::int32 adj_137 = {};
    wp::float64 adj_138 = {};
    wp::int32 adj_139 = {};
    wp::float64 adj_140 = {};
    wp::float64 adj_141 = {};
    wp::float64 adj_142 = {};
    wp::int32 adj_143 = {};
    wp::float64 adj_144 = {};
    wp::int32 adj_145 = {};
    wp::float64 adj_146 = {};
    wp::float64 adj_147 = {};
    wp::float64 adj_148 = {};
    wp::int32 adj_149 = {};
    wp::vec_t<3,wp::float64> adj_150 = {};
    wp::float64 adj_151 = {};
    wp::int32 adj_152 = {};
    wp::float64 adj_153 = {};
    wp::int32 adj_154 = {};
    wp::int32 adj_155 = {};
    wp::float64 adj_156 = {};
    wp::int32 adj_157 = {};
    wp::float64 adj_158 = {};
    wp::int32 adj_159 = {};
    wp::float64 adj_160 = {};
    wp::int32 adj_161 = {};
    wp::float64 adj_162 = {};
    wp::int32 adj_163 = {};
    wp::float64 adj_164 = {};
    wp::float64 adj_165 = {};
    wp::float64 adj_166 = {};
    wp::float32 adj_167 = {};
    wp::float64 adj_168 = {};
    bool adj_169 = {};
    wp::float32 adj_170 = {};
    wp::float64 adj_171 = {};
    bool adj_172 = {};
    wp::float32 adj_173 = {};
    wp::float64 adj_174 = {};
    bool adj_175 = {};
    bool adj_176 = {};
    wp::int32 adj_177 = {};
    wp::vec_t<3,wp::float64> adj_178 = {};
    wp::vec_t<3,wp::float64> adj_179 = {};
    wp::int32 adj_180 = {};
    wp::float64 adj_181 = {};
    wp::int32 adj_182 = {};
    wp::float64 adj_183 = {};
    wp::float64 adj_184 = {};
    wp::int32 adj_185 = {};
    wp::float64 adj_186 = {};
    wp::int32 adj_187 = {};
    wp::float64 adj_188 = {};
    wp::float64 adj_189 = {};
    wp::float64 adj_190 = {};
    wp::int32 adj_191 = {};
    wp::float64 adj_192 = {};
    wp::int32 adj_193 = {};
    wp::float64 adj_194 = {};
    wp::float64 adj_195 = {};
    wp::float64 adj_196 = {};
    wp::int32 adj_197 = {};
    wp::int32 adj_198 = {};
    wp::float64 adj_199 = {};
    wp::int32 adj_200 = {};
    wp::float64 adj_201 = {};
    wp::float64 adj_202 = {};
    wp::int32 adj_203 = {};
    wp::float64 adj_204 = {};
    wp::int32 adj_205 = {};
    wp::float64 adj_206 = {};
    wp::float64 adj_207 = {};
    wp::float64 adj_208 = {};
    wp::int32 adj_209 = {};
    wp::float64 adj_210 = {};
    wp::int32 adj_211 = {};
    wp::float64 adj_212 = {};
    wp::float64 adj_213 = {};
    wp::float64 adj_214 = {};
    wp::int32 adj_215 = {};
    wp::int32 adj_216 = {};
    wp::float64 adj_217 = {};
    wp::int32 adj_218 = {};
    wp::int32 adj_219 = {};
    wp::float64 adj_220 = {};
    wp::int32 adj_221 = {};
    wp::float64 adj_222 = {};
    wp::float64 adj_223 = {};
    wp::int32 adj_224 = {};
    wp::float64 adj_225 = {};
    wp::int32 adj_226 = {};
    wp::float64 adj_227 = {};
    wp::float64 adj_228 = {};
    wp::float64 adj_229 = {};
    wp::int32 adj_230 = {};
    wp::float64 adj_231 = {};
    wp::int32 adj_232 = {};
    wp::float64 adj_233 = {};
    wp::float64 adj_234 = {};
    wp::float64 adj_235 = {};
    wp::int32 adj_236 = {};
    wp::vec_t<3,wp::float64> adj_237 = {};
    wp::float64 adj_238 = {};
    wp::int32 adj_239 = {};
    wp::float64 adj_240 = {};
    wp::int32 adj_241 = {};
    wp::int32 adj_242 = {};
    wp::float64 adj_243 = {};
    wp::int32 adj_244 = {};
    wp::float64 adj_245 = {};
    wp::int32 adj_246 = {};
    wp::float64 adj_247 = {};
    wp::int32 adj_248 = {};
    wp::float64 adj_249 = {};
    wp::int32 adj_250 = {};
    wp::float64 adj_251 = {};
    wp::float64 adj_252 = {};
    wp::float64 adj_253 = {};
    wp::float32 adj_254 = {};
    wp::float64 adj_255 = {};
    bool adj_256 = {};
    wp::float32 adj_257 = {};
    wp::float64 adj_258 = {};
    bool adj_259 = {};
    wp::float32 adj_260 = {};
    wp::float64 adj_261 = {};
    bool adj_262 = {};
    bool adj_263 = {};
    wp::int32 adj_264 = {};
    wp::float32 adj_265 = {};
    wp::float64 adj_266 = {};
    bool adj_267 = {};
    wp::float32 adj_268 = {};
    wp::float64 adj_269 = {};
    bool adj_270 = {};
    bool adj_271 = {};
    wp::int32 adj_272 = {};
    wp::float32 adj_273 = {};
    wp::float64 adj_274 = {};
    bool adj_275 = {};
    wp::float32 adj_276 = {};
    wp::float64 adj_277 = {};
    bool adj_278 = {};
    bool adj_279 = {};
    wp::int32 adj_280 = {};
    wp::float32 adj_281 = {};
    wp::float64 adj_282 = {};
    bool adj_283 = {};
    wp::float32 adj_284 = {};
    wp::float64 adj_285 = {};
    bool adj_286 = {};
    bool adj_287 = {};
    wp::int32 adj_288 = {};
    wp::int32 adj_289 = {};
    wp::vec_t<3,wp::float64> adj_290 = {};
    wp::vec_t<3,wp::float64> adj_291 = {};
    wp::vec_t<3,wp::float64> adj_292 = {};
    wp::vec_t<3,wp::float64> adj_293 = {};
    wp::vec_t<3,wp::float64> adj_294 = {};
    wp::vec_t<3,wp::float64> adj_295 = {};
    //---------
    // forward
    // def point_triangle_distance_type(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d) -> int:       <L 19>
    // basis_row0 = t1 - t0                                                                   <L 20>
    var_0 = wp::sub(var_t1, var_t0);
    // basis_row1 = t2 - t0                                                                   <L 21>
    var_1 = wp::sub(var_t2, var_t0);
    // nVec = wp.cross(basis_row0, basis_row1)                                                <L 22>
    var_2 = wp::cross(var_0, var_1);
    // basis_row1 = wp.cross(basis_row0, nVec)                                                <L 23>
    var_3 = wp::cross(var_0, var_2);
    // sys = wp.vec4d()                                                                       <L 25>
    var_4 = wp::vec_t<4,wp::float64>();
    // rhs = wp.vec2d()                                                                       <L 26>
    var_5 = wp::vec_t<2,wp::float64>();
    // sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]       <L 27>
    var_7 = wp::extract(var_0, var_6);
    var_9 = wp::extract(var_0, var_8);
    var_10 = wp::mul(var_7, var_9);
    var_12 = wp::extract(var_0, var_11);
    var_14 = wp::extract(var_0, var_13);
    var_15 = wp::mul(var_12, var_14);
    var_16 = wp::add(var_10, var_15);
    var_18 = wp::extract(var_0, var_17);
    var_20 = wp::extract(var_0, var_19);
    var_21 = wp::mul(var_18, var_20);
    var_22 = wp::add(var_16, var_21);
    wp::assign_inplace(var_4, var_23, var_22);
    // sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]       <L 28>
    var_25 = wp::extract(var_0, var_24);
    var_27 = wp::extract(var_3, var_26);
    var_28 = wp::mul(var_25, var_27);
    var_30 = wp::extract(var_0, var_29);
    var_32 = wp::extract(var_3, var_31);
    var_33 = wp::mul(var_30, var_32);
    var_34 = wp::add(var_28, var_33);
    var_36 = wp::extract(var_0, var_35);
    var_38 = wp::extract(var_3, var_37);
    var_39 = wp::mul(var_36, var_38);
    var_40 = wp::add(var_34, var_39);
    wp::assign_inplace(var_4, var_41, var_40);
    // sys[1] = sys[2]                                                                        <L 29>
    var_43 = wp::extract(var_4, var_42);
    wp::assign_inplace(var_4, var_44, var_43);
    // sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]       <L 30>
    var_46 = wp::extract(var_3, var_45);
    var_48 = wp::extract(var_3, var_47);
    var_49 = wp::mul(var_46, var_48);
    var_51 = wp::extract(var_3, var_50);
    var_53 = wp::extract(var_3, var_52);
    var_54 = wp::mul(var_51, var_53);
    var_55 = wp::add(var_49, var_54);
    var_57 = wp::extract(var_3, var_56);
    var_59 = wp::extract(var_3, var_58);
    var_60 = wp::mul(var_57, var_59);
    var_61 = wp::add(var_55, var_60);
    wp::assign_inplace(var_4, var_62, var_61);
    // b = p - t0                                                                             <L 32>
    var_63 = wp::sub(var_p, var_t0);
    // rhs[0] = wp.dot(basis_row0, b)                                                         <L 33>
    var_64 = wp::dot(var_0, var_63);
    wp::assign_inplace(var_5, var_65, var_64);
    // rhs[1] = wp.dot(basis_row1, b)                                                         <L 34>
    var_66 = wp::dot(var_3, var_63);
    wp::assign_inplace(var_5, var_67, var_66);
    // param_col00, param_col01 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])         <L 35>
    var_69 = wp::extract(var_4, var_68);
    var_71 = wp::extract(var_4, var_70);
    var_73 = wp::extract(var_4, var_72);
    var_75 = wp::extract(var_5, var_74);
    var_77 = wp::extract(var_5, var_76);
    solveLdlt2D_0(var_69, var_71, var_73, var_75, var_77, var_78, var_79);
    // if param_col00 > wp.float64(0.0) and param_col00 < wp.float64(1.0) and param_col01 >= wp.float64(0.0):       <L 36>
    var_81 = wp::float64(var_80);
    var_82 = (var_78 > var_81);
    var_84 = wp::float64(var_83);
    var_85 = (var_78 < var_84);
    var_87 = wp::float64(var_86);
    var_88 = (var_79 >= var_87);
    var_89 = var_82 && var_85 && var_88;
    if (var_89) {
        // return 3  # PE t0t1                                                                <L 37>
        goto label0;
    }
    if (!var_89) {
        // basis_row0 = t2 - t1                                                               <L 39>
        var_91 = wp::sub(var_t2, var_t1);
        // basis_row1 = wp.cross(basis_row0, nVec)                                            <L 40>
        var_92 = wp::cross(var_91, var_2);
        // sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]       <L 42>
        var_94 = wp::extract(var_91, var_93);
        var_96 = wp::extract(var_91, var_95);
        var_97 = wp::mul(var_94, var_96);
        var_99 = wp::extract(var_91, var_98);
        var_101 = wp::extract(var_91, var_100);
        var_102 = wp::mul(var_99, var_101);
        var_103 = wp::add(var_97, var_102);
        var_105 = wp::extract(var_91, var_104);
        var_107 = wp::extract(var_91, var_106);
        var_108 = wp::mul(var_105, var_107);
        var_109 = wp::add(var_103, var_108);
        wp::assign_inplace(var_4, var_110, var_109);
        // sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]       <L 43>
        var_112 = wp::extract(var_91, var_111);
        var_114 = wp::extract(var_92, var_113);
        var_115 = wp::mul(var_112, var_114);
        var_117 = wp::extract(var_91, var_116);
        var_119 = wp::extract(var_92, var_118);
        var_120 = wp::mul(var_117, var_119);
        var_121 = wp::add(var_115, var_120);
        var_123 = wp::extract(var_91, var_122);
        var_125 = wp::extract(var_92, var_124);
        var_126 = wp::mul(var_123, var_125);
        var_127 = wp::add(var_121, var_126);
        wp::assign_inplace(var_4, var_128, var_127);
        // sys[1] = sys[2]                                                                    <L 44>
        var_130 = wp::extract(var_4, var_129);
        wp::assign_inplace(var_4, var_131, var_130);
        // sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]       <L 45>
        var_133 = wp::extract(var_92, var_132);
        var_135 = wp::extract(var_92, var_134);
        var_136 = wp::mul(var_133, var_135);
        var_138 = wp::extract(var_92, var_137);
        var_140 = wp::extract(var_92, var_139);
        var_141 = wp::mul(var_138, var_140);
        var_142 = wp::add(var_136, var_141);
        var_144 = wp::extract(var_92, var_143);
        var_146 = wp::extract(var_92, var_145);
        var_147 = wp::mul(var_144, var_146);
        var_148 = wp::add(var_142, var_147);
        wp::assign_inplace(var_4, var_149, var_148);
        // b = p - t1                                                                         <L 46>
        var_150 = wp::sub(var_p, var_t1);
        // rhs[0] = wp.dot(basis_row0, b)                                                     <L 47>
        var_151 = wp::dot(var_91, var_150);
        wp::assign_inplace(var_5, var_152, var_151);
        // rhs[1] = wp.dot(basis_row1, b)                                                     <L 48>
        var_153 = wp::dot(var_92, var_150);
        wp::assign_inplace(var_5, var_154, var_153);
        // param_col10, param_col11 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])       <L 49>
        var_156 = wp::extract(var_4, var_155);
        var_158 = wp::extract(var_4, var_157);
        var_160 = wp::extract(var_4, var_159);
        var_162 = wp::extract(var_5, var_161);
        var_164 = wp::extract(var_5, var_163);
        solveLdlt2D_0(var_156, var_158, var_160, var_162, var_164, var_165, var_166);
        // if param_col10 > wp.float64(0.0) and param_col10 < wp.float64(1.0) and param_col11 >= wp.float64(0.0):       <L 50>
        var_168 = wp::float64(var_167);
        var_169 = (var_165 > var_168);
        var_171 = wp::float64(var_170);
        var_172 = (var_165 < var_171);
        var_174 = wp::float64(var_173);
        var_175 = (var_166 >= var_174);
        var_176 = var_169 && var_172 && var_175;
        if (var_176) {
            // return 4  # PE t1t2                                                            <L 51>
            goto label1;
        }
        if (!var_176) {
            // basis_row0 = t0 - t2                                                           <L 53>
            var_178 = wp::sub(var_t0, var_t2);
            // basis_row1 = wp.cross(basis_row0, nVec)                                        <L 54>
            var_179 = wp::cross(var_178, var_2);
            // sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]       <L 55>
            var_181 = wp::extract(var_178, var_180);
            var_183 = wp::extract(var_178, var_182);
            var_184 = wp::mul(var_181, var_183);
            var_186 = wp::extract(var_178, var_185);
            var_188 = wp::extract(var_178, var_187);
            var_189 = wp::mul(var_186, var_188);
            var_190 = wp::add(var_184, var_189);
            var_192 = wp::extract(var_178, var_191);
            var_194 = wp::extract(var_178, var_193);
            var_195 = wp::mul(var_192, var_194);
            var_196 = wp::add(var_190, var_195);
            wp::assign_inplace(var_4, var_197, var_196);
            // sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]       <L 56>
            var_199 = wp::extract(var_178, var_198);
            var_201 = wp::extract(var_179, var_200);
            var_202 = wp::mul(var_199, var_201);
            var_204 = wp::extract(var_178, var_203);
            var_206 = wp::extract(var_179, var_205);
            var_207 = wp::mul(var_204, var_206);
            var_208 = wp::add(var_202, var_207);
            var_210 = wp::extract(var_178, var_209);
            var_212 = wp::extract(var_179, var_211);
            var_213 = wp::mul(var_210, var_212);
            var_214 = wp::add(var_208, var_213);
            wp::assign_inplace(var_4, var_215, var_214);
            // sys[1] = sys[2]                                                                <L 57>
            var_217 = wp::extract(var_4, var_216);
            wp::assign_inplace(var_4, var_218, var_217);
            // sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]       <L 58>
            var_220 = wp::extract(var_179, var_219);
            var_222 = wp::extract(var_179, var_221);
            var_223 = wp::mul(var_220, var_222);
            var_225 = wp::extract(var_179, var_224);
            var_227 = wp::extract(var_179, var_226);
            var_228 = wp::mul(var_225, var_227);
            var_229 = wp::add(var_223, var_228);
            var_231 = wp::extract(var_179, var_230);
            var_233 = wp::extract(var_179, var_232);
            var_234 = wp::mul(var_231, var_233);
            var_235 = wp::add(var_229, var_234);
            wp::assign_inplace(var_4, var_236, var_235);
            // b = p - t2                                                                     <L 59>
            var_237 = wp::sub(var_p, var_t2);
            // rhs[0] = wp.dot(basis_row0, b)                                                 <L 60>
            var_238 = wp::dot(var_178, var_237);
            wp::assign_inplace(var_5, var_239, var_238);
            // rhs[1] = wp.dot(basis_row1, b)                                                 <L 61>
            var_240 = wp::dot(var_179, var_237);
            wp::assign_inplace(var_5, var_241, var_240);
            // param_col20, param_col21 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])       <L 62>
            var_243 = wp::extract(var_4, var_242);
            var_245 = wp::extract(var_4, var_244);
            var_247 = wp::extract(var_4, var_246);
            var_249 = wp::extract(var_5, var_248);
            var_251 = wp::extract(var_5, var_250);
            solveLdlt2D_0(var_243, var_245, var_247, var_249, var_251, var_252, var_253);
            // if param_col20 > wp.float64(0.0) and param_col20 < wp.float64(1.0) and param_col21 >= wp.float64(0.0):       <L 64>
            var_255 = wp::float64(var_254);
            var_256 = (var_252 > var_255);
            var_258 = wp::float64(var_257);
            var_259 = (var_252 < var_258);
            var_261 = wp::float64(var_260);
            var_262 = (var_253 >= var_261);
            var_263 = var_256 && var_259 && var_262;
            if (var_263) {
                // return 5  # PE t2t0                                                        <L 65>
                goto label2;
            }
            if (!var_263) {
                // if param_col00 <= wp.float64(0.0) and param_col20 >= wp.float64(1.0):       <L 67>
                var_266 = wp::float64(var_265);
                var_267 = (var_78 <= var_266);
                var_269 = wp::float64(var_268);
                var_270 = (var_252 >= var_269);
                var_271 = var_267 && var_270;
                if (var_271) {
                    // return 0  # PP t0                                                      <L 68>
                    goto label3;
                }
                if (!var_271) {
                    // elif param_col10 <= wp.float64(0.0) and param_col00 >= wp.float64(1.0):       <L 69>
                    var_274 = wp::float64(var_273);
                    var_275 = (var_165 <= var_274);
                    var_277 = wp::float64(var_276);
                    var_278 = (var_78 >= var_277);
                    var_279 = var_275 && var_278;
                    if (var_279) {
                        // return 1  # PP t1                                                  <L 70>
                        goto label4;
                    }
                    if (!var_279) {
                        // elif param_col20 <= wp.float64(0.0) and param_col10 >= wp.float64(1.0):       <L 71>
                        var_282 = wp::float64(var_281);
                        var_283 = (var_252 <= var_282);
                        var_285 = wp::float64(var_284);
                        var_286 = (var_165 >= var_285);
                        var_287 = var_283 && var_286;
                        if (var_287) {
                            // return 2  # PP t2                                              <L 72>
                            goto label5;
                        }
                        if (!var_287) {
                            // return 6  # PT                                                 <L 74>
                            goto label6;
                        }
                    }
                }
            }
        }
        var_290 = wp::where(var_176, var_91, var_178);
        var_291 = wp::where(var_176, var_92, var_179);
        var_292 = wp::where(var_176, var_150, var_237);
    }
    var_293 = wp::where(var_89, var_0, var_290);
    var_294 = wp::where(var_89, var_3, var_291);
    var_295 = wp::where(var_89, var_63, var_292);
    //---------
    // reverse
    wp::adj_where(var_89, var_63, var_292, adj_89, adj_63, adj_292, adj_295);
    wp::adj_where(var_89, var_3, var_291, adj_89, adj_3, adj_291, adj_294);
    wp::adj_where(var_89, var_0, var_290, adj_89, adj_0, adj_290, adj_293);
    if (!var_89) {
        wp::adj_where(var_176, var_150, var_237, adj_176, adj_150, adj_237, adj_292);
        wp::adj_where(var_176, var_92, var_179, adj_176, adj_92, adj_179, adj_291);
        wp::adj_where(var_176, var_91, var_178, adj_176, adj_91, adj_178, adj_290);
        if (!var_176) {
            if (!var_263) {
                if (!var_271) {
                    if (!var_279) {
                        if (!var_287) {
                            label6:;
                            adj_289 += adj_ret;
                            // adj: return 6  # PT                                            <L 74>
                        }
                        if (var_287) {
                            label5:;
                            adj_288 += adj_ret;
                            // adj: return 2  # PP t2                                         <L 72>
                        }
                        wp::adj_float64(var_284, adj_284, adj_285);
                        wp::adj_float64(var_281, adj_281, adj_282);
                        // adj: elif param_col20 <= wp.float64(0.0) and param_col10 >= wp.float64(1.0):  <L 71>
                    }
                    if (var_279) {
                        label4:;
                        adj_280 += adj_ret;
                        // adj: return 1  # PP t1                                             <L 70>
                    }
                    wp::adj_float64(var_276, adj_276, adj_277);
                    wp::adj_float64(var_273, adj_273, adj_274);
                    // adj: elif param_col10 <= wp.float64(0.0) and param_col00 >= wp.float64(1.0):  <L 69>
                }
                if (var_271) {
                    label3:;
                    adj_272 += adj_ret;
                    // adj: return 0  # PP t0                                                 <L 68>
                }
                wp::adj_float64(var_268, adj_268, adj_269);
                wp::adj_float64(var_265, adj_265, adj_266);
                // adj: if param_col00 <= wp.float64(0.0) and param_col20 >= wp.float64(1.0):  <L 67>
            }
            if (var_263) {
                label2:;
                adj_264 += adj_ret;
                // adj: return 5  # PE t2t0                                                   <L 65>
            }
            wp::adj_float64(var_260, adj_260, adj_261);
            wp::adj_float64(var_257, adj_257, adj_258);
            wp::adj_float64(var_254, adj_254, adj_255);
            // adj: if param_col20 > wp.float64(0.0) and param_col20 < wp.float64(1.0) and param_col21 >= wp.float64(0.0):  <L 64>
            adj_solveLdlt2D_0(var_243, var_245, var_247, var_249, var_251, var_252, var_253, adj_243, adj_245, adj_247, adj_249, adj_251, adj_252, adj_253);
            wp::adj_extract(var_5, var_250, adj_5, adj_250, adj_251);
            wp::adj_extract(var_5, var_248, adj_5, adj_248, adj_249);
            wp::adj_extract(var_4, var_246, adj_4, adj_246, adj_247);
            wp::adj_extract(var_4, var_244, adj_4, adj_244, adj_245);
            wp::adj_extract(var_4, var_242, adj_4, adj_242, adj_243);
            // adj: param_col20, param_col21 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])  <L 62>
            wp::adj_assign_inplace(var_5, var_241, var_240, adj_5, adj_241, adj_240);
            wp::adj_dot(var_179, var_237, adj_179, adj_237, adj_240);
            // adj: rhs[1] = wp.dot(basis_row1, b)                                            <L 61>
            wp::adj_assign_inplace(var_5, var_239, var_238, adj_5, adj_239, adj_238);
            wp::adj_dot(var_178, var_237, adj_178, adj_237, adj_238);
            // adj: rhs[0] = wp.dot(basis_row0, b)                                            <L 60>
            wp::adj_sub(var_p, var_t2, adj_p, adj_t2, adj_237);
            // adj: b = p - t2                                                                <L 59>
            wp::adj_assign_inplace(var_4, var_236, var_235, adj_4, adj_236, adj_235);
            wp::adj_add(var_229, var_234, adj_229, adj_234, adj_235);
            wp::adj_mul(var_231, var_233, adj_231, adj_233, adj_234);
            wp::adj_extract(var_179, var_232, adj_179, adj_232, adj_233);
            wp::adj_extract(var_179, var_230, adj_179, adj_230, adj_231);
            wp::adj_add(var_223, var_228, adj_223, adj_228, adj_229);
            wp::adj_mul(var_225, var_227, adj_225, adj_227, adj_228);
            wp::adj_extract(var_179, var_226, adj_179, adj_226, adj_227);
            wp::adj_extract(var_179, var_224, adj_179, adj_224, adj_225);
            wp::adj_mul(var_220, var_222, adj_220, adj_222, adj_223);
            wp::adj_extract(var_179, var_221, adj_179, adj_221, adj_222);
            wp::adj_extract(var_179, var_219, adj_179, adj_219, adj_220);
            // adj: sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]  <L 58>
            wp::adj_assign_inplace(var_4, var_218, var_217, adj_4, adj_218, adj_217);
            wp::adj_extract(var_4, var_216, adj_4, adj_216, adj_217);
            // adj: sys[1] = sys[2]                                                           <L 57>
            wp::adj_assign_inplace(var_4, var_215, var_214, adj_4, adj_215, adj_214);
            wp::adj_add(var_208, var_213, adj_208, adj_213, adj_214);
            wp::adj_mul(var_210, var_212, adj_210, adj_212, adj_213);
            wp::adj_extract(var_179, var_211, adj_179, adj_211, adj_212);
            wp::adj_extract(var_178, var_209, adj_178, adj_209, adj_210);
            wp::adj_add(var_202, var_207, adj_202, adj_207, adj_208);
            wp::adj_mul(var_204, var_206, adj_204, adj_206, adj_207);
            wp::adj_extract(var_179, var_205, adj_179, adj_205, adj_206);
            wp::adj_extract(var_178, var_203, adj_178, adj_203, adj_204);
            wp::adj_mul(var_199, var_201, adj_199, adj_201, adj_202);
            wp::adj_extract(var_179, var_200, adj_179, adj_200, adj_201);
            wp::adj_extract(var_178, var_198, adj_178, adj_198, adj_199);
            // adj: sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]  <L 56>
            wp::adj_assign_inplace(var_4, var_197, var_196, adj_4, adj_197, adj_196);
            wp::adj_add(var_190, var_195, adj_190, adj_195, adj_196);
            wp::adj_mul(var_192, var_194, adj_192, adj_194, adj_195);
            wp::adj_extract(var_178, var_193, adj_178, adj_193, adj_194);
            wp::adj_extract(var_178, var_191, adj_178, adj_191, adj_192);
            wp::adj_add(var_184, var_189, adj_184, adj_189, adj_190);
            wp::adj_mul(var_186, var_188, adj_186, adj_188, adj_189);
            wp::adj_extract(var_178, var_187, adj_178, adj_187, adj_188);
            wp::adj_extract(var_178, var_185, adj_178, adj_185, adj_186);
            wp::adj_mul(var_181, var_183, adj_181, adj_183, adj_184);
            wp::adj_extract(var_178, var_182, adj_178, adj_182, adj_183);
            wp::adj_extract(var_178, var_180, adj_178, adj_180, adj_181);
            // adj: sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]  <L 55>
            wp::adj_cross(var_178, var_2, adj_178, adj_2, adj_179);
            // adj: basis_row1 = wp.cross(basis_row0, nVec)                                   <L 54>
            wp::adj_sub(var_t0, var_t2, adj_t0, adj_t2, adj_178);
            // adj: basis_row0 = t0 - t2                                                      <L 53>
        }
        if (var_176) {
            label1:;
            adj_177 += adj_ret;
            // adj: return 4  # PE t1t2                                                       <L 51>
        }
        wp::adj_float64(var_173, adj_173, adj_174);
        wp::adj_float64(var_170, adj_170, adj_171);
        wp::adj_float64(var_167, adj_167, adj_168);
        // adj: if param_col10 > wp.float64(0.0) and param_col10 < wp.float64(1.0) and param_col11 >= wp.float64(0.0):  <L 50>
        adj_solveLdlt2D_0(var_156, var_158, var_160, var_162, var_164, var_165, var_166, adj_156, adj_158, adj_160, adj_162, adj_164, adj_165, adj_166);
        wp::adj_extract(var_5, var_163, adj_5, adj_163, adj_164);
        wp::adj_extract(var_5, var_161, adj_5, adj_161, adj_162);
        wp::adj_extract(var_4, var_159, adj_4, adj_159, adj_160);
        wp::adj_extract(var_4, var_157, adj_4, adj_157, adj_158);
        wp::adj_extract(var_4, var_155, adj_4, adj_155, adj_156);
        // adj: param_col10, param_col11 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])  <L 49>
        wp::adj_assign_inplace(var_5, var_154, var_153, adj_5, adj_154, adj_153);
        wp::adj_dot(var_92, var_150, adj_92, adj_150, adj_153);
        // adj: rhs[1] = wp.dot(basis_row1, b)                                                <L 48>
        wp::adj_assign_inplace(var_5, var_152, var_151, adj_5, adj_152, adj_151);
        wp::adj_dot(var_91, var_150, adj_91, adj_150, adj_151);
        // adj: rhs[0] = wp.dot(basis_row0, b)                                                <L 47>
        wp::adj_sub(var_p, var_t1, adj_p, adj_t1, adj_150);
        // adj: b = p - t1                                                                    <L 46>
        wp::adj_assign_inplace(var_4, var_149, var_148, adj_4, adj_149, adj_148);
        wp::adj_add(var_142, var_147, adj_142, adj_147, adj_148);
        wp::adj_mul(var_144, var_146, adj_144, adj_146, adj_147);
        wp::adj_extract(var_92, var_145, adj_92, adj_145, adj_146);
        wp::adj_extract(var_92, var_143, adj_92, adj_143, adj_144);
        wp::adj_add(var_136, var_141, adj_136, adj_141, adj_142);
        wp::adj_mul(var_138, var_140, adj_138, adj_140, adj_141);
        wp::adj_extract(var_92, var_139, adj_92, adj_139, adj_140);
        wp::adj_extract(var_92, var_137, adj_92, adj_137, adj_138);
        wp::adj_mul(var_133, var_135, adj_133, adj_135, adj_136);
        wp::adj_extract(var_92, var_134, adj_92, adj_134, adj_135);
        wp::adj_extract(var_92, var_132, adj_92, adj_132, adj_133);
        // adj: sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]  <L 45>
        wp::adj_assign_inplace(var_4, var_131, var_130, adj_4, adj_131, adj_130);
        wp::adj_extract(var_4, var_129, adj_4, adj_129, adj_130);
        // adj: sys[1] = sys[2]                                                               <L 44>
        wp::adj_assign_inplace(var_4, var_128, var_127, adj_4, adj_128, adj_127);
        wp::adj_add(var_121, var_126, adj_121, adj_126, adj_127);
        wp::adj_mul(var_123, var_125, adj_123, adj_125, adj_126);
        wp::adj_extract(var_92, var_124, adj_92, adj_124, adj_125);
        wp::adj_extract(var_91, var_122, adj_91, adj_122, adj_123);
        wp::adj_add(var_115, var_120, adj_115, adj_120, adj_121);
        wp::adj_mul(var_117, var_119, adj_117, adj_119, adj_120);
        wp::adj_extract(var_92, var_118, adj_92, adj_118, adj_119);
        wp::adj_extract(var_91, var_116, adj_91, adj_116, adj_117);
        wp::adj_mul(var_112, var_114, adj_112, adj_114, adj_115);
        wp::adj_extract(var_92, var_113, adj_92, adj_113, adj_114);
        wp::adj_extract(var_91, var_111, adj_91, adj_111, adj_112);
        // adj: sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]  <L 43>
        wp::adj_assign_inplace(var_4, var_110, var_109, adj_4, adj_110, adj_109);
        wp::adj_add(var_103, var_108, adj_103, adj_108, adj_109);
        wp::adj_mul(var_105, var_107, adj_105, adj_107, adj_108);
        wp::adj_extract(var_91, var_106, adj_91, adj_106, adj_107);
        wp::adj_extract(var_91, var_104, adj_91, adj_104, adj_105);
        wp::adj_add(var_97, var_102, adj_97, adj_102, adj_103);
        wp::adj_mul(var_99, var_101, adj_99, adj_101, adj_102);
        wp::adj_extract(var_91, var_100, adj_91, adj_100, adj_101);
        wp::adj_extract(var_91, var_98, adj_91, adj_98, adj_99);
        wp::adj_mul(var_94, var_96, adj_94, adj_96, adj_97);
        wp::adj_extract(var_91, var_95, adj_91, adj_95, adj_96);
        wp::adj_extract(var_91, var_93, adj_91, adj_93, adj_94);
        // adj: sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]  <L 42>
        wp::adj_cross(var_91, var_2, adj_91, adj_2, adj_92);
        // adj: basis_row1 = wp.cross(basis_row0, nVec)                                       <L 40>
        wp::adj_sub(var_t2, var_t1, adj_t2, adj_t1, adj_91);
        // adj: basis_row0 = t2 - t1                                                          <L 39>
    }
    if (var_89) {
        label0:;
        adj_90 += adj_ret;
        // adj: return 3  # PE t0t1                                                           <L 37>
    }
    wp::adj_float64(var_86, adj_86, adj_87);
    wp::adj_float64(var_83, adj_83, adj_84);
    wp::adj_float64(var_80, adj_80, adj_81);
    // adj: if param_col00 > wp.float64(0.0) and param_col00 < wp.float64(1.0) and param_col01 >= wp.float64(0.0):  <L 36>
    adj_solveLdlt2D_0(var_69, var_71, var_73, var_75, var_77, var_78, var_79, adj_69, adj_71, adj_73, adj_75, adj_77, adj_78, adj_79);
    wp::adj_extract(var_5, var_76, adj_5, adj_76, adj_77);
    wp::adj_extract(var_5, var_74, adj_5, adj_74, adj_75);
    wp::adj_extract(var_4, var_72, adj_4, adj_72, adj_73);
    wp::adj_extract(var_4, var_70, adj_4, adj_70, adj_71);
    wp::adj_extract(var_4, var_68, adj_4, adj_68, adj_69);
    // adj: param_col00, param_col01 = solveLdlt2D(sys[0], sys[1], sys[3], rhs[0], rhs[1])    <L 35>
    wp::adj_assign_inplace(var_5, var_67, var_66, adj_5, adj_67, adj_66);
    wp::adj_dot(var_3, var_63, adj_3, adj_63, adj_66);
    // adj: rhs[1] = wp.dot(basis_row1, b)                                                    <L 34>
    wp::adj_assign_inplace(var_5, var_65, var_64, adj_5, adj_65, adj_64);
    wp::adj_dot(var_0, var_63, adj_0, adj_63, adj_64);
    // adj: rhs[0] = wp.dot(basis_row0, b)                                                    <L 33>
    wp::adj_sub(var_p, var_t0, adj_p, adj_t0, adj_63);
    // adj: b = p - t0                                                                        <L 32>
    wp::adj_assign_inplace(var_4, var_62, var_61, adj_4, adj_62, adj_61);
    wp::adj_add(var_55, var_60, adj_55, adj_60, adj_61);
    wp::adj_mul(var_57, var_59, adj_57, adj_59, adj_60);
    wp::adj_extract(var_3, var_58, adj_3, adj_58, adj_59);
    wp::adj_extract(var_3, var_56, adj_3, adj_56, adj_57);
    wp::adj_add(var_49, var_54, adj_49, adj_54, adj_55);
    wp::adj_mul(var_51, var_53, adj_51, adj_53, adj_54);
    wp::adj_extract(var_3, var_52, adj_3, adj_52, adj_53);
    wp::adj_extract(var_3, var_50, adj_3, adj_50, adj_51);
    wp::adj_mul(var_46, var_48, adj_46, adj_48, adj_49);
    wp::adj_extract(var_3, var_47, adj_3, adj_47, adj_48);
    wp::adj_extract(var_3, var_45, adj_3, adj_45, adj_46);
    // adj: sys[3] = basis_row1[0] * basis_row1[0] + basis_row1[1] * basis_row1[1] + basis_row1[2] * basis_row1[2]  <L 30>
    wp::adj_assign_inplace(var_4, var_44, var_43, adj_4, adj_44, adj_43);
    wp::adj_extract(var_4, var_42, adj_4, adj_42, adj_43);
    // adj: sys[1] = sys[2]                                                                   <L 29>
    wp::adj_assign_inplace(var_4, var_41, var_40, adj_4, adj_41, adj_40);
    wp::adj_add(var_34, var_39, adj_34, adj_39, adj_40);
    wp::adj_mul(var_36, var_38, adj_36, adj_38, adj_39);
    wp::adj_extract(var_3, var_37, adj_3, adj_37, adj_38);
    wp::adj_extract(var_0, var_35, adj_0, adj_35, adj_36);
    wp::adj_add(var_28, var_33, adj_28, adj_33, adj_34);
    wp::adj_mul(var_30, var_32, adj_30, adj_32, adj_33);
    wp::adj_extract(var_3, var_31, adj_3, adj_31, adj_32);
    wp::adj_extract(var_0, var_29, adj_0, adj_29, adj_30);
    wp::adj_mul(var_25, var_27, adj_25, adj_27, adj_28);
    wp::adj_extract(var_3, var_26, adj_3, adj_26, adj_27);
    wp::adj_extract(var_0, var_24, adj_0, adj_24, adj_25);
    // adj: sys[2] = basis_row0[0] * basis_row1[0] + basis_row0[1] * basis_row1[1] + basis_row0[2] * basis_row1[2]  <L 28>
    wp::adj_assign_inplace(var_4, var_23, var_22, adj_4, adj_23, adj_22);
    wp::adj_add(var_16, var_21, adj_16, adj_21, adj_22);
    wp::adj_mul(var_18, var_20, adj_18, adj_20, adj_21);
    wp::adj_extract(var_0, var_19, adj_0, adj_19, adj_20);
    wp::adj_extract(var_0, var_17, adj_0, adj_17, adj_18);
    wp::adj_add(var_10, var_15, adj_10, adj_15, adj_16);
    wp::adj_mul(var_12, var_14, adj_12, adj_14, adj_15);
    wp::adj_extract(var_0, var_13, adj_0, adj_13, adj_14);
    wp::adj_extract(var_0, var_11, adj_0, adj_11, adj_12);
    wp::adj_mul(var_7, var_9, adj_7, adj_9, adj_10);
    wp::adj_extract(var_0, var_8, adj_0, adj_8, adj_9);
    wp::adj_extract(var_0, var_6, adj_0, adj_6, adj_7);
    // adj: sys[0] = basis_row0[0] * basis_row0[0] + basis_row0[1] * basis_row0[1] + basis_row0[2] * basis_row0[2]  <L 27>
    // adj: rhs = wp.vec2d()                                                                  <L 26>
    // adj: sys = wp.vec4d()                                                                  <L 25>
    wp::adj_cross(var_0, var_2, adj_0, adj_2, adj_3);
    // adj: basis_row1 = wp.cross(basis_row0, nVec)                                           <L 23>
    wp::adj_cross(var_0, var_1, adj_0, adj_1, adj_2);
    // adj: nVec = wp.cross(basis_row0, basis_row1)                                           <L 22>
    wp::adj_sub(var_t2, var_t0, adj_t2, adj_t0, adj_1);
    // adj: basis_row1 = t2 - t0                                                              <L 21>
    wp::adj_sub(var_t1, var_t0, adj_t1, adj_t0, adj_0);
    // adj: basis_row0 = t1 - t0                                                              <L 20>
    // adj: def point_triangle_distance_type(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d) -> int:  <L 19>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_point_distance.py:4
static CUDA_CALLABLE wp::float64 point_point_distance_0(
    wp::vec_t<3,wp::float64> var_a,
    wp::vec_t<3,wp::float64> var_b)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::float64 var_1;
    //---------
    // forward
    // def point_point_distance(a: wp.vec3d, b: wp.vec3d):                                    <L 5>
    // return wp.length_sq(a - b)                                                             <L 6>
    var_0 = wp::sub(var_a, var_b);
    var_1 = wp::length_sq(var_0);
    return var_1;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_point_distance.py:4
static CUDA_CALLABLE void adj_point_point_distance_0(
    wp::vec_t<3,wp::float64> var_a,
    wp::vec_t<3,wp::float64> var_b,
    wp::vec_t<3,wp::float64> & adj_a,
    wp::vec_t<3,wp::float64> & adj_b,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::float64 var_1;
    //---------
    // dual vars
    wp::vec_t<3,wp::float64> adj_0 = {};
    wp::float64 adj_1 = {};
    //---------
    // forward
    // def point_point_distance(a: wp.vec3d, b: wp.vec3d):                                    <L 5>
    // return wp.length_sq(a - b)                                                             <L 6>
    var_0 = wp::sub(var_a, var_b);
    var_1 = wp::length_sq(var_0);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_1 += adj_ret;
    wp::adj_length_sq(var_0, adj_0, adj_1);
    wp::adj_sub(var_a, var_b, adj_a, adj_b, adj_0);
    // adj: return wp.length_sq(a - b)                                                        <L 6>
    // adj: def point_point_distance(a: wp.vec3d, b: wp.vec3d):                               <L 5>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_line_distance.py:287
static CUDA_CALLABLE wp::float64 point_line_distance_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_e0,
    wp::vec_t<3,wp::float64> var_e1)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    //---------
    // forward
    // def point_line_distance(p: wp.vec3d, e0: wp.vec3d, e1: wp.vec3d):                      <L 288>
    // pe0 = e0 - p                                                                           <L 290>
    var_0 = wp::sub(var_e0, var_p);
    // pe1 = e1 - p                                                                           <L 291>
    var_1 = wp::sub(var_e1, var_p);
    // e0e1 = e1 - e0                                                                         <L 292>
    var_2 = wp::sub(var_e1, var_e0);
    // nor = wp.cross(pe0, pe1)                                                               <L 293>
    var_3 = wp::cross(var_0, var_1);
    // return wp.length_sq(nor) / wp.length_sq(e0e1)                                          <L 294>
    var_4 = wp::length_sq(var_3);
    var_5 = wp::length_sq(var_2);
    var_6 = wp::div(var_4, var_5);
    return var_6;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_line_distance.py:287
static CUDA_CALLABLE void adj_point_line_distance_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_e0,
    wp::vec_t<3,wp::float64> var_e1,
    wp::vec_t<3,wp::float64> & adj_p,
    wp::vec_t<3,wp::float64> & adj_e0,
    wp::vec_t<3,wp::float64> & adj_e1,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    //---------
    // dual vars
    wp::vec_t<3,wp::float64> adj_0 = {};
    wp::vec_t<3,wp::float64> adj_1 = {};
    wp::vec_t<3,wp::float64> adj_2 = {};
    wp::vec_t<3,wp::float64> adj_3 = {};
    wp::float64 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::float64 adj_6 = {};
    //---------
    // forward
    // def point_line_distance(p: wp.vec3d, e0: wp.vec3d, e1: wp.vec3d):                      <L 288>
    // pe0 = e0 - p                                                                           <L 290>
    var_0 = wp::sub(var_e0, var_p);
    // pe1 = e1 - p                                                                           <L 291>
    var_1 = wp::sub(var_e1, var_p);
    // e0e1 = e1 - e0                                                                         <L 292>
    var_2 = wp::sub(var_e1, var_e0);
    // nor = wp.cross(pe0, pe1)                                                               <L 293>
    var_3 = wp::cross(var_0, var_1);
    // return wp.length_sq(nor) / wp.length_sq(e0e1)                                          <L 294>
    var_4 = wp::length_sq(var_3);
    var_5 = wp::length_sq(var_2);
    var_6 = wp::div(var_4, var_5);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_6 += adj_ret;
    wp::adj_div(var_4, var_5, var_6, adj_4, adj_5, adj_6);
    wp::adj_length_sq(var_2, adj_2, adj_5);
    wp::adj_length_sq(var_3, adj_3, adj_4);
    // adj: return wp.length_sq(nor) / wp.length_sq(e0e1)                                     <L 294>
    wp::adj_cross(var_0, var_1, adj_0, adj_1, adj_3);
    // adj: nor = wp.cross(pe0, pe1)                                                          <L 293>
    wp::adj_sub(var_e1, var_e0, adj_e1, adj_e0, adj_2);
    // adj: e0e1 = e1 - e0                                                                    <L 292>
    wp::adj_sub(var_e1, var_p, adj_e1, adj_p, adj_1);
    // adj: pe1 = e1 - p                                                                      <L 291>
    wp::adj_sub(var_e0, var_p, adj_e0, adj_p, adj_0);
    // adj: pe0 = e0 - p                                                                      <L 290>
    // adj: def point_line_distance(p: wp.vec3d, e0: wp.vec3d, e1: wp.vec3d):                 <L 288>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_plane_distance.py:555
static CUDA_CALLABLE wp::float64 point_plane_distance_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_t0,
    wp::vec_t<3,wp::float64> var_t1,
    wp::vec_t<3,wp::float64> var_t2)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    //---------
    // forward
    // def point_plane_distance(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d):       <L 556>
    // t0t1 = t1 - t0                                                                         <L 561>
    var_0 = wp::sub(var_t1, var_t0);
    // t0t2 = t2 - t0                                                                         <L 562>
    var_1 = wp::sub(var_t2, var_t0);
    // t0p = p - t0                                                                           <L 563>
    var_2 = wp::sub(var_p, var_t0);
    // b = wp.cross(t0t1, t0t2)                                                               <L 564>
    var_3 = wp::cross(var_0, var_1);
    // aTb = wp.dot(t0p, b)                                                                   <L 565>
    var_4 = wp::dot(var_2, var_3);
    // return aTb * aTb / wp.length_sq(b)                                                     <L 566>
    var_5 = wp::mul(var_4, var_4);
    var_6 = wp::length_sq(var_3);
    var_7 = wp::div(var_5, var_6);
    return var_7;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_plane_distance.py:555
static CUDA_CALLABLE void adj_point_plane_distance_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_t0,
    wp::vec_t<3,wp::float64> var_t1,
    wp::vec_t<3,wp::float64> var_t2,
    wp::vec_t<3,wp::float64> & adj_p,
    wp::vec_t<3,wp::float64> & adj_t0,
    wp::vec_t<3,wp::float64> & adj_t1,
    wp::vec_t<3,wp::float64> & adj_t2,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    //---------
    // dual vars
    wp::vec_t<3,wp::float64> adj_0 = {};
    wp::vec_t<3,wp::float64> adj_1 = {};
    wp::vec_t<3,wp::float64> adj_2 = {};
    wp::vec_t<3,wp::float64> adj_3 = {};
    wp::float64 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::float64 adj_6 = {};
    wp::float64 adj_7 = {};
    //---------
    // forward
    // def point_plane_distance(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d):       <L 556>
    // t0t1 = t1 - t0                                                                         <L 561>
    var_0 = wp::sub(var_t1, var_t0);
    // t0t2 = t2 - t0                                                                         <L 562>
    var_1 = wp::sub(var_t2, var_t0);
    // t0p = p - t0                                                                           <L 563>
    var_2 = wp::sub(var_p, var_t0);
    // b = wp.cross(t0t1, t0t2)                                                               <L 564>
    var_3 = wp::cross(var_0, var_1);
    // aTb = wp.dot(t0p, b)                                                                   <L 565>
    var_4 = wp::dot(var_2, var_3);
    // return aTb * aTb / wp.length_sq(b)                                                     <L 566>
    var_5 = wp::mul(var_4, var_4);
    var_6 = wp::length_sq(var_3);
    var_7 = wp::div(var_5, var_6);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_7 += adj_ret;
    wp::adj_div(var_5, var_6, var_7, adj_5, adj_6, adj_7);
    wp::adj_length_sq(var_3, adj_3, adj_6);
    wp::adj_mul(var_4, var_4, adj_4, adj_4, adj_5);
    // adj: return aTb * aTb / wp.length_sq(b)                                                <L 566>
    wp::adj_dot(var_2, var_3, adj_2, adj_3, adj_4);
    // adj: aTb = wp.dot(t0p, b)                                                              <L 565>
    wp::adj_cross(var_0, var_1, adj_0, adj_1, adj_3);
    // adj: b = wp.cross(t0t1, t0t2)                                                          <L 564>
    wp::adj_sub(var_p, var_t0, adj_p, adj_t0, adj_2);
    // adj: t0p = p - t0                                                                      <L 563>
    wp::adj_sub(var_t2, var_t0, adj_t2, adj_t0, adj_1);
    // adj: t0t2 = t2 - t0                                                                    <L 562>
    wp::adj_sub(var_t1, var_t0, adj_t1, adj_t0, adj_0);
    // adj: t0t1 = t1 - t0                                                                    <L 561>
    // adj: def point_plane_distance(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d):  <L 556>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_triangle_distance.py:9
static CUDA_CALLABLE wp::float64 point_triangle_distance_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_t0,
    wp::vec_t<3,wp::float64> var_t1,
    wp::vec_t<3,wp::float64> var_t2)
{
    //---------
    // primal vars
    wp::int32 var_0;
    const wp::int32 var_1 = 0;
    bool var_2;
    wp::float64 var_3;
    const wp::int32 var_4 = 1;
    bool var_5;
    wp::float64 var_6;
    const wp::int32 var_7 = 2;
    bool var_8;
    wp::float64 var_9;
    const wp::int32 var_10 = 3;
    bool var_11;
    wp::float64 var_12;
    const wp::int32 var_13 = 4;
    bool var_14;
    wp::float64 var_15;
    const wp::int32 var_16 = 5;
    bool var_17;
    wp::float64 var_18;
    const wp::int32 var_19 = 6;
    bool var_20;
    wp::float64 var_21;
    const wp::float32 var_22 = 1e+20;
    wp::float64 var_23;
    //---------
    // forward
    // def point_triangle_distance(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d) -> wp.float64:       <L 10>
    // dt = point_triangle_distance_type(p, t0, t1, t2)                                       <L 11>
    var_0 = point_triangle_distance_type_0(var_p, var_t0, var_t1, var_t2);
    // if dt == 0:                                                                            <L 12>
    var_2 = (var_0 == var_1);
    if (var_2) {
        // return point_point_distance(p, t0)                                                 <L 13>
        var_3 = point_point_distance_0(var_p, var_t0);
        return var_3;
    }
    if (!var_2) {
        // elif dt == 1:                                                                      <L 14>
        var_5 = (var_0 == var_4);
        if (var_5) {
            // return point_point_distance(p, t1)                                             <L 15>
            var_6 = point_point_distance_0(var_p, var_t1);
            return var_6;
        }
        if (!var_5) {
            // elif dt == 2:                                                                  <L 16>
            var_8 = (var_0 == var_7);
            if (var_8) {
                // return point_point_distance(p, t2)                                         <L 17>
                var_9 = point_point_distance_0(var_p, var_t2);
                return var_9;
            }
            if (!var_8) {
                // elif dt == 3:                                                              <L 18>
                var_11 = (var_0 == var_10);
                if (var_11) {
                    // return point_line_distance(p, t0, t1)                                  <L 19>
                    var_12 = point_line_distance_0(var_p, var_t0, var_t1);
                    return var_12;
                }
                if (!var_11) {
                    // elif dt == 4:                                                          <L 20>
                    var_14 = (var_0 == var_13);
                    if (var_14) {
                        // return point_line_distance(p, t1, t2)                              <L 21>
                        var_15 = point_line_distance_0(var_p, var_t1, var_t2);
                        return var_15;
                    }
                    if (!var_14) {
                        // elif dt == 5:                                                      <L 22>
                        var_17 = (var_0 == var_16);
                        if (var_17) {
                            // return point_line_distance(p, t2, t0)                          <L 23>
                            var_18 = point_line_distance_0(var_p, var_t2, var_t0);
                            return var_18;
                        }
                        if (!var_17) {
                            // elif dt == 6:                                                  <L 24>
                            var_20 = (var_0 == var_19);
                            if (var_20) {
                                // return point_plane_distance(p, t0, t1, t2)                 <L 25>
                                var_21 = point_plane_distance_0(var_p, var_t0, var_t1, var_t2);
                                return var_21;
                            }
                            if (!var_20) {
                                // return wp.float64(1e20)                                    <L 27>
                                var_23 = wp::float64(var_22);
                                return var_23;
                            }
                        }
                    }
                }
            }
        }
    }
    return {};
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/point_triangle_distance.py:9
static CUDA_CALLABLE void adj_point_triangle_distance_0(
    wp::vec_t<3,wp::float64> var_p,
    wp::vec_t<3,wp::float64> var_t0,
    wp::vec_t<3,wp::float64> var_t1,
    wp::vec_t<3,wp::float64> var_t2,
    wp::vec_t<3,wp::float64> & adj_p,
    wp::vec_t<3,wp::float64> & adj_t0,
    wp::vec_t<3,wp::float64> & adj_t1,
    wp::vec_t<3,wp::float64> & adj_t2,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::int32 var_0;
    const wp::int32 var_1 = 0;
    bool var_2;
    wp::float64 var_3;
    const wp::int32 var_4 = 1;
    bool var_5;
    wp::float64 var_6;
    const wp::int32 var_7 = 2;
    bool var_8;
    wp::float64 var_9;
    const wp::int32 var_10 = 3;
    bool var_11;
    wp::float64 var_12;
    const wp::int32 var_13 = 4;
    bool var_14;
    wp::float64 var_15;
    const wp::int32 var_16 = 5;
    bool var_17;
    wp::float64 var_18;
    const wp::int32 var_19 = 6;
    bool var_20;
    wp::float64 var_21;
    const wp::float32 var_22 = 1e+20;
    wp::float64 var_23;
    //---------
    // dual vars
    wp::int32 adj_0 = {};
    wp::int32 adj_1 = {};
    bool adj_2 = {};
    wp::float64 adj_3 = {};
    wp::int32 adj_4 = {};
    bool adj_5 = {};
    wp::float64 adj_6 = {};
    wp::int32 adj_7 = {};
    bool adj_8 = {};
    wp::float64 adj_9 = {};
    wp::int32 adj_10 = {};
    bool adj_11 = {};
    wp::float64 adj_12 = {};
    wp::int32 adj_13 = {};
    bool adj_14 = {};
    wp::float64 adj_15 = {};
    wp::int32 adj_16 = {};
    bool adj_17 = {};
    wp::float64 adj_18 = {};
    wp::int32 adj_19 = {};
    bool adj_20 = {};
    wp::float64 adj_21 = {};
    wp::float32 adj_22 = {};
    wp::float64 adj_23 = {};
    //---------
    // forward
    // def point_triangle_distance(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d) -> wp.float64:       <L 10>
    // dt = point_triangle_distance_type(p, t0, t1, t2)                                       <L 11>
    var_0 = point_triangle_distance_type_0(var_p, var_t0, var_t1, var_t2);
    // if dt == 0:                                                                            <L 12>
    var_2 = (var_0 == var_1);
    if (var_2) {
        // return point_point_distance(p, t0)                                                 <L 13>
        var_3 = point_point_distance_0(var_p, var_t0);
        goto label0;
    }
    if (!var_2) {
        // elif dt == 1:                                                                      <L 14>
        var_5 = (var_0 == var_4);
        if (var_5) {
            // return point_point_distance(p, t1)                                             <L 15>
            var_6 = point_point_distance_0(var_p, var_t1);
            goto label1;
        }
        if (!var_5) {
            // elif dt == 2:                                                                  <L 16>
            var_8 = (var_0 == var_7);
            if (var_8) {
                // return point_point_distance(p, t2)                                         <L 17>
                var_9 = point_point_distance_0(var_p, var_t2);
                goto label2;
            }
            if (!var_8) {
                // elif dt == 3:                                                              <L 18>
                var_11 = (var_0 == var_10);
                if (var_11) {
                    // return point_line_distance(p, t0, t1)                                  <L 19>
                    var_12 = point_line_distance_0(var_p, var_t0, var_t1);
                    goto label3;
                }
                if (!var_11) {
                    // elif dt == 4:                                                          <L 20>
                    var_14 = (var_0 == var_13);
                    if (var_14) {
                        // return point_line_distance(p, t1, t2)                              <L 21>
                        var_15 = point_line_distance_0(var_p, var_t1, var_t2);
                        goto label4;
                    }
                    if (!var_14) {
                        // elif dt == 5:                                                      <L 22>
                        var_17 = (var_0 == var_16);
                        if (var_17) {
                            // return point_line_distance(p, t2, t0)                          <L 23>
                            var_18 = point_line_distance_0(var_p, var_t2, var_t0);
                            goto label5;
                        }
                        if (!var_17) {
                            // elif dt == 6:                                                  <L 24>
                            var_20 = (var_0 == var_19);
                            if (var_20) {
                                // return point_plane_distance(p, t0, t1, t2)                 <L 25>
                                var_21 = point_plane_distance_0(var_p, var_t0, var_t1, var_t2);
                                goto label6;
                            }
                            if (!var_20) {
                                // return wp.float64(1e20)                                    <L 27>
                                var_23 = wp::float64(var_22);
                                goto label7;
                            }
                        }
                    }
                }
            }
        }
    }
    //---------
    // reverse
    if (!var_2) {
        if (!var_5) {
            if (!var_8) {
                if (!var_11) {
                    if (!var_14) {
                        if (!var_17) {
                            if (!var_20) {
                                label7:;
                                adj_23 += adj_ret;
                                wp::adj_float64(var_22, adj_22, adj_23);
                                // adj: return wp.float64(1e20)                               <L 27>
                            }
                            if (var_20) {
                                label6:;
                                adj_21 += adj_ret;
                                adj_point_plane_distance_0(var_p, var_t0, var_t1, var_t2, adj_p, adj_t0, adj_t1, adj_t2, adj_21);
                                // adj: return point_plane_distance(p, t0, t1, t2)            <L 25>
                            }
                            // adj: elif dt == 6:                                             <L 24>
                        }
                        if (var_17) {
                            label5:;
                            adj_18 += adj_ret;
                            adj_point_line_distance_0(var_p, var_t2, var_t0, adj_p, adj_t2, adj_t0, adj_18);
                            // adj: return point_line_distance(p, t2, t0)                     <L 23>
                        }
                        // adj: elif dt == 5:                                                 <L 22>
                    }
                    if (var_14) {
                        label4:;
                        adj_15 += adj_ret;
                        adj_point_line_distance_0(var_p, var_t1, var_t2, adj_p, adj_t1, adj_t2, adj_15);
                        // adj: return point_line_distance(p, t1, t2)                         <L 21>
                    }
                    // adj: elif dt == 4:                                                     <L 20>
                }
                if (var_11) {
                    label3:;
                    adj_12 += adj_ret;
                    adj_point_line_distance_0(var_p, var_t0, var_t1, adj_p, adj_t0, adj_t1, adj_12);
                    // adj: return point_line_distance(p, t0, t1)                             <L 19>
                }
                // adj: elif dt == 3:                                                         <L 18>
            }
            if (var_8) {
                label2:;
                adj_9 += adj_ret;
                adj_point_point_distance_0(var_p, var_t2, adj_p, adj_t2, adj_9);
                // adj: return point_point_distance(p, t2)                                    <L 17>
            }
            // adj: elif dt == 2:                                                             <L 16>
        }
        if (var_5) {
            label1:;
            adj_6 += adj_ret;
            adj_point_point_distance_0(var_p, var_t1, adj_p, adj_t1, adj_6);
            // adj: return point_point_distance(p, t1)                                        <L 15>
        }
        // adj: elif dt == 1:                                                                 <L 14>
    }
    if (var_2) {
        label0:;
        adj_3 += adj_ret;
        adj_point_point_distance_0(var_p, var_t0, adj_p, adj_t0, adj_3);
        // adj: return point_point_distance(p, t0)                                            <L 13>
    }
    // adj: if dt == 0:                                                                       <L 12>
    adj_point_triangle_distance_type_0(var_p, var_t0, var_t1, var_t2, adj_p, adj_t0, adj_t1, adj_t2, adj_0);
    // adj: dt = point_triangle_distance_type(p, t0, t1, t2)                                  <L 11>
    // adj: def point_triangle_distance(p: wp.vec3d, t0: wp.vec3d, t1: wp.vec3d, t2: wp.vec3d) -> wp.float64:  <L 10>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/distance_type.py:78
static CUDA_CALLABLE wp::int32 edge_edge_distance_type_0(
    wp::vec_t<3,wp::float64> var_ea0,
    wp::vec_t<3,wp::float64> var_ea1,
    wp::vec_t<3,wp::float64> var_eb0,
    wp::vec_t<3,wp::float64> var_eb1)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::float64 var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    wp::float64 var_8;
    wp::float64 var_9;
    wp::float64 var_10;
    wp::float64 var_11;
    const wp::int32 var_12 = 8;
    const wp::float32 var_13 = 0.0;
    wp::float64 var_14;
    const wp::float32 var_15 = 0.0;
    wp::float64 var_16;
    wp::float64 var_17;
    wp::float64 var_18;
    wp::float64 var_19;
    const wp::float32 var_20 = 0.0;
    wp::float64 var_21;
    bool var_22;
    wp::float64 var_23;
    wp::float64 var_24;
    const wp::int32 var_25 = 2;
    wp::float64 var_26;
    wp::int32 var_27;
    wp::float64 var_28;
    bool var_29;
    wp::float64 var_30;
    wp::float64 var_31;
    const wp::int32 var_32 = 5;
    wp::float64 var_33;
    wp::int32 var_34;
    wp::float64 var_35;
    wp::float64 var_36;
    wp::float64 var_37;
    wp::float64 var_38;
    wp::vec_t<3,wp::float64> var_39;
    const wp::float32 var_40 = 0.0;
    wp::float64 var_41;
    bool var_42;
    bool var_43;
    wp::float64 var_44;
    const wp::float32 var_45 = 0.0;
    wp::float64 var_46;
    bool var_47;
    wp::float64 var_48;
    const wp::float32 var_49 = 1e-20;
    wp::float64 var_50;
    wp::float64 var_51;
    wp::float64 var_52;
    bool var_53;
    bool var_54;
    bool var_55;
    const wp::float32 var_56 = 2.0;
    wp::float64 var_57;
    wp::float64 var_58;
    bool var_59;
    wp::float64 var_60;
    wp::float64 var_61;
    const wp::int32 var_62 = 2;
    wp::float64 var_63;
    wp::int32 var_64;
    wp::float64 var_65;
    wp::float64 var_66;
    wp::float64 var_67;
    const wp::int32 var_68 = 5;
    wp::float64 var_69;
    wp::int32 var_70;
    wp::float64 var_71;
    wp::float64 var_72;
    wp::int32 var_73;
    wp::float64 var_74;
    wp::float64 var_75;
    wp::int32 var_76;
    wp::float64 var_77;
    wp::float64 var_78;
    wp::int32 var_79;
    wp::float64 var_80;
    const wp::float32 var_81 = 0.0;
    wp::float64 var_82;
    bool var_83;
    wp::float64 var_84;
    const wp::float32 var_85 = 0.0;
    wp::float64 var_86;
    bool var_87;
    const wp::int32 var_88 = 0;
    wp::float64 var_89;
    bool var_90;
    const wp::int32 var_91 = 3;
    const wp::int32 var_92 = 6;
    bool var_93;
    wp::float64 var_94;
    wp::float64 var_95;
    const wp::float32 var_96 = 0.0;
    wp::float64 var_97;
    bool var_98;
    const wp::int32 var_99 = 1;
    wp::float64 var_100;
    wp::float64 var_101;
    bool var_102;
    const wp::int32 var_103 = 4;
    const wp::int32 var_104 = 7;
    //---------
    // forward
    // def edge_edge_distance_type(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d) -> int:       <L 79>
    // u = ea1 - ea0                                                                          <L 80>
    var_0 = wp::sub(var_ea1, var_ea0);
    // v = eb1 - eb0                                                                          <L 81>
    var_1 = wp::sub(var_eb1, var_eb0);
    // w = ea0 - eb0                                                                          <L 82>
    var_2 = wp::sub(var_ea0, var_eb0);
    // a = wp.length_sq(u)  # always >= 0                                                     <L 84>
    var_3 = wp::length_sq(var_0);
    // b = wp.dot(u, v)                                                                       <L 85>
    var_4 = wp::dot(var_0, var_1);
    // c = wp.length_sq(v)  # always >= 0                                                     <L 86>
    var_5 = wp::length_sq(var_1);
    // d = wp.dot(u, w)                                                                       <L 87>
    var_6 = wp::dot(var_0, var_2);
    // e = wp.dot(v, w)                                                                       <L 88>
    var_7 = wp::dot(var_1, var_2);
    // D = a * c - b * b  # always >= 0                                                       <L 89>
    var_8 = wp::mul(var_3, var_5);
    var_9 = wp::mul(var_4, var_4);
    var_10 = wp::sub(var_8, var_9);
    // tD = D  # tc = tN / tD, default tD = D >= 0                                            <L 90>
    var_11 = wp::copy(var_10);
    // defaultCase = 8                                                                        <L 91>
    // sN, tN = wp.float64(0.0), wp.float64(0.0)                                              <L 92>
    var_14 = wp::float64(var_13);
    var_16 = wp::float64(var_15);
    // sN = b * e - c * d                                                                     <L 95>
    var_17 = wp::mul(var_4, var_7);
    var_18 = wp::mul(var_5, var_6);
    var_19 = wp::sub(var_17, var_18);
    // if sN <= wp.float64(0.0):  # sc < 0 => the s=0 edge is visible                         <L 96>
    var_21 = wp::float64(var_20);
    var_22 = (var_19 <= var_21);
    if (var_22) {
        // tN = e                                                                             <L 97>
        var_23 = wp::copy(var_7);
        // tD = c                                                                             <L 98>
        var_24 = wp::copy(var_5);
        // defaultCase = 2                                                                    <L 99>
    }
    var_26 = wp::where(var_22, var_24, var_11);
    var_27 = wp::where(var_22, var_25, var_12);
    var_28 = wp::where(var_22, var_23, var_16);
    if (!var_22) {
        // elif sN >= D:  # sc > 1  => the s=1 edge is visible                                <L 100>
        var_29 = (var_19 >= var_10);
        if (var_29) {
            // tN = e + b                                                                     <L 101>
            var_30 = wp::add(var_7, var_4);
            // tD = c                                                                         <L 102>
            var_31 = wp::copy(var_5);
            // defaultCase = 5                                                                <L 103>
        }
        var_33 = wp::where(var_29, var_31, var_26);
        var_34 = wp::where(var_29, var_32, var_27);
        var_35 = wp::where(var_29, var_30, var_28);
        if (!var_29) {
            // tN = a * e - b * d                                                             <L 105>
            var_36 = wp::mul(var_3, var_7);
            var_37 = wp::mul(var_4, var_6);
            var_38 = wp::sub(var_36, var_37);
            // uv = wp.cross(u, v)                                                            <L 106>
            var_39 = wp::cross(var_0, var_1);
            // if tN > wp.float64(0.0) and tN < tD and (wp.dot(uv, w) == wp.float64(0.0) or wp.length_sq(uv) < wp.float64(1e-20) * a * c):       <L 108>
            var_41 = wp::float64(var_40);
            var_42 = (var_38 > var_41);
            var_43 = (var_38 < var_33);
            var_44 = wp::dot(var_39, var_2);
            var_46 = wp::float64(var_45);
            var_47 = (var_44 == var_46);
            var_48 = wp::length_sq(var_39);
            var_50 = wp::float64(var_49);
            var_51 = wp::mul(var_50, var_3);
            var_52 = wp::mul(var_51, var_5);
            var_53 = (var_48 < var_52);
            var_54 = var_47 || var_53;
            var_55 = var_42 && var_43 && var_54;
            if (var_55) {
                // if sN < D / wp.float64(2.0):                                               <L 113>
                var_57 = wp::float64(var_56);
                var_58 = wp::div(var_10, var_57);
                var_59 = (var_19 < var_58);
                if (var_59) {
                    // tN = e                                                                 <L 114>
                    var_60 = wp::copy(var_7);
                    // tD = c                                                                 <L 115>
                    var_61 = wp::copy(var_5);
                    // defaultCase = 2                                                        <L 116>
                }
                var_63 = wp::where(var_59, var_61, var_33);
                var_64 = wp::where(var_59, var_62, var_34);
                var_65 = wp::where(var_59, var_60, var_38);
                if (!var_59) {
                    // tN = e + b                                                             <L 118>
                    var_66 = wp::add(var_7, var_4);
                    // tD = c                                                                 <L 119>
                    var_67 = wp::copy(var_5);
                    // defaultCase = 5                                                        <L 120>
                }
                var_69 = wp::where(var_59, var_63, var_67);
                var_70 = wp::where(var_59, var_64, var_68);
                var_71 = wp::where(var_59, var_65, var_66);
            }
            var_72 = wp::where(var_55, var_69, var_33);
            var_73 = wp::where(var_55, var_70, var_34);
            var_74 = wp::where(var_55, var_71, var_38);
        }
        var_75 = wp::where(var_29, var_33, var_72);
        var_76 = wp::where(var_29, var_34, var_73);
        var_77 = wp::where(var_29, var_35, var_74);
    }
    var_78 = wp::where(var_22, var_26, var_75);
    var_79 = wp::where(var_22, var_27, var_76);
    var_80 = wp::where(var_22, var_28, var_77);
    // if tN <= wp.float64(0.0):  # tc < 0 => the t=0 edge is visible                         <L 123>
    var_82 = wp::float64(var_81);
    var_83 = (var_80 <= var_82);
    if (var_83) {
        // if -d <= wp.float64(0.0):                                                          <L 125>
        var_84 = wp::neg(var_6);
        var_86 = wp::float64(var_85);
        var_87 = (var_84 <= var_86);
        if (var_87) {
            // return 0                                                                       <L 126>
            return var_88;
        }
        if (!var_87) {
            // elif -d >= a:                                                                  <L 127>
            var_89 = wp::neg(var_6);
            var_90 = (var_89 >= var_3);
            if (var_90) {
                // return 3                                                                   <L 128>
                return var_91;
            }
            if (!var_90) {
                // return 6                                                                   <L 130>
                return var_92;
            }
        }
    }
    if (!var_83) {
        // elif tN >= tD:  # tc > 1  => the t=1 edge is visible                               <L 131>
        var_93 = (var_80 >= var_78);
        if (var_93) {
            // if (-d + b) <= wp.float64(0.0):                                                <L 133>
            var_94 = wp::neg(var_6);
            var_95 = wp::add(var_94, var_4);
            var_97 = wp::float64(var_96);
            var_98 = (var_95 <= var_97);
            if (var_98) {
                // return 1                                                                   <L 134>
                return var_99;
            }
            if (!var_98) {
                // elif (-d + b) >= a:                                                        <L 135>
                var_100 = wp::neg(var_6);
                var_101 = wp::add(var_100, var_4);
                var_102 = (var_101 >= var_3);
                if (var_102) {
                    // return 4                                                               <L 136>
                    return var_103;
                }
                if (!var_102) {
                    // return 7                                                               <L 138>
                    return var_104;
                }
            }
        }
    }
    // return defaultCase                                                                     <L 140>
    return var_79;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/distance_type.py:78
static CUDA_CALLABLE void adj_edge_edge_distance_type_0(
    wp::vec_t<3,wp::float64> var_ea0,
    wp::vec_t<3,wp::float64> var_ea1,
    wp::vec_t<3,wp::float64> var_eb0,
    wp::vec_t<3,wp::float64> var_eb1,
    wp::vec_t<3,wp::float64> & adj_ea0,
    wp::vec_t<3,wp::float64> & adj_ea1,
    wp::vec_t<3,wp::float64> & adj_eb0,
    wp::vec_t<3,wp::float64> & adj_eb1,
    wp::int32 & adj_ret)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::float64 var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    wp::float64 var_8;
    wp::float64 var_9;
    wp::float64 var_10;
    wp::float64 var_11;
    const wp::int32 var_12 = 8;
    const wp::float32 var_13 = 0.0;
    wp::float64 var_14;
    const wp::float32 var_15 = 0.0;
    wp::float64 var_16;
    wp::float64 var_17;
    wp::float64 var_18;
    wp::float64 var_19;
    const wp::float32 var_20 = 0.0;
    wp::float64 var_21;
    bool var_22;
    wp::float64 var_23;
    wp::float64 var_24;
    const wp::int32 var_25 = 2;
    wp::float64 var_26;
    wp::int32 var_27;
    wp::float64 var_28;
    bool var_29;
    wp::float64 var_30;
    wp::float64 var_31;
    const wp::int32 var_32 = 5;
    wp::float64 var_33;
    wp::int32 var_34;
    wp::float64 var_35;
    wp::float64 var_36;
    wp::float64 var_37;
    wp::float64 var_38;
    wp::vec_t<3,wp::float64> var_39;
    const wp::float32 var_40 = 0.0;
    wp::float64 var_41;
    bool var_42;
    bool var_43;
    wp::float64 var_44;
    const wp::float32 var_45 = 0.0;
    wp::float64 var_46;
    bool var_47;
    wp::float64 var_48;
    const wp::float32 var_49 = 1e-20;
    wp::float64 var_50;
    wp::float64 var_51;
    wp::float64 var_52;
    bool var_53;
    bool var_54;
    bool var_55;
    const wp::float32 var_56 = 2.0;
    wp::float64 var_57;
    wp::float64 var_58;
    bool var_59;
    wp::float64 var_60;
    wp::float64 var_61;
    const wp::int32 var_62 = 2;
    wp::float64 var_63;
    wp::int32 var_64;
    wp::float64 var_65;
    wp::float64 var_66;
    wp::float64 var_67;
    const wp::int32 var_68 = 5;
    wp::float64 var_69;
    wp::int32 var_70;
    wp::float64 var_71;
    wp::float64 var_72;
    wp::int32 var_73;
    wp::float64 var_74;
    wp::float64 var_75;
    wp::int32 var_76;
    wp::float64 var_77;
    wp::float64 var_78;
    wp::int32 var_79;
    wp::float64 var_80;
    const wp::float32 var_81 = 0.0;
    wp::float64 var_82;
    bool var_83;
    wp::float64 var_84;
    const wp::float32 var_85 = 0.0;
    wp::float64 var_86;
    bool var_87;
    const wp::int32 var_88 = 0;
    wp::float64 var_89;
    bool var_90;
    const wp::int32 var_91 = 3;
    const wp::int32 var_92 = 6;
    bool var_93;
    wp::float64 var_94;
    wp::float64 var_95;
    const wp::float32 var_96 = 0.0;
    wp::float64 var_97;
    bool var_98;
    const wp::int32 var_99 = 1;
    wp::float64 var_100;
    wp::float64 var_101;
    bool var_102;
    const wp::int32 var_103 = 4;
    const wp::int32 var_104 = 7;
    //---------
    // dual vars
    wp::vec_t<3,wp::float64> adj_0 = {};
    wp::vec_t<3,wp::float64> adj_1 = {};
    wp::vec_t<3,wp::float64> adj_2 = {};
    wp::float64 adj_3 = {};
    wp::float64 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::float64 adj_6 = {};
    wp::float64 adj_7 = {};
    wp::float64 adj_8 = {};
    wp::float64 adj_9 = {};
    wp::float64 adj_10 = {};
    wp::float64 adj_11 = {};
    wp::int32 adj_12 = {};
    wp::float32 adj_13 = {};
    wp::float64 adj_14 = {};
    wp::float32 adj_15 = {};
    wp::float64 adj_16 = {};
    wp::float64 adj_17 = {};
    wp::float64 adj_18 = {};
    wp::float64 adj_19 = {};
    wp::float32 adj_20 = {};
    wp::float64 adj_21 = {};
    bool adj_22 = {};
    wp::float64 adj_23 = {};
    wp::float64 adj_24 = {};
    wp::int32 adj_25 = {};
    wp::float64 adj_26 = {};
    wp::int32 adj_27 = {};
    wp::float64 adj_28 = {};
    bool adj_29 = {};
    wp::float64 adj_30 = {};
    wp::float64 adj_31 = {};
    wp::int32 adj_32 = {};
    wp::float64 adj_33 = {};
    wp::int32 adj_34 = {};
    wp::float64 adj_35 = {};
    wp::float64 adj_36 = {};
    wp::float64 adj_37 = {};
    wp::float64 adj_38 = {};
    wp::vec_t<3,wp::float64> adj_39 = {};
    wp::float32 adj_40 = {};
    wp::float64 adj_41 = {};
    bool adj_42 = {};
    bool adj_43 = {};
    wp::float64 adj_44 = {};
    wp::float32 adj_45 = {};
    wp::float64 adj_46 = {};
    bool adj_47 = {};
    wp::float64 adj_48 = {};
    wp::float32 adj_49 = {};
    wp::float64 adj_50 = {};
    wp::float64 adj_51 = {};
    wp::float64 adj_52 = {};
    bool adj_53 = {};
    bool adj_54 = {};
    bool adj_55 = {};
    wp::float32 adj_56 = {};
    wp::float64 adj_57 = {};
    wp::float64 adj_58 = {};
    bool adj_59 = {};
    wp::float64 adj_60 = {};
    wp::float64 adj_61 = {};
    wp::int32 adj_62 = {};
    wp::float64 adj_63 = {};
    wp::int32 adj_64 = {};
    wp::float64 adj_65 = {};
    wp::float64 adj_66 = {};
    wp::float64 adj_67 = {};
    wp::int32 adj_68 = {};
    wp::float64 adj_69 = {};
    wp::int32 adj_70 = {};
    wp::float64 adj_71 = {};
    wp::float64 adj_72 = {};
    wp::int32 adj_73 = {};
    wp::float64 adj_74 = {};
    wp::float64 adj_75 = {};
    wp::int32 adj_76 = {};
    wp::float64 adj_77 = {};
    wp::float64 adj_78 = {};
    wp::int32 adj_79 = {};
    wp::float64 adj_80 = {};
    wp::float32 adj_81 = {};
    wp::float64 adj_82 = {};
    bool adj_83 = {};
    wp::float64 adj_84 = {};
    wp::float32 adj_85 = {};
    wp::float64 adj_86 = {};
    bool adj_87 = {};
    wp::int32 adj_88 = {};
    wp::float64 adj_89 = {};
    bool adj_90 = {};
    wp::int32 adj_91 = {};
    wp::int32 adj_92 = {};
    bool adj_93 = {};
    wp::float64 adj_94 = {};
    wp::float64 adj_95 = {};
    wp::float32 adj_96 = {};
    wp::float64 adj_97 = {};
    bool adj_98 = {};
    wp::int32 adj_99 = {};
    wp::float64 adj_100 = {};
    wp::float64 adj_101 = {};
    bool adj_102 = {};
    wp::int32 adj_103 = {};
    wp::int32 adj_104 = {};
    //---------
    // forward
    // def edge_edge_distance_type(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d) -> int:       <L 79>
    // u = ea1 - ea0                                                                          <L 80>
    var_0 = wp::sub(var_ea1, var_ea0);
    // v = eb1 - eb0                                                                          <L 81>
    var_1 = wp::sub(var_eb1, var_eb0);
    // w = ea0 - eb0                                                                          <L 82>
    var_2 = wp::sub(var_ea0, var_eb0);
    // a = wp.length_sq(u)  # always >= 0                                                     <L 84>
    var_3 = wp::length_sq(var_0);
    // b = wp.dot(u, v)                                                                       <L 85>
    var_4 = wp::dot(var_0, var_1);
    // c = wp.length_sq(v)  # always >= 0                                                     <L 86>
    var_5 = wp::length_sq(var_1);
    // d = wp.dot(u, w)                                                                       <L 87>
    var_6 = wp::dot(var_0, var_2);
    // e = wp.dot(v, w)                                                                       <L 88>
    var_7 = wp::dot(var_1, var_2);
    // D = a * c - b * b  # always >= 0                                                       <L 89>
    var_8 = wp::mul(var_3, var_5);
    var_9 = wp::mul(var_4, var_4);
    var_10 = wp::sub(var_8, var_9);
    // tD = D  # tc = tN / tD, default tD = D >= 0                                            <L 90>
    var_11 = wp::copy(var_10);
    // defaultCase = 8                                                                        <L 91>
    // sN, tN = wp.float64(0.0), wp.float64(0.0)                                              <L 92>
    var_14 = wp::float64(var_13);
    var_16 = wp::float64(var_15);
    // sN = b * e - c * d                                                                     <L 95>
    var_17 = wp::mul(var_4, var_7);
    var_18 = wp::mul(var_5, var_6);
    var_19 = wp::sub(var_17, var_18);
    // if sN <= wp.float64(0.0):  # sc < 0 => the s=0 edge is visible                         <L 96>
    var_21 = wp::float64(var_20);
    var_22 = (var_19 <= var_21);
    if (var_22) {
        // tN = e                                                                             <L 97>
        var_23 = wp::copy(var_7);
        // tD = c                                                                             <L 98>
        var_24 = wp::copy(var_5);
        // defaultCase = 2                                                                    <L 99>
    }
    var_26 = wp::where(var_22, var_24, var_11);
    var_27 = wp::where(var_22, var_25, var_12);
    var_28 = wp::where(var_22, var_23, var_16);
    if (!var_22) {
        // elif sN >= D:  # sc > 1  => the s=1 edge is visible                                <L 100>
        var_29 = (var_19 >= var_10);
        if (var_29) {
            // tN = e + b                                                                     <L 101>
            var_30 = wp::add(var_7, var_4);
            // tD = c                                                                         <L 102>
            var_31 = wp::copy(var_5);
            // defaultCase = 5                                                                <L 103>
        }
        var_33 = wp::where(var_29, var_31, var_26);
        var_34 = wp::where(var_29, var_32, var_27);
        var_35 = wp::where(var_29, var_30, var_28);
        if (!var_29) {
            // tN = a * e - b * d                                                             <L 105>
            var_36 = wp::mul(var_3, var_7);
            var_37 = wp::mul(var_4, var_6);
            var_38 = wp::sub(var_36, var_37);
            // uv = wp.cross(u, v)                                                            <L 106>
            var_39 = wp::cross(var_0, var_1);
            // if tN > wp.float64(0.0) and tN < tD and (wp.dot(uv, w) == wp.float64(0.0) or wp.length_sq(uv) < wp.float64(1e-20) * a * c):       <L 108>
            var_41 = wp::float64(var_40);
            var_42 = (var_38 > var_41);
            var_43 = (var_38 < var_33);
            var_44 = wp::dot(var_39, var_2);
            var_46 = wp::float64(var_45);
            var_47 = (var_44 == var_46);
            var_48 = wp::length_sq(var_39);
            var_50 = wp::float64(var_49);
            var_51 = wp::mul(var_50, var_3);
            var_52 = wp::mul(var_51, var_5);
            var_53 = (var_48 < var_52);
            var_54 = var_47 || var_53;
            var_55 = var_42 && var_43 && var_54;
            if (var_55) {
                // if sN < D / wp.float64(2.0):                                               <L 113>
                var_57 = wp::float64(var_56);
                var_58 = wp::div(var_10, var_57);
                var_59 = (var_19 < var_58);
                if (var_59) {
                    // tN = e                                                                 <L 114>
                    var_60 = wp::copy(var_7);
                    // tD = c                                                                 <L 115>
                    var_61 = wp::copy(var_5);
                    // defaultCase = 2                                                        <L 116>
                }
                var_63 = wp::where(var_59, var_61, var_33);
                var_64 = wp::where(var_59, var_62, var_34);
                var_65 = wp::where(var_59, var_60, var_38);
                if (!var_59) {
                    // tN = e + b                                                             <L 118>
                    var_66 = wp::add(var_7, var_4);
                    // tD = c                                                                 <L 119>
                    var_67 = wp::copy(var_5);
                    // defaultCase = 5                                                        <L 120>
                }
                var_69 = wp::where(var_59, var_63, var_67);
                var_70 = wp::where(var_59, var_64, var_68);
                var_71 = wp::where(var_59, var_65, var_66);
            }
            var_72 = wp::where(var_55, var_69, var_33);
            var_73 = wp::where(var_55, var_70, var_34);
            var_74 = wp::where(var_55, var_71, var_38);
        }
        var_75 = wp::where(var_29, var_33, var_72);
        var_76 = wp::where(var_29, var_34, var_73);
        var_77 = wp::where(var_29, var_35, var_74);
    }
    var_78 = wp::where(var_22, var_26, var_75);
    var_79 = wp::where(var_22, var_27, var_76);
    var_80 = wp::where(var_22, var_28, var_77);
    // if tN <= wp.float64(0.0):  # tc < 0 => the t=0 edge is visible                         <L 123>
    var_82 = wp::float64(var_81);
    var_83 = (var_80 <= var_82);
    if (var_83) {
        // if -d <= wp.float64(0.0):                                                          <L 125>
        var_84 = wp::neg(var_6);
        var_86 = wp::float64(var_85);
        var_87 = (var_84 <= var_86);
        if (var_87) {
            // return 0                                                                       <L 126>
            goto label0;
        }
        if (!var_87) {
            // elif -d >= a:                                                                  <L 127>
            var_89 = wp::neg(var_6);
            var_90 = (var_89 >= var_3);
            if (var_90) {
                // return 3                                                                   <L 128>
                goto label1;
            }
            if (!var_90) {
                // return 6                                                                   <L 130>
                goto label2;
            }
        }
    }
    if (!var_83) {
        // elif tN >= tD:  # tc > 1  => the t=1 edge is visible                               <L 131>
        var_93 = (var_80 >= var_78);
        if (var_93) {
            // if (-d + b) <= wp.float64(0.0):                                                <L 133>
            var_94 = wp::neg(var_6);
            var_95 = wp::add(var_94, var_4);
            var_97 = wp::float64(var_96);
            var_98 = (var_95 <= var_97);
            if (var_98) {
                // return 1                                                                   <L 134>
                goto label3;
            }
            if (!var_98) {
                // elif (-d + b) >= a:                                                        <L 135>
                var_100 = wp::neg(var_6);
                var_101 = wp::add(var_100, var_4);
                var_102 = (var_101 >= var_3);
                if (var_102) {
                    // return 4                                                               <L 136>
                    goto label4;
                }
                if (!var_102) {
                    // return 7                                                               <L 138>
                    goto label5;
                }
            }
        }
    }
    // return defaultCase                                                                     <L 140>
    goto label6;
    //---------
    // reverse
    label6:;
    adj_79 += adj_ret;
    // adj: return defaultCase                                                                <L 140>
    if (!var_83) {
        if (var_93) {
            if (!var_98) {
                if (!var_102) {
                    label5:;
                    adj_104 += adj_ret;
                    // adj: return 7                                                          <L 138>
                }
                if (var_102) {
                    label4:;
                    adj_103 += adj_ret;
                    // adj: return 4                                                          <L 136>
                }
                wp::adj_add(var_100, var_4, adj_100, adj_4, adj_101);
                wp::adj_neg(var_6, adj_6, adj_100);
                // adj: elif (-d + b) >= a:                                                   <L 135>
            }
            if (var_98) {
                label3:;
                adj_99 += adj_ret;
                // adj: return 1                                                              <L 134>
            }
            wp::adj_float64(var_96, adj_96, adj_97);
            wp::adj_add(var_94, var_4, adj_94, adj_4, adj_95);
            wp::adj_neg(var_6, adj_6, adj_94);
            // adj: if (-d + b) <= wp.float64(0.0):                                           <L 133>
        }
        // adj: elif tN >= tD:  # tc > 1  => the t=1 edge is visible                          <L 131>
    }
    if (var_83) {
        if (!var_87) {
            if (!var_90) {
                label2:;
                adj_92 += adj_ret;
                // adj: return 6                                                              <L 130>
            }
            if (var_90) {
                label1:;
                adj_91 += adj_ret;
                // adj: return 3                                                              <L 128>
            }
            wp::adj_neg(var_6, adj_6, adj_89);
            // adj: elif -d >= a:                                                             <L 127>
        }
        if (var_87) {
            label0:;
            adj_88 += adj_ret;
            // adj: return 0                                                                  <L 126>
        }
        wp::adj_float64(var_85, adj_85, adj_86);
        wp::adj_neg(var_6, adj_6, adj_84);
        // adj: if -d <= wp.float64(0.0):                                                     <L 125>
    }
    wp::adj_float64(var_81, adj_81, adj_82);
    // adj: if tN <= wp.float64(0.0):  # tc < 0 => the t=0 edge is visible                    <L 123>
    wp::adj_where(var_22, var_28, var_77, adj_22, adj_28, adj_77, adj_80);
    wp::adj_where(var_22, var_27, var_76, adj_22, adj_27, adj_76, adj_79);
    wp::adj_where(var_22, var_26, var_75, adj_22, adj_26, adj_75, adj_78);
    if (!var_22) {
        wp::adj_where(var_29, var_35, var_74, adj_29, adj_35, adj_74, adj_77);
        wp::adj_where(var_29, var_34, var_73, adj_29, adj_34, adj_73, adj_76);
        wp::adj_where(var_29, var_33, var_72, adj_29, adj_33, adj_72, adj_75);
        if (!var_29) {
            wp::adj_where(var_55, var_71, var_38, adj_55, adj_71, adj_38, adj_74);
            wp::adj_where(var_55, var_70, var_34, adj_55, adj_70, adj_34, adj_73);
            wp::adj_where(var_55, var_69, var_33, adj_55, adj_69, adj_33, adj_72);
            if (var_55) {
                wp::adj_where(var_59, var_65, var_66, adj_59, adj_65, adj_66, adj_71);
                wp::adj_where(var_59, var_64, var_68, adj_59, adj_64, adj_68, adj_70);
                wp::adj_where(var_59, var_63, var_67, adj_59, adj_63, adj_67, adj_69);
                if (!var_59) {
                    // adj: defaultCase = 5                                                   <L 120>
                    wp::adj_copy(var_5, adj_5, adj_67);
                    // adj: tD = c                                                            <L 119>
                    wp::adj_add(var_7, var_4, adj_7, adj_4, adj_66);
                    // adj: tN = e + b                                                        <L 118>
                }
                wp::adj_where(var_59, var_60, var_38, adj_59, adj_60, adj_38, adj_65);
                wp::adj_where(var_59, var_62, var_34, adj_59, adj_62, adj_34, adj_64);
                wp::adj_where(var_59, var_61, var_33, adj_59, adj_61, adj_33, adj_63);
                if (var_59) {
                    // adj: defaultCase = 2                                                   <L 116>
                    wp::adj_copy(var_5, adj_5, adj_61);
                    // adj: tD = c                                                            <L 115>
                    wp::adj_copy(var_7, adj_7, adj_60);
                    // adj: tN = e                                                            <L 114>
                }
                wp::adj_div(var_10, var_57, var_58, adj_10, adj_57, adj_58);
                wp::adj_float64(var_56, adj_56, adj_57);
                // adj: if sN < D / wp.float64(2.0):                                          <L 113>
            }
            wp::adj_mul(var_51, var_5, adj_51, adj_5, adj_52);
            wp::adj_mul(var_50, var_3, adj_50, adj_3, adj_51);
            wp::adj_float64(var_49, adj_49, adj_50);
            wp::adj_length_sq(var_39, adj_39, adj_48);
            wp::adj_float64(var_45, adj_45, adj_46);
            wp::adj_dot(var_39, var_2, adj_39, adj_2, adj_44);
            wp::adj_float64(var_40, adj_40, adj_41);
            // adj: if tN > wp.float64(0.0) and tN < tD and (wp.dot(uv, w) == wp.float64(0.0) or wp.length_sq(uv) < wp.float64(1e-20) * a * c):  <L 108>
            wp::adj_cross(var_0, var_1, adj_0, adj_1, adj_39);
            // adj: uv = wp.cross(u, v)                                                       <L 106>
            wp::adj_sub(var_36, var_37, adj_36, adj_37, adj_38);
            wp::adj_mul(var_4, var_6, adj_4, adj_6, adj_37);
            wp::adj_mul(var_3, var_7, adj_3, adj_7, adj_36);
            // adj: tN = a * e - b * d                                                        <L 105>
        }
        wp::adj_where(var_29, var_30, var_28, adj_29, adj_30, adj_28, adj_35);
        wp::adj_where(var_29, var_32, var_27, adj_29, adj_32, adj_27, adj_34);
        wp::adj_where(var_29, var_31, var_26, adj_29, adj_31, adj_26, adj_33);
        if (var_29) {
            // adj: defaultCase = 5                                                           <L 103>
            wp::adj_copy(var_5, adj_5, adj_31);
            // adj: tD = c                                                                    <L 102>
            wp::adj_add(var_7, var_4, adj_7, adj_4, adj_30);
            // adj: tN = e + b                                                                <L 101>
        }
        // adj: elif sN >= D:  # sc > 1  => the s=1 edge is visible                           <L 100>
    }
    wp::adj_where(var_22, var_23, var_16, adj_22, adj_23, adj_16, adj_28);
    wp::adj_where(var_22, var_25, var_12, adj_22, adj_25, adj_12, adj_27);
    wp::adj_where(var_22, var_24, var_11, adj_22, adj_24, adj_11, adj_26);
    if (var_22) {
        // adj: defaultCase = 2                                                               <L 99>
        wp::adj_copy(var_5, adj_5, adj_24);
        // adj: tD = c                                                                        <L 98>
        wp::adj_copy(var_7, adj_7, adj_23);
        // adj: tN = e                                                                        <L 97>
    }
    wp::adj_float64(var_20, adj_20, adj_21);
    // adj: if sN <= wp.float64(0.0):  # sc < 0 => the s=0 edge is visible                    <L 96>
    wp::adj_sub(var_17, var_18, adj_17, adj_18, adj_19);
    wp::adj_mul(var_5, var_6, adj_5, adj_6, adj_18);
    wp::adj_mul(var_4, var_7, adj_4, adj_7, adj_17);
    // adj: sN = b * e - c * d                                                                <L 95>
    wp::adj_float64(var_15, adj_15, adj_16);
    wp::adj_float64(var_13, adj_13, adj_14);
    // adj: sN, tN = wp.float64(0.0), wp.float64(0.0)                                         <L 92>
    // adj: defaultCase = 8                                                                   <L 91>
    wp::adj_copy(var_10, adj_10, adj_11);
    // adj: tD = D  # tc = tN / tD, default tD = D >= 0                                       <L 90>
    wp::adj_sub(var_8, var_9, adj_8, adj_9, adj_10);
    wp::adj_mul(var_4, var_4, adj_4, adj_4, adj_9);
    wp::adj_mul(var_3, var_5, adj_3, adj_5, adj_8);
    // adj: D = a * c - b * b  # always >= 0                                                  <L 89>
    wp::adj_dot(var_1, var_2, adj_1, adj_2, adj_7);
    // adj: e = wp.dot(v, w)                                                                  <L 88>
    wp::adj_dot(var_0, var_2, adj_0, adj_2, adj_6);
    // adj: d = wp.dot(u, w)                                                                  <L 87>
    wp::adj_length_sq(var_1, adj_1, adj_5);
    // adj: c = wp.length_sq(v)  # always >= 0                                                <L 86>
    wp::adj_dot(var_0, var_1, adj_0, adj_1, adj_4);
    // adj: b = wp.dot(u, v)                                                                  <L 85>
    wp::adj_length_sq(var_0, adj_0, adj_3);
    // adj: a = wp.length_sq(u)  # always >= 0                                                <L 84>
    wp::adj_sub(var_ea0, var_eb0, adj_ea0, adj_eb0, adj_2);
    // adj: w = ea0 - eb0                                                                     <L 82>
    wp::adj_sub(var_eb1, var_eb0, adj_eb1, adj_eb0, adj_1);
    // adj: v = eb1 - eb0                                                                     <L 81>
    wp::adj_sub(var_ea1, var_ea0, adj_ea1, adj_ea0, adj_0);
    // adj: u = ea1 - ea0                                                                     <L 80>
    // adj: def edge_edge_distance_type(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d) -> int:  <L 79>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/line_line_distance.py:507
static CUDA_CALLABLE wp::float64 line_line_distance_0(
    wp::vec_t<3,wp::float64> var_ea0,
    wp::vec_t<3,wp::float64> var_ea1,
    wp::vec_t<3,wp::float64> var_eb0,
    wp::vec_t<3,wp::float64> var_eb1)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    //---------
    // forward
    // def line_line_distance(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d):       <L 508>
    // da = ea1 - ea0                                                                         <L 509>
    var_0 = wp::sub(var_ea1, var_ea0);
    // db = eb1 - eb0                                                                         <L 510>
    var_1 = wp::sub(var_eb1, var_eb0);
    // d = eb0 - ea0                                                                          <L 511>
    var_2 = wp::sub(var_eb0, var_ea0);
    // b = wp.cross(da, db)                                                                   <L 513>
    var_3 = wp::cross(var_0, var_1);
    // aTb = wp.dot(d, b)                                                                     <L 514>
    var_4 = wp::dot(var_2, var_3);
    // return aTb * aTb / wp.length_sq(b)                                                     <L 515>
    var_5 = wp::mul(var_4, var_4);
    var_6 = wp::length_sq(var_3);
    var_7 = wp::div(var_5, var_6);
    return var_7;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/line_line_distance.py:507
static CUDA_CALLABLE void adj_line_line_distance_0(
    wp::vec_t<3,wp::float64> var_ea0,
    wp::vec_t<3,wp::float64> var_ea1,
    wp::vec_t<3,wp::float64> var_eb0,
    wp::vec_t<3,wp::float64> var_eb1,
    wp::vec_t<3,wp::float64> & adj_ea0,
    wp::vec_t<3,wp::float64> & adj_ea1,
    wp::vec_t<3,wp::float64> & adj_eb0,
    wp::vec_t<3,wp::float64> & adj_eb1,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::vec_t<3,wp::float64> var_0;
    wp::vec_t<3,wp::float64> var_1;
    wp::vec_t<3,wp::float64> var_2;
    wp::vec_t<3,wp::float64> var_3;
    wp::float64 var_4;
    wp::float64 var_5;
    wp::float64 var_6;
    wp::float64 var_7;
    //---------
    // dual vars
    wp::vec_t<3,wp::float64> adj_0 = {};
    wp::vec_t<3,wp::float64> adj_1 = {};
    wp::vec_t<3,wp::float64> adj_2 = {};
    wp::vec_t<3,wp::float64> adj_3 = {};
    wp::float64 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::float64 adj_6 = {};
    wp::float64 adj_7 = {};
    //---------
    // forward
    // def line_line_distance(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d):       <L 508>
    // da = ea1 - ea0                                                                         <L 509>
    var_0 = wp::sub(var_ea1, var_ea0);
    // db = eb1 - eb0                                                                         <L 510>
    var_1 = wp::sub(var_eb1, var_eb0);
    // d = eb0 - ea0                                                                          <L 511>
    var_2 = wp::sub(var_eb0, var_ea0);
    // b = wp.cross(da, db)                                                                   <L 513>
    var_3 = wp::cross(var_0, var_1);
    // aTb = wp.dot(d, b)                                                                     <L 514>
    var_4 = wp::dot(var_2, var_3);
    // return aTb * aTb / wp.length_sq(b)                                                     <L 515>
    var_5 = wp::mul(var_4, var_4);
    var_6 = wp::length_sq(var_3);
    var_7 = wp::div(var_5, var_6);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_7 += adj_ret;
    wp::adj_div(var_5, var_6, var_7, adj_5, adj_6, adj_7);
    wp::adj_length_sq(var_3, adj_3, adj_6);
    wp::adj_mul(var_4, var_4, adj_4, adj_4, adj_5);
    // adj: return aTb * aTb / wp.length_sq(b)                                                <L 515>
    wp::adj_dot(var_2, var_3, adj_2, adj_3, adj_4);
    // adj: aTb = wp.dot(d, b)                                                                <L 514>
    wp::adj_cross(var_0, var_1, adj_0, adj_1, adj_3);
    // adj: b = wp.cross(da, db)                                                              <L 513>
    wp::adj_sub(var_eb0, var_ea0, adj_eb0, adj_ea0, adj_2);
    // adj: d = eb0 - ea0                                                                     <L 511>
    wp::adj_sub(var_eb1, var_eb0, adj_eb1, adj_eb0, adj_1);
    // adj: db = eb1 - eb0                                                                    <L 510>
    wp::adj_sub(var_ea1, var_ea0, adj_ea1, adj_ea0, adj_0);
    // adj: da = ea1 - ea0                                                                    <L 509>
    // adj: def line_line_distance(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d):  <L 508>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/edge_edge_distance.py:9
static CUDA_CALLABLE wp::float64 edge_edge_distance_0(
    wp::vec_t<3,wp::float64> var_ea0,
    wp::vec_t<3,wp::float64> var_ea1,
    wp::vec_t<3,wp::float64> var_eb0,
    wp::vec_t<3,wp::float64> var_eb1)
{
    //---------
    // primal vars
    wp::int32 var_0;
    const wp::int32 var_1 = 0;
    bool var_2;
    wp::float64 var_3;
    const wp::int32 var_4 = 1;
    bool var_5;
    wp::float64 var_6;
    const wp::int32 var_7 = 2;
    bool var_8;
    wp::float64 var_9;
    const wp::int32 var_10 = 3;
    bool var_11;
    wp::float64 var_12;
    const wp::int32 var_13 = 4;
    bool var_14;
    wp::float64 var_15;
    const wp::int32 var_16 = 5;
    bool var_17;
    wp::float64 var_18;
    const wp::int32 var_19 = 6;
    bool var_20;
    wp::float64 var_21;
    const wp::int32 var_22 = 7;
    bool var_23;
    wp::float64 var_24;
    const wp::int32 var_25 = 8;
    bool var_26;
    wp::float64 var_27;
    const wp::float32 var_28 = 1e+20;
    wp::float64 var_29;
    //---------
    // forward
    // def edge_edge_distance(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d):       <L 10>
    // dt = edge_edge_distance_type(ea0, ea1, eb0, eb1)                                       <L 11>
    var_0 = edge_edge_distance_type_0(var_ea0, var_ea1, var_eb0, var_eb1);
    // if dt == 0:                                                                            <L 12>
    var_2 = (var_0 == var_1);
    if (var_2) {
        // return point_point_distance(ea0, eb0)                                              <L 13>
        var_3 = point_point_distance_0(var_ea0, var_eb0);
        return var_3;
    }
    if (!var_2) {
        // elif dt == 1:                                                                      <L 14>
        var_5 = (var_0 == var_4);
        if (var_5) {
            // return point_point_distance(ea0, eb1)                                          <L 15>
            var_6 = point_point_distance_0(var_ea0, var_eb1);
            return var_6;
        }
        if (!var_5) {
            // elif dt == 2:                                                                  <L 16>
            var_8 = (var_0 == var_7);
            if (var_8) {
                // return point_line_distance(ea0, eb0, eb1)                                  <L 17>
                var_9 = point_line_distance_0(var_ea0, var_eb0, var_eb1);
                return var_9;
            }
            if (!var_8) {
                // elif dt == 3:                                                              <L 18>
                var_11 = (var_0 == var_10);
                if (var_11) {
                    // return point_point_distance(ea1, eb0)                                  <L 19>
                    var_12 = point_point_distance_0(var_ea1, var_eb0);
                    return var_12;
                }
                if (!var_11) {
                    // elif dt == 4:                                                          <L 20>
                    var_14 = (var_0 == var_13);
                    if (var_14) {
                        // return point_point_distance(ea1, eb1)                              <L 21>
                        var_15 = point_point_distance_0(var_ea1, var_eb1);
                        return var_15;
                    }
                    if (!var_14) {
                        // elif dt == 5:                                                      <L 22>
                        var_17 = (var_0 == var_16);
                        if (var_17) {
                            // return point_line_distance(ea1, eb0, eb1)                      <L 23>
                            var_18 = point_line_distance_0(var_ea1, var_eb0, var_eb1);
                            return var_18;
                        }
                        if (!var_17) {
                            // elif dt == 6:                                                  <L 24>
                            var_20 = (var_0 == var_19);
                            if (var_20) {
                                // return point_line_distance(eb0, ea0, ea1)                  <L 25>
                                var_21 = point_line_distance_0(var_eb0, var_ea0, var_ea1);
                                return var_21;
                            }
                            if (!var_20) {
                                // elif dt == 7:                                              <L 26>
                                var_23 = (var_0 == var_22);
                                if (var_23) {
                                    // return point_line_distance(eb1, ea0, ea1)              <L 27>
                                    var_24 = point_line_distance_0(var_eb1, var_ea0, var_ea1);
                                    return var_24;
                                }
                                if (!var_23) {
                                    // elif dt == 8:                                          <L 28>
                                    var_26 = (var_0 == var_25);
                                    if (var_26) {
                                        // return line_line_distance(ea0, ea1, eb0, eb1)       <L 29>
                                        var_27 = line_line_distance_0(var_ea0, var_ea1, var_eb0, var_eb1);
                                        return var_27;
                                    }
                                    if (!var_26) {
                                        // return wp.float64(1e20)                            <L 31>
                                        var_29 = wp::float64(var_28);
                                        return var_29;
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    return {};
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/contact/edge_edge_distance.py:9
static CUDA_CALLABLE void adj_edge_edge_distance_0(
    wp::vec_t<3,wp::float64> var_ea0,
    wp::vec_t<3,wp::float64> var_ea1,
    wp::vec_t<3,wp::float64> var_eb0,
    wp::vec_t<3,wp::float64> var_eb1,
    wp::vec_t<3,wp::float64> & adj_ea0,
    wp::vec_t<3,wp::float64> & adj_ea1,
    wp::vec_t<3,wp::float64> & adj_eb0,
    wp::vec_t<3,wp::float64> & adj_eb1,
    wp::float64 & adj_ret)
{
    //---------
    // primal vars
    wp::int32 var_0;
    const wp::int32 var_1 = 0;
    bool var_2;
    wp::float64 var_3;
    const wp::int32 var_4 = 1;
    bool var_5;
    wp::float64 var_6;
    const wp::int32 var_7 = 2;
    bool var_8;
    wp::float64 var_9;
    const wp::int32 var_10 = 3;
    bool var_11;
    wp::float64 var_12;
    const wp::int32 var_13 = 4;
    bool var_14;
    wp::float64 var_15;
    const wp::int32 var_16 = 5;
    bool var_17;
    wp::float64 var_18;
    const wp::int32 var_19 = 6;
    bool var_20;
    wp::float64 var_21;
    const wp::int32 var_22 = 7;
    bool var_23;
    wp::float64 var_24;
    const wp::int32 var_25 = 8;
    bool var_26;
    wp::float64 var_27;
    const wp::float32 var_28 = 1e+20;
    wp::float64 var_29;
    //---------
    // dual vars
    wp::int32 adj_0 = {};
    wp::int32 adj_1 = {};
    bool adj_2 = {};
    wp::float64 adj_3 = {};
    wp::int32 adj_4 = {};
    bool adj_5 = {};
    wp::float64 adj_6 = {};
    wp::int32 adj_7 = {};
    bool adj_8 = {};
    wp::float64 adj_9 = {};
    wp::int32 adj_10 = {};
    bool adj_11 = {};
    wp::float64 adj_12 = {};
    wp::int32 adj_13 = {};
    bool adj_14 = {};
    wp::float64 adj_15 = {};
    wp::int32 adj_16 = {};
    bool adj_17 = {};
    wp::float64 adj_18 = {};
    wp::int32 adj_19 = {};
    bool adj_20 = {};
    wp::float64 adj_21 = {};
    wp::int32 adj_22 = {};
    bool adj_23 = {};
    wp::float64 adj_24 = {};
    wp::int32 adj_25 = {};
    bool adj_26 = {};
    wp::float64 adj_27 = {};
    wp::float32 adj_28 = {};
    wp::float64 adj_29 = {};
    //---------
    // forward
    // def edge_edge_distance(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d):       <L 10>
    // dt = edge_edge_distance_type(ea0, ea1, eb0, eb1)                                       <L 11>
    var_0 = edge_edge_distance_type_0(var_ea0, var_ea1, var_eb0, var_eb1);
    // if dt == 0:                                                                            <L 12>
    var_2 = (var_0 == var_1);
    if (var_2) {
        // return point_point_distance(ea0, eb0)                                              <L 13>
        var_3 = point_point_distance_0(var_ea0, var_eb0);
        goto label0;
    }
    if (!var_2) {
        // elif dt == 1:                                                                      <L 14>
        var_5 = (var_0 == var_4);
        if (var_5) {
            // return point_point_distance(ea0, eb1)                                          <L 15>
            var_6 = point_point_distance_0(var_ea0, var_eb1);
            goto label1;
        }
        if (!var_5) {
            // elif dt == 2:                                                                  <L 16>
            var_8 = (var_0 == var_7);
            if (var_8) {
                // return point_line_distance(ea0, eb0, eb1)                                  <L 17>
                var_9 = point_line_distance_0(var_ea0, var_eb0, var_eb1);
                goto label2;
            }
            if (!var_8) {
                // elif dt == 3:                                                              <L 18>
                var_11 = (var_0 == var_10);
                if (var_11) {
                    // return point_point_distance(ea1, eb0)                                  <L 19>
                    var_12 = point_point_distance_0(var_ea1, var_eb0);
                    goto label3;
                }
                if (!var_11) {
                    // elif dt == 4:                                                          <L 20>
                    var_14 = (var_0 == var_13);
                    if (var_14) {
                        // return point_point_distance(ea1, eb1)                              <L 21>
                        var_15 = point_point_distance_0(var_ea1, var_eb1);
                        goto label4;
                    }
                    if (!var_14) {
                        // elif dt == 5:                                                      <L 22>
                        var_17 = (var_0 == var_16);
                        if (var_17) {
                            // return point_line_distance(ea1, eb0, eb1)                      <L 23>
                            var_18 = point_line_distance_0(var_ea1, var_eb0, var_eb1);
                            goto label5;
                        }
                        if (!var_17) {
                            // elif dt == 6:                                                  <L 24>
                            var_20 = (var_0 == var_19);
                            if (var_20) {
                                // return point_line_distance(eb0, ea0, ea1)                  <L 25>
                                var_21 = point_line_distance_0(var_eb0, var_ea0, var_ea1);
                                goto label6;
                            }
                            if (!var_20) {
                                // elif dt == 7:                                              <L 26>
                                var_23 = (var_0 == var_22);
                                if (var_23) {
                                    // return point_line_distance(eb1, ea0, ea1)              <L 27>
                                    var_24 = point_line_distance_0(var_eb1, var_ea0, var_ea1);
                                    goto label7;
                                }
                                if (!var_23) {
                                    // elif dt == 8:                                          <L 28>
                                    var_26 = (var_0 == var_25);
                                    if (var_26) {
                                        // return line_line_distance(ea0, ea1, eb0, eb1)       <L 29>
                                        var_27 = line_line_distance_0(var_ea0, var_ea1, var_eb0, var_eb1);
                                        goto label8;
                                    }
                                    if (!var_26) {
                                        // return wp.float64(1e20)                            <L 31>
                                        var_29 = wp::float64(var_28);
                                        goto label9;
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    //---------
    // reverse
    if (!var_2) {
        if (!var_5) {
            if (!var_8) {
                if (!var_11) {
                    if (!var_14) {
                        if (!var_17) {
                            if (!var_20) {
                                if (!var_23) {
                                    if (!var_26) {
                                        label9:;
                                        adj_29 += adj_ret;
                                        wp::adj_float64(var_28, adj_28, adj_29);
                                        // adj: return wp.float64(1e20)                       <L 31>
                                    }
                                    if (var_26) {
                                        label8:;
                                        adj_27 += adj_ret;
                                        adj_line_line_distance_0(var_ea0, var_ea1, var_eb0, var_eb1, adj_ea0, adj_ea1, adj_eb0, adj_eb1, adj_27);
                                        // adj: return line_line_distance(ea0, ea1, eb0, eb1)  <L 29>
                                    }
                                    // adj: elif dt == 8:                                     <L 28>
                                }
                                if (var_23) {
                                    label7:;
                                    adj_24 += adj_ret;
                                    adj_point_line_distance_0(var_eb1, var_ea0, var_ea1, adj_eb1, adj_ea0, adj_ea1, adj_24);
                                    // adj: return point_line_distance(eb1, ea0, ea1)         <L 27>
                                }
                                // adj: elif dt == 7:                                         <L 26>
                            }
                            if (var_20) {
                                label6:;
                                adj_21 += adj_ret;
                                adj_point_line_distance_0(var_eb0, var_ea0, var_ea1, adj_eb0, adj_ea0, adj_ea1, adj_21);
                                // adj: return point_line_distance(eb0, ea0, ea1)             <L 25>
                            }
                            // adj: elif dt == 6:                                             <L 24>
                        }
                        if (var_17) {
                            label5:;
                            adj_18 += adj_ret;
                            adj_point_line_distance_0(var_ea1, var_eb0, var_eb1, adj_ea1, adj_eb0, adj_eb1, adj_18);
                            // adj: return point_line_distance(ea1, eb0, eb1)                 <L 23>
                        }
                        // adj: elif dt == 5:                                                 <L 22>
                    }
                    if (var_14) {
                        label4:;
                        adj_15 += adj_ret;
                        adj_point_point_distance_0(var_ea1, var_eb1, adj_ea1, adj_eb1, adj_15);
                        // adj: return point_point_distance(ea1, eb1)                         <L 21>
                    }
                    // adj: elif dt == 4:                                                     <L 20>
                }
                if (var_11) {
                    label3:;
                    adj_12 += adj_ret;
                    adj_point_point_distance_0(var_ea1, var_eb0, adj_ea1, adj_eb0, adj_12);
                    // adj: return point_point_distance(ea1, eb0)                             <L 19>
                }
                // adj: elif dt == 3:                                                         <L 18>
            }
            if (var_8) {
                label2:;
                adj_9 += adj_ret;
                adj_point_line_distance_0(var_ea0, var_eb0, var_eb1, adj_ea0, adj_eb0, adj_eb1, adj_9);
                // adj: return point_line_distance(ea0, eb0, eb1)                             <L 17>
            }
            // adj: elif dt == 2:                                                             <L 16>
        }
        if (var_5) {
            label1:;
            adj_6 += adj_ret;
            adj_point_point_distance_0(var_ea0, var_eb1, adj_ea0, adj_eb1, adj_6);
            // adj: return point_point_distance(ea0, eb1)                                     <L 15>
        }
        // adj: elif dt == 1:                                                                 <L 14>
    }
    if (var_2) {
        label0:;
        adj_3 += adj_ret;
        adj_point_point_distance_0(var_ea0, var_eb0, adj_ea0, adj_eb0, adj_3);
        // adj: return point_point_distance(ea0, eb0)                                         <L 13>
    }
    // adj: if dt == 0:                                                                       <L 12>
    adj_edge_edge_distance_type_0(var_ea0, var_ea1, var_eb0, var_eb1, adj_ea0, adj_ea1, adj_eb0, adj_eb1, adj_0);
    // adj: dt = edge_edge_distance_type(ea0, ea1, eb0, eb1)                                  <L 11>
    // adj: def edge_edge_distance(ea0: wp.vec3d, ea1: wp.vec3d, eb0: wp.vec3d, eb1: wp.vec3d):  <L 10>
    return;
}



extern "C" __global__ void dist_IPC_hs_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy_x,
    wp::array_t<wp::int32> var_hs_node,
    wp::array_t<wp::int32> var_hs_ground,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::int32> var_surf_vi,
    wp::array_t<wp::float64> var_node_xi,
    wp::array_t<wp::vec_t<3,wp::float64>> var_half_space_n,
    wp::array_t<wp::vec_t<3,wp::float64>> var_half_space_o)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        wp::float64* var_4;
        wp::float64 var_5;
        wp::float64 var_6;
        wp::int32* var_7;
        wp::int32 var_8;
        wp::int32 var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        wp::int32 var_12;
        wp::vec_t<3,wp::float64>* var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64> var_15;
        wp::vec_t<3,wp::float64>* var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::vec_t<3,wp::float64> var_18;
        wp::vec_t<3,wp::float64>* var_19;
        wp::vec_t<3,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::vec_t<3,wp::float64> var_22;
        wp::float64 var_23;
        wp::float64 var_24;
        //---------
        // forward
        // def dist_IPC_hs(                                                                       <L 18>
        // idx = wp.tid()                                                                         <L 28>
        var_0 = builtin_tid1d();
        // svi = hs_node[idx]                                                                     <L 29>
        var_1 = wp::address(var_hs_node, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // xi = node_xi[svi]                                                                      <L 30>
        var_4 = wp::address(var_node_xi, var_2);
        var_6 = wp::load(var_4);
        var_5 = wp::copy(var_6);
        // xI = surf_vi[svi]                                                                      <L 31>
        var_7 = wp::address(var_surf_vi, var_2);
        var_9 = wp::load(var_7);
        var_8 = wp::copy(var_9);
        // hs = hs_ground[idx]                                                                    <L 32>
        var_10 = wp::address(var_hs_ground, var_0);
        var_12 = wp::load(var_10);
        var_11 = wp::copy(var_12);
        // n = half_space_n[hs]                                                                   <L 33>
        var_13 = wp::address(var_half_space_n, var_11);
        var_15 = wp::load(var_13);
        var_14 = wp::copy(var_15);
        // o = half_space_o[hs]                                                                   <L 34>
        var_16 = wp::address(var_half_space_o, var_11);
        var_18 = wp::load(var_16);
        var_17 = wp::copy(var_18);
        // xIcoord = x[xI]                                                                        <L 35>
        var_19 = wp::address(var_x, var_8);
        var_21 = wp::load(var_19);
        var_20 = wp::copy(var_21);
        // d = wp.dot(xIcoord - o, n) - xi                                                        <L 36>
        var_22 = wp::sub(var_20, var_17);
        var_23 = wp::dot(var_22, var_14);
        var_24 = wp::sub(var_23, var_5);
        // energy_x[idx] = d                                                                      <L 37>
        wp::array_store(var_energy_x, var_0, var_24);
    }
}



extern "C" __global__ void dist_IPC_hs_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy_x,
    wp::array_t<wp::int32> var_hs_node,
    wp::array_t<wp::int32> var_hs_ground,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::int32> var_surf_vi,
    wp::array_t<wp::float64> var_node_xi,
    wp::array_t<wp::vec_t<3,wp::float64>> var_half_space_n,
    wp::array_t<wp::vec_t<3,wp::float64>> var_half_space_o,
    wp::array_t<wp::float64> adj_energy_x,
    wp::array_t<wp::int32> adj_hs_node,
    wp::array_t<wp::int32> adj_hs_ground,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::int32> adj_surf_vi,
    wp::array_t<wp::float64> adj_node_xi,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_half_space_n,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_half_space_o)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        wp::float64* var_4;
        wp::float64 var_5;
        wp::float64 var_6;
        wp::int32* var_7;
        wp::int32 var_8;
        wp::int32 var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        wp::int32 var_12;
        wp::vec_t<3,wp::float64>* var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64> var_15;
        wp::vec_t<3,wp::float64>* var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::vec_t<3,wp::float64> var_18;
        wp::vec_t<3,wp::float64>* var_19;
        wp::vec_t<3,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::vec_t<3,wp::float64> var_22;
        wp::float64 var_23;
        wp::float64 var_24;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::float64 adj_4 = {};
        wp::float64 adj_5 = {};
        wp::float64 adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int32 adj_12 = {};
        wp::vec_t<3,wp::float64> adj_13 = {};
        wp::vec_t<3,wp::float64> adj_14 = {};
        wp::vec_t<3,wp::float64> adj_15 = {};
        wp::vec_t<3,wp::float64> adj_16 = {};
        wp::vec_t<3,wp::float64> adj_17 = {};
        wp::vec_t<3,wp::float64> adj_18 = {};
        wp::vec_t<3,wp::float64> adj_19 = {};
        wp::vec_t<3,wp::float64> adj_20 = {};
        wp::vec_t<3,wp::float64> adj_21 = {};
        wp::vec_t<3,wp::float64> adj_22 = {};
        wp::float64 adj_23 = {};
        wp::float64 adj_24 = {};
        //---------
        // forward
        // def dist_IPC_hs(                                                                       <L 18>
        // idx = wp.tid()                                                                         <L 28>
        var_0 = builtin_tid1d();
        // svi = hs_node[idx]                                                                     <L 29>
        var_1 = wp::address(var_hs_node, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // xi = node_xi[svi]                                                                      <L 30>
        var_4 = wp::address(var_node_xi, var_2);
        var_6 = wp::load(var_4);
        var_5 = wp::copy(var_6);
        // xI = surf_vi[svi]                                                                      <L 31>
        var_7 = wp::address(var_surf_vi, var_2);
        var_9 = wp::load(var_7);
        var_8 = wp::copy(var_9);
        // hs = hs_ground[idx]                                                                    <L 32>
        var_10 = wp::address(var_hs_ground, var_0);
        var_12 = wp::load(var_10);
        var_11 = wp::copy(var_12);
        // n = half_space_n[hs]                                                                   <L 33>
        var_13 = wp::address(var_half_space_n, var_11);
        var_15 = wp::load(var_13);
        var_14 = wp::copy(var_15);
        // o = half_space_o[hs]                                                                   <L 34>
        var_16 = wp::address(var_half_space_o, var_11);
        var_18 = wp::load(var_16);
        var_17 = wp::copy(var_18);
        // xIcoord = x[xI]                                                                        <L 35>
        var_19 = wp::address(var_x, var_8);
        var_21 = wp::load(var_19);
        var_20 = wp::copy(var_21);
        // d = wp.dot(xIcoord - o, n) - xi                                                        <L 36>
        var_22 = wp::sub(var_20, var_17);
        var_23 = wp::dot(var_22, var_14);
        var_24 = wp::sub(var_23, var_5);
        // energy_x[idx] = d                                                                      <L 37>
        // wp::array_store(var_energy_x, var_0, var_24);
        //---------
        // reverse
        wp::adj_array_store(var_energy_x, var_0, var_24, adj_energy_x, adj_0, adj_24);
        // adj: energy_x[idx] = d                                                                 <L 37>
        wp::adj_sub(var_23, var_5, adj_23, adj_5, adj_24);
        wp::adj_dot(var_22, var_14, adj_22, adj_14, adj_23);
        wp::adj_sub(var_20, var_17, adj_20, adj_17, adj_22);
        // adj: d = wp.dot(xIcoord - o, n) - xi                                                   <L 36>
        wp::adj_copy(var_21, adj_19, adj_20);
        wp::adj_load(var_19, adj_19, adj_21);
        wp::adj_address(var_x, var_8, adj_x, adj_8, adj_19);
        // adj: xIcoord = x[xI]                                                                   <L 35>
        wp::adj_copy(var_18, adj_16, adj_17);
        wp::adj_load(var_16, adj_16, adj_18);
        wp::adj_address(var_half_space_o, var_11, adj_half_space_o, adj_11, adj_16);
        // adj: o = half_space_o[hs]                                                              <L 34>
        wp::adj_copy(var_15, adj_13, adj_14);
        wp::adj_load(var_13, adj_13, adj_15);
        wp::adj_address(var_half_space_n, var_11, adj_half_space_n, adj_11, adj_13);
        // adj: n = half_space_n[hs]                                                              <L 33>
        wp::adj_copy(var_12, adj_10, adj_11);
        wp::adj_load(var_10, adj_10, adj_12);
        wp::adj_address(var_hs_ground, var_0, adj_hs_ground, adj_0, adj_10);
        // adj: hs = hs_ground[idx]                                                               <L 32>
        wp::adj_copy(var_9, adj_7, adj_8);
        wp::adj_load(var_7, adj_7, adj_9);
        wp::adj_address(var_surf_vi, var_2, adj_surf_vi, adj_2, adj_7);
        // adj: xI = surf_vi[svi]                                                                 <L 31>
        wp::adj_copy(var_6, adj_4, adj_5);
        wp::adj_load(var_4, adj_4, adj_6);
        wp::adj_address(var_node_xi, var_2, adj_node_xi, adj_2, adj_4);
        // adj: xi = node_xi[svi]                                                                 <L 30>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_hs_node, var_0, adj_hs_node, adj_0, adj_1);
        // adj: svi = hs_node[idx]                                                                <L 29>
        // adj: idx = wp.tid()                                                                    <L 28>
        // adj: def dist_IPC_hs(                                                                  <L 18>
        continue;
    }
}



extern "C" __global__ void dist_IPC_collisions_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy_x,
    wp::array_t<wp::int32> var_nodeI,
    wp::array_t<wp::int32> var_nodeJ,
    wp::array_t<wp::int32> var_collision_type,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::int32> var_surf_vi,
    wp::array_t<wp::float64> var_node_xi,
    wp::array_t<wp::float64> var_edge_xi,
    wp::array_t<wp::float64> var_face_xi,
    wp::array_t<wp::vec_t<2,wp::int32>> var_edge,
    wp::array_t<wp::vec_t<3,wp::int32>> var_face)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        const wp::int32 var_2 = 0;
        bool var_3;
        wp::int32 var_4;
        wp::int32* var_5;
        const wp::int32 var_6 = 1;
        bool var_7;
        wp::int32 var_8;
        bool var_9;
        wp::int32* var_10;
        bool var_11;
        wp::int32 var_12;
        wp::int32* var_13;
        wp::int32* var_14;
        wp::int32 var_15;
        wp::int32 var_16;
        wp::int32 var_17;
        wp::int32* var_18;
        bool var_19;
        wp::int32 var_20;
        wp::int32* var_21;
        wp::int32* var_22;
        wp::int32 var_23;
        wp::int32 var_24;
        wp::int32 var_25;
        wp::vec_t<3,wp::int32>* var_26;
        wp::vec_t<3,wp::int32> var_27;
        wp::vec_t<3,wp::int32> var_28;
        wp::int32* var_29;
        wp::int32 var_30;
        wp::int32 var_31;
        wp::vec_t<3,wp::float64>* var_32;
        wp::vec_t<3,wp::float64> var_33;
        wp::vec_t<3,wp::float64> var_34;
        const wp::int32 var_35 = 0;
        wp::int32 var_36;
        wp::vec_t<3,wp::float64>* var_37;
        wp::vec_t<3,wp::float64> var_38;
        wp::vec_t<3,wp::float64> var_39;
        const wp::int32 var_40 = 1;
        wp::int32 var_41;
        wp::vec_t<3,wp::float64>* var_42;
        wp::vec_t<3,wp::float64> var_43;
        wp::vec_t<3,wp::float64> var_44;
        const wp::int32 var_45 = 2;
        wp::int32 var_46;
        wp::vec_t<3,wp::float64>* var_47;
        wp::vec_t<3,wp::float64> var_48;
        wp::vec_t<3,wp::float64> var_49;
        wp::float64* var_50;
        wp::float64* var_51;
        wp::float64 var_52;
        wp::float64 var_53;
        wp::float64 var_54;
        wp::float64 var_55;
        wp::float64 var_56;
        wp::float64 var_57;
        wp::float64 var_58;
        wp::int32* var_59;
        wp::int32 var_60;
        wp::int32 var_61;
        wp::int32* var_62;
        wp::int32 var_63;
        wp::int32 var_64;
        wp::vec_t<2,wp::int32>* var_65;
        wp::vec_t<2,wp::int32> var_66;
        wp::vec_t<2,wp::int32> var_67;
        wp::vec_t<2,wp::int32>* var_68;
        wp::vec_t<2,wp::int32> var_69;
        wp::vec_t<2,wp::int32> var_70;
        wp::float64* var_71;
        wp::float64* var_72;
        wp::float64 var_73;
        wp::float64 var_74;
        wp::float64 var_75;
        const wp::int32 var_76 = 0;
        wp::int32 var_77;
        wp::vec_t<3,wp::float64>* var_78;
        wp::vec_t<3,wp::float64> var_79;
        wp::vec_t<3,wp::float64> var_80;
        const wp::int32 var_81 = 1;
        wp::int32 var_82;
        wp::vec_t<3,wp::float64>* var_83;
        wp::vec_t<3,wp::float64> var_84;
        wp::vec_t<3,wp::float64> var_85;
        const wp::int32 var_86 = 0;
        wp::int32 var_87;
        wp::vec_t<3,wp::float64>* var_88;
        wp::vec_t<3,wp::float64> var_89;
        wp::vec_t<3,wp::float64> var_90;
        const wp::int32 var_91 = 1;
        wp::int32 var_92;
        wp::vec_t<3,wp::float64>* var_93;
        wp::vec_t<3,wp::float64> var_94;
        wp::vec_t<3,wp::float64> var_95;
        wp::float64 var_96;
        wp::float64 var_97;
        wp::float64 var_98;
        wp::float64 var_99;
        wp::float64 var_100;
        wp::float64 var_101;
        //---------
        // forward
        // def dist_IPC_collisions(                                                               <L 41>
        // idx = wp.tid()                                                                         <L 54>
        var_0 = builtin_tid1d();
        // if collision_type[idx] == TRIANGLE_POINT_PAIR or collision_type[idx] == POINT_TRIANGLE_PAIR:       <L 56>
        var_1 = wp::address(var_collision_type, var_0);
        var_4 = wp::load(var_1);
        var_3 = (var_4 == var_2);
        var_5 = wp::address(var_collision_type, var_0);
        var_8 = wp::load(var_5);
        var_7 = (var_8 == var_6);
        var_9 = var_3 || var_7;
        if (var_9) {
            // svi = wp.select(collision_type[idx] == POINT_TRIANGLE_PAIR, nodeJ[idx], nodeI[idx])       <L 57>
            var_10 = wp::address(var_collision_type, var_0);
            var_12 = wp::load(var_10);
            var_11 = (var_12 == var_6);
            var_13 = wp::address(var_nodeJ, var_0);
            var_14 = wp::address(var_nodeI, var_0);
            var_16 = wp::load(var_13);
            var_17 = wp::load(var_14);
            var_15 = wp::select(var_11, var_16, var_17);
            // faceJ = wp.select(collision_type[idx] == POINT_TRIANGLE_PAIR, nodeI[idx], nodeJ[idx])       <L 58>
            var_18 = wp::address(var_collision_type, var_0);
            var_20 = wp::load(var_18);
            var_19 = (var_20 == var_6);
            var_21 = wp::address(var_nodeI, var_0);
            var_22 = wp::address(var_nodeJ, var_0);
            var_24 = wp::load(var_21);
            var_25 = wp::load(var_22);
            var_23 = wp::select(var_19, var_24, var_25);
            // faceVInd = face[faceJ]                                                             <L 59>
            var_26 = wp::address(var_face, var_23);
            var_28 = wp::load(var_26);
            var_27 = wp::copy(var_28);
            // xI = surf_vi[svi]                                                                  <L 60>
            var_29 = wp::address(var_surf_vi, var_15);
            var_31 = wp::load(var_29);
            var_30 = wp::copy(var_31);
            // p = x[xI]                                                                          <L 61>
            var_32 = wp::address(var_x, var_30);
            var_34 = wp::load(var_32);
            var_33 = wp::copy(var_34);
            // t0 = x[faceVInd[0]]                                                                <L 62>
            var_36 = wp::extract(var_27, var_35);
            var_37 = wp::address(var_x, var_36);
            var_39 = wp::load(var_37);
            var_38 = wp::copy(var_39);
            // t1 = x[faceVInd[1]]                                                                <L 63>
            var_41 = wp::extract(var_27, var_40);
            var_42 = wp::address(var_x, var_41);
            var_44 = wp::load(var_42);
            var_43 = wp::copy(var_44);
            // t2 = x[faceVInd[2]]                                                                <L 64>
            var_46 = wp::extract(var_27, var_45);
            var_47 = wp::address(var_x, var_46);
            var_49 = wp::load(var_47);
            var_48 = wp::copy(var_49);
            // xi = node_xi[svi] + face_xi[faceJ]                                                 <L 65>
            var_50 = wp::address(var_node_xi, var_15);
            var_51 = wp::address(var_face_xi, var_23);
            var_53 = wp::load(var_50);
            var_54 = wp::load(var_51);
            var_52 = wp::add(var_53, var_54);
            // dsq = point_triangle_distance(p, t0, t1, t2) - xi * xi                             <L 66>
            var_55 = point_triangle_distance_0(var_33, var_38, var_43, var_48);
            var_56 = wp::mul(var_52, var_52);
            var_57 = wp::sub(var_55, var_56);
            // energy_x[idx] = wp.sqrt(dsq)                                                       <L 67>
            var_58 = wp::sqrt(var_57);
            wp::array_store(var_energy_x, var_0, var_58);
        }
        if (!var_9) {
            // edgeI = nodeI[idx]                                                                 <L 69>
            var_59 = wp::address(var_nodeI, var_0);
            var_61 = wp::load(var_59);
            var_60 = wp::copy(var_61);
            // edgeJ = nodeJ[idx]                                                                 <L 70>
            var_62 = wp::address(var_nodeJ, var_0);
            var_64 = wp::load(var_62);
            var_63 = wp::copy(var_64);
            // edgeVIndI = edge[edgeI]                                                            <L 71>
            var_65 = wp::address(var_edge, var_60);
            var_67 = wp::load(var_65);
            var_66 = wp::copy(var_67);
            // edgeVIndJ = edge[edgeJ]                                                            <L 72>
            var_68 = wp::address(var_edge, var_63);
            var_70 = wp::load(var_68);
            var_69 = wp::copy(var_70);
            // xi = edge_xi[edgeI] + edge_xi[edgeJ]                                               <L 73>
            var_71 = wp::address(var_edge_xi, var_60);
            var_72 = wp::address(var_edge_xi, var_63);
            var_74 = wp::load(var_71);
            var_75 = wp::load(var_72);
            var_73 = wp::add(var_74, var_75);
            // ea0 = x[edgeVIndI[0]]                                                              <L 74>
            var_77 = wp::extract(var_66, var_76);
            var_78 = wp::address(var_x, var_77);
            var_80 = wp::load(var_78);
            var_79 = wp::copy(var_80);
            // ea1 = x[edgeVIndI[1]]                                                              <L 75>
            var_82 = wp::extract(var_66, var_81);
            var_83 = wp::address(var_x, var_82);
            var_85 = wp::load(var_83);
            var_84 = wp::copy(var_85);
            // eb0 = x[edgeVIndJ[0]]                                                              <L 76>
            var_87 = wp::extract(var_69, var_86);
            var_88 = wp::address(var_x, var_87);
            var_90 = wp::load(var_88);
            var_89 = wp::copy(var_90);
            // eb1 = x[edgeVIndJ[1]]                                                              <L 77>
            var_92 = wp::extract(var_69, var_91);
            var_93 = wp::address(var_x, var_92);
            var_95 = wp::load(var_93);
            var_94 = wp::copy(var_95);
            // dsq = edge_edge_distance(ea0, ea1, eb0, eb1) - xi * xi                             <L 78>
            var_96 = edge_edge_distance_0(var_79, var_84, var_89, var_94);
            var_97 = wp::mul(var_73, var_73);
            var_98 = wp::sub(var_96, var_97);
            // energy_x[idx] = wp.sqrt(dsq)                                                       <L 79>
            var_99 = wp::sqrt(var_98);
            wp::array_store(var_energy_x, var_0, var_99);
        }
        var_100 = wp::where(var_9, var_52, var_73);
        var_101 = wp::where(var_9, var_57, var_98);
    }
}



extern "C" __global__ void dist_IPC_collisions_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy_x,
    wp::array_t<wp::int32> var_nodeI,
    wp::array_t<wp::int32> var_nodeJ,
    wp::array_t<wp::int32> var_collision_type,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::int32> var_surf_vi,
    wp::array_t<wp::float64> var_node_xi,
    wp::array_t<wp::float64> var_edge_xi,
    wp::array_t<wp::float64> var_face_xi,
    wp::array_t<wp::vec_t<2,wp::int32>> var_edge,
    wp::array_t<wp::vec_t<3,wp::int32>> var_face,
    wp::array_t<wp::float64> adj_energy_x,
    wp::array_t<wp::int32> adj_nodeI,
    wp::array_t<wp::int32> adj_nodeJ,
    wp::array_t<wp::int32> adj_collision_type,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::int32> adj_surf_vi,
    wp::array_t<wp::float64> adj_node_xi,
    wp::array_t<wp::float64> adj_edge_xi,
    wp::array_t<wp::float64> adj_face_xi,
    wp::array_t<wp::vec_t<2,wp::int32>> adj_edge,
    wp::array_t<wp::vec_t<3,wp::int32>> adj_face)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        const wp::int32 var_2 = 0;
        bool var_3;
        wp::int32 var_4;
        wp::int32* var_5;
        const wp::int32 var_6 = 1;
        bool var_7;
        wp::int32 var_8;
        bool var_9;
        wp::int32* var_10;
        bool var_11;
        wp::int32 var_12;
        wp::int32* var_13;
        wp::int32* var_14;
        wp::int32 var_15;
        wp::int32 var_16;
        wp::int32 var_17;
        wp::int32* var_18;
        bool var_19;
        wp::int32 var_20;
        wp::int32* var_21;
        wp::int32* var_22;
        wp::int32 var_23;
        wp::int32 var_24;
        wp::int32 var_25;
        wp::vec_t<3,wp::int32>* var_26;
        wp::vec_t<3,wp::int32> var_27;
        wp::vec_t<3,wp::int32> var_28;
        wp::int32* var_29;
        wp::int32 var_30;
        wp::int32 var_31;
        wp::vec_t<3,wp::float64>* var_32;
        wp::vec_t<3,wp::float64> var_33;
        wp::vec_t<3,wp::float64> var_34;
        const wp::int32 var_35 = 0;
        wp::int32 var_36;
        wp::vec_t<3,wp::float64>* var_37;
        wp::vec_t<3,wp::float64> var_38;
        wp::vec_t<3,wp::float64> var_39;
        const wp::int32 var_40 = 1;
        wp::int32 var_41;
        wp::vec_t<3,wp::float64>* var_42;
        wp::vec_t<3,wp::float64> var_43;
        wp::vec_t<3,wp::float64> var_44;
        const wp::int32 var_45 = 2;
        wp::int32 var_46;
        wp::vec_t<3,wp::float64>* var_47;
        wp::vec_t<3,wp::float64> var_48;
        wp::vec_t<3,wp::float64> var_49;
        wp::float64* var_50;
        wp::float64* var_51;
        wp::float64 var_52;
        wp::float64 var_53;
        wp::float64 var_54;
        wp::float64 var_55;
        wp::float64 var_56;
        wp::float64 var_57;
        wp::float64 var_58;
        wp::int32* var_59;
        wp::int32 var_60;
        wp::int32 var_61;
        wp::int32* var_62;
        wp::int32 var_63;
        wp::int32 var_64;
        wp::vec_t<2,wp::int32>* var_65;
        wp::vec_t<2,wp::int32> var_66;
        wp::vec_t<2,wp::int32> var_67;
        wp::vec_t<2,wp::int32>* var_68;
        wp::vec_t<2,wp::int32> var_69;
        wp::vec_t<2,wp::int32> var_70;
        wp::float64* var_71;
        wp::float64* var_72;
        wp::float64 var_73;
        wp::float64 var_74;
        wp::float64 var_75;
        const wp::int32 var_76 = 0;
        wp::int32 var_77;
        wp::vec_t<3,wp::float64>* var_78;
        wp::vec_t<3,wp::float64> var_79;
        wp::vec_t<3,wp::float64> var_80;
        const wp::int32 var_81 = 1;
        wp::int32 var_82;
        wp::vec_t<3,wp::float64>* var_83;
        wp::vec_t<3,wp::float64> var_84;
        wp::vec_t<3,wp::float64> var_85;
        const wp::int32 var_86 = 0;
        wp::int32 var_87;
        wp::vec_t<3,wp::float64>* var_88;
        wp::vec_t<3,wp::float64> var_89;
        wp::vec_t<3,wp::float64> var_90;
        const wp::int32 var_91 = 1;
        wp::int32 var_92;
        wp::vec_t<3,wp::float64>* var_93;
        wp::vec_t<3,wp::float64> var_94;
        wp::vec_t<3,wp::float64> var_95;
        wp::float64 var_96;
        wp::float64 var_97;
        wp::float64 var_98;
        wp::float64 var_99;
        wp::float64 var_100;
        wp::float64 var_101;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        bool adj_3 = {};
        wp::int32 adj_4 = {};
        wp::int32 adj_5 = {};
        wp::int32 adj_6 = {};
        bool adj_7 = {};
        wp::int32 adj_8 = {};
        bool adj_9 = {};
        wp::int32 adj_10 = {};
        bool adj_11 = {};
        wp::int32 adj_12 = {};
        wp::int32 adj_13 = {};
        wp::int32 adj_14 = {};
        wp::int32 adj_15 = {};
        wp::int32 adj_16 = {};
        wp::int32 adj_17 = {};
        wp::int32 adj_18 = {};
        bool adj_19 = {};
        wp::int32 adj_20 = {};
        wp::int32 adj_21 = {};
        wp::int32 adj_22 = {};
        wp::int32 adj_23 = {};
        wp::int32 adj_24 = {};
        wp::int32 adj_25 = {};
        wp::vec_t<3,wp::int32> adj_26 = {};
        wp::vec_t<3,wp::int32> adj_27 = {};
        wp::vec_t<3,wp::int32> adj_28 = {};
        wp::int32 adj_29 = {};
        wp::int32 adj_30 = {};
        wp::int32 adj_31 = {};
        wp::vec_t<3,wp::float64> adj_32 = {};
        wp::vec_t<3,wp::float64> adj_33 = {};
        wp::vec_t<3,wp::float64> adj_34 = {};
        wp::int32 adj_35 = {};
        wp::int32 adj_36 = {};
        wp::vec_t<3,wp::float64> adj_37 = {};
        wp::vec_t<3,wp::float64> adj_38 = {};
        wp::vec_t<3,wp::float64> adj_39 = {};
        wp::int32 adj_40 = {};
        wp::int32 adj_41 = {};
        wp::vec_t<3,wp::float64> adj_42 = {};
        wp::vec_t<3,wp::float64> adj_43 = {};
        wp::vec_t<3,wp::float64> adj_44 = {};
        wp::int32 adj_45 = {};
        wp::int32 adj_46 = {};
        wp::vec_t<3,wp::float64> adj_47 = {};
        wp::vec_t<3,wp::float64> adj_48 = {};
        wp::vec_t<3,wp::float64> adj_49 = {};
        wp::float64 adj_50 = {};
        wp::float64 adj_51 = {};
        wp::float64 adj_52 = {};
        wp::float64 adj_53 = {};
        wp::float64 adj_54 = {};
        wp::float64 adj_55 = {};
        wp::float64 adj_56 = {};
        wp::float64 adj_57 = {};
        wp::float64 adj_58 = {};
        wp::int32 adj_59 = {};
        wp::int32 adj_60 = {};
        wp::int32 adj_61 = {};
        wp::int32 adj_62 = {};
        wp::int32 adj_63 = {};
        wp::int32 adj_64 = {};
        wp::vec_t<2,wp::int32> adj_65 = {};
        wp::vec_t<2,wp::int32> adj_66 = {};
        wp::vec_t<2,wp::int32> adj_67 = {};
        wp::vec_t<2,wp::int32> adj_68 = {};
        wp::vec_t<2,wp::int32> adj_69 = {};
        wp::vec_t<2,wp::int32> adj_70 = {};
        wp::float64 adj_71 = {};
        wp::float64 adj_72 = {};
        wp::float64 adj_73 = {};
        wp::float64 adj_74 = {};
        wp::float64 adj_75 = {};
        wp::int32 adj_76 = {};
        wp::int32 adj_77 = {};
        wp::vec_t<3,wp::float64> adj_78 = {};
        wp::vec_t<3,wp::float64> adj_79 = {};
        wp::vec_t<3,wp::float64> adj_80 = {};
        wp::int32 adj_81 = {};
        wp::int32 adj_82 = {};
        wp::vec_t<3,wp::float64> adj_83 = {};
        wp::vec_t<3,wp::float64> adj_84 = {};
        wp::vec_t<3,wp::float64> adj_85 = {};
        wp::int32 adj_86 = {};
        wp::int32 adj_87 = {};
        wp::vec_t<3,wp::float64> adj_88 = {};
        wp::vec_t<3,wp::float64> adj_89 = {};
        wp::vec_t<3,wp::float64> adj_90 = {};
        wp::int32 adj_91 = {};
        wp::int32 adj_92 = {};
        wp::vec_t<3,wp::float64> adj_93 = {};
        wp::vec_t<3,wp::float64> adj_94 = {};
        wp::vec_t<3,wp::float64> adj_95 = {};
        wp::float64 adj_96 = {};
        wp::float64 adj_97 = {};
        wp::float64 adj_98 = {};
        wp::float64 adj_99 = {};
        wp::float64 adj_100 = {};
        wp::float64 adj_101 = {};
        //---------
        // forward
        // def dist_IPC_collisions(                                                               <L 41>
        // idx = wp.tid()                                                                         <L 54>
        var_0 = builtin_tid1d();
        // if collision_type[idx] == TRIANGLE_POINT_PAIR or collision_type[idx] == POINT_TRIANGLE_PAIR:       <L 56>
        var_1 = wp::address(var_collision_type, var_0);
        var_4 = wp::load(var_1);
        var_3 = (var_4 == var_2);
        var_5 = wp::address(var_collision_type, var_0);
        var_8 = wp::load(var_5);
        var_7 = (var_8 == var_6);
        var_9 = var_3 || var_7;
        if (var_9) {
            // svi = wp.select(collision_type[idx] == POINT_TRIANGLE_PAIR, nodeJ[idx], nodeI[idx])       <L 57>
            var_10 = wp::address(var_collision_type, var_0);
            var_12 = wp::load(var_10);
            var_11 = (var_12 == var_6);
            var_13 = wp::address(var_nodeJ, var_0);
            var_14 = wp::address(var_nodeI, var_0);
            var_16 = wp::load(var_13);
            var_17 = wp::load(var_14);
            var_15 = wp::select(var_11, var_16, var_17);
            // faceJ = wp.select(collision_type[idx] == POINT_TRIANGLE_PAIR, nodeI[idx], nodeJ[idx])       <L 58>
            var_18 = wp::address(var_collision_type, var_0);
            var_20 = wp::load(var_18);
            var_19 = (var_20 == var_6);
            var_21 = wp::address(var_nodeI, var_0);
            var_22 = wp::address(var_nodeJ, var_0);
            var_24 = wp::load(var_21);
            var_25 = wp::load(var_22);
            var_23 = wp::select(var_19, var_24, var_25);
            // faceVInd = face[faceJ]                                                             <L 59>
            var_26 = wp::address(var_face, var_23);
            var_28 = wp::load(var_26);
            var_27 = wp::copy(var_28);
            // xI = surf_vi[svi]                                                                  <L 60>
            var_29 = wp::address(var_surf_vi, var_15);
            var_31 = wp::load(var_29);
            var_30 = wp::copy(var_31);
            // p = x[xI]                                                                          <L 61>
            var_32 = wp::address(var_x, var_30);
            var_34 = wp::load(var_32);
            var_33 = wp::copy(var_34);
            // t0 = x[faceVInd[0]]                                                                <L 62>
            var_36 = wp::extract(var_27, var_35);
            var_37 = wp::address(var_x, var_36);
            var_39 = wp::load(var_37);
            var_38 = wp::copy(var_39);
            // t1 = x[faceVInd[1]]                                                                <L 63>
            var_41 = wp::extract(var_27, var_40);
            var_42 = wp::address(var_x, var_41);
            var_44 = wp::load(var_42);
            var_43 = wp::copy(var_44);
            // t2 = x[faceVInd[2]]                                                                <L 64>
            var_46 = wp::extract(var_27, var_45);
            var_47 = wp::address(var_x, var_46);
            var_49 = wp::load(var_47);
            var_48 = wp::copy(var_49);
            // xi = node_xi[svi] + face_xi[faceJ]                                                 <L 65>
            var_50 = wp::address(var_node_xi, var_15);
            var_51 = wp::address(var_face_xi, var_23);
            var_53 = wp::load(var_50);
            var_54 = wp::load(var_51);
            var_52 = wp::add(var_53, var_54);
            // dsq = point_triangle_distance(p, t0, t1, t2) - xi * xi                             <L 66>
            var_55 = point_triangle_distance_0(var_33, var_38, var_43, var_48);
            var_56 = wp::mul(var_52, var_52);
            var_57 = wp::sub(var_55, var_56);
            // energy_x[idx] = wp.sqrt(dsq)                                                       <L 67>
            var_58 = wp::sqrt(var_57);
            // wp::array_store(var_energy_x, var_0, var_58);
        }
        if (!var_9) {
            // edgeI = nodeI[idx]                                                                 <L 69>
            var_59 = wp::address(var_nodeI, var_0);
            var_61 = wp::load(var_59);
            var_60 = wp::copy(var_61);
            // edgeJ = nodeJ[idx]                                                                 <L 70>
            var_62 = wp::address(var_nodeJ, var_0);
            var_64 = wp::load(var_62);
            var_63 = wp::copy(var_64);
            // edgeVIndI = edge[edgeI]                                                            <L 71>
            var_65 = wp::address(var_edge, var_60);
            var_67 = wp::load(var_65);
            var_66 = wp::copy(var_67);
            // edgeVIndJ = edge[edgeJ]                                                            <L 72>
            var_68 = wp::address(var_edge, var_63);
            var_70 = wp::load(var_68);
            var_69 = wp::copy(var_70);
            // xi = edge_xi[edgeI] + edge_xi[edgeJ]                                               <L 73>
            var_71 = wp::address(var_edge_xi, var_60);
            var_72 = wp::address(var_edge_xi, var_63);
            var_74 = wp::load(var_71);
            var_75 = wp::load(var_72);
            var_73 = wp::add(var_74, var_75);
            // ea0 = x[edgeVIndI[0]]                                                              <L 74>
            var_77 = wp::extract(var_66, var_76);
            var_78 = wp::address(var_x, var_77);
            var_80 = wp::load(var_78);
            var_79 = wp::copy(var_80);
            // ea1 = x[edgeVIndI[1]]                                                              <L 75>
            var_82 = wp::extract(var_66, var_81);
            var_83 = wp::address(var_x, var_82);
            var_85 = wp::load(var_83);
            var_84 = wp::copy(var_85);
            // eb0 = x[edgeVIndJ[0]]                                                              <L 76>
            var_87 = wp::extract(var_69, var_86);
            var_88 = wp::address(var_x, var_87);
            var_90 = wp::load(var_88);
            var_89 = wp::copy(var_90);
            // eb1 = x[edgeVIndJ[1]]                                                              <L 77>
            var_92 = wp::extract(var_69, var_91);
            var_93 = wp::address(var_x, var_92);
            var_95 = wp::load(var_93);
            var_94 = wp::copy(var_95);
            // dsq = edge_edge_distance(ea0, ea1, eb0, eb1) - xi * xi                             <L 78>
            var_96 = edge_edge_distance_0(var_79, var_84, var_89, var_94);
            var_97 = wp::mul(var_73, var_73);
            var_98 = wp::sub(var_96, var_97);
            // energy_x[idx] = wp.sqrt(dsq)                                                       <L 79>
            var_99 = wp::sqrt(var_98);
            // wp::array_store(var_energy_x, var_0, var_99);
        }
        var_100 = wp::where(var_9, var_52, var_73);
        var_101 = wp::where(var_9, var_57, var_98);
        //---------
        // reverse
        wp::adj_where(var_9, var_57, var_98, adj_9, adj_57, adj_98, adj_101);
        wp::adj_where(var_9, var_52, var_73, adj_9, adj_52, adj_73, adj_100);
        if (!var_9) {
            wp::adj_array_store(var_energy_x, var_0, var_99, adj_energy_x, adj_0, adj_99);
            wp::adj_sqrt(var_98, var_99, adj_98, adj_99);
            // adj: energy_x[idx] = wp.sqrt(dsq)                                                  <L 79>
            wp::adj_sub(var_96, var_97, adj_96, adj_97, adj_98);
            wp::adj_mul(var_73, var_73, adj_73, adj_73, adj_97);
            adj_edge_edge_distance_0(var_79, var_84, var_89, var_94, adj_79, adj_84, adj_89, adj_94, adj_96);
            // adj: dsq = edge_edge_distance(ea0, ea1, eb0, eb1) - xi * xi                        <L 78>
            wp::adj_copy(var_95, adj_93, adj_94);
            wp::adj_load(var_93, adj_93, adj_95);
            wp::adj_address(var_x, var_92, adj_x, adj_92, adj_93);
            wp::adj_extract(var_69, var_91, adj_69, adj_91, adj_92);
            // adj: eb1 = x[edgeVIndJ[1]]                                                         <L 77>
            wp::adj_copy(var_90, adj_88, adj_89);
            wp::adj_load(var_88, adj_88, adj_90);
            wp::adj_address(var_x, var_87, adj_x, adj_87, adj_88);
            wp::adj_extract(var_69, var_86, adj_69, adj_86, adj_87);
            // adj: eb0 = x[edgeVIndJ[0]]                                                         <L 76>
            wp::adj_copy(var_85, adj_83, adj_84);
            wp::adj_load(var_83, adj_83, adj_85);
            wp::adj_address(var_x, var_82, adj_x, adj_82, adj_83);
            wp::adj_extract(var_66, var_81, adj_66, adj_81, adj_82);
            // adj: ea1 = x[edgeVIndI[1]]                                                         <L 75>
            wp::adj_copy(var_80, adj_78, adj_79);
            wp::adj_load(var_78, adj_78, adj_80);
            wp::adj_address(var_x, var_77, adj_x, adj_77, adj_78);
            wp::adj_extract(var_66, var_76, adj_66, adj_76, adj_77);
            // adj: ea0 = x[edgeVIndI[0]]                                                         <L 74>
            wp::adj_add(var_74, var_75, adj_71, adj_72, adj_73);
            wp::adj_load(var_72, adj_72, adj_75);
            wp::adj_load(var_71, adj_71, adj_74);
            wp::adj_address(var_edge_xi, var_63, adj_edge_xi, adj_63, adj_72);
            wp::adj_address(var_edge_xi, var_60, adj_edge_xi, adj_60, adj_71);
            // adj: xi = edge_xi[edgeI] + edge_xi[edgeJ]                                          <L 73>
            wp::adj_copy(var_70, adj_68, adj_69);
            wp::adj_load(var_68, adj_68, adj_70);
            wp::adj_address(var_edge, var_63, adj_edge, adj_63, adj_68);
            // adj: edgeVIndJ = edge[edgeJ]                                                       <L 72>
            wp::adj_copy(var_67, adj_65, adj_66);
            wp::adj_load(var_65, adj_65, adj_67);
            wp::adj_address(var_edge, var_60, adj_edge, adj_60, adj_65);
            // adj: edgeVIndI = edge[edgeI]                                                       <L 71>
            wp::adj_copy(var_64, adj_62, adj_63);
            wp::adj_load(var_62, adj_62, adj_64);
            wp::adj_address(var_nodeJ, var_0, adj_nodeJ, adj_0, adj_62);
            // adj: edgeJ = nodeJ[idx]                                                            <L 70>
            wp::adj_copy(var_61, adj_59, adj_60);
            wp::adj_load(var_59, adj_59, adj_61);
            wp::adj_address(var_nodeI, var_0, adj_nodeI, adj_0, adj_59);
            // adj: edgeI = nodeI[idx]                                                            <L 69>
        }
        if (var_9) {
            wp::adj_array_store(var_energy_x, var_0, var_58, adj_energy_x, adj_0, adj_58);
            wp::adj_sqrt(var_57, var_58, adj_57, adj_58);
            // adj: energy_x[idx] = wp.sqrt(dsq)                                                  <L 67>
            wp::adj_sub(var_55, var_56, adj_55, adj_56, adj_57);
            wp::adj_mul(var_52, var_52, adj_52, adj_52, adj_56);
            adj_point_triangle_distance_0(var_33, var_38, var_43, var_48, adj_33, adj_38, adj_43, adj_48, adj_55);
            // adj: dsq = point_triangle_distance(p, t0, t1, t2) - xi * xi                        <L 66>
            wp::adj_add(var_53, var_54, adj_50, adj_51, adj_52);
            wp::adj_load(var_51, adj_51, adj_54);
            wp::adj_load(var_50, adj_50, adj_53);
            wp::adj_address(var_face_xi, var_23, adj_face_xi, adj_23, adj_51);
            wp::adj_address(var_node_xi, var_15, adj_node_xi, adj_15, adj_50);
            // adj: xi = node_xi[svi] + face_xi[faceJ]                                            <L 65>
            wp::adj_copy(var_49, adj_47, adj_48);
            wp::adj_load(var_47, adj_47, adj_49);
            wp::adj_address(var_x, var_46, adj_x, adj_46, adj_47);
            wp::adj_extract(var_27, var_45, adj_27, adj_45, adj_46);
            // adj: t2 = x[faceVInd[2]]                                                           <L 64>
            wp::adj_copy(var_44, adj_42, adj_43);
            wp::adj_load(var_42, adj_42, adj_44);
            wp::adj_address(var_x, var_41, adj_x, adj_41, adj_42);
            wp::adj_extract(var_27, var_40, adj_27, adj_40, adj_41);
            // adj: t1 = x[faceVInd[1]]                                                           <L 63>
            wp::adj_copy(var_39, adj_37, adj_38);
            wp::adj_load(var_37, adj_37, adj_39);
            wp::adj_address(var_x, var_36, adj_x, adj_36, adj_37);
            wp::adj_extract(var_27, var_35, adj_27, adj_35, adj_36);
            // adj: t0 = x[faceVInd[0]]                                                           <L 62>
            wp::adj_copy(var_34, adj_32, adj_33);
            wp::adj_load(var_32, adj_32, adj_34);
            wp::adj_address(var_x, var_30, adj_x, adj_30, adj_32);
            // adj: p = x[xI]                                                                     <L 61>
            wp::adj_copy(var_31, adj_29, adj_30);
            wp::adj_load(var_29, adj_29, adj_31);
            wp::adj_address(var_surf_vi, var_15, adj_surf_vi, adj_15, adj_29);
            // adj: xI = surf_vi[svi]                                                             <L 60>
            wp::adj_copy(var_28, adj_26, adj_27);
            wp::adj_load(var_26, adj_26, adj_28);
            wp::adj_address(var_face, var_23, adj_face, adj_23, adj_26);
            // adj: faceVInd = face[faceJ]                                                        <L 59>
            wp::adj_select(var_19, var_24, var_25, adj_19, adj_21, adj_22, adj_23);
            wp::adj_load(var_22, adj_22, adj_25);
            wp::adj_load(var_21, adj_21, adj_24);
            wp::adj_address(var_nodeJ, var_0, adj_nodeJ, adj_0, adj_22);
            wp::adj_address(var_nodeI, var_0, adj_nodeI, adj_0, adj_21);
            wp::adj_load(var_18, adj_18, adj_20);
            wp::adj_address(var_collision_type, var_0, adj_collision_type, adj_0, adj_18);
            // adj: faceJ = wp.select(collision_type[idx] == POINT_TRIANGLE_PAIR, nodeI[idx], nodeJ[idx])  <L 58>
            wp::adj_select(var_11, var_16, var_17, adj_11, adj_13, adj_14, adj_15);
            wp::adj_load(var_14, adj_14, adj_17);
            wp::adj_load(var_13, adj_13, adj_16);
            wp::adj_address(var_nodeI, var_0, adj_nodeI, adj_0, adj_14);
            wp::adj_address(var_nodeJ, var_0, adj_nodeJ, adj_0, adj_13);
            wp::adj_load(var_10, adj_10, adj_12);
            wp::adj_address(var_collision_type, var_0, adj_collision_type, adj_0, adj_10);
            // adj: svi = wp.select(collision_type[idx] == POINT_TRIANGLE_PAIR, nodeJ[idx], nodeI[idx])  <L 57>
        }
        wp::adj_load(var_5, adj_5, adj_8);
        wp::adj_address(var_collision_type, var_0, adj_collision_type, adj_0, adj_5);
        wp::adj_load(var_1, adj_1, adj_4);
        wp::adj_address(var_collision_type, var_0, adj_collision_type, adj_0, adj_1);
        // adj: if collision_type[idx] == TRIANGLE_POINT_PAIR or collision_type[idx] == POINT_TRIANGLE_PAIR:  <L 56>
        // adj: idx = wp.tid()                                                                    <L 54>
        // adj: def dist_IPC_collisions(                                                          <L 41>
        continue;
    }
}

