#include "hip/hip_runtime.h"

#define WP_TILE_BLOCK_DIM 256
#define WP_NO_CRT
#include "builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx, dim)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)

extern "C" {
}


extern "C" __global__ void project_system_kernel_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_offsets,
    wp::array_t<wp::int32> var_columns,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_values,
    wp::array_t<wp::vec_t<3,wp::float64>> var_gradient,
    wp::array_t<bool> var_y_target_reached,
    wp::array_t<bool> var_x_target_reached)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::int32 var_1 = 0;
        bool var_2;
        const wp::int32 var_3 = 0;
        wp::shape_t* var_4;
        wp::int32 var_5;
        wp::shape_t var_6;
        const wp::int32 var_7 = 4;
        wp::int32 var_8;
        bool var_9;
        const wp::int32 var_10 = 4;
        wp::int32 var_11;
        wp::int32 var_12;
        bool* var_13;
        bool var_14;
        bool var_15;
        bool var_16;
        const wp::int32 var_17 = 0;
        wp::shape_t* var_18;
        wp::int32 var_19;
        wp::shape_t var_20;
        const wp::int32 var_21 = 4;
        wp::int32 var_22;
        wp::int32 var_23;
        bool* var_24;
        bool var_25;
        bool var_26;
        bool var_27;
        const wp::int32 var_28 = 0;
        wp::float64 var_29;
        wp::vec_t<3,wp::float64> var_30;
        wp::int32* var_31;
        const wp::int32 var_32 = 1;
        wp::int32 var_33;
        wp::int32* var_34;
        wp::range_t var_35;
        wp::int32 var_36;
        wp::int32 var_37;
        wp::int32 var_38;
        wp::int32* var_39;
        wp::int32 var_40;
        wp::int32 var_41;
        const wp::int32 var_42 = 0;
        bool var_43;
        const wp::int32 var_44 = 0;
        wp::shape_t* var_45;
        wp::int32 var_46;
        wp::shape_t var_47;
        const wp::int32 var_48 = 4;
        wp::int32 var_49;
        bool var_50;
        const wp::int32 var_51 = 4;
        wp::int32 var_52;
        wp::int32 var_53;
        bool* var_54;
        bool var_55;
        bool var_56;
        wp::int32 var_57;
        bool var_58;
        const wp::int32 var_59 = 0;
        wp::shape_t* var_60;
        wp::int32 var_61;
        wp::shape_t var_62;
        const wp::int32 var_63 = 4;
        wp::int32 var_64;
        wp::int32 var_65;
        bool* var_66;
        bool var_67;
        bool var_68;
        wp::int32 var_69;
        bool var_70;
        bool var_71;
        const wp::int32 var_72 = 0;
        wp::float64 var_73;
        wp::mat_t<3,3,wp::float64> var_74;
        bool var_75;
        const wp::int32 var_76 = 3;
        wp::mat_t<3,3,wp::float64> var_77;
        //---------
        // forward
        // def project_system_kernel(                                                             <L 33>
        // row_idx = wp.tid()                                                                     <L 41>
        var_0 = builtin_tid1d();
        // project_row = wp.bool(0)                                                               <L 42>
        var_2 = bool(var_1);
        // if row_idx < y_target_reached.shape[0] * 4:                                            <L 43>
        var_4 = &(var_y_target_reached.shape);
        var_6 = wp::load(var_4);
        var_5 = wp::extract(var_6, var_3);
        var_8 = wp::mul(var_5, var_7);
        var_9 = (var_0 < var_8);
        if (var_9) {
            // body_id = wp.int32(row_idx // 4)                                                   <L 44>
            var_11 = wp::floordiv(var_0, var_10);
            var_12 = wp::int32(var_11);
            // project_row = y_target_reached[body_id]                                            <L 45>
            var_13 = wp::address(var_y_target_reached, var_12);
            var_15 = wp::load(var_13);
            var_14 = wp::copy(var_15);
        }
        var_16 = wp::where(var_9, var_14, var_2);
        if (!var_9) {
            // soft_dof = row_idx - y_target_reached.shape[0] * 4                                 <L 47>
            var_18 = &(var_y_target_reached.shape);
            var_20 = wp::load(var_18);
            var_19 = wp::extract(var_20, var_17);
            var_22 = wp::mul(var_19, var_21);
            var_23 = wp::sub(var_0, var_22);
            // project_row = x_target_reached[soft_dof]                                           <L 48>
            var_24 = wp::address(var_x_target_reached, var_23);
            var_26 = wp::load(var_24);
            var_25 = wp::copy(var_26);
        }
        var_27 = wp::where(var_9, var_16, var_25);
        // if project_row:                                                                        <L 49>
        if (var_27) {
            // gradient[row_idx] = wp.vec3d(wp.float64(0))                                        <L 50>
            var_29 = wp::float64(var_28);
            var_30 = wp::vec_t<3,wp::float64>(var_29);
            wp::array_store(var_gradient, var_0, var_30);
        }
        // for i in range(offsets[row_idx], offsets[row_idx + 1]):                                <L 51>
        var_31 = wp::address(var_offsets, var_0);
        var_33 = wp::add(var_0, var_32);
        var_34 = wp::address(var_offsets, var_33);
        var_36 = wp::load(var_31);
        var_37 = wp::load(var_34);
        var_35 = wp::range(var_36, var_37);
        start_for_0:;
            if (iter_cmp(var_35) == 0) goto end_for_0;
            var_38 = wp::iter_next(var_35);
            // col_idx = columns[i]                                                               <L 52>
            var_39 = wp::address(var_columns, var_38);
            var_41 = wp::load(var_39);
            var_40 = wp::copy(var_41);
            // project_col = wp.bool(0)                                                           <L 53>
            var_43 = bool(var_42);
            // if col_idx < y_target_reached.shape[0] * 4:                                        <L 54>
            var_45 = &(var_y_target_reached.shape);
            var_47 = wp::load(var_45);
            var_46 = wp::extract(var_47, var_44);
            var_49 = wp::mul(var_46, var_48);
            var_50 = (var_40 < var_49);
            if (var_50) {
                // body_id = wp.int32(col_idx // 4)                                               <L 55>
                var_52 = wp::floordiv(var_40, var_51);
                var_53 = wp::int32(var_52);
                // project_col = y_target_reached[body_id]                                        <L 56>
                var_54 = wp::address(var_y_target_reached, var_53);
                var_56 = wp::load(var_54);
                var_55 = wp::copy(var_56);
            }
            var_57 = wp::where(var_50, var_53, var_12);
            var_58 = wp::where(var_50, var_55, var_43);
            if (!var_50) {
                // soft_dof = col_idx - y_target_reached.shape[0] * 4                             <L 58>
                var_60 = &(var_y_target_reached.shape);
                var_62 = wp::load(var_60);
                var_61 = wp::extract(var_62, var_59);
                var_64 = wp::mul(var_61, var_63);
                var_65 = wp::sub(var_40, var_64);
                // project_col = x_target_reached[soft_dof]                                       <L 59>
                var_66 = wp::address(var_x_target_reached, var_65);
                var_68 = wp::load(var_66);
                var_67 = wp::copy(var_68);
            }
            var_69 = wp::where(var_50, var_23, var_65);
            var_70 = wp::where(var_50, var_58, var_67);
            // if project_row or project_col:                                                     <L 60>
            var_71 = var_27 || var_70;
            if (var_71) {
                // values[i] = wp.mat33d(wp.float64(0))                                           <L 61>
                var_73 = wp::float64(var_72);
                var_74 = wp::mat_t<3,3,wp::float64>(var_73);
                wp::array_store(var_values, var_38, var_74);
                // if row_idx == col_idx:                                                         <L 62>
                var_75 = (var_0 == var_40);
                if (var_75) {
                    // values[i] = wp.identity(n=3, dtype=wp.float64)                             <L 63>
                    var_77 = wp::identity<3,wp::float64>();
                    wp::array_store(var_values, var_38, var_77);
                }
            }
            wp::assign(var_12, var_57);
            wp::assign(var_23, var_69);
            goto start_for_0;
        end_for_0:;
    }
}



extern "C" __global__ void project_system_kernel_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_offsets,
    wp::array_t<wp::int32> var_columns,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_values,
    wp::array_t<wp::vec_t<3,wp::float64>> var_gradient,
    wp::array_t<bool> var_y_target_reached,
    wp::array_t<bool> var_x_target_reached,
    wp::array_t<wp::int32> adj_offsets,
    wp::array_t<wp::int32> adj_columns,
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_values,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_gradient,
    wp::array_t<bool> adj_y_target_reached,
    wp::array_t<bool> adj_x_target_reached)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::int32 var_1 = 0;
        bool var_2;
        const wp::int32 var_3 = 0;
        wp::shape_t* var_4;
        wp::int32 var_5;
        wp::shape_t var_6;
        const wp::int32 var_7 = 4;
        wp::int32 var_8;
        bool var_9;
        const wp::int32 var_10 = 4;
        wp::int32 var_11;
        wp::int32 var_12;
        bool* var_13;
        bool var_14;
        bool var_15;
        bool var_16;
        const wp::int32 var_17 = 0;
        wp::shape_t* var_18;
        wp::int32 var_19;
        wp::shape_t var_20;
        const wp::int32 var_21 = 4;
        wp::int32 var_22;
        wp::int32 var_23;
        bool* var_24;
        bool var_25;
        bool var_26;
        bool var_27;
        const wp::int32 var_28 = 0;
        wp::float64 var_29;
        wp::vec_t<3,wp::float64> var_30;
        wp::int32* var_31;
        const wp::int32 var_32 = 1;
        wp::int32 var_33;
        wp::int32* var_34;
        wp::range_t var_35;
        wp::int32 var_36;
        wp::int32 var_37;
        wp::int32 var_38;
        wp::int32* var_39;
        wp::int32 var_40;
        wp::int32 var_41;
        const wp::int32 var_42 = 0;
        bool var_43;
        const wp::int32 var_44 = 0;
        wp::shape_t* var_45;
        wp::int32 var_46;
        wp::shape_t var_47;
        const wp::int32 var_48 = 4;
        wp::int32 var_49;
        bool var_50;
        const wp::int32 var_51 = 4;
        wp::int32 var_52;
        wp::int32 var_53;
        bool* var_54;
        bool var_55;
        bool var_56;
        wp::int32 var_57;
        bool var_58;
        const wp::int32 var_59 = 0;
        wp::shape_t* var_60;
        wp::int32 var_61;
        wp::shape_t var_62;
        const wp::int32 var_63 = 4;
        wp::int32 var_64;
        wp::int32 var_65;
        bool* var_66;
        bool var_67;
        bool var_68;
        wp::int32 var_69;
        bool var_70;
        bool var_71;
        const wp::int32 var_72 = 0;
        wp::float64 var_73;
        wp::mat_t<3,3,wp::float64> var_74;
        bool var_75;
        const wp::int32 var_76 = 3;
        wp::mat_t<3,3,wp::float64> var_77;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        bool adj_2 = {};
        wp::int32 adj_3 = {};
        wp::shape_t adj_4 = {};
        wp::int32 adj_5 = {};
        wp::shape_t adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        bool adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int32 adj_12 = {};
        bool adj_13 = {};
        bool adj_14 = {};
        bool adj_15 = {};
        bool adj_16 = {};
        wp::int32 adj_17 = {};
        wp::shape_t adj_18 = {};
        wp::int32 adj_19 = {};
        wp::shape_t adj_20 = {};
        wp::int32 adj_21 = {};
        wp::int32 adj_22 = {};
        wp::int32 adj_23 = {};
        bool adj_24 = {};
        bool adj_25 = {};
        bool adj_26 = {};
        bool adj_27 = {};
        wp::int32 adj_28 = {};
        wp::float64 adj_29 = {};
        wp::vec_t<3,wp::float64> adj_30 = {};
        wp::int32 adj_31 = {};
        wp::int32 adj_32 = {};
        wp::int32 adj_33 = {};
        wp::int32 adj_34 = {};
        wp::range_t adj_35 = {};
        wp::int32 adj_36 = {};
        wp::int32 adj_37 = {};
        wp::int32 adj_38 = {};
        wp::int32 adj_39 = {};
        wp::int32 adj_40 = {};
        wp::int32 adj_41 = {};
        wp::int32 adj_42 = {};
        bool adj_43 = {};
        wp::int32 adj_44 = {};
        wp::shape_t adj_45 = {};
        wp::int32 adj_46 = {};
        wp::shape_t adj_47 = {};
        wp::int32 adj_48 = {};
        wp::int32 adj_49 = {};
        bool adj_50 = {};
        wp::int32 adj_51 = {};
        wp::int32 adj_52 = {};
        wp::int32 adj_53 = {};
        bool adj_54 = {};
        bool adj_55 = {};
        bool adj_56 = {};
        wp::int32 adj_57 = {};
        bool adj_58 = {};
        wp::int32 adj_59 = {};
        wp::shape_t adj_60 = {};
        wp::int32 adj_61 = {};
        wp::shape_t adj_62 = {};
        wp::int32 adj_63 = {};
        wp::int32 adj_64 = {};
        wp::int32 adj_65 = {};
        bool adj_66 = {};
        bool adj_67 = {};
        bool adj_68 = {};
        wp::int32 adj_69 = {};
        bool adj_70 = {};
        bool adj_71 = {};
        wp::int32 adj_72 = {};
        wp::float64 adj_73 = {};
        wp::mat_t<3,3,wp::float64> adj_74 = {};
        bool adj_75 = {};
        wp::int32 adj_76 = {};
        wp::mat_t<3,3,wp::float64> adj_77 = {};
        //---------
        // forward
        // def project_system_kernel(                                                             <L 33>
        // row_idx = wp.tid()                                                                     <L 41>
        var_0 = builtin_tid1d();
        // project_row = wp.bool(0)                                                               <L 42>
        var_2 = bool(var_1);
        // if row_idx < y_target_reached.shape[0] * 4:                                            <L 43>
        var_4 = &(var_y_target_reached.shape);
        var_6 = wp::load(var_4);
        var_5 = wp::extract(var_6, var_3);
        var_8 = wp::mul(var_5, var_7);
        var_9 = (var_0 < var_8);
        if (var_9) {
            // body_id = wp.int32(row_idx // 4)                                                   <L 44>
            var_11 = wp::floordiv(var_0, var_10);
            var_12 = wp::int32(var_11);
            // project_row = y_target_reached[body_id]                                            <L 45>
            var_13 = wp::address(var_y_target_reached, var_12);
            var_15 = wp::load(var_13);
            var_14 = wp::copy(var_15);
        }
        var_16 = wp::where(var_9, var_14, var_2);
        if (!var_9) {
            // soft_dof = row_idx - y_target_reached.shape[0] * 4                                 <L 47>
            var_18 = &(var_y_target_reached.shape);
            var_20 = wp::load(var_18);
            var_19 = wp::extract(var_20, var_17);
            var_22 = wp::mul(var_19, var_21);
            var_23 = wp::sub(var_0, var_22);
            // project_row = x_target_reached[soft_dof]                                           <L 48>
            var_24 = wp::address(var_x_target_reached, var_23);
            var_26 = wp::load(var_24);
            var_25 = wp::copy(var_26);
        }
        var_27 = wp::where(var_9, var_16, var_25);
        // if project_row:                                                                        <L 49>
        if (var_27) {
            // gradient[row_idx] = wp.vec3d(wp.float64(0))                                        <L 50>
            var_29 = wp::float64(var_28);
            var_30 = wp::vec_t<3,wp::float64>(var_29);
            // wp::array_store(var_gradient, var_0, var_30);
        }
        // for i in range(offsets[row_idx], offsets[row_idx + 1]):                                <L 51>
        var_31 = wp::address(var_offsets, var_0);
        var_33 = wp::add(var_0, var_32);
        var_34 = wp::address(var_offsets, var_33);
        var_36 = wp::load(var_31);
        var_37 = wp::load(var_34);
        var_35 = wp::range(var_36, var_37);
        //---------
        // reverse
        var_35 = wp::iter_reverse(var_35);
        start_for_0:;
            if (iter_cmp(var_35) == 0) goto end_for_0;
            var_38 = wp::iter_next(var_35);
        	adj_39 = {};
        	adj_40 = {};
        	adj_41 = {};
        	adj_42 = {};
        	adj_43 = {};
        	adj_44 = {};
        	adj_45 = {};
        	adj_46 = {};
        	adj_47 = {};
        	adj_48 = {};
        	adj_49 = {};
        	adj_50 = {};
        	adj_51 = {};
        	adj_52 = {};
        	adj_53 = {};
        	adj_54 = {};
        	adj_55 = {};
        	adj_56 = {};
        	adj_57 = {};
        	adj_58 = {};
        	adj_59 = {};
        	adj_60 = {};
        	adj_61 = {};
        	adj_62 = {};
        	adj_63 = {};
        	adj_64 = {};
        	adj_65 = {};
        	adj_66 = {};
        	adj_67 = {};
        	adj_68 = {};
        	adj_69 = {};
        	adj_70 = {};
        	adj_71 = {};
        	adj_72 = {};
        	adj_73 = {};
        	adj_74 = {};
        	adj_75 = {};
        	adj_76 = {};
        	adj_77 = {};
            // col_idx = columns[i]                                                               <L 52>
            var_39 = wp::address(var_columns, var_38);
            var_41 = wp::load(var_39);
            var_40 = wp::copy(var_41);
            // project_col = wp.bool(0)                                                           <L 53>
            var_43 = bool(var_42);
            // if col_idx < y_target_reached.shape[0] * 4:                                        <L 54>
            var_45 = &(var_y_target_reached.shape);
            var_47 = wp::load(var_45);
            var_46 = wp::extract(var_47, var_44);
            var_49 = wp::mul(var_46, var_48);
            var_50 = (var_40 < var_49);
            if (var_50) {
                // body_id = wp.int32(col_idx // 4)                                               <L 55>
                var_52 = wp::floordiv(var_40, var_51);
                var_53 = wp::int32(var_52);
                // project_col = y_target_reached[body_id]                                        <L 56>
                var_54 = wp::address(var_y_target_reached, var_53);
                var_56 = wp::load(var_54);
                var_55 = wp::copy(var_56);
            }
            var_57 = wp::where(var_50, var_53, var_12);
            var_58 = wp::where(var_50, var_55, var_43);
            if (!var_50) {
                // soft_dof = col_idx - y_target_reached.shape[0] * 4                             <L 58>
                var_60 = &(var_y_target_reached.shape);
                var_62 = wp::load(var_60);
                var_61 = wp::extract(var_62, var_59);
                var_64 = wp::mul(var_61, var_63);
                var_65 = wp::sub(var_40, var_64);
                // project_col = x_target_reached[soft_dof]                                       <L 59>
                var_66 = wp::address(var_x_target_reached, var_65);
                var_68 = wp::load(var_66);
                var_67 = wp::copy(var_68);
            }
            var_69 = wp::where(var_50, var_23, var_65);
            var_70 = wp::where(var_50, var_58, var_67);
            // if project_row or project_col:                                                     <L 60>
            var_71 = var_27 || var_70;
            if (var_71) {
                // values[i] = wp.mat33d(wp.float64(0))                                           <L 61>
                var_73 = wp::float64(var_72);
                var_74 = wp::mat_t<3,3,wp::float64>(var_73);
                // wp::array_store(var_values, var_38, var_74);
                // if row_idx == col_idx:                                                         <L 62>
                var_75 = (var_0 == var_40);
                if (var_75) {
                    // values[i] = wp.identity(n=3, dtype=wp.float64)                             <L 63>
                    var_77 = wp::identity<3,wp::float64>();
                    // wp::array_store(var_values, var_38, var_77);
                }
            }
            wp::assign(var_12, var_57);
            wp::assign(var_23, var_69);
            wp::adj_assign(var_23, var_69, adj_23, adj_69);
            wp::adj_assign(var_12, var_57, adj_12, adj_57);
            if (var_71) {
                if (var_75) {
                    wp::adj_array_store(var_values, var_38, var_77, adj_values, adj_38, adj_77);
                    // adj: values[i] = wp.identity(n=3, dtype=wp.float64)                        <L 63>
                }
                // adj: if row_idx == col_idx:                                                    <L 62>
                wp::adj_array_store(var_values, var_38, var_74, adj_values, adj_38, adj_74);
                wp::adj_mat_t(var_73, adj_73, adj_74);
                wp::adj_float64(var_72, adj_72, adj_73);
                // adj: values[i] = wp.mat33d(wp.float64(0))                                      <L 61>
            }
            // adj: if project_row or project_col:                                                <L 60>
            wp::adj_where(var_50, var_58, var_67, adj_50, adj_58, adj_67, adj_70);
            wp::adj_where(var_50, var_23, var_65, adj_50, adj_23, adj_65, adj_69);
            if (!var_50) {
                wp::adj_copy(var_68, adj_66, adj_67);
                wp::adj_load(var_66, adj_66, adj_68);
                wp::adj_address(var_x_target_reached, var_65, adj_x_target_reached, adj_65, adj_66);
                // adj: project_col = x_target_reached[soft_dof]                                  <L 59>
                wp::adj_sub(var_40, var_64, adj_40, adj_64, adj_65);
                wp::adj_mul(var_61, var_63, adj_61, adj_63, adj_64);
                wp::adj_extract(var_62, var_59, adj_60, adj_59, adj_61);
                wp::adj_load(var_60, adj_60, adj_62);
                adj_y_target_reached.shape = adj_60;
                // adj: soft_dof = col_idx - y_target_reached.shape[0] * 4                        <L 58>
            }
            wp::adj_where(var_50, var_55, var_43, adj_50, adj_55, adj_43, adj_58);
            wp::adj_where(var_50, var_53, var_12, adj_50, adj_53, adj_12, adj_57);
            if (var_50) {
                wp::adj_copy(var_56, adj_54, adj_55);
                wp::adj_load(var_54, adj_54, adj_56);
                wp::adj_address(var_y_target_reached, var_53, adj_y_target_reached, adj_53, adj_54);
                // adj: project_col = y_target_reached[body_id]                                   <L 56>
                wp::adj_int32(var_52, adj_52, adj_53);
                wp::adj_floordiv(var_40, var_51, adj_40, adj_51, adj_52);
                // adj: body_id = wp.int32(col_idx // 4)                                          <L 55>
            }
            wp::adj_mul(var_46, var_48, adj_46, adj_48, adj_49);
            wp::adj_extract(var_47, var_44, adj_45, adj_44, adj_46);
            wp::adj_load(var_45, adj_45, adj_47);
            adj_y_target_reached.shape = adj_45;
            // adj: if col_idx < y_target_reached.shape[0] * 4:                                   <L 54>
            adj_bool(var_42, adj_42, adj_43);
            // adj: project_col = wp.bool(0)                                                      <L 53>
            wp::adj_copy(var_41, adj_39, adj_40);
            wp::adj_load(var_39, adj_39, adj_41);
            wp::adj_address(var_columns, var_38, adj_columns, adj_38, adj_39);
            // adj: col_idx = columns[i]                                                          <L 52>
        	goto start_for_0;
        end_for_0:;
        wp::adj_range(var_36, var_37, adj_31, adj_34, adj_35);
        wp::adj_load(var_34, adj_34, adj_37);
        wp::adj_load(var_31, adj_31, adj_36);
        wp::adj_address(var_offsets, var_33, adj_offsets, adj_33, adj_34);
        wp::adj_add(var_0, var_32, adj_0, adj_32, adj_33);
        wp::adj_address(var_offsets, var_0, adj_offsets, adj_0, adj_31);
        // adj: for i in range(offsets[row_idx], offsets[row_idx + 1]):                           <L 51>
        if (var_27) {
            wp::adj_array_store(var_gradient, var_0, var_30, adj_gradient, adj_0, adj_30);
            wp::adj_vec_t(var_29, adj_29, adj_30);
            wp::adj_float64(var_28, adj_28, adj_29);
            // adj: gradient[row_idx] = wp.vec3d(wp.float64(0))                                   <L 50>
        }
        // adj: if project_row:                                                                   <L 49>
        wp::adj_where(var_9, var_16, var_25, adj_9, adj_16, adj_25, adj_27);
        if (!var_9) {
            wp::adj_copy(var_26, adj_24, adj_25);
            wp::adj_load(var_24, adj_24, adj_26);
            wp::adj_address(var_x_target_reached, var_23, adj_x_target_reached, adj_23, adj_24);
            // adj: project_row = x_target_reached[soft_dof]                                      <L 48>
            wp::adj_sub(var_0, var_22, adj_0, adj_22, adj_23);
            wp::adj_mul(var_19, var_21, adj_19, adj_21, adj_22);
            wp::adj_extract(var_20, var_17, adj_18, adj_17, adj_19);
            wp::adj_load(var_18, adj_18, adj_20);
            adj_y_target_reached.shape = adj_18;
            // adj: soft_dof = row_idx - y_target_reached.shape[0] * 4                            <L 47>
        }
        wp::adj_where(var_9, var_14, var_2, adj_9, adj_14, adj_2, adj_16);
        if (var_9) {
            wp::adj_copy(var_15, adj_13, adj_14);
            wp::adj_load(var_13, adj_13, adj_15);
            wp::adj_address(var_y_target_reached, var_12, adj_y_target_reached, adj_12, adj_13);
            // adj: project_row = y_target_reached[body_id]                                       <L 45>
            wp::adj_int32(var_11, adj_11, adj_12);
            wp::adj_floordiv(var_0, var_10, adj_0, adj_10, adj_11);
            // adj: body_id = wp.int32(row_idx // 4)                                              <L 44>
        }
        wp::adj_mul(var_5, var_7, adj_5, adj_7, adj_8);
        wp::adj_extract(var_6, var_3, adj_4, adj_3, adj_5);
        wp::adj_load(var_4, adj_4, adj_6);
        adj_y_target_reached.shape = adj_4;
        // adj: if row_idx < y_target_reached.shape[0] * 4:                                       <L 43>
        adj_bool(var_1, adj_1, adj_2);
        // adj: project_row = wp.bool(0)                                                          <L 42>
        // adj: row_idx = wp.tid()                                                                <L 41>
        // adj: def project_system_kernel(                                                        <L 33>
        continue;
    }
}



extern "C" __global__ void update_dof_satisfied_kernel_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_x_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x_target,
    wp::array_t<bool> var_x_target_reached,
    wp::array_t<bool> var_y_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y_target,
    wp::array_t<bool> var_y_target_reached,
    wp::float64 var_dt,
    wp::float64 var_tol)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::int32 var_1 = 0;
        wp::shape_t* var_2;
        wp::int32 var_3;
        wp::shape_t var_4;
        bool var_5;
        bool* var_6;
        bool* var_7;
        const bool var_8 = false;
        bool var_9;
        bool var_10;
        bool var_11;
        bool var_12;
        wp::vec_t<3,wp::float64>* var_13;
        wp::vec_t<3,wp::float64>* var_14;
        wp::vec_t<3,wp::float64> var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::float64 var_18;
        wp::float64 var_19;
        bool var_20;
        const wp::int32 var_21 = 1;
        bool var_22;
        const wp::int32 var_23 = 0;
        wp::shape_t* var_24;
        wp::int32 var_25;
        wp::shape_t var_26;
        wp::int32 var_27;
        bool* var_28;
        bool* var_29;
        const bool var_30 = false;
        bool var_31;
        bool var_32;
        bool var_33;
        bool var_34;
        wp::vec_t<12,wp::float64>* var_35;
        wp::vec_t<12,wp::float64>* var_36;
        wp::vec_t<12,wp::float64> var_37;
        wp::vec_t<12,wp::float64> var_38;
        wp::vec_t<12,wp::float64> var_39;
        wp::float64 var_40;
        wp::float64 var_41;
        bool var_42;
        const wp::int32 var_43 = 1;
        bool var_44;
        wp::int32 var_45;
        //---------
        // forward
        // def update_dof_satisfied_kernel(                                                       <L 8>
        // tid = wp.tid()                                                                         <L 20>
        var_0 = builtin_tid1d();
        // if tid < x_has_constraint.shape[0]:                                                    <L 21>
        var_2 = &(var_x_has_constraint.shape);
        var_4 = wp::load(var_2);
        var_3 = wp::extract(var_4, var_1);
        var_5 = (var_0 < var_3);
        if (var_5) {
            // if x_has_constraint[tid] and x_target_reached[tid] == False:                       <L 22>
            var_6 = wp::address(var_x_has_constraint, var_0);
            var_7 = wp::address(var_x_target_reached, var_0);
            var_10 = wp::load(var_7);
            var_9 = (var_10 == var_8);
            var_11 = wp::load(var_6);
            var_12 = var_11 && var_9;
            if (var_12) {
                // if wp.length(x[tid] - x_target[tid]) < tol * dt:                               <L 23>
                var_13 = wp::address(var_x, var_0);
                var_14 = wp::address(var_x_target, var_0);
                var_16 = wp::load(var_13);
                var_17 = wp::load(var_14);
                var_15 = wp::sub(var_16, var_17);
                var_18 = wp::length(var_15);
                var_19 = wp::mul(var_tol, var_dt);
                var_20 = (var_18 < var_19);
                if (var_20) {
                    // x_target_reached[tid] = wp.bool(1)                                         <L 24>
                    var_22 = bool(var_21);
                    wp::array_store(var_x_target_reached, var_0, var_22);
                }
            }
        }
        if (!var_5) {
            // tid -= x_has_constraint.shape[0]                                                   <L 26>
            var_24 = &(var_x_has_constraint.shape);
            var_26 = wp::load(var_24);
            var_25 = wp::extract(var_26, var_23);
            var_27 = wp::sub(var_0, var_25);
            // if y_has_constraint[tid] and y_target_reached[tid] == False:                       <L 27>
            var_28 = wp::address(var_y_has_constraint, var_27);
            var_29 = wp::address(var_y_target_reached, var_27);
            var_32 = wp::load(var_29);
            var_31 = (var_32 == var_30);
            var_33 = wp::load(var_28);
            var_34 = var_33 && var_31;
            if (var_34) {
                // if wp.length(y[tid] - y_target[tid]) < tol * dt:                               <L 28>
                var_35 = wp::address(var_y, var_27);
                var_36 = wp::address(var_y_target, var_27);
                var_38 = wp::load(var_35);
                var_39 = wp::load(var_36);
                var_37 = wp::sub(var_38, var_39);
                var_40 = wp::length(var_37);
                var_41 = wp::mul(var_tol, var_dt);
                var_42 = (var_40 < var_41);
                if (var_42) {
                    // y_target_reached[tid] = wp.bool(1)                                         <L 29>
                    var_44 = bool(var_43);
                    wp::array_store(var_y_target_reached, var_27, var_44);
                }
            }
        }
        var_45 = wp::where(var_5, var_0, var_27);
    }
}



extern "C" __global__ void update_dof_satisfied_kernel_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_x_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x_target,
    wp::array_t<bool> var_x_target_reached,
    wp::array_t<bool> var_y_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y_target,
    wp::array_t<bool> var_y_target_reached,
    wp::float64 var_dt,
    wp::float64 var_tol,
    wp::array_t<bool> adj_x_has_constraint,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x_target,
    wp::array_t<bool> adj_x_target_reached,
    wp::array_t<bool> adj_y_has_constraint,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y_target,
    wp::array_t<bool> adj_y_target_reached,
    wp::float64 adj_dt,
    wp::float64 adj_tol)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::int32 var_1 = 0;
        wp::shape_t* var_2;
        wp::int32 var_3;
        wp::shape_t var_4;
        bool var_5;
        bool* var_6;
        bool* var_7;
        const bool var_8 = false;
        bool var_9;
        bool var_10;
        bool var_11;
        bool var_12;
        wp::vec_t<3,wp::float64>* var_13;
        wp::vec_t<3,wp::float64>* var_14;
        wp::vec_t<3,wp::float64> var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::float64 var_18;
        wp::float64 var_19;
        bool var_20;
        const wp::int32 var_21 = 1;
        bool var_22;
        const wp::int32 var_23 = 0;
        wp::shape_t* var_24;
        wp::int32 var_25;
        wp::shape_t var_26;
        wp::int32 var_27;
        bool* var_28;
        bool* var_29;
        const bool var_30 = false;
        bool var_31;
        bool var_32;
        bool var_33;
        bool var_34;
        wp::vec_t<12,wp::float64>* var_35;
        wp::vec_t<12,wp::float64>* var_36;
        wp::vec_t<12,wp::float64> var_37;
        wp::vec_t<12,wp::float64> var_38;
        wp::vec_t<12,wp::float64> var_39;
        wp::float64 var_40;
        wp::float64 var_41;
        bool var_42;
        const wp::int32 var_43 = 1;
        bool var_44;
        wp::int32 var_45;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::shape_t adj_2 = {};
        wp::int32 adj_3 = {};
        wp::shape_t adj_4 = {};
        bool adj_5 = {};
        bool adj_6 = {};
        bool adj_7 = {};
        bool adj_8 = {};
        bool adj_9 = {};
        bool adj_10 = {};
        bool adj_11 = {};
        bool adj_12 = {};
        wp::vec_t<3,wp::float64> adj_13 = {};
        wp::vec_t<3,wp::float64> adj_14 = {};
        wp::vec_t<3,wp::float64> adj_15 = {};
        wp::vec_t<3,wp::float64> adj_16 = {};
        wp::vec_t<3,wp::float64> adj_17 = {};
        wp::float64 adj_18 = {};
        wp::float64 adj_19 = {};
        bool adj_20 = {};
        wp::int32 adj_21 = {};
        bool adj_22 = {};
        wp::int32 adj_23 = {};
        wp::shape_t adj_24 = {};
        wp::int32 adj_25 = {};
        wp::shape_t adj_26 = {};
        wp::int32 adj_27 = {};
        bool adj_28 = {};
        bool adj_29 = {};
        bool adj_30 = {};
        bool adj_31 = {};
        bool adj_32 = {};
        bool adj_33 = {};
        bool adj_34 = {};
        wp::vec_t<12,wp::float64> adj_35 = {};
        wp::vec_t<12,wp::float64> adj_36 = {};
        wp::vec_t<12,wp::float64> adj_37 = {};
        wp::vec_t<12,wp::float64> adj_38 = {};
        wp::vec_t<12,wp::float64> adj_39 = {};
        wp::float64 adj_40 = {};
        wp::float64 adj_41 = {};
        bool adj_42 = {};
        wp::int32 adj_43 = {};
        bool adj_44 = {};
        wp::int32 adj_45 = {};
        //---------
        // forward
        // def update_dof_satisfied_kernel(                                                       <L 8>
        // tid = wp.tid()                                                                         <L 20>
        var_0 = builtin_tid1d();
        // if tid < x_has_constraint.shape[0]:                                                    <L 21>
        var_2 = &(var_x_has_constraint.shape);
        var_4 = wp::load(var_2);
        var_3 = wp::extract(var_4, var_1);
        var_5 = (var_0 < var_3);
        if (var_5) {
            // if x_has_constraint[tid] and x_target_reached[tid] == False:                       <L 22>
            var_6 = wp::address(var_x_has_constraint, var_0);
            var_7 = wp::address(var_x_target_reached, var_0);
            var_10 = wp::load(var_7);
            var_9 = (var_10 == var_8);
            var_11 = wp::load(var_6);
            var_12 = var_11 && var_9;
            if (var_12) {
                // if wp.length(x[tid] - x_target[tid]) < tol * dt:                               <L 23>
                var_13 = wp::address(var_x, var_0);
                var_14 = wp::address(var_x_target, var_0);
                var_16 = wp::load(var_13);
                var_17 = wp::load(var_14);
                var_15 = wp::sub(var_16, var_17);
                var_18 = wp::length(var_15);
                var_19 = wp::mul(var_tol, var_dt);
                var_20 = (var_18 < var_19);
                if (var_20) {
                    // x_target_reached[tid] = wp.bool(1)                                         <L 24>
                    var_22 = bool(var_21);
                    // wp::array_store(var_x_target_reached, var_0, var_22);
                }
            }
        }
        if (!var_5) {
            // tid -= x_has_constraint.shape[0]                                                   <L 26>
            var_24 = &(var_x_has_constraint.shape);
            var_26 = wp::load(var_24);
            var_25 = wp::extract(var_26, var_23);
            var_27 = wp::sub(var_0, var_25);
            // if y_has_constraint[tid] and y_target_reached[tid] == False:                       <L 27>
            var_28 = wp::address(var_y_has_constraint, var_27);
            var_29 = wp::address(var_y_target_reached, var_27);
            var_32 = wp::load(var_29);
            var_31 = (var_32 == var_30);
            var_33 = wp::load(var_28);
            var_34 = var_33 && var_31;
            if (var_34) {
                // if wp.length(y[tid] - y_target[tid]) < tol * dt:                               <L 28>
                var_35 = wp::address(var_y, var_27);
                var_36 = wp::address(var_y_target, var_27);
                var_38 = wp::load(var_35);
                var_39 = wp::load(var_36);
                var_37 = wp::sub(var_38, var_39);
                var_40 = wp::length(var_37);
                var_41 = wp::mul(var_tol, var_dt);
                var_42 = (var_40 < var_41);
                if (var_42) {
                    // y_target_reached[tid] = wp.bool(1)                                         <L 29>
                    var_44 = bool(var_43);
                    // wp::array_store(var_y_target_reached, var_27, var_44);
                }
            }
        }
        var_45 = wp::where(var_5, var_0, var_27);
        //---------
        // reverse
        wp::adj_where(var_5, var_0, var_27, adj_5, adj_0, adj_27, adj_45);
        if (!var_5) {
            if (var_34) {
                if (var_42) {
                    wp::adj_array_store(var_y_target_reached, var_27, var_44, adj_y_target_reached, adj_27, adj_44);
                    adj_bool(var_43, adj_43, adj_44);
                    // adj: y_target_reached[tid] = wp.bool(1)                                    <L 29>
                }
                wp::adj_mul(var_tol, var_dt, adj_tol, adj_dt, adj_41);
                wp::adj_length(var_37, var_40, adj_37, adj_40);
                wp::adj_sub(var_38, var_39, adj_35, adj_36, adj_37);
                wp::adj_load(var_36, adj_36, adj_39);
                wp::adj_load(var_35, adj_35, adj_38);
                wp::adj_address(var_y_target, var_27, adj_y_target, adj_27, adj_36);
                wp::adj_address(var_y, var_27, adj_y, adj_27, adj_35);
                // adj: if wp.length(y[tid] - y_target[tid]) < tol * dt:                          <L 28>
            }
            wp::adj_load(var_28, adj_28, adj_33);
            wp::adj_load(var_29, adj_29, adj_32);
            wp::adj_address(var_y_target_reached, var_27, adj_y_target_reached, adj_27, adj_29);
            wp::adj_address(var_y_has_constraint, var_27, adj_y_has_constraint, adj_27, adj_28);
            // adj: if y_has_constraint[tid] and y_target_reached[tid] == False:                  <L 27>
            wp::adj_sub(var_0, var_25, adj_0, adj_25, adj_27);
            wp::adj_extract(var_26, var_23, adj_24, adj_23, adj_25);
            wp::adj_load(var_24, adj_24, adj_26);
            adj_x_has_constraint.shape = adj_24;
            // adj: tid -= x_has_constraint.shape[0]                                              <L 26>
        }
        if (var_5) {
            if (var_12) {
                if (var_20) {
                    wp::adj_array_store(var_x_target_reached, var_0, var_22, adj_x_target_reached, adj_0, adj_22);
                    adj_bool(var_21, adj_21, adj_22);
                    // adj: x_target_reached[tid] = wp.bool(1)                                    <L 24>
                }
                wp::adj_mul(var_tol, var_dt, adj_tol, adj_dt, adj_19);
                wp::adj_length(var_15, var_18, adj_15, adj_18);
                wp::adj_sub(var_16, var_17, adj_13, adj_14, adj_15);
                wp::adj_load(var_14, adj_14, adj_17);
                wp::adj_load(var_13, adj_13, adj_16);
                wp::adj_address(var_x_target, var_0, adj_x_target, adj_0, adj_14);
                wp::adj_address(var_x, var_0, adj_x, adj_0, adj_13);
                // adj: if wp.length(x[tid] - x_target[tid]) < tol * dt:                          <L 23>
            }
            wp::adj_load(var_6, adj_6, adj_11);
            wp::adj_load(var_7, adj_7, adj_10);
            wp::adj_address(var_x_target_reached, var_0, adj_x_target_reached, adj_0, adj_7);
            wp::adj_address(var_x_has_constraint, var_0, adj_x_has_constraint, adj_0, adj_6);
            // adj: if x_has_constraint[tid] and x_target_reached[tid] == False:                  <L 22>
        }
        wp::adj_extract(var_4, var_1, adj_2, adj_1, adj_3);
        wp::adj_load(var_2, adj_2, adj_4);
        adj_x_has_constraint.shape = adj_2;
        // adj: if tid < x_has_constraint.shape[0]:                                               <L 21>
        // adj: tid = wp.tid()                                                                    <L 20>
        // adj: def update_dof_satisfied_kernel(                                                  <L 8>
        continue;
    }
}



extern "C" __global__ void check_is_satisfied_kernel_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_y_target_reached,
    wp::array_t<bool> var_y_has_constraint,
    wp::array_t<bool> var_x_target_reached,
    wp::array_t<bool> var_x_has_constraint,
    wp::array_t<bool> var_satisfied_val,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::int32 var_1 = 0;
        wp::shape_t* var_2;
        wp::int32 var_3;
        wp::shape_t var_4;
        bool var_5;
        bool* var_6;
        bool var_7;
        bool var_8;
        wp::int32* var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        const wp::int32 var_12 = 1;
        bool var_13;
        wp::int32 var_14;
        wp::int32* var_15;
        wp::int32* var_16;
        wp::int32 var_17;
        const wp::int32 var_18 = 2;
        bool var_19;
        wp::int32 var_20;
        bool var_21;
        bool var_22;
        bool* var_23;
        bool var_24;
        bool var_25;
        const bool var_26 = false;
        wp::int32* var_27;
        wp::int32 var_28;
        const wp::int32 var_29 = 0;
        wp::shape_t* var_30;
        wp::int32 var_31;
        wp::shape_t var_32;
        wp::int32 var_33;
        bool* var_34;
        bool var_35;
        bool var_36;
        wp::int32 var_37;
        wp::int32* var_38;
        wp::int32* var_39;
        wp::int32 var_40;
        bool var_41;
        wp::int32 var_42;
        wp::int32 var_43;
        wp::int32* var_44;
        wp::int32* var_45;
        wp::int32 var_46;
        bool var_47;
        wp::int32 var_48;
        bool var_49;
        bool var_50;
        bool* var_51;
        bool var_52;
        bool var_53;
        const bool var_54 = false;
        wp::int32 var_55;
        wp::int32* var_56;
        wp::int32 var_57;
        wp::int32 var_58;
        //---------
        // forward
        // def check_is_satisfied_kernel(                                                         <L 67>
        // tid = wp.tid()                                                                         <L 78>
        var_0 = builtin_tid1d();
        // if tid < y_target_reached.shape[0]:                                                    <L 79>
        var_2 = &(var_y_target_reached.shape);
        var_4 = wp::load(var_2);
        var_3 = wp::extract(var_4, var_1);
        var_5 = (var_0 < var_3);
        if (var_5) {
            // if not y_has_constraint[tid] or (                                                  <L 80>
            var_6 = wp::address(var_y_has_constraint, var_0);
            var_8 = wp::load(var_6);
            var_7 = wp::unot(var_8);
            // (env_states[body_env_id[tid]] == ENV_STATE_INVALID) or (env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED)       <L 81>
            var_9 = wp::address(var_body_env_id, var_0);
            var_11 = wp::load(var_9);
            var_10 = wp::address(var_env_states, var_11);
            var_14 = wp::load(var_10);
            var_13 = (var_14 == var_12);
            var_15 = wp::address(var_body_env_id, var_0);
            var_17 = wp::load(var_15);
            var_16 = wp::address(var_env_states, var_17);
            var_20 = wp::load(var_16);
            var_19 = (var_20 == var_18);
            var_21 = var_13 || var_19;
            var_22 = var_7 || var_21;
            if (var_22) {
                // return                                                                         <L 83>
                continue;
            }
            // if not y_target_reached[tid]:                                                      <L 84>
            var_23 = wp::address(var_y_target_reached, var_0);
            var_25 = wp::load(var_23);
            var_24 = wp::unot(var_25);
            if (var_24) {
                // satisfied_val[body_env_id[tid]] = False                                        <L 85>
                var_27 = wp::address(var_body_env_id, var_0);
                var_28 = wp::load(var_27);
                wp::array_store(var_satisfied_val, var_28, var_26);
            }
        }
        if (!var_5) {
            // tid -= y_target_reached.shape[0]                                                   <L 87>
            var_30 = &(var_y_target_reached.shape);
            var_32 = wp::load(var_30);
            var_31 = wp::extract(var_32, var_29);
            var_33 = wp::sub(var_0, var_31);
            // if not x_has_constraint[tid] or (                                                  <L 88>
            var_34 = wp::address(var_x_has_constraint, var_33);
            var_36 = wp::load(var_34);
            var_35 = wp::unot(var_36);
            // (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID)                <L 89>
            var_37 = wp::add(var_33, var_affine_verts_num);
            var_38 = wp::address(var_node2env, var_37);
            var_40 = wp::load(var_38);
            var_39 = wp::address(var_env_states, var_40);
            var_42 = wp::load(var_39);
            var_41 = (var_42 == var_12);
            // or (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED)       <L 90>
            var_43 = wp::add(var_33, var_affine_verts_num);
            var_44 = wp::address(var_node2env, var_43);
            var_46 = wp::load(var_44);
            var_45 = wp::address(var_env_states, var_46);
            var_48 = wp::load(var_45);
            var_47 = (var_48 == var_18);
            var_49 = var_41 || var_47;
            var_50 = var_35 || var_49;
            if (var_50) {
                // return                                                                         <L 92>
                continue;
            }
            // if not x_target_reached[tid]:                                                      <L 93>
            var_51 = wp::address(var_x_target_reached, var_33);
            var_53 = wp::load(var_51);
            var_52 = wp::unot(var_53);
            if (var_52) {
                // satisfied_val[node2env[tid + affine_verts_num]] = False                        <L 94>
                var_55 = wp::add(var_33, var_affine_verts_num);
                var_56 = wp::address(var_node2env, var_55);
                var_57 = wp::load(var_56);
                wp::array_store(var_satisfied_val, var_57, var_54);
            }
        }
        var_58 = wp::where(var_5, var_0, var_33);
    }
}



extern "C" __global__ void check_is_satisfied_kernel_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<bool> var_y_target_reached,
    wp::array_t<bool> var_y_has_constraint,
    wp::array_t<bool> var_x_target_reached,
    wp::array_t<bool> var_x_has_constraint,
    wp::array_t<bool> var_satisfied_val,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<bool> adj_y_target_reached,
    wp::array_t<bool> adj_y_has_constraint,
    wp::array_t<bool> adj_x_target_reached,
    wp::array_t<bool> adj_x_has_constraint,
    wp::array_t<bool> adj_satisfied_val,
    wp::int32 adj_affine_verts_num,
    wp::array_t<wp::int32> adj_body_env_id,
    wp::array_t<wp::int32> adj_node2env,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::int32 var_1 = 0;
        wp::shape_t* var_2;
        wp::int32 var_3;
        wp::shape_t var_4;
        bool var_5;
        bool* var_6;
        bool var_7;
        bool var_8;
        wp::int32* var_9;
        wp::int32* var_10;
        wp::int32 var_11;
        const wp::int32 var_12 = 1;
        bool var_13;
        wp::int32 var_14;
        wp::int32* var_15;
        wp::int32* var_16;
        wp::int32 var_17;
        const wp::int32 var_18 = 2;
        bool var_19;
        wp::int32 var_20;
        bool var_21;
        bool var_22;
        bool* var_23;
        bool var_24;
        bool var_25;
        const bool var_26 = false;
        wp::int32* var_27;
        wp::int32 var_28;
        const wp::int32 var_29 = 0;
        wp::shape_t* var_30;
        wp::int32 var_31;
        wp::shape_t var_32;
        wp::int32 var_33;
        bool* var_34;
        bool var_35;
        bool var_36;
        wp::int32 var_37;
        wp::int32* var_38;
        wp::int32* var_39;
        wp::int32 var_40;
        bool var_41;
        wp::int32 var_42;
        wp::int32 var_43;
        wp::int32* var_44;
        wp::int32* var_45;
        wp::int32 var_46;
        bool var_47;
        wp::int32 var_48;
        bool var_49;
        bool var_50;
        bool* var_51;
        bool var_52;
        bool var_53;
        const bool var_54 = false;
        wp::int32 var_55;
        wp::int32* var_56;
        wp::int32 var_57;
        wp::int32 var_58;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::shape_t adj_2 = {};
        wp::int32 adj_3 = {};
        wp::shape_t adj_4 = {};
        bool adj_5 = {};
        bool adj_6 = {};
        bool adj_7 = {};
        bool adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int32 adj_12 = {};
        bool adj_13 = {};
        wp::int32 adj_14 = {};
        wp::int32 adj_15 = {};
        wp::int32 adj_16 = {};
        wp::int32 adj_17 = {};
        wp::int32 adj_18 = {};
        bool adj_19 = {};
        wp::int32 adj_20 = {};
        bool adj_21 = {};
        bool adj_22 = {};
        bool adj_23 = {};
        bool adj_24 = {};
        bool adj_25 = {};
        bool adj_26 = {};
        wp::int32 adj_27 = {};
        wp::int32 adj_28 = {};
        wp::int32 adj_29 = {};
        wp::shape_t adj_30 = {};
        wp::int32 adj_31 = {};
        wp::shape_t adj_32 = {};
        wp::int32 adj_33 = {};
        bool adj_34 = {};
        bool adj_35 = {};
        bool adj_36 = {};
        wp::int32 adj_37 = {};
        wp::int32 adj_38 = {};
        wp::int32 adj_39 = {};
        wp::int32 adj_40 = {};
        bool adj_41 = {};
        wp::int32 adj_42 = {};
        wp::int32 adj_43 = {};
        wp::int32 adj_44 = {};
        wp::int32 adj_45 = {};
        wp::int32 adj_46 = {};
        bool adj_47 = {};
        wp::int32 adj_48 = {};
        bool adj_49 = {};
        bool adj_50 = {};
        bool adj_51 = {};
        bool adj_52 = {};
        bool adj_53 = {};
        bool adj_54 = {};
        wp::int32 adj_55 = {};
        wp::int32 adj_56 = {};
        wp::int32 adj_57 = {};
        wp::int32 adj_58 = {};
        //---------
        // forward
        // def check_is_satisfied_kernel(                                                         <L 67>
        // tid = wp.tid()                                                                         <L 78>
        var_0 = builtin_tid1d();
        // if tid < y_target_reached.shape[0]:                                                    <L 79>
        var_2 = &(var_y_target_reached.shape);
        var_4 = wp::load(var_2);
        var_3 = wp::extract(var_4, var_1);
        var_5 = (var_0 < var_3);
        if (var_5) {
            // if not y_has_constraint[tid] or (                                                  <L 80>
            var_6 = wp::address(var_y_has_constraint, var_0);
            var_8 = wp::load(var_6);
            var_7 = wp::unot(var_8);
            // (env_states[body_env_id[tid]] == ENV_STATE_INVALID) or (env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED)       <L 81>
            var_9 = wp::address(var_body_env_id, var_0);
            var_11 = wp::load(var_9);
            var_10 = wp::address(var_env_states, var_11);
            var_14 = wp::load(var_10);
            var_13 = (var_14 == var_12);
            var_15 = wp::address(var_body_env_id, var_0);
            var_17 = wp::load(var_15);
            var_16 = wp::address(var_env_states, var_17);
            var_20 = wp::load(var_16);
            var_19 = (var_20 == var_18);
            var_21 = var_13 || var_19;
            var_22 = var_7 || var_21;
            if (var_22) {
                // return                                                                         <L 83>
                goto label0;
            }
            // if not y_target_reached[tid]:                                                      <L 84>
            var_23 = wp::address(var_y_target_reached, var_0);
            var_25 = wp::load(var_23);
            var_24 = wp::unot(var_25);
            if (var_24) {
                // satisfied_val[body_env_id[tid]] = False                                        <L 85>
                var_27 = wp::address(var_body_env_id, var_0);
                var_28 = wp::load(var_27);
                // wp::array_store(var_satisfied_val, var_28, var_26);
            }
        }
        if (!var_5) {
            // tid -= y_target_reached.shape[0]                                                   <L 87>
            var_30 = &(var_y_target_reached.shape);
            var_32 = wp::load(var_30);
            var_31 = wp::extract(var_32, var_29);
            var_33 = wp::sub(var_0, var_31);
            // if not x_has_constraint[tid] or (                                                  <L 88>
            var_34 = wp::address(var_x_has_constraint, var_33);
            var_36 = wp::load(var_34);
            var_35 = wp::unot(var_36);
            // (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID)                <L 89>
            var_37 = wp::add(var_33, var_affine_verts_num);
            var_38 = wp::address(var_node2env, var_37);
            var_40 = wp::load(var_38);
            var_39 = wp::address(var_env_states, var_40);
            var_42 = wp::load(var_39);
            var_41 = (var_42 == var_12);
            // or (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED)       <L 90>
            var_43 = wp::add(var_33, var_affine_verts_num);
            var_44 = wp::address(var_node2env, var_43);
            var_46 = wp::load(var_44);
            var_45 = wp::address(var_env_states, var_46);
            var_48 = wp::load(var_45);
            var_47 = (var_48 == var_18);
            var_49 = var_41 || var_47;
            var_50 = var_35 || var_49;
            if (var_50) {
                // return                                                                         <L 92>
                goto label1;
            }
            // if not x_target_reached[tid]:                                                      <L 93>
            var_51 = wp::address(var_x_target_reached, var_33);
            var_53 = wp::load(var_51);
            var_52 = wp::unot(var_53);
            if (var_52) {
                // satisfied_val[node2env[tid + affine_verts_num]] = False                        <L 94>
                var_55 = wp::add(var_33, var_affine_verts_num);
                var_56 = wp::address(var_node2env, var_55);
                var_57 = wp::load(var_56);
                // wp::array_store(var_satisfied_val, var_57, var_54);
            }
        }
        var_58 = wp::where(var_5, var_0, var_33);
        //---------
        // reverse
        wp::adj_where(var_5, var_0, var_33, adj_5, adj_0, adj_33, adj_58);
        if (!var_5) {
            if (var_52) {
                wp::adj_array_store(var_satisfied_val, var_57, var_54, adj_satisfied_val, adj_56, adj_54);
                wp::adj_load(var_56, adj_56, adj_57);
                wp::adj_address(var_node2env, var_55, adj_node2env, adj_55, adj_56);
                wp::adj_add(var_33, var_affine_verts_num, adj_33, adj_affine_verts_num, adj_55);
                // adj: satisfied_val[node2env[tid + affine_verts_num]] = False                   <L 94>
            }
            wp::adj_unot(var_53, adj_51, adj_52);
            wp::adj_load(var_51, adj_51, adj_53);
            wp::adj_address(var_x_target_reached, var_33, adj_x_target_reached, adj_33, adj_51);
            // adj: if not x_target_reached[tid]:                                                 <L 93>
            if (var_50) {
                label1:;
                // adj: return                                                                    <L 92>
            }
            wp::adj_load(var_45, adj_45, adj_48);
            wp::adj_address(var_env_states, var_46, adj_env_states, adj_44, adj_45);
            wp::adj_load(var_44, adj_44, adj_46);
            wp::adj_address(var_node2env, var_43, adj_node2env, adj_43, adj_44);
            wp::adj_add(var_33, var_affine_verts_num, adj_33, adj_affine_verts_num, adj_43);
            // adj: or (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED)  <L 90>
            wp::adj_load(var_39, adj_39, adj_42);
            wp::adj_address(var_env_states, var_40, adj_env_states, adj_38, adj_39);
            wp::adj_load(var_38, adj_38, adj_40);
            wp::adj_address(var_node2env, var_37, adj_node2env, adj_37, adj_38);
            wp::adj_add(var_33, var_affine_verts_num, adj_33, adj_affine_verts_num, adj_37);
            // adj: (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID)           <L 89>
            wp::adj_unot(var_36, adj_34, adj_35);
            wp::adj_load(var_34, adj_34, adj_36);
            wp::adj_address(var_x_has_constraint, var_33, adj_x_has_constraint, adj_33, adj_34);
            // adj: if not x_has_constraint[tid] or (                                             <L 88>
            wp::adj_sub(var_0, var_31, adj_0, adj_31, adj_33);
            wp::adj_extract(var_32, var_29, adj_30, adj_29, adj_31);
            wp::adj_load(var_30, adj_30, adj_32);
            adj_y_target_reached.shape = adj_30;
            // adj: tid -= y_target_reached.shape[0]                                              <L 87>
        }
        if (var_5) {
            if (var_24) {
                wp::adj_array_store(var_satisfied_val, var_28, var_26, adj_satisfied_val, adj_27, adj_26);
                wp::adj_load(var_27, adj_27, adj_28);
                wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_27);
                // adj: satisfied_val[body_env_id[tid]] = False                                   <L 85>
            }
            wp::adj_unot(var_25, adj_23, adj_24);
            wp::adj_load(var_23, adj_23, adj_25);
            wp::adj_address(var_y_target_reached, var_0, adj_y_target_reached, adj_0, adj_23);
            // adj: if not y_target_reached[tid]:                                                 <L 84>
            if (var_22) {
                label0:;
                // adj: return                                                                    <L 83>
            }
            wp::adj_load(var_16, adj_16, adj_20);
            wp::adj_address(var_env_states, var_17, adj_env_states, adj_15, adj_16);
            wp::adj_load(var_15, adj_15, adj_17);
            wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_15);
            wp::adj_load(var_10, adj_10, adj_14);
            wp::adj_address(var_env_states, var_11, adj_env_states, adj_9, adj_10);
            wp::adj_load(var_9, adj_9, adj_11);
            wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_9);
            // adj: (env_states[body_env_id[tid]] == ENV_STATE_INVALID) or (env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED)  <L 81>
            wp::adj_unot(var_8, adj_6, adj_7);
            wp::adj_load(var_6, adj_6, adj_8);
            wp::adj_address(var_y_has_constraint, var_0, adj_y_has_constraint, adj_0, adj_6);
            // adj: if not y_has_constraint[tid] or (                                             <L 80>
        }
        wp::adj_extract(var_4, var_1, adj_2, adj_1, adj_3);
        wp::adj_load(var_2, adj_2, adj_4);
        adj_y_target_reached.shape = adj_2;
        // adj: if tid < y_target_reached.shape[0]:                                               <L 79>
        // adj: tid = wp.tid()                                                                    <L 78>
        // adj: def check_is_satisfied_kernel(                                                    <L 67>
        continue;
    }
}

