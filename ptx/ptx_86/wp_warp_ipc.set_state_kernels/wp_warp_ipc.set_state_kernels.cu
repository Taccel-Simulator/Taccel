#include "hip/hip_runtime.h"

#define WP_TILE_BLOCK_DIM 256
#define WP_NO_CRT
#include "builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx, dim)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)

extern "C" {
}


extern "C" __global__ void set_sim_soft_state_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_soft_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_verts_positions,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x_hat,
    wp::int32 var_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::int32 var_9;
        wp::int32 var_10;
        //---------
        // forward
        // def set_sim_soft_state(                                                                <L 83>
        // tid = wp.tid()                                                                         <L 90>
        var_0 = builtin_tid1d();
        // mask = soft_update_mask[tid]                                                           <L 91>
        var_1 = wp::address(var_soft_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 92>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 93>
            continue;
        }
        // pos = soft_verts_positions[tid]                                                        <L 94>
        var_6 = wp::address(var_soft_verts_positions, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // x[tid + affine_verts_num] = pos                                                        <L 95>
        var_9 = wp::add(var_0, var_affine_verts_num);
        wp::array_store(var_x, var_9, var_7);
        // x_hat[tid + affine_verts_num] = pos                                                    <L 96>
        var_10 = wp::add(var_0, var_affine_verts_num);
        wp::array_store(var_x_hat, var_10, var_7);
    }
}



extern "C" __global__ void set_sim_soft_state_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_soft_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_verts_positions,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x_hat,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::int32> adj_soft_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_soft_verts_positions,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x_hat,
    wp::int32 adj_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::int32 var_9;
        wp::int32 var_10;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::vec_t<3,wp::float64> adj_6 = {};
        wp::vec_t<3,wp::float64> adj_7 = {};
        wp::vec_t<3,wp::float64> adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        //---------
        // forward
        // def set_sim_soft_state(                                                                <L 83>
        // tid = wp.tid()                                                                         <L 90>
        var_0 = builtin_tid1d();
        // mask = soft_update_mask[tid]                                                           <L 91>
        var_1 = wp::address(var_soft_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 92>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 93>
            goto label0;
        }
        // pos = soft_verts_positions[tid]                                                        <L 94>
        var_6 = wp::address(var_soft_verts_positions, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // x[tid + affine_verts_num] = pos                                                        <L 95>
        var_9 = wp::add(var_0, var_affine_verts_num);
        // wp::array_store(var_x, var_9, var_7);
        // x_hat[tid + affine_verts_num] = pos                                                    <L 96>
        var_10 = wp::add(var_0, var_affine_verts_num);
        // wp::array_store(var_x_hat, var_10, var_7);
        //---------
        // reverse
        wp::adj_array_store(var_x_hat, var_10, var_7, adj_x_hat, adj_10, adj_7);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_10);
        // adj: x_hat[tid + affine_verts_num] = pos                                               <L 96>
        wp::adj_array_store(var_x, var_9, var_7, adj_x, adj_9, adj_7);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_9);
        // adj: x[tid + affine_verts_num] = pos                                                   <L 95>
        wp::adj_copy(var_8, adj_6, adj_7);
        wp::adj_load(var_6, adj_6, adj_8);
        wp::adj_address(var_soft_verts_positions, var_0, adj_soft_verts_positions, adj_0, adj_6);
        // adj: pos = soft_verts_positions[tid]                                                   <L 94>
        if (var_5) {
            label0:;
            // adj: return                                                                        <L 93>
        }
        // adj: if mask == 0:                                                                     <L 92>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_soft_update_mask, var_0, adj_soft_update_mask, adj_0, adj_1);
        // adj: mask = soft_update_mask[tid]                                                      <L 91>
        // adj: tid = wp.tid()                                                                    <L 90>
        // adj: def set_sim_soft_state(                                                           <L 83>
        continue;
    }
}



extern "C" __global__ void set_sim_affine_state_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_affine_update_mask,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_affine_rotations,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_translations,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y_hat,
    wp::array_t<wp::vec_t<3,wp::float64>> var_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ABD_centers)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::mat_t<3,3,wp::float64>* var_9;
        wp::mat_t<3,3,wp::float64> var_10;
        wp::mat_t<3,3,wp::float64> var_11;
        wp::vec_t<3,wp::float64>* var_12;
        wp::vec_t<3,wp::float64> var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64>* var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        const wp::float64 var_18 = 1.0;
        const wp::float64 var_19 = 0.0;
        wp::vec_t<3,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::vec_t<3,wp::float64> var_22;
        wp::vec_t<3,wp::float64> var_23;
        wp::vec_t<3,wp::float64> var_24;
        wp::vec_t<3,wp::float64> var_25;
        wp::vec_t<3,wp::float64> var_26;
        wp::vec_t<3,wp::float64> var_27;
        wp::vec_t<3,wp::float64> var_28;
        wp::vec_t<3,wp::float64> var_29;
        wp::vec_t<3,wp::float64> var_30;
        wp::vec_t<3,wp::float64> var_31;
        wp::vec_t<3,wp::float64> var_32;
        wp::vec_t<3,wp::float64> var_33;
        wp::vec_t<3,wp::float64> var_34;
        wp::vec_t<3,wp::float64> var_35;
        wp::vec_t<3,wp::float64> var_36;
        wp::vec_t<3,wp::float64> var_37;
        wp::vec_t<3,wp::float64> var_38;
        wp::vec_t<3,wp::float64> var_39;
        wp::vec_t<3,wp::float64> var_40;
        wp::vec_t<3,wp::float64> var_41;
        const wp::int32 var_42 = 0;
        wp::float64 var_43;
        const wp::int32 var_44 = 1;
        wp::float64 var_45;
        const wp::int32 var_46 = 2;
        wp::float64 var_47;
        const wp::int32 var_48 = 0;
        wp::float64 var_49;
        const wp::int32 var_50 = 1;
        wp::float64 var_51;
        const wp::int32 var_52 = 2;
        wp::float64 var_53;
        const wp::int32 var_54 = 0;
        wp::float64 var_55;
        const wp::int32 var_56 = 1;
        wp::float64 var_57;
        const wp::int32 var_58 = 2;
        wp::float64 var_59;
        const wp::int32 var_60 = 0;
        wp::float64 var_61;
        const wp::int32 var_62 = 1;
        wp::float64 var_63;
        const wp::int32 var_64 = 2;
        wp::float64 var_65;
        wp::vec_t<12,wp::float64> var_66;
        //---------
        // forward
        // def set_sim_affine_state(                                                              <L 8>
        // tid = wp.tid()                                                                         <L 17>
        var_0 = builtin_tid1d();
        // mask = affine_update_mask[tid]                                                         <L 18>
        var_1 = wp::address(var_affine_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 19>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 20>
            continue;
        }
        // virtual_center = virtual_object_centers[tid]                                           <L 21>
        var_6 = wp::address(var_virtual_object_centers, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // rot = affine_rotations[tid]                                                            <L 22>
        var_9 = wp::address(var_affine_rotations, var_0);
        var_11 = wp::load(var_9);
        var_10 = wp::copy(var_11);
        // trans = affine_translations[tid]                                                       <L 23>
        var_12 = wp::address(var_affine_translations, var_0);
        var_14 = wp::load(var_12);
        var_13 = wp::copy(var_14);
        // rest_O = ABD_centers[tid]                                                              <L 24>
        var_15 = wp::address(var_ABD_centers, var_0);
        var_17 = wp::load(var_15);
        var_16 = wp::copy(var_17);
        // rest_A = rest_O + wp.vec3d(_1, _0, _0)                                                 <L 25>
        var_20 = wp::vec_t<3,wp::float64>(var_18, var_19, var_19);
        var_21 = wp::add(var_16, var_20);
        // rest_B = rest_O + wp.vec3d(_0, _1, _0)                                                 <L 26>
        var_22 = wp::vec_t<3,wp::float64>(var_19, var_18, var_19);
        var_23 = wp::add(var_16, var_22);
        // rest_C = rest_O + wp.vec3d(_0, _0, _1)                                                 <L 27>
        var_24 = wp::vec_t<3,wp::float64>(var_19, var_19, var_18);
        var_25 = wp::add(var_16, var_24);
        // O = rot @ (rest_O - virtual_center) + trans + virtual_center                           <L 28>
        var_26 = wp::sub(var_16, var_7);
        var_27 = wp::mul(var_10, var_26);
        var_28 = wp::add(var_27, var_13);
        var_29 = wp::add(var_28, var_7);
        // A = rot @ (rest_A - virtual_center) + trans + virtual_center                           <L 29>
        var_30 = wp::sub(var_21, var_7);
        var_31 = wp::mul(var_10, var_30);
        var_32 = wp::add(var_31, var_13);
        var_33 = wp::add(var_32, var_7);
        // B = rot @ (rest_B - virtual_center) + trans + virtual_center                           <L 30>
        var_34 = wp::sub(var_23, var_7);
        var_35 = wp::mul(var_10, var_34);
        var_36 = wp::add(var_35, var_13);
        var_37 = wp::add(var_36, var_7);
        // C = rot @ (rest_C - virtual_center) + trans + virtual_center                           <L 31>
        var_38 = wp::sub(var_25, var_7);
        var_39 = wp::mul(var_10, var_38);
        var_40 = wp::add(var_39, var_13);
        var_41 = wp::add(var_40, var_7);
        // new_y = vec12d(O[0], O[1], O[2], A[0], A[1], A[2], B[0], B[1], B[2], C[0], C[1], C[2])       <L 32>
        var_43 = wp::extract(var_29, var_42);
        var_45 = wp::extract(var_29, var_44);
        var_47 = wp::extract(var_29, var_46);
        var_49 = wp::extract(var_33, var_48);
        var_51 = wp::extract(var_33, var_50);
        var_53 = wp::extract(var_33, var_52);
        var_55 = wp::extract(var_37, var_54);
        var_57 = wp::extract(var_37, var_56);
        var_59 = wp::extract(var_37, var_58);
        var_61 = wp::extract(var_41, var_60);
        var_63 = wp::extract(var_41, var_62);
        var_65 = wp::extract(var_41, var_64);
        var_66 = wp::vec_t<12,wp::float64>({var_43, var_45, var_47, var_49, var_51, var_53, var_55, var_57, var_59, var_61, var_63, var_65});
        // y[tid] = new_y                                                                         <L 33>
        wp::array_store(var_y, var_0, var_66);
        // y_hat[tid] = new_y                                                                     <L 34>
        wp::array_store(var_y_hat, var_0, var_66);
    }
}



extern "C" __global__ void set_sim_affine_state_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_affine_update_mask,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_affine_rotations,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_translations,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y_hat,
    wp::array_t<wp::vec_t<3,wp::float64>> var_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ABD_centers,
    wp::array_t<wp::int32> adj_affine_update_mask,
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_affine_rotations,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_affine_translations,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y_hat,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_ABD_centers)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::mat_t<3,3,wp::float64>* var_9;
        wp::mat_t<3,3,wp::float64> var_10;
        wp::mat_t<3,3,wp::float64> var_11;
        wp::vec_t<3,wp::float64>* var_12;
        wp::vec_t<3,wp::float64> var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64>* var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        const wp::float64 var_18 = 1.0;
        const wp::float64 var_19 = 0.0;
        wp::vec_t<3,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::vec_t<3,wp::float64> var_22;
        wp::vec_t<3,wp::float64> var_23;
        wp::vec_t<3,wp::float64> var_24;
        wp::vec_t<3,wp::float64> var_25;
        wp::vec_t<3,wp::float64> var_26;
        wp::vec_t<3,wp::float64> var_27;
        wp::vec_t<3,wp::float64> var_28;
        wp::vec_t<3,wp::float64> var_29;
        wp::vec_t<3,wp::float64> var_30;
        wp::vec_t<3,wp::float64> var_31;
        wp::vec_t<3,wp::float64> var_32;
        wp::vec_t<3,wp::float64> var_33;
        wp::vec_t<3,wp::float64> var_34;
        wp::vec_t<3,wp::float64> var_35;
        wp::vec_t<3,wp::float64> var_36;
        wp::vec_t<3,wp::float64> var_37;
        wp::vec_t<3,wp::float64> var_38;
        wp::vec_t<3,wp::float64> var_39;
        wp::vec_t<3,wp::float64> var_40;
        wp::vec_t<3,wp::float64> var_41;
        const wp::int32 var_42 = 0;
        wp::float64 var_43;
        const wp::int32 var_44 = 1;
        wp::float64 var_45;
        const wp::int32 var_46 = 2;
        wp::float64 var_47;
        const wp::int32 var_48 = 0;
        wp::float64 var_49;
        const wp::int32 var_50 = 1;
        wp::float64 var_51;
        const wp::int32 var_52 = 2;
        wp::float64 var_53;
        const wp::int32 var_54 = 0;
        wp::float64 var_55;
        const wp::int32 var_56 = 1;
        wp::float64 var_57;
        const wp::int32 var_58 = 2;
        wp::float64 var_59;
        const wp::int32 var_60 = 0;
        wp::float64 var_61;
        const wp::int32 var_62 = 1;
        wp::float64 var_63;
        const wp::int32 var_64 = 2;
        wp::float64 var_65;
        wp::vec_t<12,wp::float64> var_66;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::vec_t<3,wp::float64> adj_6 = {};
        wp::vec_t<3,wp::float64> adj_7 = {};
        wp::vec_t<3,wp::float64> adj_8 = {};
        wp::mat_t<3,3,wp::float64> adj_9 = {};
        wp::mat_t<3,3,wp::float64> adj_10 = {};
        wp::mat_t<3,3,wp::float64> adj_11 = {};
        wp::vec_t<3,wp::float64> adj_12 = {};
        wp::vec_t<3,wp::float64> adj_13 = {};
        wp::vec_t<3,wp::float64> adj_14 = {};
        wp::vec_t<3,wp::float64> adj_15 = {};
        wp::vec_t<3,wp::float64> adj_16 = {};
        wp::vec_t<3,wp::float64> adj_17 = {};
        wp::float64 adj_18 = {};
        wp::float64 adj_19 = {};
        wp::vec_t<3,wp::float64> adj_20 = {};
        wp::vec_t<3,wp::float64> adj_21 = {};
        wp::vec_t<3,wp::float64> adj_22 = {};
        wp::vec_t<3,wp::float64> adj_23 = {};
        wp::vec_t<3,wp::float64> adj_24 = {};
        wp::vec_t<3,wp::float64> adj_25 = {};
        wp::vec_t<3,wp::float64> adj_26 = {};
        wp::vec_t<3,wp::float64> adj_27 = {};
        wp::vec_t<3,wp::float64> adj_28 = {};
        wp::vec_t<3,wp::float64> adj_29 = {};
        wp::vec_t<3,wp::float64> adj_30 = {};
        wp::vec_t<3,wp::float64> adj_31 = {};
        wp::vec_t<3,wp::float64> adj_32 = {};
        wp::vec_t<3,wp::float64> adj_33 = {};
        wp::vec_t<3,wp::float64> adj_34 = {};
        wp::vec_t<3,wp::float64> adj_35 = {};
        wp::vec_t<3,wp::float64> adj_36 = {};
        wp::vec_t<3,wp::float64> adj_37 = {};
        wp::vec_t<3,wp::float64> adj_38 = {};
        wp::vec_t<3,wp::float64> adj_39 = {};
        wp::vec_t<3,wp::float64> adj_40 = {};
        wp::vec_t<3,wp::float64> adj_41 = {};
        wp::int32 adj_42 = {};
        wp::float64 adj_43 = {};
        wp::int32 adj_44 = {};
        wp::float64 adj_45 = {};
        wp::int32 adj_46 = {};
        wp::float64 adj_47 = {};
        wp::int32 adj_48 = {};
        wp::float64 adj_49 = {};
        wp::int32 adj_50 = {};
        wp::float64 adj_51 = {};
        wp::int32 adj_52 = {};
        wp::float64 adj_53 = {};
        wp::int32 adj_54 = {};
        wp::float64 adj_55 = {};
        wp::int32 adj_56 = {};
        wp::float64 adj_57 = {};
        wp::int32 adj_58 = {};
        wp::float64 adj_59 = {};
        wp::int32 adj_60 = {};
        wp::float64 adj_61 = {};
        wp::int32 adj_62 = {};
        wp::float64 adj_63 = {};
        wp::int32 adj_64 = {};
        wp::float64 adj_65 = {};
        wp::vec_t<12,wp::float64> adj_66 = {};
        //---------
        // forward
        // def set_sim_affine_state(                                                              <L 8>
        // tid = wp.tid()                                                                         <L 17>
        var_0 = builtin_tid1d();
        // mask = affine_update_mask[tid]                                                         <L 18>
        var_1 = wp::address(var_affine_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 19>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 20>
            goto label0;
        }
        // virtual_center = virtual_object_centers[tid]                                           <L 21>
        var_6 = wp::address(var_virtual_object_centers, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // rot = affine_rotations[tid]                                                            <L 22>
        var_9 = wp::address(var_affine_rotations, var_0);
        var_11 = wp::load(var_9);
        var_10 = wp::copy(var_11);
        // trans = affine_translations[tid]                                                       <L 23>
        var_12 = wp::address(var_affine_translations, var_0);
        var_14 = wp::load(var_12);
        var_13 = wp::copy(var_14);
        // rest_O = ABD_centers[tid]                                                              <L 24>
        var_15 = wp::address(var_ABD_centers, var_0);
        var_17 = wp::load(var_15);
        var_16 = wp::copy(var_17);
        // rest_A = rest_O + wp.vec3d(_1, _0, _0)                                                 <L 25>
        var_20 = wp::vec_t<3,wp::float64>(var_18, var_19, var_19);
        var_21 = wp::add(var_16, var_20);
        // rest_B = rest_O + wp.vec3d(_0, _1, _0)                                                 <L 26>
        var_22 = wp::vec_t<3,wp::float64>(var_19, var_18, var_19);
        var_23 = wp::add(var_16, var_22);
        // rest_C = rest_O + wp.vec3d(_0, _0, _1)                                                 <L 27>
        var_24 = wp::vec_t<3,wp::float64>(var_19, var_19, var_18);
        var_25 = wp::add(var_16, var_24);
        // O = rot @ (rest_O - virtual_center) + trans + virtual_center                           <L 28>
        var_26 = wp::sub(var_16, var_7);
        var_27 = wp::mul(var_10, var_26);
        var_28 = wp::add(var_27, var_13);
        var_29 = wp::add(var_28, var_7);
        // A = rot @ (rest_A - virtual_center) + trans + virtual_center                           <L 29>
        var_30 = wp::sub(var_21, var_7);
        var_31 = wp::mul(var_10, var_30);
        var_32 = wp::add(var_31, var_13);
        var_33 = wp::add(var_32, var_7);
        // B = rot @ (rest_B - virtual_center) + trans + virtual_center                           <L 30>
        var_34 = wp::sub(var_23, var_7);
        var_35 = wp::mul(var_10, var_34);
        var_36 = wp::add(var_35, var_13);
        var_37 = wp::add(var_36, var_7);
        // C = rot @ (rest_C - virtual_center) + trans + virtual_center                           <L 31>
        var_38 = wp::sub(var_25, var_7);
        var_39 = wp::mul(var_10, var_38);
        var_40 = wp::add(var_39, var_13);
        var_41 = wp::add(var_40, var_7);
        // new_y = vec12d(O[0], O[1], O[2], A[0], A[1], A[2], B[0], B[1], B[2], C[0], C[1], C[2])       <L 32>
        var_43 = wp::extract(var_29, var_42);
        var_45 = wp::extract(var_29, var_44);
        var_47 = wp::extract(var_29, var_46);
        var_49 = wp::extract(var_33, var_48);
        var_51 = wp::extract(var_33, var_50);
        var_53 = wp::extract(var_33, var_52);
        var_55 = wp::extract(var_37, var_54);
        var_57 = wp::extract(var_37, var_56);
        var_59 = wp::extract(var_37, var_58);
        var_61 = wp::extract(var_41, var_60);
        var_63 = wp::extract(var_41, var_62);
        var_65 = wp::extract(var_41, var_64);
        var_66 = wp::vec_t<12,wp::float64>({var_43, var_45, var_47, var_49, var_51, var_53, var_55, var_57, var_59, var_61, var_63, var_65});
        // y[tid] = new_y                                                                         <L 33>
        // wp::array_store(var_y, var_0, var_66);
        // y_hat[tid] = new_y                                                                     <L 34>
        // wp::array_store(var_y_hat, var_0, var_66);
        //---------
        // reverse
        wp::adj_array_store(var_y_hat, var_0, var_66, adj_y_hat, adj_0, adj_66);
        // adj: y_hat[tid] = new_y                                                                <L 34>
        wp::adj_array_store(var_y, var_0, var_66, adj_y, adj_0, adj_66);
        // adj: y[tid] = new_y                                                                    <L 33>
        wp::adj_vec_t({var_43, var_45, var_47, var_49, var_51, var_53, var_55, var_57, var_59, var_61, var_63, var_65}, {&adj_43, &adj_45, &adj_47, &adj_49, &adj_51, &adj_53, &adj_55, &adj_57, &adj_59, &adj_61, &adj_63, &adj_65}, adj_66);
        wp::adj_extract(var_41, var_64, adj_41, adj_64, adj_65);
        wp::adj_extract(var_41, var_62, adj_41, adj_62, adj_63);
        wp::adj_extract(var_41, var_60, adj_41, adj_60, adj_61);
        wp::adj_extract(var_37, var_58, adj_37, adj_58, adj_59);
        wp::adj_extract(var_37, var_56, adj_37, adj_56, adj_57);
        wp::adj_extract(var_37, var_54, adj_37, adj_54, adj_55);
        wp::adj_extract(var_33, var_52, adj_33, adj_52, adj_53);
        wp::adj_extract(var_33, var_50, adj_33, adj_50, adj_51);
        wp::adj_extract(var_33, var_48, adj_33, adj_48, adj_49);
        wp::adj_extract(var_29, var_46, adj_29, adj_46, adj_47);
        wp::adj_extract(var_29, var_44, adj_29, adj_44, adj_45);
        wp::adj_extract(var_29, var_42, adj_29, adj_42, adj_43);
        // adj: new_y = vec12d(O[0], O[1], O[2], A[0], A[1], A[2], B[0], B[1], B[2], C[0], C[1], C[2])  <L 32>
        wp::adj_add(var_40, var_7, adj_40, adj_7, adj_41);
        wp::adj_add(var_39, var_13, adj_39, adj_13, adj_40);
        wp::adj_mul(var_10, var_38, adj_10, adj_38, adj_39);
        wp::adj_sub(var_25, var_7, adj_25, adj_7, adj_38);
        // adj: C = rot @ (rest_C - virtual_center) + trans + virtual_center                      <L 31>
        wp::adj_add(var_36, var_7, adj_36, adj_7, adj_37);
        wp::adj_add(var_35, var_13, adj_35, adj_13, adj_36);
        wp::adj_mul(var_10, var_34, adj_10, adj_34, adj_35);
        wp::adj_sub(var_23, var_7, adj_23, adj_7, adj_34);
        // adj: B = rot @ (rest_B - virtual_center) + trans + virtual_center                      <L 30>
        wp::adj_add(var_32, var_7, adj_32, adj_7, adj_33);
        wp::adj_add(var_31, var_13, adj_31, adj_13, adj_32);
        wp::adj_mul(var_10, var_30, adj_10, adj_30, adj_31);
        wp::adj_sub(var_21, var_7, adj_21, adj_7, adj_30);
        // adj: A = rot @ (rest_A - virtual_center) + trans + virtual_center                      <L 29>
        wp::adj_add(var_28, var_7, adj_28, adj_7, adj_29);
        wp::adj_add(var_27, var_13, adj_27, adj_13, adj_28);
        wp::adj_mul(var_10, var_26, adj_10, adj_26, adj_27);
        wp::adj_sub(var_16, var_7, adj_16, adj_7, adj_26);
        // adj: O = rot @ (rest_O - virtual_center) + trans + virtual_center                      <L 28>
        wp::adj_add(var_16, var_24, adj_16, adj_24, adj_25);
        wp::adj_vec_t(var_19, var_19, var_18, adj_19, adj_19, adj_18, adj_24);
        // adj: rest_C = rest_O + wp.vec3d(_0, _0, _1)                                            <L 27>
        wp::adj_add(var_16, var_22, adj_16, adj_22, adj_23);
        wp::adj_vec_t(var_19, var_18, var_19, adj_19, adj_18, adj_19, adj_22);
        // adj: rest_B = rest_O + wp.vec3d(_0, _1, _0)                                            <L 26>
        wp::adj_add(var_16, var_20, adj_16, adj_20, adj_21);
        wp::adj_vec_t(var_18, var_19, var_19, adj_18, adj_19, adj_19, adj_20);
        // adj: rest_A = rest_O + wp.vec3d(_1, _0, _0)                                            <L 25>
        wp::adj_copy(var_17, adj_15, adj_16);
        wp::adj_load(var_15, adj_15, adj_17);
        wp::adj_address(var_ABD_centers, var_0, adj_ABD_centers, adj_0, adj_15);
        // adj: rest_O = ABD_centers[tid]                                                         <L 24>
        wp::adj_copy(var_14, adj_12, adj_13);
        wp::adj_load(var_12, adj_12, adj_14);
        wp::adj_address(var_affine_translations, var_0, adj_affine_translations, adj_0, adj_12);
        // adj: trans = affine_translations[tid]                                                  <L 23>
        wp::adj_copy(var_11, adj_9, adj_10);
        wp::adj_load(var_9, adj_9, adj_11);
        wp::adj_address(var_affine_rotations, var_0, adj_affine_rotations, adj_0, adj_9);
        // adj: rot = affine_rotations[tid]                                                       <L 22>
        wp::adj_copy(var_8, adj_6, adj_7);
        wp::adj_load(var_6, adj_6, adj_8);
        wp::adj_address(var_virtual_object_centers, var_0, adj_virtual_object_centers, adj_0, adj_6);
        // adj: virtual_center = virtual_object_centers[tid]                                      <L 21>
        if (var_5) {
            label0:;
            // adj: return                                                                        <L 20>
        }
        // adj: if mask == 0:                                                                     <L 19>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_affine_update_mask, var_0, adj_affine_update_mask, adj_0, adj_1);
        // adj: mask = affine_update_mask[tid]                                                    <L 18>
        // adj: tid = wp.tid()                                                                    <L 17>
        // adj: def set_sim_affine_state(                                                         <L 8>
        continue;
    }
}



extern "C" __global__ void set_sim_soft_vel_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_soft_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_verts_vel,
    wp::array_t<wp::vec_t<3,wp::float64>> var_v_x,
    wp::int32 var_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::int32 var_9;
        //---------
        // forward
        // def set_sim_soft_vel(                                                                  <L 100>
        // tid = wp.tid()                                                                         <L 106>
        var_0 = builtin_tid1d();
        // mask = soft_update_mask[tid]                                                           <L 107>
        var_1 = wp::address(var_soft_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 108>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 109>
            continue;
        }
        // vel = soft_verts_vel[tid]                                                              <L 110>
        var_6 = wp::address(var_soft_verts_vel, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // v_x[tid + affine_verts_num] = vel                                                      <L 111>
        var_9 = wp::add(var_0, var_affine_verts_num);
        wp::array_store(var_v_x, var_9, var_7);
    }
}



extern "C" __global__ void set_sim_soft_vel_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_soft_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> var_soft_verts_vel,
    wp::array_t<wp::vec_t<3,wp::float64>> var_v_x,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::int32> adj_soft_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_soft_verts_vel,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_v_x,
    wp::int32 adj_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::int32 var_9;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::vec_t<3,wp::float64> adj_6 = {};
        wp::vec_t<3,wp::float64> adj_7 = {};
        wp::vec_t<3,wp::float64> adj_8 = {};
        wp::int32 adj_9 = {};
        //---------
        // forward
        // def set_sim_soft_vel(                                                                  <L 100>
        // tid = wp.tid()                                                                         <L 106>
        var_0 = builtin_tid1d();
        // mask = soft_update_mask[tid]                                                           <L 107>
        var_1 = wp::address(var_soft_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 108>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 109>
            goto label0;
        }
        // vel = soft_verts_vel[tid]                                                              <L 110>
        var_6 = wp::address(var_soft_verts_vel, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // v_x[tid + affine_verts_num] = vel                                                      <L 111>
        var_9 = wp::add(var_0, var_affine_verts_num);
        // wp::array_store(var_v_x, var_9, var_7);
        //---------
        // reverse
        wp::adj_array_store(var_v_x, var_9, var_7, adj_v_x, adj_9, adj_7);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_9);
        // adj: v_x[tid + affine_verts_num] = vel                                                 <L 111>
        wp::adj_copy(var_8, adj_6, adj_7);
        wp::adj_load(var_6, adj_6, adj_8);
        wp::adj_address(var_soft_verts_vel, var_0, adj_soft_verts_vel, adj_0, adj_6);
        // adj: vel = soft_verts_vel[tid]                                                         <L 110>
        if (var_5) {
            label0:;
            // adj: return                                                                        <L 109>
        }
        // adj: if mask == 0:                                                                     <L 108>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_soft_update_mask, var_0, adj_soft_update_mask, adj_0, adj_1);
        // adj: mask = soft_update_mask[tid]                                                      <L 107>
        // adj: tid = wp.tid()                                                                    <L 106>
        // adj: def set_sim_soft_vel(                                                             <L 100>
        continue;
    }
}



extern "C" __global__ void set_sim_affine_vel_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_affine_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_linear_vel,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_angular_vel,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_v_y,
    wp::array_t<wp::vec_t<3,wp::float64>> var_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ABD_centers)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::vec_t<3,wp::float64>* var_9;
        wp::vec_t<3,wp::float64> var_10;
        wp::vec_t<3,wp::float64> var_11;
        wp::vec_t<3,wp::float64>* var_12;
        wp::vec_t<3,wp::float64> var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64>* var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::vec_t<12,wp::float64>* var_18;
        wp::vec_t<12,wp::float64> var_19;
        wp::vec_t<12,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        const wp::int32 var_22 = 0;
        wp::float64 var_23;
        const wp::int32 var_24 = 1;
        wp::float64 var_25;
        const wp::int32 var_26 = 2;
        wp::float64 var_27;
        wp::vec_t<3,wp::float64> var_28;
        const wp::int32 var_29 = 3;
        wp::float64 var_30;
        const wp::int32 var_31 = 4;
        wp::float64 var_32;
        const wp::int32 var_33 = 5;
        wp::float64 var_34;
        wp::vec_t<3,wp::float64> var_35;
        const wp::int32 var_36 = 6;
        wp::float64 var_37;
        const wp::int32 var_38 = 7;
        wp::float64 var_39;
        const wp::int32 var_40 = 8;
        wp::float64 var_41;
        wp::vec_t<3,wp::float64> var_42;
        const wp::int32 var_43 = 9;
        wp::float64 var_44;
        const wp::int32 var_45 = 10;
        wp::float64 var_46;
        const wp::int32 var_47 = 11;
        wp::float64 var_48;
        wp::vec_t<3,wp::float64> var_49;
        const wp::float64 var_50 = 1.0;
        const wp::int32 var_51 = 0;
        wp::float64 var_52;
        wp::float64 var_53;
        const wp::int32 var_54 = 1;
        wp::float64 var_55;
        wp::float64 var_56;
        const wp::int32 var_57 = 2;
        wp::float64 var_58;
        wp::float64 var_59;
        wp::vec_t<3,wp::float64> var_60;
        const wp::int32 var_61 = 0;
        wp::float64 var_62;
        wp::vec_t<3,wp::float64> var_63;
        wp::vec_t<3,wp::float64> var_64;
        const wp::int32 var_65 = 1;
        wp::float64 var_66;
        wp::vec_t<3,wp::float64> var_67;
        wp::vec_t<3,wp::float64> var_68;
        const wp::int32 var_69 = 2;
        wp::float64 var_70;
        wp::vec_t<3,wp::float64> var_71;
        wp::vec_t<3,wp::float64> var_72;
        wp::vec_t<3,wp::float64> var_73;
        wp::vec_t<3,wp::float64> var_74;
        wp::vec_t<3,wp::float64> var_75;
        wp::vec_t<3,wp::float64> var_76;
        wp::vec_t<3,wp::float64> var_77;
        wp::vec_t<3,wp::float64> var_78;
        wp::vec_t<3,wp::float64> var_79;
        wp::vec_t<3,wp::float64> var_80;
        wp::vec_t<3,wp::float64> var_81;
        wp::vec_t<3,wp::float64> var_82;
        wp::vec_t<3,wp::float64> var_83;
        wp::vec_t<3,wp::float64> var_84;
        const wp::int32 var_85 = 0;
        wp::float64 var_86;
        const wp::int32 var_87 = 1;
        wp::float64 var_88;
        const wp::int32 var_89 = 2;
        wp::float64 var_90;
        const wp::int32 var_91 = 0;
        wp::float64 var_92;
        const wp::int32 var_93 = 1;
        wp::float64 var_94;
        const wp::int32 var_95 = 2;
        wp::float64 var_96;
        const wp::int32 var_97 = 0;
        wp::float64 var_98;
        const wp::int32 var_99 = 1;
        wp::float64 var_100;
        const wp::int32 var_101 = 2;
        wp::float64 var_102;
        const wp::int32 var_103 = 0;
        wp::float64 var_104;
        const wp::int32 var_105 = 1;
        wp::float64 var_106;
        const wp::int32 var_107 = 2;
        wp::float64 var_108;
        wp::vec_t<12,wp::float64> var_109;
        //---------
        // forward
        // def set_sim_affine_vel(                                                                <L 38>
        // tid = wp.tid()                                                                         <L 47>
        var_0 = builtin_tid1d();
        // mask = affine_update_mask[tid]                                                         <L 48>
        var_1 = wp::address(var_affine_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 49>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 50>
            continue;
        }
        // linear_vel = affine_linear_vel[tid]                                                    <L 51>
        var_6 = wp::address(var_affine_linear_vel, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // angular_vel = affine_angular_vel[tid]                                                  <L 52>
        var_9 = wp::address(var_affine_angular_vel, var_0);
        var_11 = wp::load(var_9);
        var_10 = wp::copy(var_11);
        // rest_center = virtual_object_centers[tid]                                              <L 53>
        var_12 = wp::address(var_virtual_object_centers, var_0);
        var_14 = wp::load(var_12);
        var_13 = wp::copy(var_14);
        // rest_O = ABD_centers[tid]                                                              <L 54>
        var_15 = wp::address(var_ABD_centers, var_0);
        var_17 = wp::load(var_15);
        var_16 = wp::copy(var_17);
        // current_y = y[tid]                                                                     <L 55>
        var_18 = wp::address(var_y, var_0);
        var_20 = wp::load(var_18);
        var_19 = wp::copy(var_20);
        // w = rest_center - rest_O                                                               <L 56>
        var_21 = wp::sub(var_13, var_16);
        // O = wp.vec3d(current_y[0], current_y[1], current_y[2])                                 <L 57>
        var_23 = wp::extract(var_19, var_22);
        var_25 = wp::extract(var_19, var_24);
        var_27 = wp::extract(var_19, var_26);
        var_28 = wp::vec_t<3,wp::float64>(var_23, var_25, var_27);
        // A = wp.vec3d(current_y[3], current_y[4], current_y[5])                                 <L 58>
        var_30 = wp::extract(var_19, var_29);
        var_32 = wp::extract(var_19, var_31);
        var_34 = wp::extract(var_19, var_33);
        var_35 = wp::vec_t<3,wp::float64>(var_30, var_32, var_34);
        // B = wp.vec3d(current_y[6], current_y[7], current_y[8])                                 <L 59>
        var_37 = wp::extract(var_19, var_36);
        var_39 = wp::extract(var_19, var_38);
        var_41 = wp::extract(var_19, var_40);
        var_42 = wp::vec_t<3,wp::float64>(var_37, var_39, var_41);
        // C = wp.vec3d(current_y[9], current_y[10], current_y[11])                               <L 60>
        var_44 = wp::extract(var_19, var_43);
        var_46 = wp::extract(var_19, var_45);
        var_48 = wp::extract(var_19, var_47);
        var_49 = wp::vec_t<3,wp::float64>(var_44, var_46, var_48);
        // center = (_1 - w[0] - w[1] - w[2]) * O + w[0] * A + w[1] * B + w[2] * C                <L 61>
        var_52 = wp::extract(var_21, var_51);
        var_53 = wp::sub(var_50, var_52);
        var_55 = wp::extract(var_21, var_54);
        var_56 = wp::sub(var_53, var_55);
        var_58 = wp::extract(var_21, var_57);
        var_59 = wp::sub(var_56, var_58);
        var_60 = wp::mul(var_59, var_28);
        var_62 = wp::extract(var_21, var_61);
        var_63 = wp::mul(var_62, var_35);
        var_64 = wp::add(var_60, var_63);
        var_66 = wp::extract(var_21, var_65);
        var_67 = wp::mul(var_66, var_42);
        var_68 = wp::add(var_64, var_67);
        var_70 = wp::extract(var_21, var_69);
        var_71 = wp::mul(var_70, var_49);
        var_72 = wp::add(var_68, var_71);
        // vel_O = wp.cross(angular_vel, O - center) + linear_vel                                 <L 62>
        var_73 = wp::sub(var_28, var_72);
        var_74 = wp::cross(var_10, var_73);
        var_75 = wp::add(var_74, var_7);
        // vel_A = wp.cross(angular_vel, A - center) + linear_vel                                 <L 63>
        var_76 = wp::sub(var_35, var_72);
        var_77 = wp::cross(var_10, var_76);
        var_78 = wp::add(var_77, var_7);
        // vel_B = wp.cross(angular_vel, B - center) + linear_vel                                 <L 64>
        var_79 = wp::sub(var_42, var_72);
        var_80 = wp::cross(var_10, var_79);
        var_81 = wp::add(var_80, var_7);
        // vel_C = wp.cross(angular_vel, C - center) + linear_vel                                 <L 65>
        var_82 = wp::sub(var_49, var_72);
        var_83 = wp::cross(var_10, var_82);
        var_84 = wp::add(var_83, var_7);
        // v_y[tid] = vec12d(                                                                     <L 66>
        // vel_O[0],                                                                              <L 67>
        var_86 = wp::extract(var_75, var_85);
        // vel_O[1],                                                                              <L 68>
        var_88 = wp::extract(var_75, var_87);
        // vel_O[2],                                                                              <L 69>
        var_90 = wp::extract(var_75, var_89);
        // vel_A[0],                                                                              <L 70>
        var_92 = wp::extract(var_78, var_91);
        // vel_A[1],                                                                              <L 71>
        var_94 = wp::extract(var_78, var_93);
        // vel_A[2],                                                                              <L 72>
        var_96 = wp::extract(var_78, var_95);
        // vel_B[0],                                                                              <L 73>
        var_98 = wp::extract(var_81, var_97);
        // vel_B[1],                                                                              <L 74>
        var_100 = wp::extract(var_81, var_99);
        // vel_B[2],                                                                              <L 75>
        var_102 = wp::extract(var_81, var_101);
        // vel_C[0],                                                                              <L 76>
        var_104 = wp::extract(var_84, var_103);
        // vel_C[1],                                                                              <L 77>
        var_106 = wp::extract(var_84, var_105);
        // vel_C[2],                                                                              <L 78>
        var_108 = wp::extract(var_84, var_107);
        var_109 = wp::vec_t<12,wp::float64>({var_86, var_88, var_90, var_92, var_94, var_96, var_98, var_100, var_102, var_104, var_106, var_108});
        // v_y[tid] = vec12d(                                                                     <L 66>
        wp::array_store(var_v_y, var_0, var_109);
    }
}



extern "C" __global__ void set_sim_affine_vel_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::int32> var_affine_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_linear_vel,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_angular_vel,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_v_y,
    wp::array_t<wp::vec_t<3,wp::float64>> var_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ABD_centers,
    wp::array_t<wp::int32> adj_affine_update_mask,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_affine_linear_vel,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_affine_angular_vel,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_v_y,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_virtual_object_centers,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_ABD_centers)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::int32* var_1;
        wp::int32 var_2;
        wp::int32 var_3;
        const wp::int32 var_4 = 0;
        bool var_5;
        wp::vec_t<3,wp::float64>* var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::vec_t<3,wp::float64> var_8;
        wp::vec_t<3,wp::float64>* var_9;
        wp::vec_t<3,wp::float64> var_10;
        wp::vec_t<3,wp::float64> var_11;
        wp::vec_t<3,wp::float64>* var_12;
        wp::vec_t<3,wp::float64> var_13;
        wp::vec_t<3,wp::float64> var_14;
        wp::vec_t<3,wp::float64>* var_15;
        wp::vec_t<3,wp::float64> var_16;
        wp::vec_t<3,wp::float64> var_17;
        wp::vec_t<12,wp::float64>* var_18;
        wp::vec_t<12,wp::float64> var_19;
        wp::vec_t<12,wp::float64> var_20;
        wp::vec_t<3,wp::float64> var_21;
        const wp::int32 var_22 = 0;
        wp::float64 var_23;
        const wp::int32 var_24 = 1;
        wp::float64 var_25;
        const wp::int32 var_26 = 2;
        wp::float64 var_27;
        wp::vec_t<3,wp::float64> var_28;
        const wp::int32 var_29 = 3;
        wp::float64 var_30;
        const wp::int32 var_31 = 4;
        wp::float64 var_32;
        const wp::int32 var_33 = 5;
        wp::float64 var_34;
        wp::vec_t<3,wp::float64> var_35;
        const wp::int32 var_36 = 6;
        wp::float64 var_37;
        const wp::int32 var_38 = 7;
        wp::float64 var_39;
        const wp::int32 var_40 = 8;
        wp::float64 var_41;
        wp::vec_t<3,wp::float64> var_42;
        const wp::int32 var_43 = 9;
        wp::float64 var_44;
        const wp::int32 var_45 = 10;
        wp::float64 var_46;
        const wp::int32 var_47 = 11;
        wp::float64 var_48;
        wp::vec_t<3,wp::float64> var_49;
        const wp::float64 var_50 = 1.0;
        const wp::int32 var_51 = 0;
        wp::float64 var_52;
        wp::float64 var_53;
        const wp::int32 var_54 = 1;
        wp::float64 var_55;
        wp::float64 var_56;
        const wp::int32 var_57 = 2;
        wp::float64 var_58;
        wp::float64 var_59;
        wp::vec_t<3,wp::float64> var_60;
        const wp::int32 var_61 = 0;
        wp::float64 var_62;
        wp::vec_t<3,wp::float64> var_63;
        wp::vec_t<3,wp::float64> var_64;
        const wp::int32 var_65 = 1;
        wp::float64 var_66;
        wp::vec_t<3,wp::float64> var_67;
        wp::vec_t<3,wp::float64> var_68;
        const wp::int32 var_69 = 2;
        wp::float64 var_70;
        wp::vec_t<3,wp::float64> var_71;
        wp::vec_t<3,wp::float64> var_72;
        wp::vec_t<3,wp::float64> var_73;
        wp::vec_t<3,wp::float64> var_74;
        wp::vec_t<3,wp::float64> var_75;
        wp::vec_t<3,wp::float64> var_76;
        wp::vec_t<3,wp::float64> var_77;
        wp::vec_t<3,wp::float64> var_78;
        wp::vec_t<3,wp::float64> var_79;
        wp::vec_t<3,wp::float64> var_80;
        wp::vec_t<3,wp::float64> var_81;
        wp::vec_t<3,wp::float64> var_82;
        wp::vec_t<3,wp::float64> var_83;
        wp::vec_t<3,wp::float64> var_84;
        const wp::int32 var_85 = 0;
        wp::float64 var_86;
        const wp::int32 var_87 = 1;
        wp::float64 var_88;
        const wp::int32 var_89 = 2;
        wp::float64 var_90;
        const wp::int32 var_91 = 0;
        wp::float64 var_92;
        const wp::int32 var_93 = 1;
        wp::float64 var_94;
        const wp::int32 var_95 = 2;
        wp::float64 var_96;
        const wp::int32 var_97 = 0;
        wp::float64 var_98;
        const wp::int32 var_99 = 1;
        wp::float64 var_100;
        const wp::int32 var_101 = 2;
        wp::float64 var_102;
        const wp::int32 var_103 = 0;
        wp::float64 var_104;
        const wp::int32 var_105 = 1;
        wp::float64 var_106;
        const wp::int32 var_107 = 2;
        wp::float64 var_108;
        wp::vec_t<12,wp::float64> var_109;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::int32 adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::vec_t<3,wp::float64> adj_6 = {};
        wp::vec_t<3,wp::float64> adj_7 = {};
        wp::vec_t<3,wp::float64> adj_8 = {};
        wp::vec_t<3,wp::float64> adj_9 = {};
        wp::vec_t<3,wp::float64> adj_10 = {};
        wp::vec_t<3,wp::float64> adj_11 = {};
        wp::vec_t<3,wp::float64> adj_12 = {};
        wp::vec_t<3,wp::float64> adj_13 = {};
        wp::vec_t<3,wp::float64> adj_14 = {};
        wp::vec_t<3,wp::float64> adj_15 = {};
        wp::vec_t<3,wp::float64> adj_16 = {};
        wp::vec_t<3,wp::float64> adj_17 = {};
        wp::vec_t<12,wp::float64> adj_18 = {};
        wp::vec_t<12,wp::float64> adj_19 = {};
        wp::vec_t<12,wp::float64> adj_20 = {};
        wp::vec_t<3,wp::float64> adj_21 = {};
        wp::int32 adj_22 = {};
        wp::float64 adj_23 = {};
        wp::int32 adj_24 = {};
        wp::float64 adj_25 = {};
        wp::int32 adj_26 = {};
        wp::float64 adj_27 = {};
        wp::vec_t<3,wp::float64> adj_28 = {};
        wp::int32 adj_29 = {};
        wp::float64 adj_30 = {};
        wp::int32 adj_31 = {};
        wp::float64 adj_32 = {};
        wp::int32 adj_33 = {};
        wp::float64 adj_34 = {};
        wp::vec_t<3,wp::float64> adj_35 = {};
        wp::int32 adj_36 = {};
        wp::float64 adj_37 = {};
        wp::int32 adj_38 = {};
        wp::float64 adj_39 = {};
        wp::int32 adj_40 = {};
        wp::float64 adj_41 = {};
        wp::vec_t<3,wp::float64> adj_42 = {};
        wp::int32 adj_43 = {};
        wp::float64 adj_44 = {};
        wp::int32 adj_45 = {};
        wp::float64 adj_46 = {};
        wp::int32 adj_47 = {};
        wp::float64 adj_48 = {};
        wp::vec_t<3,wp::float64> adj_49 = {};
        wp::float64 adj_50 = {};
        wp::int32 adj_51 = {};
        wp::float64 adj_52 = {};
        wp::float64 adj_53 = {};
        wp::int32 adj_54 = {};
        wp::float64 adj_55 = {};
        wp::float64 adj_56 = {};
        wp::int32 adj_57 = {};
        wp::float64 adj_58 = {};
        wp::float64 adj_59 = {};
        wp::vec_t<3,wp::float64> adj_60 = {};
        wp::int32 adj_61 = {};
        wp::float64 adj_62 = {};
        wp::vec_t<3,wp::float64> adj_63 = {};
        wp::vec_t<3,wp::float64> adj_64 = {};
        wp::int32 adj_65 = {};
        wp::float64 adj_66 = {};
        wp::vec_t<3,wp::float64> adj_67 = {};
        wp::vec_t<3,wp::float64> adj_68 = {};
        wp::int32 adj_69 = {};
        wp::float64 adj_70 = {};
        wp::vec_t<3,wp::float64> adj_71 = {};
        wp::vec_t<3,wp::float64> adj_72 = {};
        wp::vec_t<3,wp::float64> adj_73 = {};
        wp::vec_t<3,wp::float64> adj_74 = {};
        wp::vec_t<3,wp::float64> adj_75 = {};
        wp::vec_t<3,wp::float64> adj_76 = {};
        wp::vec_t<3,wp::float64> adj_77 = {};
        wp::vec_t<3,wp::float64> adj_78 = {};
        wp::vec_t<3,wp::float64> adj_79 = {};
        wp::vec_t<3,wp::float64> adj_80 = {};
        wp::vec_t<3,wp::float64> adj_81 = {};
        wp::vec_t<3,wp::float64> adj_82 = {};
        wp::vec_t<3,wp::float64> adj_83 = {};
        wp::vec_t<3,wp::float64> adj_84 = {};
        wp::int32 adj_85 = {};
        wp::float64 adj_86 = {};
        wp::int32 adj_87 = {};
        wp::float64 adj_88 = {};
        wp::int32 adj_89 = {};
        wp::float64 adj_90 = {};
        wp::int32 adj_91 = {};
        wp::float64 adj_92 = {};
        wp::int32 adj_93 = {};
        wp::float64 adj_94 = {};
        wp::int32 adj_95 = {};
        wp::float64 adj_96 = {};
        wp::int32 adj_97 = {};
        wp::float64 adj_98 = {};
        wp::int32 adj_99 = {};
        wp::float64 adj_100 = {};
        wp::int32 adj_101 = {};
        wp::float64 adj_102 = {};
        wp::int32 adj_103 = {};
        wp::float64 adj_104 = {};
        wp::int32 adj_105 = {};
        wp::float64 adj_106 = {};
        wp::int32 adj_107 = {};
        wp::float64 adj_108 = {};
        wp::vec_t<12,wp::float64> adj_109 = {};
        //---------
        // forward
        // def set_sim_affine_vel(                                                                <L 38>
        // tid = wp.tid()                                                                         <L 47>
        var_0 = builtin_tid1d();
        // mask = affine_update_mask[tid]                                                         <L 48>
        var_1 = wp::address(var_affine_update_mask, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::copy(var_3);
        // if mask == 0:                                                                          <L 49>
        var_5 = (var_2 == var_4);
        if (var_5) {
            // return                                                                             <L 50>
            goto label0;
        }
        // linear_vel = affine_linear_vel[tid]                                                    <L 51>
        var_6 = wp::address(var_affine_linear_vel, var_0);
        var_8 = wp::load(var_6);
        var_7 = wp::copy(var_8);
        // angular_vel = affine_angular_vel[tid]                                                  <L 52>
        var_9 = wp::address(var_affine_angular_vel, var_0);
        var_11 = wp::load(var_9);
        var_10 = wp::copy(var_11);
        // rest_center = virtual_object_centers[tid]                                              <L 53>
        var_12 = wp::address(var_virtual_object_centers, var_0);
        var_14 = wp::load(var_12);
        var_13 = wp::copy(var_14);
        // rest_O = ABD_centers[tid]                                                              <L 54>
        var_15 = wp::address(var_ABD_centers, var_0);
        var_17 = wp::load(var_15);
        var_16 = wp::copy(var_17);
        // current_y = y[tid]                                                                     <L 55>
        var_18 = wp::address(var_y, var_0);
        var_20 = wp::load(var_18);
        var_19 = wp::copy(var_20);
        // w = rest_center - rest_O                                                               <L 56>
        var_21 = wp::sub(var_13, var_16);
        // O = wp.vec3d(current_y[0], current_y[1], current_y[2])                                 <L 57>
        var_23 = wp::extract(var_19, var_22);
        var_25 = wp::extract(var_19, var_24);
        var_27 = wp::extract(var_19, var_26);
        var_28 = wp::vec_t<3,wp::float64>(var_23, var_25, var_27);
        // A = wp.vec3d(current_y[3], current_y[4], current_y[5])                                 <L 58>
        var_30 = wp::extract(var_19, var_29);
        var_32 = wp::extract(var_19, var_31);
        var_34 = wp::extract(var_19, var_33);
        var_35 = wp::vec_t<3,wp::float64>(var_30, var_32, var_34);
        // B = wp.vec3d(current_y[6], current_y[7], current_y[8])                                 <L 59>
        var_37 = wp::extract(var_19, var_36);
        var_39 = wp::extract(var_19, var_38);
        var_41 = wp::extract(var_19, var_40);
        var_42 = wp::vec_t<3,wp::float64>(var_37, var_39, var_41);
        // C = wp.vec3d(current_y[9], current_y[10], current_y[11])                               <L 60>
        var_44 = wp::extract(var_19, var_43);
        var_46 = wp::extract(var_19, var_45);
        var_48 = wp::extract(var_19, var_47);
        var_49 = wp::vec_t<3,wp::float64>(var_44, var_46, var_48);
        // center = (_1 - w[0] - w[1] - w[2]) * O + w[0] * A + w[1] * B + w[2] * C                <L 61>
        var_52 = wp::extract(var_21, var_51);
        var_53 = wp::sub(var_50, var_52);
        var_55 = wp::extract(var_21, var_54);
        var_56 = wp::sub(var_53, var_55);
        var_58 = wp::extract(var_21, var_57);
        var_59 = wp::sub(var_56, var_58);
        var_60 = wp::mul(var_59, var_28);
        var_62 = wp::extract(var_21, var_61);
        var_63 = wp::mul(var_62, var_35);
        var_64 = wp::add(var_60, var_63);
        var_66 = wp::extract(var_21, var_65);
        var_67 = wp::mul(var_66, var_42);
        var_68 = wp::add(var_64, var_67);
        var_70 = wp::extract(var_21, var_69);
        var_71 = wp::mul(var_70, var_49);
        var_72 = wp::add(var_68, var_71);
        // vel_O = wp.cross(angular_vel, O - center) + linear_vel                                 <L 62>
        var_73 = wp::sub(var_28, var_72);
        var_74 = wp::cross(var_10, var_73);
        var_75 = wp::add(var_74, var_7);
        // vel_A = wp.cross(angular_vel, A - center) + linear_vel                                 <L 63>
        var_76 = wp::sub(var_35, var_72);
        var_77 = wp::cross(var_10, var_76);
        var_78 = wp::add(var_77, var_7);
        // vel_B = wp.cross(angular_vel, B - center) + linear_vel                                 <L 64>
        var_79 = wp::sub(var_42, var_72);
        var_80 = wp::cross(var_10, var_79);
        var_81 = wp::add(var_80, var_7);
        // vel_C = wp.cross(angular_vel, C - center) + linear_vel                                 <L 65>
        var_82 = wp::sub(var_49, var_72);
        var_83 = wp::cross(var_10, var_82);
        var_84 = wp::add(var_83, var_7);
        // v_y[tid] = vec12d(                                                                     <L 66>
        // vel_O[0],                                                                              <L 67>
        var_86 = wp::extract(var_75, var_85);
        // vel_O[1],                                                                              <L 68>
        var_88 = wp::extract(var_75, var_87);
        // vel_O[2],                                                                              <L 69>
        var_90 = wp::extract(var_75, var_89);
        // vel_A[0],                                                                              <L 70>
        var_92 = wp::extract(var_78, var_91);
        // vel_A[1],                                                                              <L 71>
        var_94 = wp::extract(var_78, var_93);
        // vel_A[2],                                                                              <L 72>
        var_96 = wp::extract(var_78, var_95);
        // vel_B[0],                                                                              <L 73>
        var_98 = wp::extract(var_81, var_97);
        // vel_B[1],                                                                              <L 74>
        var_100 = wp::extract(var_81, var_99);
        // vel_B[2],                                                                              <L 75>
        var_102 = wp::extract(var_81, var_101);
        // vel_C[0],                                                                              <L 76>
        var_104 = wp::extract(var_84, var_103);
        // vel_C[1],                                                                              <L 77>
        var_106 = wp::extract(var_84, var_105);
        // vel_C[2],                                                                              <L 78>
        var_108 = wp::extract(var_84, var_107);
        var_109 = wp::vec_t<12,wp::float64>({var_86, var_88, var_90, var_92, var_94, var_96, var_98, var_100, var_102, var_104, var_106, var_108});
        // v_y[tid] = vec12d(                                                                     <L 66>
        // wp::array_store(var_v_y, var_0, var_109);
        //---------
        // reverse
        wp::adj_array_store(var_v_y, var_0, var_109, adj_v_y, adj_0, adj_109);
        // adj: v_y[tid] = vec12d(                                                                <L 66>
        wp::adj_vec_t({var_86, var_88, var_90, var_92, var_94, var_96, var_98, var_100, var_102, var_104, var_106, var_108}, {&adj_86, &adj_88, &adj_90, &adj_92, &adj_94, &adj_96, &adj_98, &adj_100, &adj_102, &adj_104, &adj_106, &adj_108}, adj_109);
        wp::adj_extract(var_84, var_107, adj_84, adj_107, adj_108);
        // adj: vel_C[2],                                                                         <L 78>
        wp::adj_extract(var_84, var_105, adj_84, adj_105, adj_106);
        // adj: vel_C[1],                                                                         <L 77>
        wp::adj_extract(var_84, var_103, adj_84, adj_103, adj_104);
        // adj: vel_C[0],                                                                         <L 76>
        wp::adj_extract(var_81, var_101, adj_81, adj_101, adj_102);
        // adj: vel_B[2],                                                                         <L 75>
        wp::adj_extract(var_81, var_99, adj_81, adj_99, adj_100);
        // adj: vel_B[1],                                                                         <L 74>
        wp::adj_extract(var_81, var_97, adj_81, adj_97, adj_98);
        // adj: vel_B[0],                                                                         <L 73>
        wp::adj_extract(var_78, var_95, adj_78, adj_95, adj_96);
        // adj: vel_A[2],                                                                         <L 72>
        wp::adj_extract(var_78, var_93, adj_78, adj_93, adj_94);
        // adj: vel_A[1],                                                                         <L 71>
        wp::adj_extract(var_78, var_91, adj_78, adj_91, adj_92);
        // adj: vel_A[0],                                                                         <L 70>
        wp::adj_extract(var_75, var_89, adj_75, adj_89, adj_90);
        // adj: vel_O[2],                                                                         <L 69>
        wp::adj_extract(var_75, var_87, adj_75, adj_87, adj_88);
        // adj: vel_O[1],                                                                         <L 68>
        wp::adj_extract(var_75, var_85, adj_75, adj_85, adj_86);
        // adj: vel_O[0],                                                                         <L 67>
        // adj: v_y[tid] = vec12d(                                                                <L 66>
        wp::adj_add(var_83, var_7, adj_83, adj_7, adj_84);
        wp::adj_cross(var_10, var_82, adj_10, adj_82, adj_83);
        wp::adj_sub(var_49, var_72, adj_49, adj_72, adj_82);
        // adj: vel_C = wp.cross(angular_vel, C - center) + linear_vel                            <L 65>
        wp::adj_add(var_80, var_7, adj_80, adj_7, adj_81);
        wp::adj_cross(var_10, var_79, adj_10, adj_79, adj_80);
        wp::adj_sub(var_42, var_72, adj_42, adj_72, adj_79);
        // adj: vel_B = wp.cross(angular_vel, B - center) + linear_vel                            <L 64>
        wp::adj_add(var_77, var_7, adj_77, adj_7, adj_78);
        wp::adj_cross(var_10, var_76, adj_10, adj_76, adj_77);
        wp::adj_sub(var_35, var_72, adj_35, adj_72, adj_76);
        // adj: vel_A = wp.cross(angular_vel, A - center) + linear_vel                            <L 63>
        wp::adj_add(var_74, var_7, adj_74, adj_7, adj_75);
        wp::adj_cross(var_10, var_73, adj_10, adj_73, adj_74);
        wp::adj_sub(var_28, var_72, adj_28, adj_72, adj_73);
        // adj: vel_O = wp.cross(angular_vel, O - center) + linear_vel                            <L 62>
        wp::adj_add(var_68, var_71, adj_68, adj_71, adj_72);
        wp::adj_mul(var_70, var_49, adj_70, adj_49, adj_71);
        wp::adj_extract(var_21, var_69, adj_21, adj_69, adj_70);
        wp::adj_add(var_64, var_67, adj_64, adj_67, adj_68);
        wp::adj_mul(var_66, var_42, adj_66, adj_42, adj_67);
        wp::adj_extract(var_21, var_65, adj_21, adj_65, adj_66);
        wp::adj_add(var_60, var_63, adj_60, adj_63, adj_64);
        wp::adj_mul(var_62, var_35, adj_62, adj_35, adj_63);
        wp::adj_extract(var_21, var_61, adj_21, adj_61, adj_62);
        wp::adj_mul(var_59, var_28, adj_59, adj_28, adj_60);
        wp::adj_sub(var_56, var_58, adj_56, adj_58, adj_59);
        wp::adj_extract(var_21, var_57, adj_21, adj_57, adj_58);
        wp::adj_sub(var_53, var_55, adj_53, adj_55, adj_56);
        wp::adj_extract(var_21, var_54, adj_21, adj_54, adj_55);
        wp::adj_sub(var_50, var_52, adj_50, adj_52, adj_53);
        wp::adj_extract(var_21, var_51, adj_21, adj_51, adj_52);
        // adj: center = (_1 - w[0] - w[1] - w[2]) * O + w[0] * A + w[1] * B + w[2] * C           <L 61>
        wp::adj_vec_t(var_44, var_46, var_48, adj_44, adj_46, adj_48, adj_49);
        wp::adj_extract(var_19, var_47, adj_19, adj_47, adj_48);
        wp::adj_extract(var_19, var_45, adj_19, adj_45, adj_46);
        wp::adj_extract(var_19, var_43, adj_19, adj_43, adj_44);
        // adj: C = wp.vec3d(current_y[9], current_y[10], current_y[11])                          <L 60>
        wp::adj_vec_t(var_37, var_39, var_41, adj_37, adj_39, adj_41, adj_42);
        wp::adj_extract(var_19, var_40, adj_19, adj_40, adj_41);
        wp::adj_extract(var_19, var_38, adj_19, adj_38, adj_39);
        wp::adj_extract(var_19, var_36, adj_19, adj_36, adj_37);
        // adj: B = wp.vec3d(current_y[6], current_y[7], current_y[8])                            <L 59>
        wp::adj_vec_t(var_30, var_32, var_34, adj_30, adj_32, adj_34, adj_35);
        wp::adj_extract(var_19, var_33, adj_19, adj_33, adj_34);
        wp::adj_extract(var_19, var_31, adj_19, adj_31, adj_32);
        wp::adj_extract(var_19, var_29, adj_19, adj_29, adj_30);
        // adj: A = wp.vec3d(current_y[3], current_y[4], current_y[5])                            <L 58>
        wp::adj_vec_t(var_23, var_25, var_27, adj_23, adj_25, adj_27, adj_28);
        wp::adj_extract(var_19, var_26, adj_19, adj_26, adj_27);
        wp::adj_extract(var_19, var_24, adj_19, adj_24, adj_25);
        wp::adj_extract(var_19, var_22, adj_19, adj_22, adj_23);
        // adj: O = wp.vec3d(current_y[0], current_y[1], current_y[2])                            <L 57>
        wp::adj_sub(var_13, var_16, adj_13, adj_16, adj_21);
        // adj: w = rest_center - rest_O                                                          <L 56>
        wp::adj_copy(var_20, adj_18, adj_19);
        wp::adj_load(var_18, adj_18, adj_20);
        wp::adj_address(var_y, var_0, adj_y, adj_0, adj_18);
        // adj: current_y = y[tid]                                                                <L 55>
        wp::adj_copy(var_17, adj_15, adj_16);
        wp::adj_load(var_15, adj_15, adj_17);
        wp::adj_address(var_ABD_centers, var_0, adj_ABD_centers, adj_0, adj_15);
        // adj: rest_O = ABD_centers[tid]                                                         <L 54>
        wp::adj_copy(var_14, adj_12, adj_13);
        wp::adj_load(var_12, adj_12, adj_14);
        wp::adj_address(var_virtual_object_centers, var_0, adj_virtual_object_centers, adj_0, adj_12);
        // adj: rest_center = virtual_object_centers[tid]                                         <L 53>
        wp::adj_copy(var_11, adj_9, adj_10);
        wp::adj_load(var_9, adj_9, adj_11);
        wp::adj_address(var_affine_angular_vel, var_0, adj_affine_angular_vel, adj_0, adj_9);
        // adj: angular_vel = affine_angular_vel[tid]                                             <L 52>
        wp::adj_copy(var_8, adj_6, adj_7);
        wp::adj_load(var_6, adj_6, adj_8);
        wp::adj_address(var_affine_linear_vel, var_0, adj_affine_linear_vel, adj_0, adj_6);
        // adj: linear_vel = affine_linear_vel[tid]                                               <L 51>
        if (var_5) {
            label0:;
            // adj: return                                                                        <L 50>
        }
        // adj: if mask == 0:                                                                     <L 49>
        wp::adj_copy(var_3, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_affine_update_mask, var_0, adj_affine_update_mask, adj_0, adj_1);
        // adj: mask = affine_update_mask[tid]                                                    <L 48>
        // adj: tid = wp.tid()                                                                    <L 47>
        // adj: def set_sim_affine_vel(                                                           <L 38>
        continue;
    }
}

