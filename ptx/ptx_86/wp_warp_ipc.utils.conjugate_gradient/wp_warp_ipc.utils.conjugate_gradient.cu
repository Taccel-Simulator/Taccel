#include "hip/hip_runtime.h"

#define WP_TILE_BLOCK_DIM 256
#define WP_NO_CRT
#include "builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx, dim)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)

extern "C" {
}


extern "C" __global__ void array_matmul_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_A,
    wp::array_t<wp::vec_t<3,wp::float64>> var_y,
    wp::array_t<wp::vec_t<3,wp::float64>> var_z)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::mat_t<3,3,wp::float64>* var_1;
        wp::vec_t<3,wp::float64>* var_2;
        wp::vec_t<3,wp::float64> var_3;
        wp::mat_t<3,3,wp::float64> var_4;
        wp::vec_t<3,wp::float64> var_5;
        //---------
        // forward
        // def array_matmul(A: wp.array(dtype=wp.mat33d), y: wp.array(dtype=wp.vec3d), z: wp.array(dtype=wp.vec3d)):       <L 21>
        // tid = wp.tid()                                                                         <L 22>
        var_0 = builtin_tid1d();
        // z[tid] = A[tid] @ y[tid]                                                               <L 23>
        var_1 = wp::address(var_A, var_0);
        var_2 = wp::address(var_y, var_0);
        var_4 = wp::load(var_1);
        var_5 = wp::load(var_2);
        var_3 = wp::mul(var_4, var_5);
        wp::array_store(var_z, var_0, var_3);
    }
}



extern "C" __global__ void array_matmul_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_A,
    wp::array_t<wp::vec_t<3,wp::float64>> var_y,
    wp::array_t<wp::vec_t<3,wp::float64>> var_z,
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_A,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_y,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_z)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::mat_t<3,3,wp::float64>* var_1;
        wp::vec_t<3,wp::float64>* var_2;
        wp::vec_t<3,wp::float64> var_3;
        wp::mat_t<3,3,wp::float64> var_4;
        wp::vec_t<3,wp::float64> var_5;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::mat_t<3,3,wp::float64> adj_1 = {};
        wp::vec_t<3,wp::float64> adj_2 = {};
        wp::vec_t<3,wp::float64> adj_3 = {};
        wp::mat_t<3,3,wp::float64> adj_4 = {};
        wp::vec_t<3,wp::float64> adj_5 = {};
        //---------
        // forward
        // def array_matmul(A: wp.array(dtype=wp.mat33d), y: wp.array(dtype=wp.vec3d), z: wp.array(dtype=wp.vec3d)):       <L 21>
        // tid = wp.tid()                                                                         <L 22>
        var_0 = builtin_tid1d();
        // z[tid] = A[tid] @ y[tid]                                                               <L 23>
        var_1 = wp::address(var_A, var_0);
        var_2 = wp::address(var_y, var_0);
        var_4 = wp::load(var_1);
        var_5 = wp::load(var_2);
        var_3 = wp::mul(var_4, var_5);
        // wp::array_store(var_z, var_0, var_3);
        //---------
        // reverse
        wp::adj_array_store(var_z, var_0, var_3, adj_z, adj_0, adj_3);
        wp::adj_mul(var_4, var_5, adj_1, adj_2, adj_3);
        wp::adj_load(var_2, adj_2, adj_5);
        wp::adj_load(var_1, adj_1, adj_4);
        wp::adj_address(var_y, var_0, adj_y, adj_0, adj_2);
        wp::adj_address(var_A, var_0, adj_A, adj_0, adj_1);
        // adj: z[tid] = A[tid] @ y[tid]                                                          <L 23>
        // adj: tid = wp.tid()                                                                    <L 22>
        // adj: def array_matmul(A: wp.array(dtype=wp.mat33d), y: wp.array(dtype=wp.vec3d), z: wp.array(dtype=wp.vec3d)):  <L 21>
        continue;
    }
}



extern "C" __global__ void axpy_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_y,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::float64> var_env_beta,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ret,
    wp::array_t<wp::int64> var_affine_dof_env_id,
    wp::array_t<wp::int32> var_node2env,
    wp::int32 var_n_affine_dofs,
    wp::int32 var_num_constraints,
    wp::int32 var_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::float32 var_1 = 0.0;
        wp::float64 var_2;
        const wp::int32 var_3 = 3;
        wp::int32 var_4;
        bool var_5;
        const wp::int32 var_6 = 3;
        wp::int32 var_7;
        bool var_8;
        const wp::int32 var_9 = 3;
        wp::int32 var_10;
        wp::int32 var_11;
        wp::int64* var_12;
        wp::float64* var_13;
        wp::int64 var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        wp::float64 var_17;
        const wp::int32 var_18 = 3;
        wp::int32 var_19;
        wp::int32 var_20;
        wp::int32 var_21;
        wp::int32* var_22;
        wp::float64* var_23;
        wp::int32 var_24;
        wp::float64 var_25;
        wp::float64 var_26;
        wp::float64 var_27;
        wp::float64 var_28;
        wp::vec_t<3,wp::float64>* var_29;
        wp::vec_t<3,wp::float64>* var_30;
        wp::vec_t<3,wp::float64> var_31;
        wp::vec_t<3,wp::float64> var_32;
        wp::vec_t<3,wp::float64> var_33;
        wp::vec_t<3,wp::float64> var_34;
        //---------
        // forward
        // def axpy(                                                                              <L 27>
        // tid = wp.tid()                                                                         <L 38>
        var_0 = builtin_tid1d();
        // beta = wp.float64(0.0)                                                                 <L 39>
        var_2 = wp::float64(var_1);
        // if tid < num_constraints // 3:                                                         <L 40>
        var_4 = wp::floordiv(var_num_constraints, var_3);
        var_5 = (var_0 < var_4);
        if (var_5) {
            // return                                                                             <L 41>
            continue;
        }
        if (!var_5) {
            // elif tid < n_affine_dofs // 3:                                                     <L 42>
            var_7 = wp::floordiv(var_n_affine_dofs, var_6);
            var_8 = (var_0 < var_7);
            if (var_8) {
                // beta = env_beta[affine_dof_env_id[tid - num_constraints // 3]]                 <L 43>
                var_10 = wp::floordiv(var_num_constraints, var_9);
                var_11 = wp::sub(var_0, var_10);
                var_12 = wp::address(var_affine_dof_env_id, var_11);
                var_14 = wp::load(var_12);
                var_13 = wp::address(var_env_beta, var_14);
                var_16 = wp::load(var_13);
                var_15 = wp::copy(var_16);
            }
            var_17 = wp::where(var_8, var_15, var_2);
            if (!var_8) {
                // beta = env_beta[node2env[tid - n_affine_dofs // 3 + affine_verts_num]]         <L 45>
                var_19 = wp::floordiv(var_n_affine_dofs, var_18);
                var_20 = wp::sub(var_0, var_19);
                var_21 = wp::add(var_20, var_affine_verts_num);
                var_22 = wp::address(var_node2env, var_21);
                var_24 = wp::load(var_22);
                var_23 = wp::address(var_env_beta, var_24);
                var_26 = wp::load(var_23);
                var_25 = wp::copy(var_26);
            }
            var_27 = wp::where(var_8, var_17, var_25);
        }
        var_28 = wp::where(var_5, var_2, var_27);
        // ret[tid] = y[tid] + beta * x[tid]                                                      <L 46>
        var_29 = wp::address(var_y, var_0);
        var_30 = wp::address(var_x, var_0);
        var_32 = wp::load(var_30);
        var_31 = wp::mul(var_28, var_32);
        var_34 = wp::load(var_29);
        var_33 = wp::add(var_34, var_31);
        wp::array_store(var_ret, var_0, var_33);
    }
}



extern "C" __global__ void axpy_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_y,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::float64> var_env_beta,
    wp::array_t<wp::vec_t<3,wp::float64>> var_ret,
    wp::array_t<wp::int64> var_affine_dof_env_id,
    wp::array_t<wp::int32> var_node2env,
    wp::int32 var_n_affine_dofs,
    wp::int32 var_num_constraints,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_y,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::float64> adj_env_beta,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_ret,
    wp::array_t<wp::int64> adj_affine_dof_env_id,
    wp::array_t<wp::int32> adj_node2env,
    wp::int32 adj_n_affine_dofs,
    wp::int32 adj_num_constraints,
    wp::int32 adj_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::float32 var_1 = 0.0;
        wp::float64 var_2;
        const wp::int32 var_3 = 3;
        wp::int32 var_4;
        bool var_5;
        const wp::int32 var_6 = 3;
        wp::int32 var_7;
        bool var_8;
        const wp::int32 var_9 = 3;
        wp::int32 var_10;
        wp::int32 var_11;
        wp::int64* var_12;
        wp::float64* var_13;
        wp::int64 var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        wp::float64 var_17;
        const wp::int32 var_18 = 3;
        wp::int32 var_19;
        wp::int32 var_20;
        wp::int32 var_21;
        wp::int32* var_22;
        wp::float64* var_23;
        wp::int32 var_24;
        wp::float64 var_25;
        wp::float64 var_26;
        wp::float64 var_27;
        wp::float64 var_28;
        wp::vec_t<3,wp::float64>* var_29;
        wp::vec_t<3,wp::float64>* var_30;
        wp::vec_t<3,wp::float64> var_31;
        wp::vec_t<3,wp::float64> var_32;
        wp::vec_t<3,wp::float64> var_33;
        wp::vec_t<3,wp::float64> var_34;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::float32 adj_1 = {};
        wp::float64 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::int32 adj_6 = {};
        wp::int32 adj_7 = {};
        bool adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int64 adj_12 = {};
        wp::float64 adj_13 = {};
        wp::int64 adj_14 = {};
        wp::float64 adj_15 = {};
        wp::float64 adj_16 = {};
        wp::float64 adj_17 = {};
        wp::int32 adj_18 = {};
        wp::int32 adj_19 = {};
        wp::int32 adj_20 = {};
        wp::int32 adj_21 = {};
        wp::int32 adj_22 = {};
        wp::float64 adj_23 = {};
        wp::int32 adj_24 = {};
        wp::float64 adj_25 = {};
        wp::float64 adj_26 = {};
        wp::float64 adj_27 = {};
        wp::float64 adj_28 = {};
        wp::vec_t<3,wp::float64> adj_29 = {};
        wp::vec_t<3,wp::float64> adj_30 = {};
        wp::vec_t<3,wp::float64> adj_31 = {};
        wp::vec_t<3,wp::float64> adj_32 = {};
        wp::vec_t<3,wp::float64> adj_33 = {};
        wp::vec_t<3,wp::float64> adj_34 = {};
        //---------
        // forward
        // def axpy(                                                                              <L 27>
        // tid = wp.tid()                                                                         <L 38>
        var_0 = builtin_tid1d();
        // beta = wp.float64(0.0)                                                                 <L 39>
        var_2 = wp::float64(var_1);
        // if tid < num_constraints // 3:                                                         <L 40>
        var_4 = wp::floordiv(var_num_constraints, var_3);
        var_5 = (var_0 < var_4);
        if (var_5) {
            // return                                                                             <L 41>
            goto label0;
        }
        if (!var_5) {
            // elif tid < n_affine_dofs // 3:                                                     <L 42>
            var_7 = wp::floordiv(var_n_affine_dofs, var_6);
            var_8 = (var_0 < var_7);
            if (var_8) {
                // beta = env_beta[affine_dof_env_id[tid - num_constraints // 3]]                 <L 43>
                var_10 = wp::floordiv(var_num_constraints, var_9);
                var_11 = wp::sub(var_0, var_10);
                var_12 = wp::address(var_affine_dof_env_id, var_11);
                var_14 = wp::load(var_12);
                var_13 = wp::address(var_env_beta, var_14);
                var_16 = wp::load(var_13);
                var_15 = wp::copy(var_16);
            }
            var_17 = wp::where(var_8, var_15, var_2);
            if (!var_8) {
                // beta = env_beta[node2env[tid - n_affine_dofs // 3 + affine_verts_num]]         <L 45>
                var_19 = wp::floordiv(var_n_affine_dofs, var_18);
                var_20 = wp::sub(var_0, var_19);
                var_21 = wp::add(var_20, var_affine_verts_num);
                var_22 = wp::address(var_node2env, var_21);
                var_24 = wp::load(var_22);
                var_23 = wp::address(var_env_beta, var_24);
                var_26 = wp::load(var_23);
                var_25 = wp::copy(var_26);
            }
            var_27 = wp::where(var_8, var_17, var_25);
        }
        var_28 = wp::where(var_5, var_2, var_27);
        // ret[tid] = y[tid] + beta * x[tid]                                                      <L 46>
        var_29 = wp::address(var_y, var_0);
        var_30 = wp::address(var_x, var_0);
        var_32 = wp::load(var_30);
        var_31 = wp::mul(var_28, var_32);
        var_34 = wp::load(var_29);
        var_33 = wp::add(var_34, var_31);
        // wp::array_store(var_ret, var_0, var_33);
        //---------
        // reverse
        wp::adj_array_store(var_ret, var_0, var_33, adj_ret, adj_0, adj_33);
        wp::adj_add(var_34, var_31, adj_29, adj_31, adj_33);
        wp::adj_load(var_29, adj_29, adj_34);
        wp::adj_mul(var_28, var_32, adj_28, adj_30, adj_31);
        wp::adj_load(var_30, adj_30, adj_32);
        wp::adj_address(var_x, var_0, adj_x, adj_0, adj_30);
        wp::adj_address(var_y, var_0, adj_y, adj_0, adj_29);
        // adj: ret[tid] = y[tid] + beta * x[tid]                                                 <L 46>
        wp::adj_where(var_5, var_2, var_27, adj_5, adj_2, adj_27, adj_28);
        if (!var_5) {
            wp::adj_where(var_8, var_17, var_25, adj_8, adj_17, adj_25, adj_27);
            if (!var_8) {
                wp::adj_copy(var_26, adj_23, adj_25);
                wp::adj_load(var_23, adj_23, adj_26);
                wp::adj_address(var_env_beta, var_24, adj_env_beta, adj_22, adj_23);
                wp::adj_load(var_22, adj_22, adj_24);
                wp::adj_address(var_node2env, var_21, adj_node2env, adj_21, adj_22);
                wp::adj_add(var_20, var_affine_verts_num, adj_20, adj_affine_verts_num, adj_21);
                wp::adj_sub(var_0, var_19, adj_0, adj_19, adj_20);
                wp::adj_floordiv(var_n_affine_dofs, var_18, adj_n_affine_dofs, adj_18, adj_19);
                // adj: beta = env_beta[node2env[tid - n_affine_dofs // 3 + affine_verts_num]]    <L 45>
            }
            wp::adj_where(var_8, var_15, var_2, adj_8, adj_15, adj_2, adj_17);
            if (var_8) {
                wp::adj_copy(var_16, adj_13, adj_15);
                wp::adj_load(var_13, adj_13, adj_16);
                wp::adj_address(var_env_beta, var_14, adj_env_beta, adj_12, adj_13);
                wp::adj_load(var_12, adj_12, adj_14);
                wp::adj_address(var_affine_dof_env_id, var_11, adj_affine_dof_env_id, adj_11, adj_12);
                wp::adj_sub(var_0, var_10, adj_0, adj_10, adj_11);
                wp::adj_floordiv(var_num_constraints, var_9, adj_num_constraints, adj_9, adj_10);
                // adj: beta = env_beta[affine_dof_env_id[tid - num_constraints // 3]]            <L 43>
            }
            wp::adj_floordiv(var_n_affine_dofs, var_6, adj_n_affine_dofs, adj_6, adj_7);
            // adj: elif tid < n_affine_dofs // 3:                                                <L 42>
        }
        if (var_5) {
            label0:;
            // adj: return                                                                        <L 41>
        }
        wp::adj_floordiv(var_num_constraints, var_3, adj_num_constraints, adj_3, adj_4);
        // adj: if tid < num_constraints // 3:                                                    <L 40>
        wp::adj_float64(var_1, adj_1, adj_2);
        // adj: beta = wp.float64(0.0)                                                            <L 39>
        // adj: tid = wp.tid()                                                                    <L 38>
        // adj: def axpy(                                                                         <L 27>
        continue;
    }
}



extern "C" __global__ void array_inv_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_x)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::mat_t<3,3,wp::float64>* var_1;
        wp::mat_t<3,3,wp::float64> var_2;
        wp::mat_t<3,3,wp::float64> var_3;
        //---------
        // forward
        // def array_inv(x: wp.array(dtype=wp.mat33d)):                                           <L 15>
        // tid = wp.tid()                                                                         <L 16>
        var_0 = builtin_tid1d();
        // x[tid] = wp.inverse(x[tid])                                                            <L 17>
        var_1 = wp::address(var_x, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::inverse(var_3);
        wp::array_store(var_x, var_0, var_2);
    }
}



extern "C" __global__ void array_inv_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_x,
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_x)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        wp::mat_t<3,3,wp::float64>* var_1;
        wp::mat_t<3,3,wp::float64> var_2;
        wp::mat_t<3,3,wp::float64> var_3;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::mat_t<3,3,wp::float64> adj_1 = {};
        wp::mat_t<3,3,wp::float64> adj_2 = {};
        wp::mat_t<3,3,wp::float64> adj_3 = {};
        //---------
        // forward
        // def array_inv(x: wp.array(dtype=wp.mat33d)):                                           <L 15>
        // tid = wp.tid()                                                                         <L 16>
        var_0 = builtin_tid1d();
        // x[tid] = wp.inverse(x[tid])                                                            <L 17>
        var_1 = wp::address(var_x, var_0);
        var_3 = wp::load(var_1);
        var_2 = wp::inverse(var_3);
        // wp::array_store(var_x, var_0, var_2);
        //---------
        // reverse
        wp::adj_array_store(var_x, var_0, var_2, adj_x, adj_0, adj_2);
        wp::adj_inverse(var_3, var_2, adj_1, adj_2);
        wp::adj_load(var_1, adj_1, adj_3);
        wp::adj_address(var_x, var_0, adj_x, adj_0, adj_1);
        // adj: x[tid] = wp.inverse(x[tid])                                                       <L 17>
        // adj: tid = wp.tid()                                                                    <L 16>
        // adj: def array_inv(x: wp.array(dtype=wp.mat33d)):                                      <L 15>
        continue;
    }
}



extern "C" __global__ void cg_one_iter_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::vec_t<3,wp::float64>> var_r,
    wp::array_t<wp::vec_t<3,wp::float64>> var_q,
    wp::array_t<wp::float64> var_env_alpha,
    wp::array_t<wp::vec_t<3,wp::float64>> var_p,
    wp::array_t<wp::vec_t<3,wp::float64>> var_Ap,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_diag_inv,
    wp::array_t<wp::int64> var_affine_dof_env_id,
    wp::array_t<wp::int32> var_node2env,
    wp::int32 var_n_affine_dofs,
    wp::int32 var_num_constraints,
    wp::int32 var_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::float32 var_1 = 0.0;
        wp::float64 var_2;
        const wp::int32 var_3 = 3;
        wp::int32 var_4;
        bool var_5;
        const wp::int32 var_6 = 3;
        wp::int32 var_7;
        bool var_8;
        const wp::int32 var_9 = 3;
        wp::int32 var_10;
        wp::int32 var_11;
        wp::int64* var_12;
        wp::float64* var_13;
        wp::int64 var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        wp::float64 var_17;
        const wp::int32 var_18 = 3;
        wp::int32 var_19;
        wp::int32 var_20;
        wp::int32 var_21;
        wp::int32* var_22;
        wp::float64* var_23;
        wp::int32 var_24;
        wp::float64 var_25;
        wp::float64 var_26;
        wp::float64 var_27;
        wp::float64 var_28;
        wp::vec_t<3,wp::float64>* var_29;
        wp::vec_t<3,wp::float64> var_30;
        wp::vec_t<3,wp::float64> var_31;
        wp::vec_t<3,wp::float64> var_32;
        wp::vec_t<3,wp::float64>* var_33;
        wp::vec_t<3,wp::float64> var_34;
        wp::vec_t<3,wp::float64> var_35;
        wp::vec_t<3,wp::float64> var_36;
        wp::mat_t<3,3,wp::float64>* var_37;
        wp::vec_t<3,wp::float64>* var_38;
        wp::vec_t<3,wp::float64> var_39;
        wp::mat_t<3,3,wp::float64> var_40;
        wp::vec_t<3,wp::float64> var_41;
        //---------
        // forward
        // def cg_one_iter(                                                                       <L 50>
        // tid = wp.tid()                                                                         <L 64>
        var_0 = builtin_tid1d();
        // alpha = wp.float64(0.0)                                                                <L 65>
        var_2 = wp::float64(var_1);
        // if tid < num_constraints // 3:                                                         <L 66>
        var_4 = wp::floordiv(var_num_constraints, var_3);
        var_5 = (var_0 < var_4);
        if (var_5) {
            // return                                                                             <L 67>
            continue;
        }
        if (!var_5) {
            // elif tid < n_affine_dofs // 3:                                                     <L 68>
            var_7 = wp::floordiv(var_n_affine_dofs, var_6);
            var_8 = (var_0 < var_7);
            if (var_8) {
                // alpha = env_alpha[affine_dof_env_id[tid - num_constraints // 3]]               <L 69>
                var_10 = wp::floordiv(var_num_constraints, var_9);
                var_11 = wp::sub(var_0, var_10);
                var_12 = wp::address(var_affine_dof_env_id, var_11);
                var_14 = wp::load(var_12);
                var_13 = wp::address(var_env_alpha, var_14);
                var_16 = wp::load(var_13);
                var_15 = wp::copy(var_16);
            }
            var_17 = wp::where(var_8, var_15, var_2);
            if (!var_8) {
                // alpha = env_alpha[node2env[tid - n_affine_dofs // 3 + affine_verts_num]]       <L 71>
                var_19 = wp::floordiv(var_n_affine_dofs, var_18);
                var_20 = wp::sub(var_0, var_19);
                var_21 = wp::add(var_20, var_affine_verts_num);
                var_22 = wp::address(var_node2env, var_21);
                var_24 = wp::load(var_22);
                var_23 = wp::address(var_env_alpha, var_24);
                var_26 = wp::load(var_23);
                var_25 = wp::copy(var_26);
            }
            var_27 = wp::where(var_8, var_17, var_25);
        }
        var_28 = wp::where(var_5, var_2, var_27);
        // x[tid] += alpha * p[tid]                                                               <L 72>
        var_29 = wp::address(var_p, var_0);
        var_31 = wp::load(var_29);
        var_30 = wp::mul(var_28, var_31);
        var_32 = wp::atomic_add(var_x, var_0, var_30);
        // r[tid] -= alpha * Ap[tid]                                                              <L 73>
        var_33 = wp::address(var_Ap, var_0);
        var_35 = wp::load(var_33);
        var_34 = wp::mul(var_28, var_35);
        var_36 = wp::atomic_sub(var_r, var_0, var_34);
        // q[tid] = diag_inv[tid] @ r[tid]                                                        <L 74>
        var_37 = wp::address(var_diag_inv, var_0);
        var_38 = wp::address(var_r, var_0);
        var_40 = wp::load(var_37);
        var_41 = wp::load(var_38);
        var_39 = wp::mul(var_40, var_41);
        wp::array_store(var_q, var_0, var_39);
    }
}



extern "C" __global__ void cg_one_iter_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::vec_t<3,wp::float64>> var_r,
    wp::array_t<wp::vec_t<3,wp::float64>> var_q,
    wp::array_t<wp::float64> var_env_alpha,
    wp::array_t<wp::vec_t<3,wp::float64>> var_p,
    wp::array_t<wp::vec_t<3,wp::float64>> var_Ap,
    wp::array_t<wp::mat_t<3,3,wp::float64>> var_diag_inv,
    wp::array_t<wp::int64> var_affine_dof_env_id,
    wp::array_t<wp::int32> var_node2env,
    wp::int32 var_n_affine_dofs,
    wp::int32 var_num_constraints,
    wp::int32 var_affine_verts_num,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_r,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_q,
    wp::array_t<wp::float64> adj_env_alpha,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_p,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_Ap,
    wp::array_t<wp::mat_t<3,3,wp::float64>> adj_diag_inv,
    wp::array_t<wp::int64> adj_affine_dof_env_id,
    wp::array_t<wp::int32> adj_node2env,
    wp::int32 adj_n_affine_dofs,
    wp::int32 adj_num_constraints,
    wp::int32 adj_affine_verts_num)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        const wp::float32 var_1 = 0.0;
        wp::float64 var_2;
        const wp::int32 var_3 = 3;
        wp::int32 var_4;
        bool var_5;
        const wp::int32 var_6 = 3;
        wp::int32 var_7;
        bool var_8;
        const wp::int32 var_9 = 3;
        wp::int32 var_10;
        wp::int32 var_11;
        wp::int64* var_12;
        wp::float64* var_13;
        wp::int64 var_14;
        wp::float64 var_15;
        wp::float64 var_16;
        wp::float64 var_17;
        const wp::int32 var_18 = 3;
        wp::int32 var_19;
        wp::int32 var_20;
        wp::int32 var_21;
        wp::int32* var_22;
        wp::float64* var_23;
        wp::int32 var_24;
        wp::float64 var_25;
        wp::float64 var_26;
        wp::float64 var_27;
        wp::float64 var_28;
        wp::vec_t<3,wp::float64>* var_29;
        wp::vec_t<3,wp::float64> var_30;
        wp::vec_t<3,wp::float64> var_31;
        wp::vec_t<3,wp::float64> var_32;
        wp::vec_t<3,wp::float64>* var_33;
        wp::vec_t<3,wp::float64> var_34;
        wp::vec_t<3,wp::float64> var_35;
        wp::vec_t<3,wp::float64> var_36;
        wp::mat_t<3,3,wp::float64>* var_37;
        wp::vec_t<3,wp::float64>* var_38;
        wp::vec_t<3,wp::float64> var_39;
        wp::mat_t<3,3,wp::float64> var_40;
        wp::vec_t<3,wp::float64> var_41;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        wp::float32 adj_1 = {};
        wp::float64 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        bool adj_5 = {};
        wp::int32 adj_6 = {};
        wp::int32 adj_7 = {};
        bool adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int64 adj_12 = {};
        wp::float64 adj_13 = {};
        wp::int64 adj_14 = {};
        wp::float64 adj_15 = {};
        wp::float64 adj_16 = {};
        wp::float64 adj_17 = {};
        wp::int32 adj_18 = {};
        wp::int32 adj_19 = {};
        wp::int32 adj_20 = {};
        wp::int32 adj_21 = {};
        wp::int32 adj_22 = {};
        wp::float64 adj_23 = {};
        wp::int32 adj_24 = {};
        wp::float64 adj_25 = {};
        wp::float64 adj_26 = {};
        wp::float64 adj_27 = {};
        wp::float64 adj_28 = {};
        wp::vec_t<3,wp::float64> adj_29 = {};
        wp::vec_t<3,wp::float64> adj_30 = {};
        wp::vec_t<3,wp::float64> adj_31 = {};
        wp::vec_t<3,wp::float64> adj_32 = {};
        wp::vec_t<3,wp::float64> adj_33 = {};
        wp::vec_t<3,wp::float64> adj_34 = {};
        wp::vec_t<3,wp::float64> adj_35 = {};
        wp::vec_t<3,wp::float64> adj_36 = {};
        wp::mat_t<3,3,wp::float64> adj_37 = {};
        wp::vec_t<3,wp::float64> adj_38 = {};
        wp::vec_t<3,wp::float64> adj_39 = {};
        wp::mat_t<3,3,wp::float64> adj_40 = {};
        wp::vec_t<3,wp::float64> adj_41 = {};
        //---------
        // forward
        // def cg_one_iter(                                                                       <L 50>
        // tid = wp.tid()                                                                         <L 64>
        var_0 = builtin_tid1d();
        // alpha = wp.float64(0.0)                                                                <L 65>
        var_2 = wp::float64(var_1);
        // if tid < num_constraints // 3:                                                         <L 66>
        var_4 = wp::floordiv(var_num_constraints, var_3);
        var_5 = (var_0 < var_4);
        if (var_5) {
            // return                                                                             <L 67>
            goto label0;
        }
        if (!var_5) {
            // elif tid < n_affine_dofs // 3:                                                     <L 68>
            var_7 = wp::floordiv(var_n_affine_dofs, var_6);
            var_8 = (var_0 < var_7);
            if (var_8) {
                // alpha = env_alpha[affine_dof_env_id[tid - num_constraints // 3]]               <L 69>
                var_10 = wp::floordiv(var_num_constraints, var_9);
                var_11 = wp::sub(var_0, var_10);
                var_12 = wp::address(var_affine_dof_env_id, var_11);
                var_14 = wp::load(var_12);
                var_13 = wp::address(var_env_alpha, var_14);
                var_16 = wp::load(var_13);
                var_15 = wp::copy(var_16);
            }
            var_17 = wp::where(var_8, var_15, var_2);
            if (!var_8) {
                // alpha = env_alpha[node2env[tid - n_affine_dofs // 3 + affine_verts_num]]       <L 71>
                var_19 = wp::floordiv(var_n_affine_dofs, var_18);
                var_20 = wp::sub(var_0, var_19);
                var_21 = wp::add(var_20, var_affine_verts_num);
                var_22 = wp::address(var_node2env, var_21);
                var_24 = wp::load(var_22);
                var_23 = wp::address(var_env_alpha, var_24);
                var_26 = wp::load(var_23);
                var_25 = wp::copy(var_26);
            }
            var_27 = wp::where(var_8, var_17, var_25);
        }
        var_28 = wp::where(var_5, var_2, var_27);
        // x[tid] += alpha * p[tid]                                                               <L 72>
        var_29 = wp::address(var_p, var_0);
        var_31 = wp::load(var_29);
        var_30 = wp::mul(var_28, var_31);
        // var_32 = wp::atomic_add(var_x, var_0, var_30);
        // r[tid] -= alpha * Ap[tid]                                                              <L 73>
        var_33 = wp::address(var_Ap, var_0);
        var_35 = wp::load(var_33);
        var_34 = wp::mul(var_28, var_35);
        // var_36 = wp::atomic_sub(var_r, var_0, var_34);
        // q[tid] = diag_inv[tid] @ r[tid]                                                        <L 74>
        var_37 = wp::address(var_diag_inv, var_0);
        var_38 = wp::address(var_r, var_0);
        var_40 = wp::load(var_37);
        var_41 = wp::load(var_38);
        var_39 = wp::mul(var_40, var_41);
        // wp::array_store(var_q, var_0, var_39);
        //---------
        // reverse
        wp::adj_array_store(var_q, var_0, var_39, adj_q, adj_0, adj_39);
        wp::adj_mul(var_40, var_41, adj_37, adj_38, adj_39);
        wp::adj_load(var_38, adj_38, adj_41);
        wp::adj_load(var_37, adj_37, adj_40);
        wp::adj_address(var_r, var_0, adj_r, adj_0, adj_38);
        wp::adj_address(var_diag_inv, var_0, adj_diag_inv, adj_0, adj_37);
        // adj: q[tid] = diag_inv[tid] @ r[tid]                                                   <L 74>
        wp::adj_atomic_sub(var_r, var_0, var_34, adj_r, adj_0, adj_34, adj_36);
        wp::adj_mul(var_28, var_35, adj_28, adj_33, adj_34);
        wp::adj_load(var_33, adj_33, adj_35);
        wp::adj_address(var_Ap, var_0, adj_Ap, adj_0, adj_33);
        // adj: r[tid] -= alpha * Ap[tid]                                                         <L 73>
        wp::adj_atomic_add(var_x, var_0, var_30, adj_x, adj_0, adj_30, adj_32);
        wp::adj_mul(var_28, var_31, adj_28, adj_29, adj_30);
        wp::adj_load(var_29, adj_29, adj_31);
        wp::adj_address(var_p, var_0, adj_p, adj_0, adj_29);
        // adj: x[tid] += alpha * p[tid]                                                          <L 72>
        wp::adj_where(var_5, var_2, var_27, adj_5, adj_2, adj_27, adj_28);
        if (!var_5) {
            wp::adj_where(var_8, var_17, var_25, adj_8, adj_17, adj_25, adj_27);
            if (!var_8) {
                wp::adj_copy(var_26, adj_23, adj_25);
                wp::adj_load(var_23, adj_23, adj_26);
                wp::adj_address(var_env_alpha, var_24, adj_env_alpha, adj_22, adj_23);
                wp::adj_load(var_22, adj_22, adj_24);
                wp::adj_address(var_node2env, var_21, adj_node2env, adj_21, adj_22);
                wp::adj_add(var_20, var_affine_verts_num, adj_20, adj_affine_verts_num, adj_21);
                wp::adj_sub(var_0, var_19, adj_0, adj_19, adj_20);
                wp::adj_floordiv(var_n_affine_dofs, var_18, adj_n_affine_dofs, adj_18, adj_19);
                // adj: alpha = env_alpha[node2env[tid - n_affine_dofs // 3 + affine_verts_num]]  <L 71>
            }
            wp::adj_where(var_8, var_15, var_2, adj_8, adj_15, adj_2, adj_17);
            if (var_8) {
                wp::adj_copy(var_16, adj_13, adj_15);
                wp::adj_load(var_13, adj_13, adj_16);
                wp::adj_address(var_env_alpha, var_14, adj_env_alpha, adj_12, adj_13);
                wp::adj_load(var_12, adj_12, adj_14);
                wp::adj_address(var_affine_dof_env_id, var_11, adj_affine_dof_env_id, adj_11, adj_12);
                wp::adj_sub(var_0, var_10, adj_0, adj_10, adj_11);
                wp::adj_floordiv(var_num_constraints, var_9, adj_num_constraints, adj_9, adj_10);
                // adj: alpha = env_alpha[affine_dof_env_id[tid - num_constraints // 3]]          <L 69>
            }
            wp::adj_floordiv(var_n_affine_dofs, var_6, adj_n_affine_dofs, adj_6, adj_7);
            // adj: elif tid < n_affine_dofs // 3:                                                <L 68>
        }
        if (var_5) {
            label0:;
            // adj: return                                                                        <L 67>
        }
        wp::adj_floordiv(var_num_constraints, var_3, adj_num_constraints, adj_3, adj_4);
        // adj: if tid < num_constraints // 3:                                                    <L 66>
        wp::adj_float64(var_1, adj_1, adj_2);
        // adj: alpha = wp.float64(0.0)                                                           <L 65>
        // adj: tid = wp.tid()                                                                    <L 64>
        // adj: def cg_one_iter(                                                                  <L 50>
        continue;
    }
}

