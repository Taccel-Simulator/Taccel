#include "hip/hip_runtime.h"

#define WP_TILE_BLOCK_DIM 256
#define WP_NO_CRT
#include "builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)

#define builtin_tid1d() wp::tid(_idx, dim)
#define builtin_tid2d(x, y) wp::tid(x, y, _idx, dim)
#define builtin_tid3d(x, y, z) wp::tid(x, y, z, _idx, dim)
#define builtin_tid4d(x, y, z, w) wp::tid(x, y, z, w, _idx, dim)

extern "C" {
}

// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/wp_math.py:187
static CUDA_CALLABLE wp::mat_t<3,3,wp::float64> col_stack3_1(
    wp::vec_t<3,wp::float64> var_vec0,
    wp::vec_t<3,wp::float64> var_vec1,
    wp::vec_t<3,wp::float64> var_vec2)
{
    //---------
    // primal vars
    const wp::int32 var_0 = 0;
    wp::float64 var_1;
    const wp::int32 var_2 = 0;
    wp::float64 var_3;
    const wp::int32 var_4 = 0;
    wp::float64 var_5;
    const wp::int32 var_6 = 1;
    wp::float64 var_7;
    const wp::int32 var_8 = 1;
    wp::float64 var_9;
    const wp::int32 var_10 = 1;
    wp::float64 var_11;
    const wp::int32 var_12 = 2;
    wp::float64 var_13;
    const wp::int32 var_14 = 2;
    wp::float64 var_15;
    const wp::int32 var_16 = 2;
    wp::float64 var_17;
    wp::mat_t<3,3,wp::float64> var_18;
    //---------
    // forward
    // def col_stack3(vec0: wp.vec3d, vec1: wp.vec3d, vec2: wp.vec3d):                        <L 188>
    // return mat33d(vec0[0], vec1[0], vec2[0], vec0[1], vec1[1], vec2[1], vec0[2], vec1[2], vec2[2])       <L 189>
    var_1 = wp::extract(var_vec0, var_0);
    var_3 = wp::extract(var_vec1, var_2);
    var_5 = wp::extract(var_vec2, var_4);
    var_7 = wp::extract(var_vec0, var_6);
    var_9 = wp::extract(var_vec1, var_8);
    var_11 = wp::extract(var_vec2, var_10);
    var_13 = wp::extract(var_vec0, var_12);
    var_15 = wp::extract(var_vec1, var_14);
    var_17 = wp::extract(var_vec2, var_16);
    var_18 = wp::mat_t<3,3,wp::float64>(var_1, var_3, var_5, var_7, var_9, var_11, var_13, var_15, var_17);
    return var_18;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/wp_math.py:187
static CUDA_CALLABLE void adj_col_stack3_1(
    wp::vec_t<3,wp::float64> var_vec0,
    wp::vec_t<3,wp::float64> var_vec1,
    wp::vec_t<3,wp::float64> var_vec2,
    wp::vec_t<3,wp::float64> & adj_vec0,
    wp::vec_t<3,wp::float64> & adj_vec1,
    wp::vec_t<3,wp::float64> & adj_vec2,
    wp::mat_t<3,3,wp::float64> & adj_ret)
{
    //---------
    // primal vars
    const wp::int32 var_0 = 0;
    wp::float64 var_1;
    const wp::int32 var_2 = 0;
    wp::float64 var_3;
    const wp::int32 var_4 = 0;
    wp::float64 var_5;
    const wp::int32 var_6 = 1;
    wp::float64 var_7;
    const wp::int32 var_8 = 1;
    wp::float64 var_9;
    const wp::int32 var_10 = 1;
    wp::float64 var_11;
    const wp::int32 var_12 = 2;
    wp::float64 var_13;
    const wp::int32 var_14 = 2;
    wp::float64 var_15;
    const wp::int32 var_16 = 2;
    wp::float64 var_17;
    wp::mat_t<3,3,wp::float64> var_18;
    //---------
    // dual vars
    wp::int32 adj_0 = {};
    wp::float64 adj_1 = {};
    wp::int32 adj_2 = {};
    wp::float64 adj_3 = {};
    wp::int32 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::int32 adj_6 = {};
    wp::float64 adj_7 = {};
    wp::int32 adj_8 = {};
    wp::float64 adj_9 = {};
    wp::int32 adj_10 = {};
    wp::float64 adj_11 = {};
    wp::int32 adj_12 = {};
    wp::float64 adj_13 = {};
    wp::int32 adj_14 = {};
    wp::float64 adj_15 = {};
    wp::int32 adj_16 = {};
    wp::float64 adj_17 = {};
    wp::mat_t<3,3,wp::float64> adj_18 = {};
    //---------
    // forward
    // def col_stack3(vec0: wp.vec3d, vec1: wp.vec3d, vec2: wp.vec3d):                        <L 188>
    // return mat33d(vec0[0], vec1[0], vec2[0], vec0[1], vec1[1], vec2[1], vec0[2], vec1[2], vec2[2])       <L 189>
    var_1 = wp::extract(var_vec0, var_0);
    var_3 = wp::extract(var_vec1, var_2);
    var_5 = wp::extract(var_vec2, var_4);
    var_7 = wp::extract(var_vec0, var_6);
    var_9 = wp::extract(var_vec1, var_8);
    var_11 = wp::extract(var_vec2, var_10);
    var_13 = wp::extract(var_vec0, var_12);
    var_15 = wp::extract(var_vec1, var_14);
    var_17 = wp::extract(var_vec2, var_16);
    var_18 = wp::mat_t<3,3,wp::float64>(var_1, var_3, var_5, var_7, var_9, var_11, var_13, var_15, var_17);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_18 += adj_ret;
    wp::adj_mat_t(var_1, var_3, var_5, var_7, var_9, var_11, var_13, var_15, var_17, adj_1, adj_3, adj_5, adj_7, adj_9, adj_11, adj_13, adj_15, adj_17, adj_18);
    wp::adj_extract(var_vec2, var_16, adj_vec2, adj_16, adj_17);
    wp::adj_extract(var_vec1, var_14, adj_vec1, adj_14, adj_15);
    wp::adj_extract(var_vec0, var_12, adj_vec0, adj_12, adj_13);
    wp::adj_extract(var_vec2, var_10, adj_vec2, adj_10, adj_11);
    wp::adj_extract(var_vec1, var_8, adj_vec1, adj_8, adj_9);
    wp::adj_extract(var_vec0, var_6, adj_vec0, adj_6, adj_7);
    wp::adj_extract(var_vec2, var_4, adj_vec2, adj_4, adj_5);
    wp::adj_extract(var_vec1, var_2, adj_vec1, adj_2, adj_3);
    wp::adj_extract(var_vec0, var_0, adj_vec0, adj_0, adj_1);
    // adj: return mat33d(vec0[0], vec1[0], vec2[0], vec0[1], vec1[1], vec2[1], vec0[2], vec1[2], vec2[2])  <L 189>
    // adj: def col_stack3(vec0: wp.vec3d, vec1: wp.vec3d, vec2: wp.vec3d):                   <L 188>
    return;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/wp_math.py:92
static CUDA_CALLABLE wp::vec_t<12,wp::float64> cat_4_vec3d_0(
    wp::vec_t<3,wp::float64> var_a,
    wp::vec_t<3,wp::float64> var_b,
    wp::vec_t<3,wp::float64> var_c,
    wp::vec_t<3,wp::float64> var_d)
{
    //---------
    // primal vars
    const wp::int32 var_0 = 0;
    wp::float64 var_1;
    const wp::int32 var_2 = 1;
    wp::float64 var_3;
    const wp::int32 var_4 = 2;
    wp::float64 var_5;
    const wp::int32 var_6 = 0;
    wp::float64 var_7;
    const wp::int32 var_8 = 1;
    wp::float64 var_9;
    const wp::int32 var_10 = 2;
    wp::float64 var_11;
    const wp::int32 var_12 = 0;
    wp::float64 var_13;
    const wp::int32 var_14 = 1;
    wp::float64 var_15;
    const wp::int32 var_16 = 2;
    wp::float64 var_17;
    const wp::int32 var_18 = 0;
    wp::float64 var_19;
    const wp::int32 var_20 = 1;
    wp::float64 var_21;
    const wp::int32 var_22 = 2;
    wp::float64 var_23;
    wp::vec_t<12,wp::float64> var_24;
    //---------
    // forward
    // def cat_4_vec3d(                                                                       <L 93>
    // return vec12d(a[0], a[1], a[2], b[0], b[1], b[2], c[0], c[1], c[2], d[0], d[1], d[2])       <L 99>
    var_1 = wp::extract(var_a, var_0);
    var_3 = wp::extract(var_a, var_2);
    var_5 = wp::extract(var_a, var_4);
    var_7 = wp::extract(var_b, var_6);
    var_9 = wp::extract(var_b, var_8);
    var_11 = wp::extract(var_b, var_10);
    var_13 = wp::extract(var_c, var_12);
    var_15 = wp::extract(var_c, var_14);
    var_17 = wp::extract(var_c, var_16);
    var_19 = wp::extract(var_d, var_18);
    var_21 = wp::extract(var_d, var_20);
    var_23 = wp::extract(var_d, var_22);
    var_24 = wp::vec_t<12,wp::float64>({var_1, var_3, var_5, var_7, var_9, var_11, var_13, var_15, var_17, var_19, var_21, var_23});
    return var_24;
}


// /home/chang/actions-runner/_work/Taccel-Action/Taccel-Action/turbo/warp_ipc/utils/wp_math.py:92
static CUDA_CALLABLE void adj_cat_4_vec3d_0(
    wp::vec_t<3,wp::float64> var_a,
    wp::vec_t<3,wp::float64> var_b,
    wp::vec_t<3,wp::float64> var_c,
    wp::vec_t<3,wp::float64> var_d,
    wp::vec_t<3,wp::float64> & adj_a,
    wp::vec_t<3,wp::float64> & adj_b,
    wp::vec_t<3,wp::float64> & adj_c,
    wp::vec_t<3,wp::float64> & adj_d,
    wp::vec_t<12,wp::float64> & adj_ret)
{
    //---------
    // primal vars
    const wp::int32 var_0 = 0;
    wp::float64 var_1;
    const wp::int32 var_2 = 1;
    wp::float64 var_3;
    const wp::int32 var_4 = 2;
    wp::float64 var_5;
    const wp::int32 var_6 = 0;
    wp::float64 var_7;
    const wp::int32 var_8 = 1;
    wp::float64 var_9;
    const wp::int32 var_10 = 2;
    wp::float64 var_11;
    const wp::int32 var_12 = 0;
    wp::float64 var_13;
    const wp::int32 var_14 = 1;
    wp::float64 var_15;
    const wp::int32 var_16 = 2;
    wp::float64 var_17;
    const wp::int32 var_18 = 0;
    wp::float64 var_19;
    const wp::int32 var_20 = 1;
    wp::float64 var_21;
    const wp::int32 var_22 = 2;
    wp::float64 var_23;
    wp::vec_t<12,wp::float64> var_24;
    //---------
    // dual vars
    wp::int32 adj_0 = {};
    wp::float64 adj_1 = {};
    wp::int32 adj_2 = {};
    wp::float64 adj_3 = {};
    wp::int32 adj_4 = {};
    wp::float64 adj_5 = {};
    wp::int32 adj_6 = {};
    wp::float64 adj_7 = {};
    wp::int32 adj_8 = {};
    wp::float64 adj_9 = {};
    wp::int32 adj_10 = {};
    wp::float64 adj_11 = {};
    wp::int32 adj_12 = {};
    wp::float64 adj_13 = {};
    wp::int32 adj_14 = {};
    wp::float64 adj_15 = {};
    wp::int32 adj_16 = {};
    wp::float64 adj_17 = {};
    wp::int32 adj_18 = {};
    wp::float64 adj_19 = {};
    wp::int32 adj_20 = {};
    wp::float64 adj_21 = {};
    wp::int32 adj_22 = {};
    wp::float64 adj_23 = {};
    wp::vec_t<12,wp::float64> adj_24 = {};
    //---------
    // forward
    // def cat_4_vec3d(                                                                       <L 93>
    // return vec12d(a[0], a[1], a[2], b[0], b[1], b[2], c[0], c[1], c[2], d[0], d[1], d[2])       <L 99>
    var_1 = wp::extract(var_a, var_0);
    var_3 = wp::extract(var_a, var_2);
    var_5 = wp::extract(var_a, var_4);
    var_7 = wp::extract(var_b, var_6);
    var_9 = wp::extract(var_b, var_8);
    var_11 = wp::extract(var_b, var_10);
    var_13 = wp::extract(var_c, var_12);
    var_15 = wp::extract(var_c, var_14);
    var_17 = wp::extract(var_c, var_16);
    var_19 = wp::extract(var_d, var_18);
    var_21 = wp::extract(var_d, var_20);
    var_23 = wp::extract(var_d, var_22);
    var_24 = wp::vec_t<12,wp::float64>({var_1, var_3, var_5, var_7, var_9, var_11, var_13, var_15, var_17, var_19, var_21, var_23});
    goto label0;
    //---------
    // reverse
    label0:;
    adj_24 += adj_ret;
    wp::adj_vec_t({var_1, var_3, var_5, var_7, var_9, var_11, var_13, var_15, var_17, var_19, var_21, var_23}, {&adj_1, &adj_3, &adj_5, &adj_7, &adj_9, &adj_11, &adj_13, &adj_15, &adj_17, &adj_19, &adj_21, &adj_23}, adj_24);
    wp::adj_extract(var_d, var_22, adj_d, adj_22, adj_23);
    wp::adj_extract(var_d, var_20, adj_d, adj_20, adj_21);
    wp::adj_extract(var_d, var_18, adj_d, adj_18, adj_19);
    wp::adj_extract(var_c, var_16, adj_c, adj_16, adj_17);
    wp::adj_extract(var_c, var_14, adj_c, adj_14, adj_15);
    wp::adj_extract(var_c, var_12, adj_c, adj_12, adj_13);
    wp::adj_extract(var_b, var_10, adj_b, adj_10, adj_11);
    wp::adj_extract(var_b, var_8, adj_b, adj_8, adj_9);
    wp::adj_extract(var_b, var_6, adj_b, adj_6, adj_7);
    wp::adj_extract(var_a, var_4, adj_a, adj_4, adj_5);
    wp::adj_extract(var_a, var_2, adj_a, adj_2, adj_3);
    wp::adj_extract(var_a, var_0, adj_a, adj_0, adj_1);
    // adj: return vec12d(a[0], a[1], a[2], b[0], b[1], b[2], c[0], c[1], c[2], d[0], d[1], d[2])  <L 99>
    // adj: def cat_4_vec3d(                                                                  <L 93>
    return;
}



extern "C" __global__ void compute_body_force_energy_grad_affine_y_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<12,wp::float64>> var_gradient,
    wp::array_t<wp::vec_t<12,wp::float64>> var_hat_y,
    wp::array_t<wp::mat_t<4,4,wp::float64>> var_mass_matrix,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_ext_force,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_ext_y_force,
    wp::float64 var_scale,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        wp::int32* var_2;
        wp::int32* var_3;
        wp::int32 var_4;
        const wp::int32 var_5 = 1;
        bool var_6;
        wp::int32 var_7;
        wp::int32* var_8;
        wp::int32* var_9;
        wp::int32 var_10;
        const wp::int32 var_11 = 2;
        bool var_12;
        wp::int32 var_13;
        bool var_14;
        bool var_15;
        bool var_16;
        wp::vec_t<12,wp::float64> var_17;
        const wp::int32 var_18 = 3;
        const wp::int32 var_19 = 1;
        wp::int32 var_20;
        wp::int32 var_21;
        wp::mat_t<4,4,wp::float64>* var_22;
        wp::mat_t<4,4,wp::float64> var_23;
        wp::mat_t<4,4,wp::float64> var_24;
        wp::vec_t<3,wp::float64>* var_25;
        wp::vec_t<3,wp::float64> var_26;
        wp::vec_t<3,wp::float64> var_27;
        wp::range_t var_28;
        wp::int32 var_29;
        const wp::int32 var_30 = 1;
        wp::int32 var_31;
        wp::range_t var_32;
        wp::int32 var_33;
        wp::int32 var_34;
        wp::float64 var_35;
        wp::int32 var_36;
        wp::float64 var_37;
        wp::float64 var_38;
        wp::float64 var_39;
        wp::int32 var_40;
        wp::float64 var_41;
        wp::float64 var_42;
        wp::float64 var_43;
        wp::int32 var_44;
        wp::int32 var_45;
        wp::int32 var_46;
        wp::vec_t<12,wp::float64>* var_47;
        wp::vec_t<12,wp::float64> var_48;
        wp::vec_t<12,wp::float64> var_49;
        wp::vec_t<12,wp::float64>* var_50;
        wp::vec_t<12,wp::float64> var_51;
        wp::vec_t<12,wp::float64> var_52;
        const wp::int32 var_53 = 9;
        wp::float64 var_54;
        const wp::int32 var_55 = 10;
        wp::float64 var_56;
        const wp::int32 var_57 = 11;
        wp::float64 var_58;
        wp::vec_t<3,wp::float64> var_59;
        const wp::int32 var_60 = 0;
        wp::float64 var_61;
        const wp::int32 var_62 = 1;
        wp::float64 var_63;
        const wp::int32 var_64 = 2;
        wp::float64 var_65;
        wp::vec_t<3,wp::float64> var_66;
        const wp::int32 var_67 = 3;
        wp::float64 var_68;
        const wp::int32 var_69 = 4;
        wp::float64 var_70;
        const wp::int32 var_71 = 5;
        wp::float64 var_72;
        wp::vec_t<3,wp::float64> var_73;
        const wp::int32 var_74 = 6;
        wp::float64 var_75;
        const wp::int32 var_76 = 7;
        wp::float64 var_77;
        const wp::int32 var_78 = 8;
        wp::float64 var_79;
        wp::vec_t<3,wp::float64> var_80;
        wp::vec_t<3,wp::float64> var_81;
        wp::vec_t<3,wp::float64> var_82;
        wp::vec_t<3,wp::float64> var_83;
        wp::mat_t<3,3,wp::float64> var_84;
        const wp::int32 var_85 = 0;
        wp::float64 var_86;
        const wp::int32 var_87 = 1;
        wp::float64 var_88;
        const wp::int32 var_89 = 2;
        wp::float64 var_90;
        wp::vec_t<3,wp::float64> var_91;
        wp::vec_t<3,wp::float64> var_92;
        const wp::int32 var_93 = 3;
        wp::float64 var_94;
        const wp::int32 var_95 = 4;
        wp::float64 var_96;
        const wp::int32 var_97 = 5;
        wp::float64 var_98;
        wp::vec_t<3,wp::float64> var_99;
        wp::vec_t<3,wp::float64> var_100;
        const wp::int32 var_101 = 6;
        wp::float64 var_102;
        const wp::int32 var_103 = 7;
        wp::float64 var_104;
        const wp::int32 var_105 = 8;
        wp::float64 var_106;
        wp::vec_t<3,wp::float64> var_107;
        wp::vec_t<3,wp::float64> var_108;
        const wp::int32 var_109 = 9;
        wp::float64 var_110;
        const wp::int32 var_111 = 10;
        wp::float64 var_112;
        const wp::int32 var_113 = 11;
        wp::float64 var_114;
        wp::vec_t<3,wp::float64> var_115;
        wp::vec_t<3,wp::float64> var_116;
        wp::vec_t<12,wp::float64> var_117;
        const wp::int32 var_118 = 0;
        wp::float64 var_119;
        wp::float64 var_120;
        wp::float64 var_121;
        const wp::int32 var_122 = 1;
        wp::float64 var_123;
        wp::float64 var_124;
        wp::float64 var_125;
        const wp::int32 var_126 = 2;
        wp::float64 var_127;
        wp::float64 var_128;
        wp::float64 var_129;
        const wp::int32 var_130 = 3;
        wp::float64 var_131;
        wp::float64 var_132;
        wp::float64 var_133;
        const wp::int32 var_134 = 4;
        wp::float64 var_135;
        wp::float64 var_136;
        wp::float64 var_137;
        const wp::int32 var_138 = 5;
        wp::float64 var_139;
        wp::float64 var_140;
        wp::float64 var_141;
        const wp::int32 var_142 = 6;
        wp::float64 var_143;
        wp::float64 var_144;
        wp::float64 var_145;
        const wp::int32 var_146 = 7;
        wp::float64 var_147;
        wp::float64 var_148;
        wp::float64 var_149;
        const wp::int32 var_150 = 8;
        wp::float64 var_151;
        wp::float64 var_152;
        wp::float64 var_153;
        const wp::int32 var_154 = 9;
        wp::float64 var_155;
        wp::float64 var_156;
        wp::float64 var_157;
        const wp::int32 var_158 = 10;
        wp::float64 var_159;
        wp::float64 var_160;
        wp::float64 var_161;
        const wp::int32 var_162 = 11;
        wp::float64 var_163;
        wp::float64 var_164;
        wp::float64 var_165;
        wp::vec_t<12,wp::float64> var_166;
        //---------
        // forward
        // def compute_body_force_energy_grad_affine_y(                                           <L 72>
        // tid = wp.tid()                                                                         <L 84>
        var_0 = builtin_tid1d();
        // if affine_has_constraint[tid] or (                                                     <L 85>
        var_1 = wp::address(var_affine_has_constraint, var_0);
        // (env_states[body_env_id[tid]] == ENV_STATE_INVALID) or (env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED)       <L 86>
        var_2 = wp::address(var_body_env_id, var_0);
        var_4 = wp::load(var_2);
        var_3 = wp::address(var_env_states, var_4);
        var_7 = wp::load(var_3);
        var_6 = (var_7 == var_5);
        var_8 = wp::address(var_body_env_id, var_0);
        var_10 = wp::load(var_8);
        var_9 = wp::address(var_env_states, var_10);
        var_13 = wp::load(var_9);
        var_12 = (var_13 == var_11);
        var_14 = var_6 || var_12;
        var_15 = wp::load(var_1);
        var_16 = var_15 || var_14;
        if (var_16) {
            // return                                                                             <L 88>
            continue;
        }
        // local_gradient = vec12d()                                                              <L 90>
        var_17 = wp::vec_t<12,wp::float64>();
        // dim = 3                                                                                <L 91>
        // n_dof = (dim + 1) * dim                                                                <L 92>
        var_20 = wp::add(var_18, var_19);
        var_21 = wp::mul(var_20, var_18);
        // mass = mass_matrix[tid]                                                                <L 93>
        var_22 = wp::address(var_mass_matrix, var_0);
        var_24 = wp::load(var_22);
        var_23 = wp::copy(var_24);
        // ext_force = affine_ext_force[tid]                                                      <L 94>
        var_25 = wp::address(var_affine_ext_force, var_0);
        var_27 = wp::load(var_25);
        var_26 = wp::copy(var_27);
        // for i in range(n_dof):                                                                 <L 95>
        var_28 = wp::range(var_21);
        start_for_1:;
            if (iter_cmp(var_28) == 0) goto end_for_1;
            var_29 = wp::iter_next(var_28);
            // for j in range(dim + 1):                                                           <L 96>
            var_31 = wp::add(var_18, var_30);
            var_32 = wp::range(var_31);
            start_for_3:;
                if (iter_cmp(var_32) == 0) goto end_for_3;
                var_33 = wp::iter_next(var_32);
                // local_gradient[j * dim + i % dim] += -(gravity[i % dim] + ext_force[i % dim]) * mass[(i / dim), j] * scale       <L 97>
                var_34 = wp::mod(var_29, var_18);
                var_35 = wp::extract(var_gravity, var_34);
                var_36 = wp::mod(var_29, var_18);
                var_37 = wp::extract(var_26, var_36);
                var_38 = wp::add(var_35, var_37);
                var_39 = wp::neg(var_38);
                var_40 = wp::div(var_29, var_18);
                var_41 = wp::extract(var_23, var_40, var_33);
                var_42 = wp::mul(var_39, var_41);
                var_43 = wp::mul(var_42, var_scale);
                var_44 = wp::mul(var_33, var_18);
                var_45 = wp::mod(var_29, var_18);
                var_46 = wp::add(var_44, var_45);
                wp::add_inplace(var_17, var_46, var_43);
                goto start_for_3;
            end_for_3:;
            goto start_for_1;
        end_for_1:;
        // rest_y_force = affine_ext_y_force[tid]                                                 <L 98>
        var_47 = wp::address(var_affine_ext_y_force, var_0);
        var_49 = wp::load(var_47);
        var_48 = wp::copy(var_49);
        // hat_yi = hat_y[tid]                                                                    <L 99>
        var_50 = wp::address(var_hat_y, var_0);
        var_52 = wp::load(var_50);
        var_51 = wp::copy(var_52);
        // O = wp.vec3d(hat_yi[9], hat_yi[10], hat_yi[11])                                        <L 100>
        var_54 = wp::extract(var_51, var_53);
        var_56 = wp::extract(var_51, var_55);
        var_58 = wp::extract(var_51, var_57);
        var_59 = wp::vec_t<3,wp::float64>(var_54, var_56, var_58);
        // A = wp.vec3d(hat_yi[0], hat_yi[1], hat_yi[2])                                          <L 101>
        var_61 = wp::extract(var_51, var_60);
        var_63 = wp::extract(var_51, var_62);
        var_65 = wp::extract(var_51, var_64);
        var_66 = wp::vec_t<3,wp::float64>(var_61, var_63, var_65);
        // B = wp.vec3d(hat_yi[3], hat_yi[4], hat_yi[5])                                          <L 102>
        var_68 = wp::extract(var_51, var_67);
        var_70 = wp::extract(var_51, var_69);
        var_72 = wp::extract(var_51, var_71);
        var_73 = wp::vec_t<3,wp::float64>(var_68, var_70, var_72);
        // C = wp.vec3d(hat_yi[6], hat_yi[7], hat_yi[8])                                          <L 103>
        var_75 = wp::extract(var_51, var_74);
        var_77 = wp::extract(var_51, var_76);
        var_79 = wp::extract(var_51, var_78);
        var_80 = wp::vec_t<3,wp::float64>(var_75, var_77, var_79);
        // R = col_stack3(A - O, B - O, C - O)                                                    <L 104>
        var_81 = wp::sub(var_66, var_59);
        var_82 = wp::sub(var_73, var_59);
        var_83 = wp::sub(var_80, var_59);
        var_84 = col_stack3_1(var_81, var_82, var_83);
        // y_force = cat_4_vec3d(                                                                 <L 105>
        // R * wp.vec3d(rest_y_force[0], rest_y_force[1], rest_y_force[2]),                       <L 106>
        var_86 = wp::extract(var_48, var_85);
        var_88 = wp::extract(var_48, var_87);
        var_90 = wp::extract(var_48, var_89);
        var_91 = wp::vec_t<3,wp::float64>(var_86, var_88, var_90);
        var_92 = wp::mul(var_84, var_91);
        // R * wp.vec3d(rest_y_force[3], rest_y_force[4], rest_y_force[5]),                       <L 107>
        var_94 = wp::extract(var_48, var_93);
        var_96 = wp::extract(var_48, var_95);
        var_98 = wp::extract(var_48, var_97);
        var_99 = wp::vec_t<3,wp::float64>(var_94, var_96, var_98);
        var_100 = wp::mul(var_84, var_99);
        // R * wp.vec3d(rest_y_force[6], rest_y_force[7], rest_y_force[8]),                       <L 108>
        var_102 = wp::extract(var_48, var_101);
        var_104 = wp::extract(var_48, var_103);
        var_106 = wp::extract(var_48, var_105);
        var_107 = wp::vec_t<3,wp::float64>(var_102, var_104, var_106);
        var_108 = wp::mul(var_84, var_107);
        // R * wp.vec3d(rest_y_force[9], rest_y_force[10], rest_y_force[11]),                     <L 109>
        var_110 = wp::extract(var_48, var_109);
        var_112 = wp::extract(var_48, var_111);
        var_114 = wp::extract(var_48, var_113);
        var_115 = wp::vec_t<3,wp::float64>(var_110, var_112, var_114);
        var_116 = wp::mul(var_84, var_115);
        var_117 = cat_4_vec3d_0(var_92, var_100, var_108, var_116);
        // for d in range(12):                                                                    <L 111>
        // local_gradient[d] += -y_force[d] * scale                                               <L 112>
        var_119 = wp::extract(var_117, var_118);
        var_120 = wp::neg(var_119);
        var_121 = wp::mul(var_120, var_scale);
        wp::add_inplace(var_17, var_118, var_121);
        var_123 = wp::extract(var_117, var_122);
        var_124 = wp::neg(var_123);
        var_125 = wp::mul(var_124, var_scale);
        wp::add_inplace(var_17, var_122, var_125);
        var_127 = wp::extract(var_117, var_126);
        var_128 = wp::neg(var_127);
        var_129 = wp::mul(var_128, var_scale);
        wp::add_inplace(var_17, var_126, var_129);
        var_131 = wp::extract(var_117, var_130);
        var_132 = wp::neg(var_131);
        var_133 = wp::mul(var_132, var_scale);
        wp::add_inplace(var_17, var_130, var_133);
        var_135 = wp::extract(var_117, var_134);
        var_136 = wp::neg(var_135);
        var_137 = wp::mul(var_136, var_scale);
        wp::add_inplace(var_17, var_134, var_137);
        var_139 = wp::extract(var_117, var_138);
        var_140 = wp::neg(var_139);
        var_141 = wp::mul(var_140, var_scale);
        wp::add_inplace(var_17, var_138, var_141);
        var_143 = wp::extract(var_117, var_142);
        var_144 = wp::neg(var_143);
        var_145 = wp::mul(var_144, var_scale);
        wp::add_inplace(var_17, var_142, var_145);
        var_147 = wp::extract(var_117, var_146);
        var_148 = wp::neg(var_147);
        var_149 = wp::mul(var_148, var_scale);
        wp::add_inplace(var_17, var_146, var_149);
        var_151 = wp::extract(var_117, var_150);
        var_152 = wp::neg(var_151);
        var_153 = wp::mul(var_152, var_scale);
        wp::add_inplace(var_17, var_150, var_153);
        var_155 = wp::extract(var_117, var_154);
        var_156 = wp::neg(var_155);
        var_157 = wp::mul(var_156, var_scale);
        wp::add_inplace(var_17, var_154, var_157);
        var_159 = wp::extract(var_117, var_158);
        var_160 = wp::neg(var_159);
        var_161 = wp::mul(var_160, var_scale);
        wp::add_inplace(var_17, var_158, var_161);
        var_163 = wp::extract(var_117, var_162);
        var_164 = wp::neg(var_163);
        var_165 = wp::mul(var_164, var_scale);
        wp::add_inplace(var_17, var_162, var_165);
        // gradient[tid] += local_gradient                                                        <L 114>
        var_166 = wp::atomic_add(var_gradient, var_0, var_17);
    }
}



extern "C" __global__ void compute_body_force_energy_grad_affine_y_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<12,wp::float64>> var_gradient,
    wp::array_t<wp::vec_t<12,wp::float64>> var_hat_y,
    wp::array_t<wp::mat_t<4,4,wp::float64>> var_mass_matrix,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_ext_force,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_ext_y_force,
    wp::float64 var_scale,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::int32> var_body_env_id,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_gradient,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_hat_y,
    wp::array_t<wp::mat_t<4,4,wp::float64>> adj_mass_matrix,
    wp::vec_t<3,wp::float64> adj_gravity,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_affine_ext_force,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_ext_y_force,
    wp::float64 adj_scale,
    wp::array_t<bool> adj_affine_has_constraint,
    wp::array_t<wp::int32> adj_body_env_id,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        wp::int32* var_2;
        wp::int32* var_3;
        wp::int32 var_4;
        const wp::int32 var_5 = 1;
        bool var_6;
        wp::int32 var_7;
        wp::int32* var_8;
        wp::int32* var_9;
        wp::int32 var_10;
        const wp::int32 var_11 = 2;
        bool var_12;
        wp::int32 var_13;
        bool var_14;
        bool var_15;
        bool var_16;
        wp::vec_t<12,wp::float64> var_17;
        const wp::int32 var_18 = 3;
        const wp::int32 var_19 = 1;
        wp::int32 var_20;
        wp::int32 var_21;
        wp::mat_t<4,4,wp::float64>* var_22;
        wp::mat_t<4,4,wp::float64> var_23;
        wp::mat_t<4,4,wp::float64> var_24;
        wp::vec_t<3,wp::float64>* var_25;
        wp::vec_t<3,wp::float64> var_26;
        wp::vec_t<3,wp::float64> var_27;
        wp::range_t var_28;
        wp::int32 var_29;
        const wp::int32 var_30 = 1;
        wp::int32 var_31;
        wp::range_t var_32;
        wp::int32 var_33;
        wp::int32 var_34;
        wp::float64 var_35;
        wp::int32 var_36;
        wp::float64 var_37;
        wp::float64 var_38;
        wp::float64 var_39;
        wp::int32 var_40;
        wp::float64 var_41;
        wp::float64 var_42;
        wp::float64 var_43;
        wp::int32 var_44;
        wp::int32 var_45;
        wp::int32 var_46;
        wp::vec_t<12,wp::float64>* var_47;
        wp::vec_t<12,wp::float64> var_48;
        wp::vec_t<12,wp::float64> var_49;
        wp::vec_t<12,wp::float64>* var_50;
        wp::vec_t<12,wp::float64> var_51;
        wp::vec_t<12,wp::float64> var_52;
        const wp::int32 var_53 = 9;
        wp::float64 var_54;
        const wp::int32 var_55 = 10;
        wp::float64 var_56;
        const wp::int32 var_57 = 11;
        wp::float64 var_58;
        wp::vec_t<3,wp::float64> var_59;
        const wp::int32 var_60 = 0;
        wp::float64 var_61;
        const wp::int32 var_62 = 1;
        wp::float64 var_63;
        const wp::int32 var_64 = 2;
        wp::float64 var_65;
        wp::vec_t<3,wp::float64> var_66;
        const wp::int32 var_67 = 3;
        wp::float64 var_68;
        const wp::int32 var_69 = 4;
        wp::float64 var_70;
        const wp::int32 var_71 = 5;
        wp::float64 var_72;
        wp::vec_t<3,wp::float64> var_73;
        const wp::int32 var_74 = 6;
        wp::float64 var_75;
        const wp::int32 var_76 = 7;
        wp::float64 var_77;
        const wp::int32 var_78 = 8;
        wp::float64 var_79;
        wp::vec_t<3,wp::float64> var_80;
        wp::vec_t<3,wp::float64> var_81;
        wp::vec_t<3,wp::float64> var_82;
        wp::vec_t<3,wp::float64> var_83;
        wp::mat_t<3,3,wp::float64> var_84;
        const wp::int32 var_85 = 0;
        wp::float64 var_86;
        const wp::int32 var_87 = 1;
        wp::float64 var_88;
        const wp::int32 var_89 = 2;
        wp::float64 var_90;
        wp::vec_t<3,wp::float64> var_91;
        wp::vec_t<3,wp::float64> var_92;
        const wp::int32 var_93 = 3;
        wp::float64 var_94;
        const wp::int32 var_95 = 4;
        wp::float64 var_96;
        const wp::int32 var_97 = 5;
        wp::float64 var_98;
        wp::vec_t<3,wp::float64> var_99;
        wp::vec_t<3,wp::float64> var_100;
        const wp::int32 var_101 = 6;
        wp::float64 var_102;
        const wp::int32 var_103 = 7;
        wp::float64 var_104;
        const wp::int32 var_105 = 8;
        wp::float64 var_106;
        wp::vec_t<3,wp::float64> var_107;
        wp::vec_t<3,wp::float64> var_108;
        const wp::int32 var_109 = 9;
        wp::float64 var_110;
        const wp::int32 var_111 = 10;
        wp::float64 var_112;
        const wp::int32 var_113 = 11;
        wp::float64 var_114;
        wp::vec_t<3,wp::float64> var_115;
        wp::vec_t<3,wp::float64> var_116;
        wp::vec_t<12,wp::float64> var_117;
        const wp::int32 var_118 = 0;
        wp::float64 var_119;
        wp::float64 var_120;
        wp::float64 var_121;
        const wp::int32 var_122 = 1;
        wp::float64 var_123;
        wp::float64 var_124;
        wp::float64 var_125;
        const wp::int32 var_126 = 2;
        wp::float64 var_127;
        wp::float64 var_128;
        wp::float64 var_129;
        const wp::int32 var_130 = 3;
        wp::float64 var_131;
        wp::float64 var_132;
        wp::float64 var_133;
        const wp::int32 var_134 = 4;
        wp::float64 var_135;
        wp::float64 var_136;
        wp::float64 var_137;
        const wp::int32 var_138 = 5;
        wp::float64 var_139;
        wp::float64 var_140;
        wp::float64 var_141;
        const wp::int32 var_142 = 6;
        wp::float64 var_143;
        wp::float64 var_144;
        wp::float64 var_145;
        const wp::int32 var_146 = 7;
        wp::float64 var_147;
        wp::float64 var_148;
        wp::float64 var_149;
        const wp::int32 var_150 = 8;
        wp::float64 var_151;
        wp::float64 var_152;
        wp::float64 var_153;
        const wp::int32 var_154 = 9;
        wp::float64 var_155;
        wp::float64 var_156;
        wp::float64 var_157;
        const wp::int32 var_158 = 10;
        wp::float64 var_159;
        wp::float64 var_160;
        wp::float64 var_161;
        const wp::int32 var_162 = 11;
        wp::float64 var_163;
        wp::float64 var_164;
        wp::float64 var_165;
        wp::vec_t<12,wp::float64> var_166;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        bool adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        wp::int32 adj_5 = {};
        bool adj_6 = {};
        wp::int32 adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        bool adj_12 = {};
        wp::int32 adj_13 = {};
        bool adj_14 = {};
        bool adj_15 = {};
        bool adj_16 = {};
        wp::vec_t<12,wp::float64> adj_17 = {};
        wp::int32 adj_18 = {};
        wp::int32 adj_19 = {};
        wp::int32 adj_20 = {};
        wp::int32 adj_21 = {};
        wp::mat_t<4,4,wp::float64> adj_22 = {};
        wp::mat_t<4,4,wp::float64> adj_23 = {};
        wp::mat_t<4,4,wp::float64> adj_24 = {};
        wp::vec_t<3,wp::float64> adj_25 = {};
        wp::vec_t<3,wp::float64> adj_26 = {};
        wp::vec_t<3,wp::float64> adj_27 = {};
        wp::range_t adj_28 = {};
        wp::int32 adj_29 = {};
        wp::int32 adj_30 = {};
        wp::int32 adj_31 = {};
        wp::range_t adj_32 = {};
        wp::int32 adj_33 = {};
        wp::int32 adj_34 = {};
        wp::float64 adj_35 = {};
        wp::int32 adj_36 = {};
        wp::float64 adj_37 = {};
        wp::float64 adj_38 = {};
        wp::float64 adj_39 = {};
        wp::int32 adj_40 = {};
        wp::float64 adj_41 = {};
        wp::float64 adj_42 = {};
        wp::float64 adj_43 = {};
        wp::int32 adj_44 = {};
        wp::int32 adj_45 = {};
        wp::int32 adj_46 = {};
        wp::vec_t<12,wp::float64> adj_47 = {};
        wp::vec_t<12,wp::float64> adj_48 = {};
        wp::vec_t<12,wp::float64> adj_49 = {};
        wp::vec_t<12,wp::float64> adj_50 = {};
        wp::vec_t<12,wp::float64> adj_51 = {};
        wp::vec_t<12,wp::float64> adj_52 = {};
        wp::int32 adj_53 = {};
        wp::float64 adj_54 = {};
        wp::int32 adj_55 = {};
        wp::float64 adj_56 = {};
        wp::int32 adj_57 = {};
        wp::float64 adj_58 = {};
        wp::vec_t<3,wp::float64> adj_59 = {};
        wp::int32 adj_60 = {};
        wp::float64 adj_61 = {};
        wp::int32 adj_62 = {};
        wp::float64 adj_63 = {};
        wp::int32 adj_64 = {};
        wp::float64 adj_65 = {};
        wp::vec_t<3,wp::float64> adj_66 = {};
        wp::int32 adj_67 = {};
        wp::float64 adj_68 = {};
        wp::int32 adj_69 = {};
        wp::float64 adj_70 = {};
        wp::int32 adj_71 = {};
        wp::float64 adj_72 = {};
        wp::vec_t<3,wp::float64> adj_73 = {};
        wp::int32 adj_74 = {};
        wp::float64 adj_75 = {};
        wp::int32 adj_76 = {};
        wp::float64 adj_77 = {};
        wp::int32 adj_78 = {};
        wp::float64 adj_79 = {};
        wp::vec_t<3,wp::float64> adj_80 = {};
        wp::vec_t<3,wp::float64> adj_81 = {};
        wp::vec_t<3,wp::float64> adj_82 = {};
        wp::vec_t<3,wp::float64> adj_83 = {};
        wp::mat_t<3,3,wp::float64> adj_84 = {};
        wp::int32 adj_85 = {};
        wp::float64 adj_86 = {};
        wp::int32 adj_87 = {};
        wp::float64 adj_88 = {};
        wp::int32 adj_89 = {};
        wp::float64 adj_90 = {};
        wp::vec_t<3,wp::float64> adj_91 = {};
        wp::vec_t<3,wp::float64> adj_92 = {};
        wp::int32 adj_93 = {};
        wp::float64 adj_94 = {};
        wp::int32 adj_95 = {};
        wp::float64 adj_96 = {};
        wp::int32 adj_97 = {};
        wp::float64 adj_98 = {};
        wp::vec_t<3,wp::float64> adj_99 = {};
        wp::vec_t<3,wp::float64> adj_100 = {};
        wp::int32 adj_101 = {};
        wp::float64 adj_102 = {};
        wp::int32 adj_103 = {};
        wp::float64 adj_104 = {};
        wp::int32 adj_105 = {};
        wp::float64 adj_106 = {};
        wp::vec_t<3,wp::float64> adj_107 = {};
        wp::vec_t<3,wp::float64> adj_108 = {};
        wp::int32 adj_109 = {};
        wp::float64 adj_110 = {};
        wp::int32 adj_111 = {};
        wp::float64 adj_112 = {};
        wp::int32 adj_113 = {};
        wp::float64 adj_114 = {};
        wp::vec_t<3,wp::float64> adj_115 = {};
        wp::vec_t<3,wp::float64> adj_116 = {};
        wp::vec_t<12,wp::float64> adj_117 = {};
        wp::int32 adj_118 = {};
        wp::float64 adj_119 = {};
        wp::float64 adj_120 = {};
        wp::float64 adj_121 = {};
        wp::int32 adj_122 = {};
        wp::float64 adj_123 = {};
        wp::float64 adj_124 = {};
        wp::float64 adj_125 = {};
        wp::int32 adj_126 = {};
        wp::float64 adj_127 = {};
        wp::float64 adj_128 = {};
        wp::float64 adj_129 = {};
        wp::int32 adj_130 = {};
        wp::float64 adj_131 = {};
        wp::float64 adj_132 = {};
        wp::float64 adj_133 = {};
        wp::int32 adj_134 = {};
        wp::float64 adj_135 = {};
        wp::float64 adj_136 = {};
        wp::float64 adj_137 = {};
        wp::int32 adj_138 = {};
        wp::float64 adj_139 = {};
        wp::float64 adj_140 = {};
        wp::float64 adj_141 = {};
        wp::int32 adj_142 = {};
        wp::float64 adj_143 = {};
        wp::float64 adj_144 = {};
        wp::float64 adj_145 = {};
        wp::int32 adj_146 = {};
        wp::float64 adj_147 = {};
        wp::float64 adj_148 = {};
        wp::float64 adj_149 = {};
        wp::int32 adj_150 = {};
        wp::float64 adj_151 = {};
        wp::float64 adj_152 = {};
        wp::float64 adj_153 = {};
        wp::int32 adj_154 = {};
        wp::float64 adj_155 = {};
        wp::float64 adj_156 = {};
        wp::float64 adj_157 = {};
        wp::int32 adj_158 = {};
        wp::float64 adj_159 = {};
        wp::float64 adj_160 = {};
        wp::float64 adj_161 = {};
        wp::int32 adj_162 = {};
        wp::float64 adj_163 = {};
        wp::float64 adj_164 = {};
        wp::float64 adj_165 = {};
        wp::vec_t<12,wp::float64> adj_166 = {};
        //---------
        // forward
        // def compute_body_force_energy_grad_affine_y(                                           <L 72>
        // tid = wp.tid()                                                                         <L 84>
        var_0 = builtin_tid1d();
        // if affine_has_constraint[tid] or (                                                     <L 85>
        var_1 = wp::address(var_affine_has_constraint, var_0);
        // (env_states[body_env_id[tid]] == ENV_STATE_INVALID) or (env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED)       <L 86>
        var_2 = wp::address(var_body_env_id, var_0);
        var_4 = wp::load(var_2);
        var_3 = wp::address(var_env_states, var_4);
        var_7 = wp::load(var_3);
        var_6 = (var_7 == var_5);
        var_8 = wp::address(var_body_env_id, var_0);
        var_10 = wp::load(var_8);
        var_9 = wp::address(var_env_states, var_10);
        var_13 = wp::load(var_9);
        var_12 = (var_13 == var_11);
        var_14 = var_6 || var_12;
        var_15 = wp::load(var_1);
        var_16 = var_15 || var_14;
        if (var_16) {
            // return                                                                             <L 88>
            goto label0;
        }
        // local_gradient = vec12d()                                                              <L 90>
        var_17 = wp::vec_t<12,wp::float64>();
        // dim = 3                                                                                <L 91>
        // n_dof = (dim + 1) * dim                                                                <L 92>
        var_20 = wp::add(var_18, var_19);
        var_21 = wp::mul(var_20, var_18);
        // mass = mass_matrix[tid]                                                                <L 93>
        var_22 = wp::address(var_mass_matrix, var_0);
        var_24 = wp::load(var_22);
        var_23 = wp::copy(var_24);
        // ext_force = affine_ext_force[tid]                                                      <L 94>
        var_25 = wp::address(var_affine_ext_force, var_0);
        var_27 = wp::load(var_25);
        var_26 = wp::copy(var_27);
        // for i in range(n_dof):                                                                 <L 95>
        var_28 = wp::range(var_21);
        // rest_y_force = affine_ext_y_force[tid]                                                 <L 98>
        var_47 = wp::address(var_affine_ext_y_force, var_0);
        var_49 = wp::load(var_47);
        var_48 = wp::copy(var_49);
        // hat_yi = hat_y[tid]                                                                    <L 99>
        var_50 = wp::address(var_hat_y, var_0);
        var_52 = wp::load(var_50);
        var_51 = wp::copy(var_52);
        // O = wp.vec3d(hat_yi[9], hat_yi[10], hat_yi[11])                                        <L 100>
        var_54 = wp::extract(var_51, var_53);
        var_56 = wp::extract(var_51, var_55);
        var_58 = wp::extract(var_51, var_57);
        var_59 = wp::vec_t<3,wp::float64>(var_54, var_56, var_58);
        // A = wp.vec3d(hat_yi[0], hat_yi[1], hat_yi[2])                                          <L 101>
        var_61 = wp::extract(var_51, var_60);
        var_63 = wp::extract(var_51, var_62);
        var_65 = wp::extract(var_51, var_64);
        var_66 = wp::vec_t<3,wp::float64>(var_61, var_63, var_65);
        // B = wp.vec3d(hat_yi[3], hat_yi[4], hat_yi[5])                                          <L 102>
        var_68 = wp::extract(var_51, var_67);
        var_70 = wp::extract(var_51, var_69);
        var_72 = wp::extract(var_51, var_71);
        var_73 = wp::vec_t<3,wp::float64>(var_68, var_70, var_72);
        // C = wp.vec3d(hat_yi[6], hat_yi[7], hat_yi[8])                                          <L 103>
        var_75 = wp::extract(var_51, var_74);
        var_77 = wp::extract(var_51, var_76);
        var_79 = wp::extract(var_51, var_78);
        var_80 = wp::vec_t<3,wp::float64>(var_75, var_77, var_79);
        // R = col_stack3(A - O, B - O, C - O)                                                    <L 104>
        var_81 = wp::sub(var_66, var_59);
        var_82 = wp::sub(var_73, var_59);
        var_83 = wp::sub(var_80, var_59);
        var_84 = col_stack3_1(var_81, var_82, var_83);
        // y_force = cat_4_vec3d(                                                                 <L 105>
        // R * wp.vec3d(rest_y_force[0], rest_y_force[1], rest_y_force[2]),                       <L 106>
        var_86 = wp::extract(var_48, var_85);
        var_88 = wp::extract(var_48, var_87);
        var_90 = wp::extract(var_48, var_89);
        var_91 = wp::vec_t<3,wp::float64>(var_86, var_88, var_90);
        var_92 = wp::mul(var_84, var_91);
        // R * wp.vec3d(rest_y_force[3], rest_y_force[4], rest_y_force[5]),                       <L 107>
        var_94 = wp::extract(var_48, var_93);
        var_96 = wp::extract(var_48, var_95);
        var_98 = wp::extract(var_48, var_97);
        var_99 = wp::vec_t<3,wp::float64>(var_94, var_96, var_98);
        var_100 = wp::mul(var_84, var_99);
        // R * wp.vec3d(rest_y_force[6], rest_y_force[7], rest_y_force[8]),                       <L 108>
        var_102 = wp::extract(var_48, var_101);
        var_104 = wp::extract(var_48, var_103);
        var_106 = wp::extract(var_48, var_105);
        var_107 = wp::vec_t<3,wp::float64>(var_102, var_104, var_106);
        var_108 = wp::mul(var_84, var_107);
        // R * wp.vec3d(rest_y_force[9], rest_y_force[10], rest_y_force[11]),                     <L 109>
        var_110 = wp::extract(var_48, var_109);
        var_112 = wp::extract(var_48, var_111);
        var_114 = wp::extract(var_48, var_113);
        var_115 = wp::vec_t<3,wp::float64>(var_110, var_112, var_114);
        var_116 = wp::mul(var_84, var_115);
        var_117 = cat_4_vec3d_0(var_92, var_100, var_108, var_116);
        // for d in range(12):                                                                    <L 111>
        // local_gradient[d] += -y_force[d] * scale                                               <L 112>
        var_119 = wp::extract(var_117, var_118);
        var_120 = wp::neg(var_119);
        var_121 = wp::mul(var_120, var_scale);
        wp::add_inplace(var_17, var_118, var_121);
        var_123 = wp::extract(var_117, var_122);
        var_124 = wp::neg(var_123);
        var_125 = wp::mul(var_124, var_scale);
        wp::add_inplace(var_17, var_122, var_125);
        var_127 = wp::extract(var_117, var_126);
        var_128 = wp::neg(var_127);
        var_129 = wp::mul(var_128, var_scale);
        wp::add_inplace(var_17, var_126, var_129);
        var_131 = wp::extract(var_117, var_130);
        var_132 = wp::neg(var_131);
        var_133 = wp::mul(var_132, var_scale);
        wp::add_inplace(var_17, var_130, var_133);
        var_135 = wp::extract(var_117, var_134);
        var_136 = wp::neg(var_135);
        var_137 = wp::mul(var_136, var_scale);
        wp::add_inplace(var_17, var_134, var_137);
        var_139 = wp::extract(var_117, var_138);
        var_140 = wp::neg(var_139);
        var_141 = wp::mul(var_140, var_scale);
        wp::add_inplace(var_17, var_138, var_141);
        var_143 = wp::extract(var_117, var_142);
        var_144 = wp::neg(var_143);
        var_145 = wp::mul(var_144, var_scale);
        wp::add_inplace(var_17, var_142, var_145);
        var_147 = wp::extract(var_117, var_146);
        var_148 = wp::neg(var_147);
        var_149 = wp::mul(var_148, var_scale);
        wp::add_inplace(var_17, var_146, var_149);
        var_151 = wp::extract(var_117, var_150);
        var_152 = wp::neg(var_151);
        var_153 = wp::mul(var_152, var_scale);
        wp::add_inplace(var_17, var_150, var_153);
        var_155 = wp::extract(var_117, var_154);
        var_156 = wp::neg(var_155);
        var_157 = wp::mul(var_156, var_scale);
        wp::add_inplace(var_17, var_154, var_157);
        var_159 = wp::extract(var_117, var_158);
        var_160 = wp::neg(var_159);
        var_161 = wp::mul(var_160, var_scale);
        wp::add_inplace(var_17, var_158, var_161);
        var_163 = wp::extract(var_117, var_162);
        var_164 = wp::neg(var_163);
        var_165 = wp::mul(var_164, var_scale);
        wp::add_inplace(var_17, var_162, var_165);
        // gradient[tid] += local_gradient                                                        <L 114>
        // var_166 = wp::atomic_add(var_gradient, var_0, var_17);
        //---------
        // reverse
        wp::adj_atomic_add(var_gradient, var_0, var_17, adj_gradient, adj_0, adj_17, adj_166);
        // adj: gradient[tid] += local_gradient                                                   <L 114>
        wp::adj_add_inplace(var_17, var_162, var_165, adj_17, adj_162, adj_165);
        wp::adj_mul(var_164, var_scale, adj_164, adj_scale, adj_165);
        wp::adj_neg(var_163, adj_163, adj_164);
        wp::adj_extract(var_117, var_162, adj_117, adj_162, adj_163);
        wp::adj_add_inplace(var_17, var_158, var_161, adj_17, adj_158, adj_161);
        wp::adj_mul(var_160, var_scale, adj_160, adj_scale, adj_161);
        wp::adj_neg(var_159, adj_159, adj_160);
        wp::adj_extract(var_117, var_158, adj_117, adj_158, adj_159);
        wp::adj_add_inplace(var_17, var_154, var_157, adj_17, adj_154, adj_157);
        wp::adj_mul(var_156, var_scale, adj_156, adj_scale, adj_157);
        wp::adj_neg(var_155, adj_155, adj_156);
        wp::adj_extract(var_117, var_154, adj_117, adj_154, adj_155);
        wp::adj_add_inplace(var_17, var_150, var_153, adj_17, adj_150, adj_153);
        wp::adj_mul(var_152, var_scale, adj_152, adj_scale, adj_153);
        wp::adj_neg(var_151, adj_151, adj_152);
        wp::adj_extract(var_117, var_150, adj_117, adj_150, adj_151);
        wp::adj_add_inplace(var_17, var_146, var_149, adj_17, adj_146, adj_149);
        wp::adj_mul(var_148, var_scale, adj_148, adj_scale, adj_149);
        wp::adj_neg(var_147, adj_147, adj_148);
        wp::adj_extract(var_117, var_146, adj_117, adj_146, adj_147);
        wp::adj_add_inplace(var_17, var_142, var_145, adj_17, adj_142, adj_145);
        wp::adj_mul(var_144, var_scale, adj_144, adj_scale, adj_145);
        wp::adj_neg(var_143, adj_143, adj_144);
        wp::adj_extract(var_117, var_142, adj_117, adj_142, adj_143);
        wp::adj_add_inplace(var_17, var_138, var_141, adj_17, adj_138, adj_141);
        wp::adj_mul(var_140, var_scale, adj_140, adj_scale, adj_141);
        wp::adj_neg(var_139, adj_139, adj_140);
        wp::adj_extract(var_117, var_138, adj_117, adj_138, adj_139);
        wp::adj_add_inplace(var_17, var_134, var_137, adj_17, adj_134, adj_137);
        wp::adj_mul(var_136, var_scale, adj_136, adj_scale, adj_137);
        wp::adj_neg(var_135, adj_135, adj_136);
        wp::adj_extract(var_117, var_134, adj_117, adj_134, adj_135);
        wp::adj_add_inplace(var_17, var_130, var_133, adj_17, adj_130, adj_133);
        wp::adj_mul(var_132, var_scale, adj_132, adj_scale, adj_133);
        wp::adj_neg(var_131, adj_131, adj_132);
        wp::adj_extract(var_117, var_130, adj_117, adj_130, adj_131);
        wp::adj_add_inplace(var_17, var_126, var_129, adj_17, adj_126, adj_129);
        wp::adj_mul(var_128, var_scale, adj_128, adj_scale, adj_129);
        wp::adj_neg(var_127, adj_127, adj_128);
        wp::adj_extract(var_117, var_126, adj_117, adj_126, adj_127);
        wp::adj_add_inplace(var_17, var_122, var_125, adj_17, adj_122, adj_125);
        wp::adj_mul(var_124, var_scale, adj_124, adj_scale, adj_125);
        wp::adj_neg(var_123, adj_123, adj_124);
        wp::adj_extract(var_117, var_122, adj_117, adj_122, adj_123);
        wp::adj_add_inplace(var_17, var_118, var_121, adj_17, adj_118, adj_121);
        wp::adj_mul(var_120, var_scale, adj_120, adj_scale, adj_121);
        wp::adj_neg(var_119, adj_119, adj_120);
        wp::adj_extract(var_117, var_118, adj_117, adj_118, adj_119);
        // adj: local_gradient[d] += -y_force[d] * scale                                          <L 112>
        // adj: for d in range(12):                                                               <L 111>
        adj_cat_4_vec3d_0(var_92, var_100, var_108, var_116, adj_92, adj_100, adj_108, adj_116, adj_117);
        wp::adj_mul(var_84, var_115, adj_84, adj_115, adj_116);
        wp::adj_vec_t(var_110, var_112, var_114, adj_110, adj_112, adj_114, adj_115);
        wp::adj_extract(var_48, var_113, adj_48, adj_113, adj_114);
        wp::adj_extract(var_48, var_111, adj_48, adj_111, adj_112);
        wp::adj_extract(var_48, var_109, adj_48, adj_109, adj_110);
        // adj: R * wp.vec3d(rest_y_force[9], rest_y_force[10], rest_y_force[11]),                <L 109>
        wp::adj_mul(var_84, var_107, adj_84, adj_107, adj_108);
        wp::adj_vec_t(var_102, var_104, var_106, adj_102, adj_104, adj_106, adj_107);
        wp::adj_extract(var_48, var_105, adj_48, adj_105, adj_106);
        wp::adj_extract(var_48, var_103, adj_48, adj_103, adj_104);
        wp::adj_extract(var_48, var_101, adj_48, adj_101, adj_102);
        // adj: R * wp.vec3d(rest_y_force[6], rest_y_force[7], rest_y_force[8]),                  <L 108>
        wp::adj_mul(var_84, var_99, adj_84, adj_99, adj_100);
        wp::adj_vec_t(var_94, var_96, var_98, adj_94, adj_96, adj_98, adj_99);
        wp::adj_extract(var_48, var_97, adj_48, adj_97, adj_98);
        wp::adj_extract(var_48, var_95, adj_48, adj_95, adj_96);
        wp::adj_extract(var_48, var_93, adj_48, adj_93, adj_94);
        // adj: R * wp.vec3d(rest_y_force[3], rest_y_force[4], rest_y_force[5]),                  <L 107>
        wp::adj_mul(var_84, var_91, adj_84, adj_91, adj_92);
        wp::adj_vec_t(var_86, var_88, var_90, adj_86, adj_88, adj_90, adj_91);
        wp::adj_extract(var_48, var_89, adj_48, adj_89, adj_90);
        wp::adj_extract(var_48, var_87, adj_48, adj_87, adj_88);
        wp::adj_extract(var_48, var_85, adj_48, adj_85, adj_86);
        // adj: R * wp.vec3d(rest_y_force[0], rest_y_force[1], rest_y_force[2]),                  <L 106>
        // adj: y_force = cat_4_vec3d(                                                            <L 105>
        adj_col_stack3_1(var_81, var_82, var_83, adj_81, adj_82, adj_83, adj_84);
        wp::adj_sub(var_80, var_59, adj_80, adj_59, adj_83);
        wp::adj_sub(var_73, var_59, adj_73, adj_59, adj_82);
        wp::adj_sub(var_66, var_59, adj_66, adj_59, adj_81);
        // adj: R = col_stack3(A - O, B - O, C - O)                                               <L 104>
        wp::adj_vec_t(var_75, var_77, var_79, adj_75, adj_77, adj_79, adj_80);
        wp::adj_extract(var_51, var_78, adj_51, adj_78, adj_79);
        wp::adj_extract(var_51, var_76, adj_51, adj_76, adj_77);
        wp::adj_extract(var_51, var_74, adj_51, adj_74, adj_75);
        // adj: C = wp.vec3d(hat_yi[6], hat_yi[7], hat_yi[8])                                     <L 103>
        wp::adj_vec_t(var_68, var_70, var_72, adj_68, adj_70, adj_72, adj_73);
        wp::adj_extract(var_51, var_71, adj_51, adj_71, adj_72);
        wp::adj_extract(var_51, var_69, adj_51, adj_69, adj_70);
        wp::adj_extract(var_51, var_67, adj_51, adj_67, adj_68);
        // adj: B = wp.vec3d(hat_yi[3], hat_yi[4], hat_yi[5])                                     <L 102>
        wp::adj_vec_t(var_61, var_63, var_65, adj_61, adj_63, adj_65, adj_66);
        wp::adj_extract(var_51, var_64, adj_51, adj_64, adj_65);
        wp::adj_extract(var_51, var_62, adj_51, adj_62, adj_63);
        wp::adj_extract(var_51, var_60, adj_51, adj_60, adj_61);
        // adj: A = wp.vec3d(hat_yi[0], hat_yi[1], hat_yi[2])                                     <L 101>
        wp::adj_vec_t(var_54, var_56, var_58, adj_54, adj_56, adj_58, adj_59);
        wp::adj_extract(var_51, var_57, adj_51, adj_57, adj_58);
        wp::adj_extract(var_51, var_55, adj_51, adj_55, adj_56);
        wp::adj_extract(var_51, var_53, adj_51, adj_53, adj_54);
        // adj: O = wp.vec3d(hat_yi[9], hat_yi[10], hat_yi[11])                                   <L 100>
        wp::adj_copy(var_52, adj_50, adj_51);
        wp::adj_load(var_50, adj_50, adj_52);
        wp::adj_address(var_hat_y, var_0, adj_hat_y, adj_0, adj_50);
        // adj: hat_yi = hat_y[tid]                                                               <L 99>
        wp::adj_copy(var_49, adj_47, adj_48);
        wp::adj_load(var_47, adj_47, adj_49);
        wp::adj_address(var_affine_ext_y_force, var_0, adj_affine_ext_y_force, adj_0, adj_47);
        // adj: rest_y_force = affine_ext_y_force[tid]                                            <L 98>
        var_28 = wp::iter_reverse(var_28);
        start_for_1:;
            if (iter_cmp(var_28) == 0) goto end_for_1;
            var_29 = wp::iter_next(var_28);
        	adj_30 = {};
        	adj_31 = {};
        	adj_32 = {};
            // for j in range(dim + 1):                                                           <L 96>
            var_31 = wp::add(var_18, var_30);
            var_32 = wp::range(var_31);
            var_32 = wp::iter_reverse(var_32);
            start_for_3:;
                if (iter_cmp(var_32) == 0) goto end_for_3;
                var_33 = wp::iter_next(var_32);
            	adj_34 = {};
            	adj_35 = {};
            	adj_36 = {};
            	adj_37 = {};
            	adj_38 = {};
            	adj_39 = {};
            	adj_40 = {};
            	adj_41 = {};
            	adj_42 = {};
            	adj_43 = {};
            	adj_44 = {};
            	adj_45 = {};
            	adj_46 = {};
                // local_gradient[j * dim + i % dim] += -(gravity[i % dim] + ext_force[i % dim]) * mass[(i / dim), j] * scale       <L 97>
                var_34 = wp::mod(var_29, var_18);
                var_35 = wp::extract(var_gravity, var_34);
                var_36 = wp::mod(var_29, var_18);
                var_37 = wp::extract(var_26, var_36);
                var_38 = wp::add(var_35, var_37);
                var_39 = wp::neg(var_38);
                var_40 = wp::div(var_29, var_18);
                var_41 = wp::extract(var_23, var_40, var_33);
                var_42 = wp::mul(var_39, var_41);
                var_43 = wp::mul(var_42, var_scale);
                var_44 = wp::mul(var_33, var_18);
                var_45 = wp::mod(var_29, var_18);
                var_46 = wp::add(var_44, var_45);
                wp::add_inplace(var_17, var_46, var_43);
                wp::adj_add_inplace(var_17, var_46, var_43, adj_17, adj_46, adj_43);
                wp::adj_add(var_44, var_45, adj_44, adj_45, adj_46);
                wp::adj_mod(var_29, var_18, adj_29, adj_18, adj_45);
                wp::adj_mul(var_33, var_18, adj_33, adj_18, adj_44);
                wp::adj_mul(var_42, var_scale, adj_42, adj_scale, adj_43);
                wp::adj_mul(var_39, var_41, adj_39, adj_41, adj_42);
                wp::adj_extract(var_23, var_40, var_33, adj_23, adj_40, adj_33, adj_41);
                wp::adj_div(var_29, var_18, var_40, adj_29, adj_18, adj_40);
                wp::adj_neg(var_38, adj_38, adj_39);
                wp::adj_add(var_35, var_37, adj_35, adj_37, adj_38);
                wp::adj_extract(var_26, var_36, adj_26, adj_36, adj_37);
                wp::adj_mod(var_29, var_18, adj_29, adj_18, adj_36);
                wp::adj_extract(var_gravity, var_34, adj_gravity, adj_34, adj_35);
                wp::adj_mod(var_29, var_18, adj_29, adj_18, adj_34);
                // adj: local_gradient[j * dim + i % dim] += -(gravity[i % dim] + ext_force[i % dim]) * mass[(i / dim), j] * scale  <L 97>
            	goto start_for_3;
            end_for_3:;
            wp::adj_range(var_31, adj_31, adj_32);
            wp::adj_add(var_18, var_30, adj_18, adj_30, adj_31);
            // adj: for j in range(dim + 1):                                                      <L 96>
        	goto start_for_1;
        end_for_1:;
        wp::adj_range(var_21, adj_21, adj_28);
        // adj: for i in range(n_dof):                                                            <L 95>
        wp::adj_copy(var_27, adj_25, adj_26);
        wp::adj_load(var_25, adj_25, adj_27);
        wp::adj_address(var_affine_ext_force, var_0, adj_affine_ext_force, adj_0, adj_25);
        // adj: ext_force = affine_ext_force[tid]                                                 <L 94>
        wp::adj_copy(var_24, adj_22, adj_23);
        wp::adj_load(var_22, adj_22, adj_24);
        wp::adj_address(var_mass_matrix, var_0, adj_mass_matrix, adj_0, adj_22);
        // adj: mass = mass_matrix[tid]                                                           <L 93>
        wp::adj_mul(var_20, var_18, adj_20, adj_18, adj_21);
        wp::adj_add(var_18, var_19, adj_18, adj_19, adj_20);
        // adj: n_dof = (dim + 1) * dim                                                           <L 92>
        // adj: dim = 3                                                                           <L 91>
        // adj: local_gradient = vec12d()                                                         <L 90>
        if (var_16) {
            label0:;
            // adj: return                                                                        <L 88>
        }
        wp::adj_load(var_1, adj_1, adj_15);
        wp::adj_load(var_9, adj_9, adj_13);
        wp::adj_address(var_env_states, var_10, adj_env_states, adj_8, adj_9);
        wp::adj_load(var_8, adj_8, adj_10);
        wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_8);
        wp::adj_load(var_3, adj_3, adj_7);
        wp::adj_address(var_env_states, var_4, adj_env_states, adj_2, adj_3);
        wp::adj_load(var_2, adj_2, adj_4);
        wp::adj_address(var_body_env_id, var_0, adj_body_env_id, adj_0, adj_2);
        // adj: (env_states[body_env_id[tid]] == ENV_STATE_INVALID) or (env_states[body_env_id[tid]] == ENV_STATE_NEWTON_SOLVED)  <L 86>
        wp::adj_address(var_affine_has_constraint, var_0, adj_affine_has_constraint, adj_0, adj_1);
        // adj: if affine_has_constraint[tid] or (                                                <L 85>
        // adj: tid = wp.tid()                                                                    <L 84>
        // adj: def compute_body_force_energy_grad_affine_y(                                      <L 72>
        continue;
    }
}



extern "C" __global__ void compute_body_force_energy_val_affine_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_hat_y,
    wp::array_t<wp::mat_t<4,4,wp::float64>> var_mass_matrix,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_ext_force,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_ext_y_force,
    wp::float64 var_scale,
    wp::array_t<bool> var_affine_has_constraint)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        bool var_2;
        bool var_3;
        const wp::int32 var_4 = 3;
        const wp::int32 var_5 = 1;
        wp::int32 var_6;
        wp::int32 var_7;
        wp::mat_t<4,4,wp::float64>* var_8;
        wp::mat_t<4,4,wp::float64> var_9;
        wp::mat_t<4,4,wp::float64> var_10;
        wp::vec_t<3,wp::float64>* var_11;
        wp::vec_t<3,wp::float64> var_12;
        wp::vec_t<3,wp::float64> var_13;
        wp::range_t var_14;
        wp::int32 var_15;
        const wp::int32 var_16 = 1;
        wp::int32 var_17;
        wp::range_t var_18;
        wp::int32 var_19;
        wp::int32 var_20;
        wp::float64 var_21;
        wp::int32 var_22;
        wp::float64 var_23;
        wp::float64 var_24;
        wp::float64 var_25;
        wp::vec_t<12,wp::float64>* var_26;
        wp::float64 var_27;
        wp::vec_t<12,wp::float64> var_28;
        wp::float64 var_29;
        wp::int32 var_30;
        wp::float64 var_31;
        wp::float64 var_32;
        wp::float64 var_33;
        wp::float64 var_34;
        wp::vec_t<12,wp::float64>* var_35;
        wp::vec_t<12,wp::float64> var_36;
        wp::vec_t<12,wp::float64> var_37;
        wp::vec_t<12,wp::float64>* var_38;
        wp::vec_t<12,wp::float64> var_39;
        wp::vec_t<12,wp::float64> var_40;
        wp::vec_t<12,wp::float64>* var_41;
        wp::vec_t<12,wp::float64> var_42;
        wp::vec_t<12,wp::float64> var_43;
        const wp::int32 var_44 = 9;
        wp::float64 var_45;
        const wp::int32 var_46 = 10;
        wp::float64 var_47;
        const wp::int32 var_48 = 11;
        wp::float64 var_49;
        wp::vec_t<3,wp::float64> var_50;
        const wp::int32 var_51 = 0;
        wp::float64 var_52;
        const wp::int32 var_53 = 1;
        wp::float64 var_54;
        const wp::int32 var_55 = 2;
        wp::float64 var_56;
        wp::vec_t<3,wp::float64> var_57;
        const wp::int32 var_58 = 3;
        wp::float64 var_59;
        const wp::int32 var_60 = 4;
        wp::float64 var_61;
        const wp::int32 var_62 = 5;
        wp::float64 var_63;
        wp::vec_t<3,wp::float64> var_64;
        const wp::int32 var_65 = 6;
        wp::float64 var_66;
        const wp::int32 var_67 = 7;
        wp::float64 var_68;
        const wp::int32 var_69 = 8;
        wp::float64 var_70;
        wp::vec_t<3,wp::float64> var_71;
        wp::vec_t<3,wp::float64> var_72;
        wp::vec_t<3,wp::float64> var_73;
        wp::vec_t<3,wp::float64> var_74;
        wp::mat_t<3,3,wp::float64> var_75;
        const wp::int32 var_76 = 0;
        wp::float64 var_77;
        const wp::int32 var_78 = 1;
        wp::float64 var_79;
        const wp::int32 var_80 = 2;
        wp::float64 var_81;
        wp::vec_t<3,wp::float64> var_82;
        wp::vec_t<3,wp::float64> var_83;
        const wp::int32 var_84 = 3;
        wp::float64 var_85;
        const wp::int32 var_86 = 4;
        wp::float64 var_87;
        const wp::int32 var_88 = 5;
        wp::float64 var_89;
        wp::vec_t<3,wp::float64> var_90;
        wp::vec_t<3,wp::float64> var_91;
        const wp::int32 var_92 = 6;
        wp::float64 var_93;
        const wp::int32 var_94 = 7;
        wp::float64 var_95;
        const wp::int32 var_96 = 8;
        wp::float64 var_97;
        wp::vec_t<3,wp::float64> var_98;
        wp::vec_t<3,wp::float64> var_99;
        const wp::int32 var_100 = 9;
        wp::float64 var_101;
        const wp::int32 var_102 = 10;
        wp::float64 var_103;
        const wp::int32 var_104 = 11;
        wp::float64 var_105;
        wp::vec_t<3,wp::float64> var_106;
        wp::vec_t<3,wp::float64> var_107;
        wp::vec_t<12,wp::float64> var_108;
        const wp::int32 var_109 = 0;
        wp::float64 var_110;
        wp::float64 var_111;
        wp::float64 var_112;
        wp::float64 var_113;
        wp::float64 var_114;
        wp::float64 var_115;
        const wp::int32 var_116 = 1;
        wp::float64 var_117;
        wp::float64 var_118;
        wp::float64 var_119;
        wp::float64 var_120;
        wp::float64 var_121;
        wp::float64 var_122;
        const wp::int32 var_123 = 2;
        wp::float64 var_124;
        wp::float64 var_125;
        wp::float64 var_126;
        wp::float64 var_127;
        wp::float64 var_128;
        wp::float64 var_129;
        const wp::int32 var_130 = 3;
        wp::float64 var_131;
        wp::float64 var_132;
        wp::float64 var_133;
        wp::float64 var_134;
        wp::float64 var_135;
        wp::float64 var_136;
        const wp::int32 var_137 = 4;
        wp::float64 var_138;
        wp::float64 var_139;
        wp::float64 var_140;
        wp::float64 var_141;
        wp::float64 var_142;
        wp::float64 var_143;
        const wp::int32 var_144 = 5;
        wp::float64 var_145;
        wp::float64 var_146;
        wp::float64 var_147;
        wp::float64 var_148;
        wp::float64 var_149;
        wp::float64 var_150;
        const wp::int32 var_151 = 6;
        wp::float64 var_152;
        wp::float64 var_153;
        wp::float64 var_154;
        wp::float64 var_155;
        wp::float64 var_156;
        wp::float64 var_157;
        const wp::int32 var_158 = 7;
        wp::float64 var_159;
        wp::float64 var_160;
        wp::float64 var_161;
        wp::float64 var_162;
        wp::float64 var_163;
        wp::float64 var_164;
        const wp::int32 var_165 = 8;
        wp::float64 var_166;
        wp::float64 var_167;
        wp::float64 var_168;
        wp::float64 var_169;
        wp::float64 var_170;
        wp::float64 var_171;
        const wp::int32 var_172 = 9;
        wp::float64 var_173;
        wp::float64 var_174;
        wp::float64 var_175;
        wp::float64 var_176;
        wp::float64 var_177;
        wp::float64 var_178;
        const wp::int32 var_179 = 10;
        wp::float64 var_180;
        wp::float64 var_181;
        wp::float64 var_182;
        wp::float64 var_183;
        wp::float64 var_184;
        wp::float64 var_185;
        const wp::int32 var_186 = 11;
        wp::float64 var_187;
        wp::float64 var_188;
        wp::float64 var_189;
        wp::float64 var_190;
        wp::float64 var_191;
        wp::float64 var_192;
        //---------
        // forward
        // def compute_body_force_energy_val_affine(                                              <L 16>
        // tid = wp.tid()                                                                         <L 27>
        var_0 = builtin_tid1d();
        // if affine_has_constraint[tid]:                                                         <L 28>
        var_1 = wp::address(var_affine_has_constraint, var_0);
        var_2 = wp::load(var_1);
        if (var_2) {
            // return                                                                             <L 29>
            continue;
        }
        var_3 = wp::load(var_1);
        // dim = 3                                                                                <L 30>
        // n_dof = (dim + 1) * dim                                                                <L 31>
        var_6 = wp::add(var_4, var_5);
        var_7 = wp::mul(var_6, var_4);
        // mass = mass_matrix[tid]                                                                <L 32>
        var_8 = wp::address(var_mass_matrix, var_0);
        var_10 = wp::load(var_8);
        var_9 = wp::copy(var_10);
        // ext_force = affine_ext_force[tid]                                                      <L 33>
        var_11 = wp::address(var_affine_ext_force, var_0);
        var_13 = wp::load(var_11);
        var_12 = wp::copy(var_13);
        // for i in range(n_dof):                                                                 <L 34>
        var_14 = wp::range(var_7);
        start_for_1:;
            if (iter_cmp(var_14) == 0) goto end_for_1;
            var_15 = wp::iter_next(var_14);
            // for j in range(dim + 1):                                                           <L 35>
            var_17 = wp::add(var_4, var_16);
            var_18 = wp::range(var_17);
            start_for_3:;
                if (iter_cmp(var_18) == 0) goto end_for_3;
                var_19 = wp::iter_next(var_18);
                // energy[tid] += -(gravity[i % dim] + ext_force[i % dim]) * y[tid][i] * mass[(i / dim), j] * scale       <L 36>
                var_20 = wp::mod(var_15, var_4);
                var_21 = wp::extract(var_gravity, var_20);
                var_22 = wp::mod(var_15, var_4);
                var_23 = wp::extract(var_12, var_22);
                var_24 = wp::add(var_21, var_23);
                var_25 = wp::neg(var_24);
                var_26 = wp::address(var_y, var_0);
                var_28 = wp::load(var_26);
                var_27 = wp::extract(var_28, var_15);
                var_29 = wp::mul(var_25, var_27);
                var_30 = wp::div(var_15, var_4);
                var_31 = wp::extract(var_9, var_30, var_19);
                var_32 = wp::mul(var_29, var_31);
                var_33 = wp::mul(var_32, var_scale);
                var_34 = wp::atomic_add(var_energy, var_0, var_33);
                goto start_for_3;
            end_for_3:;
            goto start_for_1;
        end_for_1:;
        // rest_y_force = affine_ext_y_force[tid]                                                 <L 37>
        var_35 = wp::address(var_affine_ext_y_force, var_0);
        var_37 = wp::load(var_35);
        var_36 = wp::copy(var_37);
        // yi = y[tid]                                                                            <L 38>
        var_38 = wp::address(var_y, var_0);
        var_40 = wp::load(var_38);
        var_39 = wp::copy(var_40);
        // hat_yi = hat_y[tid]                                                                    <L 39>
        var_41 = wp::address(var_hat_y, var_0);
        var_43 = wp::load(var_41);
        var_42 = wp::copy(var_43);
        // O = wp.vec3d(hat_yi[9], hat_yi[10], hat_yi[11])                                        <L 40>
        var_45 = wp::extract(var_42, var_44);
        var_47 = wp::extract(var_42, var_46);
        var_49 = wp::extract(var_42, var_48);
        var_50 = wp::vec_t<3,wp::float64>(var_45, var_47, var_49);
        // A = wp.vec3d(hat_yi[0], hat_yi[1], hat_yi[2])                                          <L 41>
        var_52 = wp::extract(var_42, var_51);
        var_54 = wp::extract(var_42, var_53);
        var_56 = wp::extract(var_42, var_55);
        var_57 = wp::vec_t<3,wp::float64>(var_52, var_54, var_56);
        // B = wp.vec3d(hat_yi[3], hat_yi[4], hat_yi[5])                                          <L 42>
        var_59 = wp::extract(var_42, var_58);
        var_61 = wp::extract(var_42, var_60);
        var_63 = wp::extract(var_42, var_62);
        var_64 = wp::vec_t<3,wp::float64>(var_59, var_61, var_63);
        // C = wp.vec3d(hat_yi[6], hat_yi[7], hat_yi[8])                                          <L 43>
        var_66 = wp::extract(var_42, var_65);
        var_68 = wp::extract(var_42, var_67);
        var_70 = wp::extract(var_42, var_69);
        var_71 = wp::vec_t<3,wp::float64>(var_66, var_68, var_70);
        // R = col_stack3(A - O, B - O, C - O)                                                    <L 44>
        var_72 = wp::sub(var_57, var_50);
        var_73 = wp::sub(var_64, var_50);
        var_74 = wp::sub(var_71, var_50);
        var_75 = col_stack3_1(var_72, var_73, var_74);
        // y_force = cat_4_vec3d(                                                                 <L 45>
        // R * wp.vec3d(rest_y_force[0], rest_y_force[1], rest_y_force[2]),                       <L 46>
        var_77 = wp::extract(var_36, var_76);
        var_79 = wp::extract(var_36, var_78);
        var_81 = wp::extract(var_36, var_80);
        var_82 = wp::vec_t<3,wp::float64>(var_77, var_79, var_81);
        var_83 = wp::mul(var_75, var_82);
        // R * wp.vec3d(rest_y_force[3], rest_y_force[4], rest_y_force[5]),                       <L 47>
        var_85 = wp::extract(var_36, var_84);
        var_87 = wp::extract(var_36, var_86);
        var_89 = wp::extract(var_36, var_88);
        var_90 = wp::vec_t<3,wp::float64>(var_85, var_87, var_89);
        var_91 = wp::mul(var_75, var_90);
        // R * wp.vec3d(rest_y_force[6], rest_y_force[7], rest_y_force[8]),                       <L 48>
        var_93 = wp::extract(var_36, var_92);
        var_95 = wp::extract(var_36, var_94);
        var_97 = wp::extract(var_36, var_96);
        var_98 = wp::vec_t<3,wp::float64>(var_93, var_95, var_97);
        var_99 = wp::mul(var_75, var_98);
        // R * wp.vec3d(rest_y_force[9], rest_y_force[10], rest_y_force[11]),                     <L 49>
        var_101 = wp::extract(var_36, var_100);
        var_103 = wp::extract(var_36, var_102);
        var_105 = wp::extract(var_36, var_104);
        var_106 = wp::vec_t<3,wp::float64>(var_101, var_103, var_105);
        var_107 = wp::mul(var_75, var_106);
        var_108 = cat_4_vec3d_0(var_83, var_91, var_99, var_107);
        // for d in range(12):                                                                    <L 51>
        // energy[tid] += -y_force[d] * yi[d] * scale                                             <L 52>
        var_110 = wp::extract(var_108, var_109);
        var_111 = wp::neg(var_110);
        var_112 = wp::extract(var_39, var_109);
        var_113 = wp::mul(var_111, var_112);
        var_114 = wp::mul(var_113, var_scale);
        var_115 = wp::atomic_add(var_energy, var_0, var_114);
        var_117 = wp::extract(var_108, var_116);
        var_118 = wp::neg(var_117);
        var_119 = wp::extract(var_39, var_116);
        var_120 = wp::mul(var_118, var_119);
        var_121 = wp::mul(var_120, var_scale);
        var_122 = wp::atomic_add(var_energy, var_0, var_121);
        var_124 = wp::extract(var_108, var_123);
        var_125 = wp::neg(var_124);
        var_126 = wp::extract(var_39, var_123);
        var_127 = wp::mul(var_125, var_126);
        var_128 = wp::mul(var_127, var_scale);
        var_129 = wp::atomic_add(var_energy, var_0, var_128);
        var_131 = wp::extract(var_108, var_130);
        var_132 = wp::neg(var_131);
        var_133 = wp::extract(var_39, var_130);
        var_134 = wp::mul(var_132, var_133);
        var_135 = wp::mul(var_134, var_scale);
        var_136 = wp::atomic_add(var_energy, var_0, var_135);
        var_138 = wp::extract(var_108, var_137);
        var_139 = wp::neg(var_138);
        var_140 = wp::extract(var_39, var_137);
        var_141 = wp::mul(var_139, var_140);
        var_142 = wp::mul(var_141, var_scale);
        var_143 = wp::atomic_add(var_energy, var_0, var_142);
        var_145 = wp::extract(var_108, var_144);
        var_146 = wp::neg(var_145);
        var_147 = wp::extract(var_39, var_144);
        var_148 = wp::mul(var_146, var_147);
        var_149 = wp::mul(var_148, var_scale);
        var_150 = wp::atomic_add(var_energy, var_0, var_149);
        var_152 = wp::extract(var_108, var_151);
        var_153 = wp::neg(var_152);
        var_154 = wp::extract(var_39, var_151);
        var_155 = wp::mul(var_153, var_154);
        var_156 = wp::mul(var_155, var_scale);
        var_157 = wp::atomic_add(var_energy, var_0, var_156);
        var_159 = wp::extract(var_108, var_158);
        var_160 = wp::neg(var_159);
        var_161 = wp::extract(var_39, var_158);
        var_162 = wp::mul(var_160, var_161);
        var_163 = wp::mul(var_162, var_scale);
        var_164 = wp::atomic_add(var_energy, var_0, var_163);
        var_166 = wp::extract(var_108, var_165);
        var_167 = wp::neg(var_166);
        var_168 = wp::extract(var_39, var_165);
        var_169 = wp::mul(var_167, var_168);
        var_170 = wp::mul(var_169, var_scale);
        var_171 = wp::atomic_add(var_energy, var_0, var_170);
        var_173 = wp::extract(var_108, var_172);
        var_174 = wp::neg(var_173);
        var_175 = wp::extract(var_39, var_172);
        var_176 = wp::mul(var_174, var_175);
        var_177 = wp::mul(var_176, var_scale);
        var_178 = wp::atomic_add(var_energy, var_0, var_177);
        var_180 = wp::extract(var_108, var_179);
        var_181 = wp::neg(var_180);
        var_182 = wp::extract(var_39, var_179);
        var_183 = wp::mul(var_181, var_182);
        var_184 = wp::mul(var_183, var_scale);
        var_185 = wp::atomic_add(var_energy, var_0, var_184);
        var_187 = wp::extract(var_108, var_186);
        var_188 = wp::neg(var_187);
        var_189 = wp::extract(var_39, var_186);
        var_190 = wp::mul(var_188, var_189);
        var_191 = wp::mul(var_190, var_scale);
        var_192 = wp::atomic_add(var_energy, var_0, var_191);
    }
}



extern "C" __global__ void compute_body_force_energy_val_affine_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy,
    wp::array_t<wp::vec_t<12,wp::float64>> var_y,
    wp::array_t<wp::vec_t<12,wp::float64>> var_hat_y,
    wp::array_t<wp::mat_t<4,4,wp::float64>> var_mass_matrix,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::array_t<wp::vec_t<3,wp::float64>> var_affine_ext_force,
    wp::array_t<wp::vec_t<12,wp::float64>> var_affine_ext_y_force,
    wp::float64 var_scale,
    wp::array_t<bool> var_affine_has_constraint,
    wp::array_t<wp::float64> adj_energy,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_y,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_hat_y,
    wp::array_t<wp::mat_t<4,4,wp::float64>> adj_mass_matrix,
    wp::vec_t<3,wp::float64> adj_gravity,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_affine_ext_force,
    wp::array_t<wp::vec_t<12,wp::float64>> adj_affine_ext_y_force,
    wp::float64 adj_scale,
    wp::array_t<bool> adj_affine_has_constraint)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        bool var_2;
        bool var_3;
        const wp::int32 var_4 = 3;
        const wp::int32 var_5 = 1;
        wp::int32 var_6;
        wp::int32 var_7;
        wp::mat_t<4,4,wp::float64>* var_8;
        wp::mat_t<4,4,wp::float64> var_9;
        wp::mat_t<4,4,wp::float64> var_10;
        wp::vec_t<3,wp::float64>* var_11;
        wp::vec_t<3,wp::float64> var_12;
        wp::vec_t<3,wp::float64> var_13;
        wp::range_t var_14;
        wp::int32 var_15;
        const wp::int32 var_16 = 1;
        wp::int32 var_17;
        wp::range_t var_18;
        wp::int32 var_19;
        wp::int32 var_20;
        wp::float64 var_21;
        wp::int32 var_22;
        wp::float64 var_23;
        wp::float64 var_24;
        wp::float64 var_25;
        wp::vec_t<12,wp::float64>* var_26;
        wp::float64 var_27;
        wp::vec_t<12,wp::float64> var_28;
        wp::float64 var_29;
        wp::int32 var_30;
        wp::float64 var_31;
        wp::float64 var_32;
        wp::float64 var_33;
        wp::float64 var_34;
        wp::vec_t<12,wp::float64>* var_35;
        wp::vec_t<12,wp::float64> var_36;
        wp::vec_t<12,wp::float64> var_37;
        wp::vec_t<12,wp::float64>* var_38;
        wp::vec_t<12,wp::float64> var_39;
        wp::vec_t<12,wp::float64> var_40;
        wp::vec_t<12,wp::float64>* var_41;
        wp::vec_t<12,wp::float64> var_42;
        wp::vec_t<12,wp::float64> var_43;
        const wp::int32 var_44 = 9;
        wp::float64 var_45;
        const wp::int32 var_46 = 10;
        wp::float64 var_47;
        const wp::int32 var_48 = 11;
        wp::float64 var_49;
        wp::vec_t<3,wp::float64> var_50;
        const wp::int32 var_51 = 0;
        wp::float64 var_52;
        const wp::int32 var_53 = 1;
        wp::float64 var_54;
        const wp::int32 var_55 = 2;
        wp::float64 var_56;
        wp::vec_t<3,wp::float64> var_57;
        const wp::int32 var_58 = 3;
        wp::float64 var_59;
        const wp::int32 var_60 = 4;
        wp::float64 var_61;
        const wp::int32 var_62 = 5;
        wp::float64 var_63;
        wp::vec_t<3,wp::float64> var_64;
        const wp::int32 var_65 = 6;
        wp::float64 var_66;
        const wp::int32 var_67 = 7;
        wp::float64 var_68;
        const wp::int32 var_69 = 8;
        wp::float64 var_70;
        wp::vec_t<3,wp::float64> var_71;
        wp::vec_t<3,wp::float64> var_72;
        wp::vec_t<3,wp::float64> var_73;
        wp::vec_t<3,wp::float64> var_74;
        wp::mat_t<3,3,wp::float64> var_75;
        const wp::int32 var_76 = 0;
        wp::float64 var_77;
        const wp::int32 var_78 = 1;
        wp::float64 var_79;
        const wp::int32 var_80 = 2;
        wp::float64 var_81;
        wp::vec_t<3,wp::float64> var_82;
        wp::vec_t<3,wp::float64> var_83;
        const wp::int32 var_84 = 3;
        wp::float64 var_85;
        const wp::int32 var_86 = 4;
        wp::float64 var_87;
        const wp::int32 var_88 = 5;
        wp::float64 var_89;
        wp::vec_t<3,wp::float64> var_90;
        wp::vec_t<3,wp::float64> var_91;
        const wp::int32 var_92 = 6;
        wp::float64 var_93;
        const wp::int32 var_94 = 7;
        wp::float64 var_95;
        const wp::int32 var_96 = 8;
        wp::float64 var_97;
        wp::vec_t<3,wp::float64> var_98;
        wp::vec_t<3,wp::float64> var_99;
        const wp::int32 var_100 = 9;
        wp::float64 var_101;
        const wp::int32 var_102 = 10;
        wp::float64 var_103;
        const wp::int32 var_104 = 11;
        wp::float64 var_105;
        wp::vec_t<3,wp::float64> var_106;
        wp::vec_t<3,wp::float64> var_107;
        wp::vec_t<12,wp::float64> var_108;
        const wp::int32 var_109 = 0;
        wp::float64 var_110;
        wp::float64 var_111;
        wp::float64 var_112;
        wp::float64 var_113;
        wp::float64 var_114;
        wp::float64 var_115;
        const wp::int32 var_116 = 1;
        wp::float64 var_117;
        wp::float64 var_118;
        wp::float64 var_119;
        wp::float64 var_120;
        wp::float64 var_121;
        wp::float64 var_122;
        const wp::int32 var_123 = 2;
        wp::float64 var_124;
        wp::float64 var_125;
        wp::float64 var_126;
        wp::float64 var_127;
        wp::float64 var_128;
        wp::float64 var_129;
        const wp::int32 var_130 = 3;
        wp::float64 var_131;
        wp::float64 var_132;
        wp::float64 var_133;
        wp::float64 var_134;
        wp::float64 var_135;
        wp::float64 var_136;
        const wp::int32 var_137 = 4;
        wp::float64 var_138;
        wp::float64 var_139;
        wp::float64 var_140;
        wp::float64 var_141;
        wp::float64 var_142;
        wp::float64 var_143;
        const wp::int32 var_144 = 5;
        wp::float64 var_145;
        wp::float64 var_146;
        wp::float64 var_147;
        wp::float64 var_148;
        wp::float64 var_149;
        wp::float64 var_150;
        const wp::int32 var_151 = 6;
        wp::float64 var_152;
        wp::float64 var_153;
        wp::float64 var_154;
        wp::float64 var_155;
        wp::float64 var_156;
        wp::float64 var_157;
        const wp::int32 var_158 = 7;
        wp::float64 var_159;
        wp::float64 var_160;
        wp::float64 var_161;
        wp::float64 var_162;
        wp::float64 var_163;
        wp::float64 var_164;
        const wp::int32 var_165 = 8;
        wp::float64 var_166;
        wp::float64 var_167;
        wp::float64 var_168;
        wp::float64 var_169;
        wp::float64 var_170;
        wp::float64 var_171;
        const wp::int32 var_172 = 9;
        wp::float64 var_173;
        wp::float64 var_174;
        wp::float64 var_175;
        wp::float64 var_176;
        wp::float64 var_177;
        wp::float64 var_178;
        const wp::int32 var_179 = 10;
        wp::float64 var_180;
        wp::float64 var_181;
        wp::float64 var_182;
        wp::float64 var_183;
        wp::float64 var_184;
        wp::float64 var_185;
        const wp::int32 var_186 = 11;
        wp::float64 var_187;
        wp::float64 var_188;
        wp::float64 var_189;
        wp::float64 var_190;
        wp::float64 var_191;
        wp::float64 var_192;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        bool adj_1 = {};
        bool adj_2 = {};
        bool adj_3 = {};
        wp::int32 adj_4 = {};
        wp::int32 adj_5 = {};
        wp::int32 adj_6 = {};
        wp::int32 adj_7 = {};
        wp::mat_t<4,4,wp::float64> adj_8 = {};
        wp::mat_t<4,4,wp::float64> adj_9 = {};
        wp::mat_t<4,4,wp::float64> adj_10 = {};
        wp::vec_t<3,wp::float64> adj_11 = {};
        wp::vec_t<3,wp::float64> adj_12 = {};
        wp::vec_t<3,wp::float64> adj_13 = {};
        wp::range_t adj_14 = {};
        wp::int32 adj_15 = {};
        wp::int32 adj_16 = {};
        wp::int32 adj_17 = {};
        wp::range_t adj_18 = {};
        wp::int32 adj_19 = {};
        wp::int32 adj_20 = {};
        wp::float64 adj_21 = {};
        wp::int32 adj_22 = {};
        wp::float64 adj_23 = {};
        wp::float64 adj_24 = {};
        wp::float64 adj_25 = {};
        wp::vec_t<12,wp::float64> adj_26 = {};
        wp::float64 adj_27 = {};
        wp::vec_t<12,wp::float64> adj_28 = {};
        wp::float64 adj_29 = {};
        wp::int32 adj_30 = {};
        wp::float64 adj_31 = {};
        wp::float64 adj_32 = {};
        wp::float64 adj_33 = {};
        wp::float64 adj_34 = {};
        wp::vec_t<12,wp::float64> adj_35 = {};
        wp::vec_t<12,wp::float64> adj_36 = {};
        wp::vec_t<12,wp::float64> adj_37 = {};
        wp::vec_t<12,wp::float64> adj_38 = {};
        wp::vec_t<12,wp::float64> adj_39 = {};
        wp::vec_t<12,wp::float64> adj_40 = {};
        wp::vec_t<12,wp::float64> adj_41 = {};
        wp::vec_t<12,wp::float64> adj_42 = {};
        wp::vec_t<12,wp::float64> adj_43 = {};
        wp::int32 adj_44 = {};
        wp::float64 adj_45 = {};
        wp::int32 adj_46 = {};
        wp::float64 adj_47 = {};
        wp::int32 adj_48 = {};
        wp::float64 adj_49 = {};
        wp::vec_t<3,wp::float64> adj_50 = {};
        wp::int32 adj_51 = {};
        wp::float64 adj_52 = {};
        wp::int32 adj_53 = {};
        wp::float64 adj_54 = {};
        wp::int32 adj_55 = {};
        wp::float64 adj_56 = {};
        wp::vec_t<3,wp::float64> adj_57 = {};
        wp::int32 adj_58 = {};
        wp::float64 adj_59 = {};
        wp::int32 adj_60 = {};
        wp::float64 adj_61 = {};
        wp::int32 adj_62 = {};
        wp::float64 adj_63 = {};
        wp::vec_t<3,wp::float64> adj_64 = {};
        wp::int32 adj_65 = {};
        wp::float64 adj_66 = {};
        wp::int32 adj_67 = {};
        wp::float64 adj_68 = {};
        wp::int32 adj_69 = {};
        wp::float64 adj_70 = {};
        wp::vec_t<3,wp::float64> adj_71 = {};
        wp::vec_t<3,wp::float64> adj_72 = {};
        wp::vec_t<3,wp::float64> adj_73 = {};
        wp::vec_t<3,wp::float64> adj_74 = {};
        wp::mat_t<3,3,wp::float64> adj_75 = {};
        wp::int32 adj_76 = {};
        wp::float64 adj_77 = {};
        wp::int32 adj_78 = {};
        wp::float64 adj_79 = {};
        wp::int32 adj_80 = {};
        wp::float64 adj_81 = {};
        wp::vec_t<3,wp::float64> adj_82 = {};
        wp::vec_t<3,wp::float64> adj_83 = {};
        wp::int32 adj_84 = {};
        wp::float64 adj_85 = {};
        wp::int32 adj_86 = {};
        wp::float64 adj_87 = {};
        wp::int32 adj_88 = {};
        wp::float64 adj_89 = {};
        wp::vec_t<3,wp::float64> adj_90 = {};
        wp::vec_t<3,wp::float64> adj_91 = {};
        wp::int32 adj_92 = {};
        wp::float64 adj_93 = {};
        wp::int32 adj_94 = {};
        wp::float64 adj_95 = {};
        wp::int32 adj_96 = {};
        wp::float64 adj_97 = {};
        wp::vec_t<3,wp::float64> adj_98 = {};
        wp::vec_t<3,wp::float64> adj_99 = {};
        wp::int32 adj_100 = {};
        wp::float64 adj_101 = {};
        wp::int32 adj_102 = {};
        wp::float64 adj_103 = {};
        wp::int32 adj_104 = {};
        wp::float64 adj_105 = {};
        wp::vec_t<3,wp::float64> adj_106 = {};
        wp::vec_t<3,wp::float64> adj_107 = {};
        wp::vec_t<12,wp::float64> adj_108 = {};
        wp::int32 adj_109 = {};
        wp::float64 adj_110 = {};
        wp::float64 adj_111 = {};
        wp::float64 adj_112 = {};
        wp::float64 adj_113 = {};
        wp::float64 adj_114 = {};
        wp::float64 adj_115 = {};
        wp::int32 adj_116 = {};
        wp::float64 adj_117 = {};
        wp::float64 adj_118 = {};
        wp::float64 adj_119 = {};
        wp::float64 adj_120 = {};
        wp::float64 adj_121 = {};
        wp::float64 adj_122 = {};
        wp::int32 adj_123 = {};
        wp::float64 adj_124 = {};
        wp::float64 adj_125 = {};
        wp::float64 adj_126 = {};
        wp::float64 adj_127 = {};
        wp::float64 adj_128 = {};
        wp::float64 adj_129 = {};
        wp::int32 adj_130 = {};
        wp::float64 adj_131 = {};
        wp::float64 adj_132 = {};
        wp::float64 adj_133 = {};
        wp::float64 adj_134 = {};
        wp::float64 adj_135 = {};
        wp::float64 adj_136 = {};
        wp::int32 adj_137 = {};
        wp::float64 adj_138 = {};
        wp::float64 adj_139 = {};
        wp::float64 adj_140 = {};
        wp::float64 adj_141 = {};
        wp::float64 adj_142 = {};
        wp::float64 adj_143 = {};
        wp::int32 adj_144 = {};
        wp::float64 adj_145 = {};
        wp::float64 adj_146 = {};
        wp::float64 adj_147 = {};
        wp::float64 adj_148 = {};
        wp::float64 adj_149 = {};
        wp::float64 adj_150 = {};
        wp::int32 adj_151 = {};
        wp::float64 adj_152 = {};
        wp::float64 adj_153 = {};
        wp::float64 adj_154 = {};
        wp::float64 adj_155 = {};
        wp::float64 adj_156 = {};
        wp::float64 adj_157 = {};
        wp::int32 adj_158 = {};
        wp::float64 adj_159 = {};
        wp::float64 adj_160 = {};
        wp::float64 adj_161 = {};
        wp::float64 adj_162 = {};
        wp::float64 adj_163 = {};
        wp::float64 adj_164 = {};
        wp::int32 adj_165 = {};
        wp::float64 adj_166 = {};
        wp::float64 adj_167 = {};
        wp::float64 adj_168 = {};
        wp::float64 adj_169 = {};
        wp::float64 adj_170 = {};
        wp::float64 adj_171 = {};
        wp::int32 adj_172 = {};
        wp::float64 adj_173 = {};
        wp::float64 adj_174 = {};
        wp::float64 adj_175 = {};
        wp::float64 adj_176 = {};
        wp::float64 adj_177 = {};
        wp::float64 adj_178 = {};
        wp::int32 adj_179 = {};
        wp::float64 adj_180 = {};
        wp::float64 adj_181 = {};
        wp::float64 adj_182 = {};
        wp::float64 adj_183 = {};
        wp::float64 adj_184 = {};
        wp::float64 adj_185 = {};
        wp::int32 adj_186 = {};
        wp::float64 adj_187 = {};
        wp::float64 adj_188 = {};
        wp::float64 adj_189 = {};
        wp::float64 adj_190 = {};
        wp::float64 adj_191 = {};
        wp::float64 adj_192 = {};
        //---------
        // forward
        // def compute_body_force_energy_val_affine(                                              <L 16>
        // tid = wp.tid()                                                                         <L 27>
        var_0 = builtin_tid1d();
        // if affine_has_constraint[tid]:                                                         <L 28>
        var_1 = wp::address(var_affine_has_constraint, var_0);
        var_2 = wp::load(var_1);
        if (var_2) {
            // return                                                                             <L 29>
            goto label0;
        }
        var_3 = wp::load(var_1);
        // dim = 3                                                                                <L 30>
        // n_dof = (dim + 1) * dim                                                                <L 31>
        var_6 = wp::add(var_4, var_5);
        var_7 = wp::mul(var_6, var_4);
        // mass = mass_matrix[tid]                                                                <L 32>
        var_8 = wp::address(var_mass_matrix, var_0);
        var_10 = wp::load(var_8);
        var_9 = wp::copy(var_10);
        // ext_force = affine_ext_force[tid]                                                      <L 33>
        var_11 = wp::address(var_affine_ext_force, var_0);
        var_13 = wp::load(var_11);
        var_12 = wp::copy(var_13);
        // for i in range(n_dof):                                                                 <L 34>
        var_14 = wp::range(var_7);
        // rest_y_force = affine_ext_y_force[tid]                                                 <L 37>
        var_35 = wp::address(var_affine_ext_y_force, var_0);
        var_37 = wp::load(var_35);
        var_36 = wp::copy(var_37);
        // yi = y[tid]                                                                            <L 38>
        var_38 = wp::address(var_y, var_0);
        var_40 = wp::load(var_38);
        var_39 = wp::copy(var_40);
        // hat_yi = hat_y[tid]                                                                    <L 39>
        var_41 = wp::address(var_hat_y, var_0);
        var_43 = wp::load(var_41);
        var_42 = wp::copy(var_43);
        // O = wp.vec3d(hat_yi[9], hat_yi[10], hat_yi[11])                                        <L 40>
        var_45 = wp::extract(var_42, var_44);
        var_47 = wp::extract(var_42, var_46);
        var_49 = wp::extract(var_42, var_48);
        var_50 = wp::vec_t<3,wp::float64>(var_45, var_47, var_49);
        // A = wp.vec3d(hat_yi[0], hat_yi[1], hat_yi[2])                                          <L 41>
        var_52 = wp::extract(var_42, var_51);
        var_54 = wp::extract(var_42, var_53);
        var_56 = wp::extract(var_42, var_55);
        var_57 = wp::vec_t<3,wp::float64>(var_52, var_54, var_56);
        // B = wp.vec3d(hat_yi[3], hat_yi[4], hat_yi[5])                                          <L 42>
        var_59 = wp::extract(var_42, var_58);
        var_61 = wp::extract(var_42, var_60);
        var_63 = wp::extract(var_42, var_62);
        var_64 = wp::vec_t<3,wp::float64>(var_59, var_61, var_63);
        // C = wp.vec3d(hat_yi[6], hat_yi[7], hat_yi[8])                                          <L 43>
        var_66 = wp::extract(var_42, var_65);
        var_68 = wp::extract(var_42, var_67);
        var_70 = wp::extract(var_42, var_69);
        var_71 = wp::vec_t<3,wp::float64>(var_66, var_68, var_70);
        // R = col_stack3(A - O, B - O, C - O)                                                    <L 44>
        var_72 = wp::sub(var_57, var_50);
        var_73 = wp::sub(var_64, var_50);
        var_74 = wp::sub(var_71, var_50);
        var_75 = col_stack3_1(var_72, var_73, var_74);
        // y_force = cat_4_vec3d(                                                                 <L 45>
        // R * wp.vec3d(rest_y_force[0], rest_y_force[1], rest_y_force[2]),                       <L 46>
        var_77 = wp::extract(var_36, var_76);
        var_79 = wp::extract(var_36, var_78);
        var_81 = wp::extract(var_36, var_80);
        var_82 = wp::vec_t<3,wp::float64>(var_77, var_79, var_81);
        var_83 = wp::mul(var_75, var_82);
        // R * wp.vec3d(rest_y_force[3], rest_y_force[4], rest_y_force[5]),                       <L 47>
        var_85 = wp::extract(var_36, var_84);
        var_87 = wp::extract(var_36, var_86);
        var_89 = wp::extract(var_36, var_88);
        var_90 = wp::vec_t<3,wp::float64>(var_85, var_87, var_89);
        var_91 = wp::mul(var_75, var_90);
        // R * wp.vec3d(rest_y_force[6], rest_y_force[7], rest_y_force[8]),                       <L 48>
        var_93 = wp::extract(var_36, var_92);
        var_95 = wp::extract(var_36, var_94);
        var_97 = wp::extract(var_36, var_96);
        var_98 = wp::vec_t<3,wp::float64>(var_93, var_95, var_97);
        var_99 = wp::mul(var_75, var_98);
        // R * wp.vec3d(rest_y_force[9], rest_y_force[10], rest_y_force[11]),                     <L 49>
        var_101 = wp::extract(var_36, var_100);
        var_103 = wp::extract(var_36, var_102);
        var_105 = wp::extract(var_36, var_104);
        var_106 = wp::vec_t<3,wp::float64>(var_101, var_103, var_105);
        var_107 = wp::mul(var_75, var_106);
        var_108 = cat_4_vec3d_0(var_83, var_91, var_99, var_107);
        // for d in range(12):                                                                    <L 51>
        // energy[tid] += -y_force[d] * yi[d] * scale                                             <L 52>
        var_110 = wp::extract(var_108, var_109);
        var_111 = wp::neg(var_110);
        var_112 = wp::extract(var_39, var_109);
        var_113 = wp::mul(var_111, var_112);
        var_114 = wp::mul(var_113, var_scale);
        // var_115 = wp::atomic_add(var_energy, var_0, var_114);
        var_117 = wp::extract(var_108, var_116);
        var_118 = wp::neg(var_117);
        var_119 = wp::extract(var_39, var_116);
        var_120 = wp::mul(var_118, var_119);
        var_121 = wp::mul(var_120, var_scale);
        // var_122 = wp::atomic_add(var_energy, var_0, var_121);
        var_124 = wp::extract(var_108, var_123);
        var_125 = wp::neg(var_124);
        var_126 = wp::extract(var_39, var_123);
        var_127 = wp::mul(var_125, var_126);
        var_128 = wp::mul(var_127, var_scale);
        // var_129 = wp::atomic_add(var_energy, var_0, var_128);
        var_131 = wp::extract(var_108, var_130);
        var_132 = wp::neg(var_131);
        var_133 = wp::extract(var_39, var_130);
        var_134 = wp::mul(var_132, var_133);
        var_135 = wp::mul(var_134, var_scale);
        // var_136 = wp::atomic_add(var_energy, var_0, var_135);
        var_138 = wp::extract(var_108, var_137);
        var_139 = wp::neg(var_138);
        var_140 = wp::extract(var_39, var_137);
        var_141 = wp::mul(var_139, var_140);
        var_142 = wp::mul(var_141, var_scale);
        // var_143 = wp::atomic_add(var_energy, var_0, var_142);
        var_145 = wp::extract(var_108, var_144);
        var_146 = wp::neg(var_145);
        var_147 = wp::extract(var_39, var_144);
        var_148 = wp::mul(var_146, var_147);
        var_149 = wp::mul(var_148, var_scale);
        // var_150 = wp::atomic_add(var_energy, var_0, var_149);
        var_152 = wp::extract(var_108, var_151);
        var_153 = wp::neg(var_152);
        var_154 = wp::extract(var_39, var_151);
        var_155 = wp::mul(var_153, var_154);
        var_156 = wp::mul(var_155, var_scale);
        // var_157 = wp::atomic_add(var_energy, var_0, var_156);
        var_159 = wp::extract(var_108, var_158);
        var_160 = wp::neg(var_159);
        var_161 = wp::extract(var_39, var_158);
        var_162 = wp::mul(var_160, var_161);
        var_163 = wp::mul(var_162, var_scale);
        // var_164 = wp::atomic_add(var_energy, var_0, var_163);
        var_166 = wp::extract(var_108, var_165);
        var_167 = wp::neg(var_166);
        var_168 = wp::extract(var_39, var_165);
        var_169 = wp::mul(var_167, var_168);
        var_170 = wp::mul(var_169, var_scale);
        // var_171 = wp::atomic_add(var_energy, var_0, var_170);
        var_173 = wp::extract(var_108, var_172);
        var_174 = wp::neg(var_173);
        var_175 = wp::extract(var_39, var_172);
        var_176 = wp::mul(var_174, var_175);
        var_177 = wp::mul(var_176, var_scale);
        // var_178 = wp::atomic_add(var_energy, var_0, var_177);
        var_180 = wp::extract(var_108, var_179);
        var_181 = wp::neg(var_180);
        var_182 = wp::extract(var_39, var_179);
        var_183 = wp::mul(var_181, var_182);
        var_184 = wp::mul(var_183, var_scale);
        // var_185 = wp::atomic_add(var_energy, var_0, var_184);
        var_187 = wp::extract(var_108, var_186);
        var_188 = wp::neg(var_187);
        var_189 = wp::extract(var_39, var_186);
        var_190 = wp::mul(var_188, var_189);
        var_191 = wp::mul(var_190, var_scale);
        // var_192 = wp::atomic_add(var_energy, var_0, var_191);
        //---------
        // reverse
        wp::adj_atomic_add(var_energy, var_0, var_191, adj_energy, adj_0, adj_191, adj_192);
        wp::adj_mul(var_190, var_scale, adj_190, adj_scale, adj_191);
        wp::adj_mul(var_188, var_189, adj_188, adj_189, adj_190);
        wp::adj_extract(var_39, var_186, adj_39, adj_186, adj_189);
        wp::adj_neg(var_187, adj_187, adj_188);
        wp::adj_extract(var_108, var_186, adj_108, adj_186, adj_187);
        wp::adj_atomic_add(var_energy, var_0, var_184, adj_energy, adj_0, adj_184, adj_185);
        wp::adj_mul(var_183, var_scale, adj_183, adj_scale, adj_184);
        wp::adj_mul(var_181, var_182, adj_181, adj_182, adj_183);
        wp::adj_extract(var_39, var_179, adj_39, adj_179, adj_182);
        wp::adj_neg(var_180, adj_180, adj_181);
        wp::adj_extract(var_108, var_179, adj_108, adj_179, adj_180);
        wp::adj_atomic_add(var_energy, var_0, var_177, adj_energy, adj_0, adj_177, adj_178);
        wp::adj_mul(var_176, var_scale, adj_176, adj_scale, adj_177);
        wp::adj_mul(var_174, var_175, adj_174, adj_175, adj_176);
        wp::adj_extract(var_39, var_172, adj_39, adj_172, adj_175);
        wp::adj_neg(var_173, adj_173, adj_174);
        wp::adj_extract(var_108, var_172, adj_108, adj_172, adj_173);
        wp::adj_atomic_add(var_energy, var_0, var_170, adj_energy, adj_0, adj_170, adj_171);
        wp::adj_mul(var_169, var_scale, adj_169, adj_scale, adj_170);
        wp::adj_mul(var_167, var_168, adj_167, adj_168, adj_169);
        wp::adj_extract(var_39, var_165, adj_39, adj_165, adj_168);
        wp::adj_neg(var_166, adj_166, adj_167);
        wp::adj_extract(var_108, var_165, adj_108, adj_165, adj_166);
        wp::adj_atomic_add(var_energy, var_0, var_163, adj_energy, adj_0, adj_163, adj_164);
        wp::adj_mul(var_162, var_scale, adj_162, adj_scale, adj_163);
        wp::adj_mul(var_160, var_161, adj_160, adj_161, adj_162);
        wp::adj_extract(var_39, var_158, adj_39, adj_158, adj_161);
        wp::adj_neg(var_159, adj_159, adj_160);
        wp::adj_extract(var_108, var_158, adj_108, adj_158, adj_159);
        wp::adj_atomic_add(var_energy, var_0, var_156, adj_energy, adj_0, adj_156, adj_157);
        wp::adj_mul(var_155, var_scale, adj_155, adj_scale, adj_156);
        wp::adj_mul(var_153, var_154, adj_153, adj_154, adj_155);
        wp::adj_extract(var_39, var_151, adj_39, adj_151, adj_154);
        wp::adj_neg(var_152, adj_152, adj_153);
        wp::adj_extract(var_108, var_151, adj_108, adj_151, adj_152);
        wp::adj_atomic_add(var_energy, var_0, var_149, adj_energy, adj_0, adj_149, adj_150);
        wp::adj_mul(var_148, var_scale, adj_148, adj_scale, adj_149);
        wp::adj_mul(var_146, var_147, adj_146, adj_147, adj_148);
        wp::adj_extract(var_39, var_144, adj_39, adj_144, adj_147);
        wp::adj_neg(var_145, adj_145, adj_146);
        wp::adj_extract(var_108, var_144, adj_108, adj_144, adj_145);
        wp::adj_atomic_add(var_energy, var_0, var_142, adj_energy, adj_0, adj_142, adj_143);
        wp::adj_mul(var_141, var_scale, adj_141, adj_scale, adj_142);
        wp::adj_mul(var_139, var_140, adj_139, adj_140, adj_141);
        wp::adj_extract(var_39, var_137, adj_39, adj_137, adj_140);
        wp::adj_neg(var_138, adj_138, adj_139);
        wp::adj_extract(var_108, var_137, adj_108, adj_137, adj_138);
        wp::adj_atomic_add(var_energy, var_0, var_135, adj_energy, adj_0, adj_135, adj_136);
        wp::adj_mul(var_134, var_scale, adj_134, adj_scale, adj_135);
        wp::adj_mul(var_132, var_133, adj_132, adj_133, adj_134);
        wp::adj_extract(var_39, var_130, adj_39, adj_130, adj_133);
        wp::adj_neg(var_131, adj_131, adj_132);
        wp::adj_extract(var_108, var_130, adj_108, adj_130, adj_131);
        wp::adj_atomic_add(var_energy, var_0, var_128, adj_energy, adj_0, adj_128, adj_129);
        wp::adj_mul(var_127, var_scale, adj_127, adj_scale, adj_128);
        wp::adj_mul(var_125, var_126, adj_125, adj_126, adj_127);
        wp::adj_extract(var_39, var_123, adj_39, adj_123, adj_126);
        wp::adj_neg(var_124, adj_124, adj_125);
        wp::adj_extract(var_108, var_123, adj_108, adj_123, adj_124);
        wp::adj_atomic_add(var_energy, var_0, var_121, adj_energy, adj_0, adj_121, adj_122);
        wp::adj_mul(var_120, var_scale, adj_120, adj_scale, adj_121);
        wp::adj_mul(var_118, var_119, adj_118, adj_119, adj_120);
        wp::adj_extract(var_39, var_116, adj_39, adj_116, adj_119);
        wp::adj_neg(var_117, adj_117, adj_118);
        wp::adj_extract(var_108, var_116, adj_108, adj_116, adj_117);
        wp::adj_atomic_add(var_energy, var_0, var_114, adj_energy, adj_0, adj_114, adj_115);
        wp::adj_mul(var_113, var_scale, adj_113, adj_scale, adj_114);
        wp::adj_mul(var_111, var_112, adj_111, adj_112, adj_113);
        wp::adj_extract(var_39, var_109, adj_39, adj_109, adj_112);
        wp::adj_neg(var_110, adj_110, adj_111);
        wp::adj_extract(var_108, var_109, adj_108, adj_109, adj_110);
        // adj: energy[tid] += -y_force[d] * yi[d] * scale                                        <L 52>
        // adj: for d in range(12):                                                               <L 51>
        adj_cat_4_vec3d_0(var_83, var_91, var_99, var_107, adj_83, adj_91, adj_99, adj_107, adj_108);
        wp::adj_mul(var_75, var_106, adj_75, adj_106, adj_107);
        wp::adj_vec_t(var_101, var_103, var_105, adj_101, adj_103, adj_105, adj_106);
        wp::adj_extract(var_36, var_104, adj_36, adj_104, adj_105);
        wp::adj_extract(var_36, var_102, adj_36, adj_102, adj_103);
        wp::adj_extract(var_36, var_100, adj_36, adj_100, adj_101);
        // adj: R * wp.vec3d(rest_y_force[9], rest_y_force[10], rest_y_force[11]),                <L 49>
        wp::adj_mul(var_75, var_98, adj_75, adj_98, adj_99);
        wp::adj_vec_t(var_93, var_95, var_97, adj_93, adj_95, adj_97, adj_98);
        wp::adj_extract(var_36, var_96, adj_36, adj_96, adj_97);
        wp::adj_extract(var_36, var_94, adj_36, adj_94, adj_95);
        wp::adj_extract(var_36, var_92, adj_36, adj_92, adj_93);
        // adj: R * wp.vec3d(rest_y_force[6], rest_y_force[7], rest_y_force[8]),                  <L 48>
        wp::adj_mul(var_75, var_90, adj_75, adj_90, adj_91);
        wp::adj_vec_t(var_85, var_87, var_89, adj_85, adj_87, adj_89, adj_90);
        wp::adj_extract(var_36, var_88, adj_36, adj_88, adj_89);
        wp::adj_extract(var_36, var_86, adj_36, adj_86, adj_87);
        wp::adj_extract(var_36, var_84, adj_36, adj_84, adj_85);
        // adj: R * wp.vec3d(rest_y_force[3], rest_y_force[4], rest_y_force[5]),                  <L 47>
        wp::adj_mul(var_75, var_82, adj_75, adj_82, adj_83);
        wp::adj_vec_t(var_77, var_79, var_81, adj_77, adj_79, adj_81, adj_82);
        wp::adj_extract(var_36, var_80, adj_36, adj_80, adj_81);
        wp::adj_extract(var_36, var_78, adj_36, adj_78, adj_79);
        wp::adj_extract(var_36, var_76, adj_36, adj_76, adj_77);
        // adj: R * wp.vec3d(rest_y_force[0], rest_y_force[1], rest_y_force[2]),                  <L 46>
        // adj: y_force = cat_4_vec3d(                                                            <L 45>
        adj_col_stack3_1(var_72, var_73, var_74, adj_72, adj_73, adj_74, adj_75);
        wp::adj_sub(var_71, var_50, adj_71, adj_50, adj_74);
        wp::adj_sub(var_64, var_50, adj_64, adj_50, adj_73);
        wp::adj_sub(var_57, var_50, adj_57, adj_50, adj_72);
        // adj: R = col_stack3(A - O, B - O, C - O)                                               <L 44>
        wp::adj_vec_t(var_66, var_68, var_70, adj_66, adj_68, adj_70, adj_71);
        wp::adj_extract(var_42, var_69, adj_42, adj_69, adj_70);
        wp::adj_extract(var_42, var_67, adj_42, adj_67, adj_68);
        wp::adj_extract(var_42, var_65, adj_42, adj_65, adj_66);
        // adj: C = wp.vec3d(hat_yi[6], hat_yi[7], hat_yi[8])                                     <L 43>
        wp::adj_vec_t(var_59, var_61, var_63, adj_59, adj_61, adj_63, adj_64);
        wp::adj_extract(var_42, var_62, adj_42, adj_62, adj_63);
        wp::adj_extract(var_42, var_60, adj_42, adj_60, adj_61);
        wp::adj_extract(var_42, var_58, adj_42, adj_58, adj_59);
        // adj: B = wp.vec3d(hat_yi[3], hat_yi[4], hat_yi[5])                                     <L 42>
        wp::adj_vec_t(var_52, var_54, var_56, adj_52, adj_54, adj_56, adj_57);
        wp::adj_extract(var_42, var_55, adj_42, adj_55, adj_56);
        wp::adj_extract(var_42, var_53, adj_42, adj_53, adj_54);
        wp::adj_extract(var_42, var_51, adj_42, adj_51, adj_52);
        // adj: A = wp.vec3d(hat_yi[0], hat_yi[1], hat_yi[2])                                     <L 41>
        wp::adj_vec_t(var_45, var_47, var_49, adj_45, adj_47, adj_49, adj_50);
        wp::adj_extract(var_42, var_48, adj_42, adj_48, adj_49);
        wp::adj_extract(var_42, var_46, adj_42, adj_46, adj_47);
        wp::adj_extract(var_42, var_44, adj_42, adj_44, adj_45);
        // adj: O = wp.vec3d(hat_yi[9], hat_yi[10], hat_yi[11])                                   <L 40>
        wp::adj_copy(var_43, adj_41, adj_42);
        wp::adj_load(var_41, adj_41, adj_43);
        wp::adj_address(var_hat_y, var_0, adj_hat_y, adj_0, adj_41);
        // adj: hat_yi = hat_y[tid]                                                               <L 39>
        wp::adj_copy(var_40, adj_38, adj_39);
        wp::adj_load(var_38, adj_38, adj_40);
        wp::adj_address(var_y, var_0, adj_y, adj_0, adj_38);
        // adj: yi = y[tid]                                                                       <L 38>
        wp::adj_copy(var_37, adj_35, adj_36);
        wp::adj_load(var_35, adj_35, adj_37);
        wp::adj_address(var_affine_ext_y_force, var_0, adj_affine_ext_y_force, adj_0, adj_35);
        // adj: rest_y_force = affine_ext_y_force[tid]                                            <L 37>
        var_14 = wp::iter_reverse(var_14);
        start_for_1:;
            if (iter_cmp(var_14) == 0) goto end_for_1;
            var_15 = wp::iter_next(var_14);
        	adj_16 = {};
        	adj_17 = {};
        	adj_18 = {};
            // for j in range(dim + 1):                                                           <L 35>
            var_17 = wp::add(var_4, var_16);
            var_18 = wp::range(var_17);
            var_18 = wp::iter_reverse(var_18);
            start_for_3:;
                if (iter_cmp(var_18) == 0) goto end_for_3;
                var_19 = wp::iter_next(var_18);
            	adj_20 = {};
            	adj_21 = {};
            	adj_22 = {};
            	adj_23 = {};
            	adj_24 = {};
            	adj_25 = {};
            	adj_26 = {};
            	adj_27 = {};
            	adj_28 = {};
            	adj_29 = {};
            	adj_30 = {};
            	adj_31 = {};
            	adj_32 = {};
            	adj_33 = {};
            	adj_34 = {};
                // energy[tid] += -(gravity[i % dim] + ext_force[i % dim]) * y[tid][i] * mass[(i / dim), j] * scale       <L 36>
                var_20 = wp::mod(var_15, var_4);
                var_21 = wp::extract(var_gravity, var_20);
                var_22 = wp::mod(var_15, var_4);
                var_23 = wp::extract(var_12, var_22);
                var_24 = wp::add(var_21, var_23);
                var_25 = wp::neg(var_24);
                var_26 = wp::address(var_y, var_0);
                var_28 = wp::load(var_26);
                var_27 = wp::extract(var_28, var_15);
                var_29 = wp::mul(var_25, var_27);
                var_30 = wp::div(var_15, var_4);
                var_31 = wp::extract(var_9, var_30, var_19);
                var_32 = wp::mul(var_29, var_31);
                var_33 = wp::mul(var_32, var_scale);
                // var_34 = wp::atomic_add(var_energy, var_0, var_33);
                wp::adj_atomic_add(var_energy, var_0, var_33, adj_energy, adj_0, adj_33, adj_34);
                wp::adj_mul(var_32, var_scale, adj_32, adj_scale, adj_33);
                wp::adj_mul(var_29, var_31, adj_29, adj_31, adj_32);
                wp::adj_extract(var_9, var_30, var_19, adj_9, adj_30, adj_19, adj_31);
                wp::adj_div(var_15, var_4, var_30, adj_15, adj_4, adj_30);
                wp::adj_mul(var_25, var_27, adj_25, adj_27, adj_29);
                wp::adj_extract(var_28, var_15, adj_26, adj_15, adj_27);
                wp::adj_load(var_26, adj_26, adj_28);
                wp::adj_address(var_y, var_0, adj_y, adj_0, adj_26);
                wp::adj_neg(var_24, adj_24, adj_25);
                wp::adj_add(var_21, var_23, adj_21, adj_23, adj_24);
                wp::adj_extract(var_12, var_22, adj_12, adj_22, adj_23);
                wp::adj_mod(var_15, var_4, adj_15, adj_4, adj_22);
                wp::adj_extract(var_gravity, var_20, adj_gravity, adj_20, adj_21);
                wp::adj_mod(var_15, var_4, adj_15, adj_4, adj_20);
                // adj: energy[tid] += -(gravity[i % dim] + ext_force[i % dim]) * y[tid][i] * mass[(i / dim), j] * scale  <L 36>
            	goto start_for_3;
            end_for_3:;
            wp::adj_range(var_17, adj_17, adj_18);
            wp::adj_add(var_4, var_16, adj_4, adj_16, adj_17);
            // adj: for j in range(dim + 1):                                                      <L 35>
        	goto start_for_1;
        end_for_1:;
        wp::adj_range(var_7, adj_7, adj_14);
        // adj: for i in range(n_dof):                                                            <L 34>
        wp::adj_copy(var_13, adj_11, adj_12);
        wp::adj_load(var_11, adj_11, adj_13);
        wp::adj_address(var_affine_ext_force, var_0, adj_affine_ext_force, adj_0, adj_11);
        // adj: ext_force = affine_ext_force[tid]                                                 <L 33>
        wp::adj_copy(var_10, adj_8, adj_9);
        wp::adj_load(var_8, adj_8, adj_10);
        wp::adj_address(var_mass_matrix, var_0, adj_mass_matrix, adj_0, adj_8);
        // adj: mass = mass_matrix[tid]                                                           <L 32>
        wp::adj_mul(var_6, var_4, adj_6, adj_4, adj_7);
        wp::adj_add(var_4, var_5, adj_4, adj_5, adj_6);
        // adj: n_dof = (dim + 1) * dim                                                           <L 31>
        // adj: dim = 3                                                                           <L 30>
        if (var_3) {
        wp::adj_load(var_1, adj_1, adj_3);
            label0:;
            // adj: return                                                                        <L 29>
        }
        wp::adj_load(var_1, adj_1, adj_2);
        wp::adj_address(var_affine_has_constraint, var_0, adj_affine_has_constraint, adj_0, adj_1);
        // adj: if affine_has_constraint[tid]:                                                    <L 28>
        // adj: tid = wp.tid()                                                                    <L 27>
        // adj: def compute_body_force_energy_val_affine(                                         <L 16>
        continue;
    }
}



extern "C" __global__ void compute_body_force_energy_grad_soft_x_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_gradient,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::float64 var_scale,
    wp::int32 var_affine_verts_num,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        wp::int32 var_2;
        wp::int32* var_3;
        wp::int32* var_4;
        wp::int32 var_5;
        const wp::int32 var_6 = 1;
        bool var_7;
        wp::int32 var_8;
        wp::int32 var_9;
        wp::int32* var_10;
        wp::int32* var_11;
        wp::int32 var_12;
        const wp::int32 var_13 = 2;
        bool var_14;
        wp::int32 var_15;
        bool var_16;
        bool var_17;
        bool var_18;
        wp::vec_t<3,wp::float64> var_19;
        wp::float64* var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::float64 var_22;
        wp::vec_t<3,wp::float64> var_23;
        wp::vec_t<3,wp::float64> var_24;
        //---------
        // forward
        // def compute_body_force_energy_grad_soft_x(                                             <L 118>
        // tid = wp.tid()                                                                         <L 128>
        var_0 = builtin_tid1d();
        // if soft_has_constraint[tid] or (                                                       <L 129>
        var_1 = wp::address(var_soft_has_constraint, var_0);
        // (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID)                    <L 130>
        var_2 = wp::add(var_0, var_affine_verts_num);
        var_3 = wp::address(var_node2env, var_2);
        var_5 = wp::load(var_3);
        var_4 = wp::address(var_env_states, var_5);
        var_8 = wp::load(var_4);
        var_7 = (var_8 == var_6);
        // or (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED)           <L 131>
        var_9 = wp::add(var_0, var_affine_verts_num);
        var_10 = wp::address(var_node2env, var_9);
        var_12 = wp::load(var_10);
        var_11 = wp::address(var_env_states, var_12);
        var_15 = wp::load(var_11);
        var_14 = (var_15 == var_13);
        var_16 = var_7 || var_14;
        var_17 = wp::load(var_1);
        var_18 = var_17 || var_16;
        if (var_18) {
            // return                                                                             <L 133>
            continue;
        }
        // wp.atomic_add(gradient, tid, -gravity * soft_verts_mass[tid] * scale)                  <L 134>
        var_19 = wp::neg(var_gravity);
        var_20 = wp::address(var_soft_verts_mass, var_0);
        var_22 = wp::load(var_20);
        var_21 = wp::mul(var_19, var_22);
        var_23 = wp::mul(var_21, var_scale);
        var_24 = wp::atomic_add(var_gradient, var_0, var_23);
    }
}



extern "C" __global__ void compute_body_force_energy_grad_soft_x_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::vec_t<3,wp::float64>> var_gradient,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::float64 var_scale,
    wp::int32 var_affine_verts_num,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::int32> var_node2env,
    wp::array_t<wp::int32> var_env_states,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_gradient,
    wp::array_t<wp::float64> adj_soft_verts_mass,
    wp::vec_t<3,wp::float64> adj_gravity,
    wp::float64 adj_scale,
    wp::int32 adj_affine_verts_num,
    wp::array_t<bool> adj_soft_has_constraint,
    wp::array_t<wp::int32> adj_node2env,
    wp::array_t<wp::int32> adj_env_states)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        wp::int32 var_2;
        wp::int32* var_3;
        wp::int32* var_4;
        wp::int32 var_5;
        const wp::int32 var_6 = 1;
        bool var_7;
        wp::int32 var_8;
        wp::int32 var_9;
        wp::int32* var_10;
        wp::int32* var_11;
        wp::int32 var_12;
        const wp::int32 var_13 = 2;
        bool var_14;
        wp::int32 var_15;
        bool var_16;
        bool var_17;
        bool var_18;
        wp::vec_t<3,wp::float64> var_19;
        wp::float64* var_20;
        wp::vec_t<3,wp::float64> var_21;
        wp::float64 var_22;
        wp::vec_t<3,wp::float64> var_23;
        wp::vec_t<3,wp::float64> var_24;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        bool adj_1 = {};
        wp::int32 adj_2 = {};
        wp::int32 adj_3 = {};
        wp::int32 adj_4 = {};
        wp::int32 adj_5 = {};
        wp::int32 adj_6 = {};
        bool adj_7 = {};
        wp::int32 adj_8 = {};
        wp::int32 adj_9 = {};
        wp::int32 adj_10 = {};
        wp::int32 adj_11 = {};
        wp::int32 adj_12 = {};
        wp::int32 adj_13 = {};
        bool adj_14 = {};
        wp::int32 adj_15 = {};
        bool adj_16 = {};
        bool adj_17 = {};
        bool adj_18 = {};
        wp::vec_t<3,wp::float64> adj_19 = {};
        wp::float64 adj_20 = {};
        wp::vec_t<3,wp::float64> adj_21 = {};
        wp::float64 adj_22 = {};
        wp::vec_t<3,wp::float64> adj_23 = {};
        wp::vec_t<3,wp::float64> adj_24 = {};
        //---------
        // forward
        // def compute_body_force_energy_grad_soft_x(                                             <L 118>
        // tid = wp.tid()                                                                         <L 128>
        var_0 = builtin_tid1d();
        // if soft_has_constraint[tid] or (                                                       <L 129>
        var_1 = wp::address(var_soft_has_constraint, var_0);
        // (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID)                    <L 130>
        var_2 = wp::add(var_0, var_affine_verts_num);
        var_3 = wp::address(var_node2env, var_2);
        var_5 = wp::load(var_3);
        var_4 = wp::address(var_env_states, var_5);
        var_8 = wp::load(var_4);
        var_7 = (var_8 == var_6);
        // or (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED)           <L 131>
        var_9 = wp::add(var_0, var_affine_verts_num);
        var_10 = wp::address(var_node2env, var_9);
        var_12 = wp::load(var_10);
        var_11 = wp::address(var_env_states, var_12);
        var_15 = wp::load(var_11);
        var_14 = (var_15 == var_13);
        var_16 = var_7 || var_14;
        var_17 = wp::load(var_1);
        var_18 = var_17 || var_16;
        if (var_18) {
            // return                                                                             <L 133>
            goto label0;
        }
        // wp.atomic_add(gradient, tid, -gravity * soft_verts_mass[tid] * scale)                  <L 134>
        var_19 = wp::neg(var_gravity);
        var_20 = wp::address(var_soft_verts_mass, var_0);
        var_22 = wp::load(var_20);
        var_21 = wp::mul(var_19, var_22);
        var_23 = wp::mul(var_21, var_scale);
        // var_24 = wp::atomic_add(var_gradient, var_0, var_23);
        //---------
        // reverse
        wp::adj_atomic_add(var_gradient, var_0, var_23, adj_gradient, adj_0, adj_23, adj_24);
        wp::adj_mul(var_21, var_scale, adj_21, adj_scale, adj_23);
        wp::adj_mul(var_19, var_22, adj_19, adj_20, adj_21);
        wp::adj_load(var_20, adj_20, adj_22);
        wp::adj_address(var_soft_verts_mass, var_0, adj_soft_verts_mass, adj_0, adj_20);
        wp::adj_neg(var_gravity, adj_gravity, adj_19);
        // adj: wp.atomic_add(gradient, tid, -gravity * soft_verts_mass[tid] * scale)             <L 134>
        if (var_18) {
            label0:;
            // adj: return                                                                        <L 133>
        }
        wp::adj_load(var_1, adj_1, adj_17);
        wp::adj_load(var_11, adj_11, adj_15);
        wp::adj_address(var_env_states, var_12, adj_env_states, adj_10, adj_11);
        wp::adj_load(var_10, adj_10, adj_12);
        wp::adj_address(var_node2env, var_9, adj_node2env, adj_9, adj_10);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_9);
        // adj: or (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_NEWTON_SOLVED)      <L 131>
        wp::adj_load(var_4, adj_4, adj_8);
        wp::adj_address(var_env_states, var_5, adj_env_states, adj_3, adj_4);
        wp::adj_load(var_3, adj_3, adj_5);
        wp::adj_address(var_node2env, var_2, adj_node2env, adj_2, adj_3);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_2);
        // adj: (env_states[node2env[tid + affine_verts_num]] == ENV_STATE_INVALID)               <L 130>
        wp::adj_address(var_soft_has_constraint, var_0, adj_soft_has_constraint, adj_0, adj_1);
        // adj: if soft_has_constraint[tid] or (                                                  <L 129>
        // adj: tid = wp.tid()                                                                    <L 128>
        // adj: def compute_body_force_energy_grad_soft_x(                                        <L 118>
        continue;
    }
}



extern "C" __global__ void compute_body_force_energy_val_soft_cuda_kernel_forward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::float64 var_scale,
    wp::int32 var_affine_verts_num,
    wp::array_t<bool> var_soft_has_constraint)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        bool var_2;
        bool var_3;
        wp::int32 var_4;
        wp::vec_t<3,wp::float64>* var_5;
        wp::float64 var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::float64 var_8;
        wp::float64 var_9;
        wp::float64* var_10;
        wp::float64 var_11;
        wp::float64 var_12;
        wp::float64 var_13;
        //---------
        // forward
        // def compute_body_force_energy_val_soft(                                                <L 56>
        // tid = wp.tid()                                                                         <L 65>
        var_0 = builtin_tid1d();
        // if soft_has_constraint[tid]:                                                           <L 66>
        var_1 = wp::address(var_soft_has_constraint, var_0);
        var_2 = wp::load(var_1);
        if (var_2) {
            // return                                                                             <L 67>
            continue;
        }
        var_3 = wp::load(var_1);
        // energy[tid] += -wp.dot(gravity, x[tid + affine_verts_num]) * scale * soft_verts_mass[tid]       <L 68>
        var_4 = wp::add(var_0, var_affine_verts_num);
        var_5 = wp::address(var_x, var_4);
        var_7 = wp::load(var_5);
        var_6 = wp::dot(var_gravity, var_7);
        var_8 = wp::neg(var_6);
        var_9 = wp::mul(var_8, var_scale);
        var_10 = wp::address(var_soft_verts_mass, var_0);
        var_12 = wp::load(var_10);
        var_11 = wp::mul(var_9, var_12);
        var_13 = wp::atomic_add(var_energy, var_0, var_11);
    }
}



extern "C" __global__ void compute_body_force_energy_val_soft_cuda_kernel_backward(
    wp::launch_bounds_t dim,
    wp::array_t<wp::float64> var_energy,
    wp::array_t<wp::vec_t<3,wp::float64>> var_x,
    wp::array_t<wp::float64> var_soft_verts_mass,
    wp::vec_t<3,wp::float64> var_gravity,
    wp::float64 var_scale,
    wp::int32 var_affine_verts_num,
    wp::array_t<bool> var_soft_has_constraint,
    wp::array_t<wp::float64> adj_energy,
    wp::array_t<wp::vec_t<3,wp::float64>> adj_x,
    wp::array_t<wp::float64> adj_soft_verts_mass,
    wp::vec_t<3,wp::float64> adj_gravity,
    wp::float64 adj_scale,
    wp::int32 adj_affine_verts_num,
    wp::array_t<bool> adj_soft_has_constraint)
{
    for (size_t _idx = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
         _idx < dim.size;
         _idx += static_cast<size_t>(blockDim.x) * static_cast<size_t>(gridDim.x))
    {
        // reset shared memory allocator
        wp::tile_alloc_shared(0, true);

        //---------
        // primal vars
        wp::int32 var_0;
        bool* var_1;
        bool var_2;
        bool var_3;
        wp::int32 var_4;
        wp::vec_t<3,wp::float64>* var_5;
        wp::float64 var_6;
        wp::vec_t<3,wp::float64> var_7;
        wp::float64 var_8;
        wp::float64 var_9;
        wp::float64* var_10;
        wp::float64 var_11;
        wp::float64 var_12;
        wp::float64 var_13;
        //---------
        // dual vars
        wp::int32 adj_0 = {};
        bool adj_1 = {};
        bool adj_2 = {};
        bool adj_3 = {};
        wp::int32 adj_4 = {};
        wp::vec_t<3,wp::float64> adj_5 = {};
        wp::float64 adj_6 = {};
        wp::vec_t<3,wp::float64> adj_7 = {};
        wp::float64 adj_8 = {};
        wp::float64 adj_9 = {};
        wp::float64 adj_10 = {};
        wp::float64 adj_11 = {};
        wp::float64 adj_12 = {};
        wp::float64 adj_13 = {};
        //---------
        // forward
        // def compute_body_force_energy_val_soft(                                                <L 56>
        // tid = wp.tid()                                                                         <L 65>
        var_0 = builtin_tid1d();
        // if soft_has_constraint[tid]:                                                           <L 66>
        var_1 = wp::address(var_soft_has_constraint, var_0);
        var_2 = wp::load(var_1);
        if (var_2) {
            // return                                                                             <L 67>
            goto label0;
        }
        var_3 = wp::load(var_1);
        // energy[tid] += -wp.dot(gravity, x[tid + affine_verts_num]) * scale * soft_verts_mass[tid]       <L 68>
        var_4 = wp::add(var_0, var_affine_verts_num);
        var_5 = wp::address(var_x, var_4);
        var_7 = wp::load(var_5);
        var_6 = wp::dot(var_gravity, var_7);
        var_8 = wp::neg(var_6);
        var_9 = wp::mul(var_8, var_scale);
        var_10 = wp::address(var_soft_verts_mass, var_0);
        var_12 = wp::load(var_10);
        var_11 = wp::mul(var_9, var_12);
        // var_13 = wp::atomic_add(var_energy, var_0, var_11);
        //---------
        // reverse
        wp::adj_atomic_add(var_energy, var_0, var_11, adj_energy, adj_0, adj_11, adj_13);
        wp::adj_mul(var_9, var_12, adj_9, adj_10, adj_11);
        wp::adj_load(var_10, adj_10, adj_12);
        wp::adj_address(var_soft_verts_mass, var_0, adj_soft_verts_mass, adj_0, adj_10);
        wp::adj_mul(var_8, var_scale, adj_8, adj_scale, adj_9);
        wp::adj_neg(var_6, adj_6, adj_8);
        wp::adj_dot(var_gravity, var_7, adj_gravity, adj_5, adj_6);
        wp::adj_load(var_5, adj_5, adj_7);
        wp::adj_address(var_x, var_4, adj_x, adj_4, adj_5);
        wp::adj_add(var_0, var_affine_verts_num, adj_0, adj_affine_verts_num, adj_4);
        // adj: energy[tid] += -wp.dot(gravity, x[tid + affine_verts_num]) * scale * soft_verts_mass[tid]  <L 68>
        if (var_3) {
        wp::adj_load(var_1, adj_1, adj_3);
            label0:;
            // adj: return                                                                        <L 67>
        }
        wp::adj_load(var_1, adj_1, adj_2);
        wp::adj_address(var_soft_has_constraint, var_0, adj_soft_has_constraint, adj_0, adj_1);
        // adj: if soft_has_constraint[tid]:                                                      <L 66>
        // adj: tid = wp.tid()                                                                    <L 65>
        // adj: def compute_body_force_energy_val_soft(                                           <L 56>
        continue;
    }
}

